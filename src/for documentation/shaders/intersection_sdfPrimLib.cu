#include "hip/hip_runtime.h"
#include <optix_math.h>

rtDeclareVariable(float, spRad, , );
rtDeclareVariable(float3, center, , );

rtTextureSampler<float, 3> texSDF0;
rtDeclareVariable(float, shift0, , );
rtDeclareVariable(float, size0, , );

rtTextureSampler<float, 3> texSDF1;
rtDeclareVariable(float, shift1, , );
rtDeclareVariable(float, size1, , );

rtTextureSampler<float, 3> texSDF2;
rtDeclareVariable(float, shift2, , );
rtDeclareVariable(float, size2, , );

rtDeclareVariable(int, numTexDefined, , );
rtDeclareVariable(float, TimeSound, , );
//for SDF

typedef rtCallableProgramId<float(float3, float3)> callT;
rtDeclareVariable(callT, sdfOpPrim, , );

typedef rtCallableProgramId<float(float3, float3)> callT;
rtDeclareVariable(callT, sdfOpPrim2, , );

//------------------------------------
// list of all used or unsued parameters

rtDeclareVariable(float3, varRadius, , );
rtDeclareVariable(float3, varCenter, , );
rtDeclareVariable(float, varK, , );
rtDeclareVariable(float2, varT, , );

__device__
inline float3 max(float3 a, float3 b)
{
    return make_float3(max(a.x, b.x), max(a.y, b.y), max(a.z, b.z));
}

__device__
inline float3 abs(float3 a)
{
    return max(-a, a);
}
//for molecules
RT_CALLABLE_PROGRAM float sdfSphere(float3 p, float3 rad)
{
    return length(p) - rad.x;
}
//for molecules
RT_CALLABLE_PROGRAM float bond(float3 p, float3 a, float3 b, float r)
{
    float3 pa = p - a;
    float3 ba = b - a;
    float h = clamp(dot(pa, ba) / dot(ba, ba), 0.0, 1.0);
    return length(pa - ba*h) - r;
}

RT_CALLABLE_PROGRAM float sdf_opSmoothUnion(float d1, float d2) {
    float k = 0.7;

    float val = 0.5f + 0.5f*(d2 - d1) / k;
    float h = clamp(val, 0.0f, 1.0f);
    float mix_res = (1.0 - h)*d2 + d1*h;
    return mix_res - k*h*(1.0 - h);
}

RT_CALLABLE_PROGRAM float sdfBondSphere(float3 p, float3 a, float3 b, float rad1, float rad2)
{
    //float bf = bond(p, a, b, 0.1);
    float f1 = length(p - a) - rad1;
    float f2 = length(p - b) - rad2;

    //float f = sdf_opSmoothUnion(f1, bf);
    //return sdf_opSmoothUnion(f2, f);
    return sdf_opSmoothUnion(f2, f1);
}

RT_CALLABLE_PROGRAM float sdfDynSphere(float3 p, float3 p2, float3 rad)
{
    //float3 pp = p2*TimeSound + (1.0 - TimeSound)*p;

    float f1 = length(p) - rad.x;
    float f2 = length(p2) - rad.x;// - make_float3(0.5, 0, 0)
    float f = sdf_opSmoothUnion(f1, f2);
    for (int i = 0; i <= 10; i++)
    {
        float3 pp = p2*i / 10.0 + (1.0 - i / 10.0)*p;
        f1 = length(pp) - rad.x;
        f = sdf_opSmoothUnion(f, f1);
    }
    return  f;//f2*TimeSound + (1.0 - TimeSound)*f1;
}

RT_CALLABLE_PROGRAM float sdfDynSphere2(float3 p, float3 p2, float3 rad)
{
    float3 pp = p2*TimeSound + (1.0 - TimeSound)*p;
    return  length(pp) - rad.x;
}

//primitives
RT_CALLABLE_PROGRAM float sdSphere(float3 p, float3 rad)
{
    return length(p - varCenter) - varRadius.x;
}

RT_CALLABLE_PROGRAM float sdfBox(float3 p, float3 rad)
{
    float3 b = varRadius;// optix::make_float3(rad);
    float3 d = abs(p - varCenter) - b;
    return length(max(d, make_float3(0.0f)))
        + min(max(d.x, max(d.y, d.z)), 0.0f); // remove this line for an only partially signed sdf
}
RT_CALLABLE_PROGRAM float sdfRoundBox(float3 p, float3 rad)
{
    float3 b = rad;// optix::make_float3(rad);
    float r = 0.1;
    float3 d = abs(p - varCenter) - b;
    return length(max(d, make_float3(0.0f))) - r
        + min(max(d.x, max(d.y, d.z)), 0.0f); // remove this line for an only partially signed sdf
}

RT_CALLABLE_PROGRAM float sdfTorus(float3  p, float3 rad)
{
    float2 t = varT;
    float3 pp = p - varCenter;
    float2 xz = make_float2(pp.x, pp.z);
    float2 q = make_float2(length(xz) - t.x, pp.y);
    return length(q) - t.y;
}

//operation
RT_CALLABLE_PROGRAM float opRound(float3 p, float3 rad)
{
    //TODO: the rad should be from primitive
    return sdfOpPrim(p, rad) - varK;
}

RT_CALLABLE_PROGRAM float opElongate(float3 p, float3 rad)
{
    float3 h = varRadius;
    float3 q = p - clamp(p, -h, h);
    //should be initial primitive Rad
    return sdfOpPrim(q, rad);
}

RT_CALLABLE_PROGRAM float opSmoothUnion(float3 p, float3 rad) {
    float k = varK;
    float d1 = sdfOpPrim(p, rad);
    float d2 = sdfOpPrim2(p, rad);
    float val = 0.5f + 0.5f*(d2 - d1) / k;
    float h = clamp(val, 0.0f, 1.0f);
    float mix_res = (1.0 - h)*d2 + d1*h;
    return mix_res - k*h*(1.0 - h);
}

RT_CALLABLE_PROGRAM float opSmoothIntersection(float3 p, float3 rad) {
    float k = varK;
    float d1 = sdfOpPrim(p, rad);
    float d2 = sdfOpPrim2(p, rad);
    float val = 0.5f - 0.5f*(d2 - d1) / k;
    float h = clamp(val, 0.0f, 1.0f);
    float mix_res = (1.0 - h)*d2 + d1*h;
    return mix_res + k*h*(1.0 - h);
}

RT_CALLABLE_PROGRAM float opSmoothSubtraction(float3 p, float3 rad) {
    float k = varK;
    float d1 = sdfOpPrim(p, rad);
    float d2 = sdfOpPrim2(p, rad);
    float val = 0.5f - 0.5f*(d2 - d1) / k;
    float h = clamp(val, 0.0f, 1.0f);
    float mix_res = (1.0 - h)*d2 - d1*h;
    return mix_res + k*h*(1.0 - h);
}

RT_CALLABLE_PROGRAM float opSubtraction(float3 p, float3 rad)
{
    float d1 = sdfOpPrim2(p, rad);
    float d2 = sdfOpPrim(p, rad);
    return max(-d1, d2);
}

RT_CALLABLE_PROGRAM float sdfField(float3 p, float3 rad) {
    float s1;

    switch (numTexDefined) {
      case 1:
      {
    s1 = tex3D<float>(texSDF0, p.x*0.5f + 0.5f, p.y*0.5f + 0.5f, p.z*0.5f + 0.5f);
    //trace back to iso value and shift

    s1 += shift0;
    //dist=index*spacing
    // and devide by texture size so 0,1 range

    //Chamber distance neglect spacing.
    //Distance is computed in voxels

    s1 /= size0;// 138.0f;
                //s1 /= size;
    s1 *= 2.0f;
     break;
   }
    case 2:
    {
        float s_min = tex3D(texSDF0, p.x*0.5f + 0.5f, p.y*0.5f + 0.5f, p.z*0.5f + 0.5f);
        float s_max = tex3D(texSDF1, p.x*0.5f + 0.5f, p.y*0.5f + 0.5f, p.z*0.5f + 0.5f);

        s1 = TimeSound*s_max + (1.0 - TimeSound)*s_min;//trace back to iso value and shift
        float shift = TimeSound*shift1 + (1 - TimeSound)*shift0;
        float size = TimeSound*size1 + (1 - TimeSound)*size0;

        s1 += shift;
        //dist=index*spacing
        // and devide by texture size so 0,1 range

        //Chamber distance neglect spacing.
        //Distance is computed in voxels

        //s1 /= 139.0f;
        s1 /= size;
        s1 *= 2.0f;
        break;
    }
    }
    //float size of texture is currently unsused

    return s1;
}