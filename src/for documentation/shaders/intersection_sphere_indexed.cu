#include "hip/hip_runtime.h"
/*
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_math.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "attributeInfo.h"

using namespace optix;

rtBuffer<float3>    Positions;
rtBuffer<float>    BSRadius;
rtBuffer<int>    BSType;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, pr_pos, attribute primitive_pos, );
rtDeclareVariable(float, pr_rad, attribute primitive_rad, );
//rtDeclareVariable(int, pr_type, attribute primitive_type, );

//rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, ); //for direct tracing of sdf spheres, or defined set of primitives with definde BB

//sets Material index to call
//0 - optical type
//1 - auditory type

rtDeclareVariable(int, MaterialIndex, , );
rtDeclareVariable(int, PNum, , );
rtDeclareVariable(int, numFrames, , );
rtDeclareVariable(float, TimeSound, , );
rtDeclareVariable(attributeInfo, info, attribute info, );

//TODO:for SDF
//still have to be implemented as PTX should be generated automatically
//PTX can automatically generated for some primitives like spheres, boxes and etc
//consider further integration with python

//for SDF

typedef rtCallableProgramId<float(float3, float)> callT;
rtDeclareVariable(callT, sdfPrim, , );

//for accel build
RT_PROGRAM void primitive_bounds(int primIdx, float result[6])
{
    const float3 position = Positions[primIdx];
    //we compute only bounding sphere like parameters
    //the exact bounding box of SDF primitive can be much smaller
    const float radius = BSRadius[primIdx];

    optix::Aabb *aabb = (optix::Aabb *) result;

    aabb->m_min.x = position.x - radius - 1.0;
    aabb->m_min.y = position.y - radius - 1.0;
    aabb->m_min.z = position.z - radius - 1.0;

    aabb->m_max.x = position.x + radius + 1.0;
    aabb->m_max.y = position.y + radius + 1.0;
    aabb->m_max.z = position.z + radius + 1.0;
}

//intersection for molecular data

RT_PROGRAM void intersection_mol(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;

    int time = int(floorf(TimeSound)); //integer part
    if (time > numFrames) time = numFrames;

    float fract = TimeSound - float(time); //interpolation part
    float3 pos2 = make_float3(0);
    float3 pos = make_float3(0);
    float3 pos1 = Positions[primIdx + time*PNum]; //getting correct frame
    if (numFrames > 0) //dynamic
    {
        if ((time + 1) < numFrames) //set frames count
        {
            pos2 = Positions[primIdx + (time + 1)*PNum];
        }
        else {
            pos2 = pos1;
        }
        pos = fract*pos2 + (1.0 - fract)*pos1; //time interpolation
    }
    else
        pos = pos; //static

    const float rad = BSRadius[primIdx];
    const int type = BSType[primIdx];
    const float t = length(pos - ray.origin);
    const float3 pos_along_ray = ray.origin + ray.direction * t;
    tmax = t + 2 * rad;
    if (length(pos - pos_along_ray) < (rad))
    { //TDO: return it && rtPotentialIntersection(t)) {
            //float4 result = hit_hook(x, max_iterations, global_t);
          //  dist = sdfPrim(x - pos, rad);

            //TDO: return it && rtPotentialIntersection(t)) {
              //tmin = t;
              // === Raymarching (Sphere Tracing) Procedure ===
        optix::float3 ray_direction = ray.direction;
        optix::float3 eye = ray.origin;
        //    eye.x -= global_t * 1.2f;
        optix::float3 x = eye;// +tmin * ray_direction;

        const float epsilon = 0.002;//delta;
        const float eps = 0.0001;
        float dist = 0;

        float totalDistance = 0.0;//Jitter * tea<4>(current_prd.seed, frame_number);
        int i = 0;
        bool stop = false;
        while (!stop)
        {
            //dist = sdfPrim(x - pos, x - pos2, rad);
            dist = sdfPrim(x - pos, rad);
            // Step along the ray and accumulate the distance from the origin.
            x += abs(dist) * ray_direction;
            //dist_from_origin += dist * fudgeFactor;
            totalDistance += abs(dist);

            // Check if we're close enough or too far.
            if (dist < epsilon || totalDistance > tmax)
            {
                stop = true;
            }
            else i++;
        }

        // Found intersection?
        if (abs(dist) < epsilon)
        {
            if (rtPotentialIntersection(totalDistance))
            {
                //compute normal for primitive
                float dx = sdfPrim(x - pos + make_float3(eps, 0, 0), rad) - sdfPrim(x - pos - make_float3(eps, 0, 0), rad);
                float dy = sdfPrim(x - pos + make_float3(0, eps, 0), rad) - sdfPrim(x - pos - make_float3(0, eps, 0), rad);
                float dz = sdfPrim(x - pos + make_float3(0, 0, eps), rad) - sdfPrim(x - pos - make_float3(0, 0, eps), rad);

                //varNormal = normalize(make_float3(dx, dy, dz));

                //for material
                //fill attribute data for material
                float2 inf = make_float2(totalDistance, __int_as_float(type));
                info.primInfo = inf;
                info.type = type;
                info.normal = normalize(make_float3(dx, dy, dz));
                info.hit_point = ray.origin + ray.direction * (totalDistance);
                pr_pos = pos;
                pr_rad = rad;
                //pr_type = type;

                rtReportIntersection(MaterialIndex);
            }
        }
    }
}

//bounding box
RT_PROGRAM void boundingbox_mol(int primIdx, float result[6])
{
    const float3 pos = Positions[primIdx];
    const float rad = BSRadius[primIdx];

    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->m_min = pos - make_float3(4 * (rad + 1));
    aabb->m_max = pos + make_float3(4 * (rad + 1));
}