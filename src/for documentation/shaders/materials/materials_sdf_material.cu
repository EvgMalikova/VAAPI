#include "hip/hip_runtime.h"
/*
*/
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../renderer/rt_function.h"
#include "../renderer/per_ray_data.h"
#include "transferFunction.h"
#include "procedures.h"
#include "../attributeInfo.h"
using namespace optix;

rtDeclareVariable(attributeInfo, info, attribute info, );
// the attribute is shared with the material
                                                           //the main purpose of the material is to sort all primitives along the ray
                                                           //the actual mapping to material happens in raytracing function,
                                                           //thus sdf surfaces and volume objects are rendered on the same basis
rtDeclareVariable(float3, pr_pos, attribute primitive_pos, );
rtDeclareVariable(float, pr_rad, attribute primitive_rad, );
//rtDeclareVariable(int, pr_type, attribute primitive_type, );

//rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, ); //for direct tracing of sdf spheres, or defined set of primitives with definde BB
rtDeclareVariable(PerRayData, prd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, rayDist, rtIntersectionDistance, );
rtDeclareVariable(rtObject, sysTopObject, , );




typedef rtCallableProgramId<float2(float, float, float)> callT;
rtDeclareVariable(callT, dynamic, , );

RT_CALLABLE_PROGRAM optix::float2 highlight(float pr, float trbf, float TimeSound)
{
    //no perRayData or attributes
    float r = pr;
    float multOp = 1.0;
    // if (prd.isSoundRay)
    {
        if (fabs(TimeSound - trbf / 10) < pr) //we hit this sphere in time
        {
            //Was particle reached
            float Dur = fabs(TimeSound - trbf / 10); // TimeSound / SoundTimeCoef - trbf;
                                                         //printf(" koef %f \n", TimeSound / SoundTimeCoef);
                                                         //float ArbDur = 4.0 / SoundTimeCoef; //duration of each atom oscilattion
                                                         //if (Dur > 0.0f)
            {
                float mult = 1.0;// exp(ArbDur - Dur);
                r *= cos(Dur)*1.2*mult;
                multOp = 5.4*(cos(Dur)*mult) + 0.1;
            }
            //multOp = 0.5; //highlightment coefficients
            //r *= 1.2;

            // printf("%f ", multOp);
        }
    }
    return optix::make_float2(r, multOp);
}

inline __device__ float3 transfer_function(int t)
{
    // return TFBuffer[t];
    switch (t)
    {
    case 1: //H
        return make_float3(1, 1, 1);
        break;
    case 2: //C
        return make_float3(0.5);
        break;
    case 3: //N
        return make_float3(0, 0, 0.5);
        break;
    case 4: //S
        return make_float3(1, 1, 0);
        break;
    case 5: //O
        return make_float3(1, 0, 0);
        break;
    case 6: //P
        return make_float3(1, 0.5, 0);
        break;
    }
    return make_float3(0, 0, 0);
}

RT_PROGRAM void volume_any_hit()
{
    float trbf = info.primInfo.x;

    // int idx = __float_as_int(primInfo.y);
    float3 hit_sample = ray.origin + ray.direction * trbf;

    float3 pos = pr_pos;
    float r = pr_rad;
    float multOp = 1.0;

    //highlightment in time
    //--------------------------------------------------
    float2 hi = dynamic(r, trbf, prd.TimeSound);
    r = hi.x;
    multOp = hi.y;
    //-----------------------------

    float4 color_sample = make_float4(0.2f);// , 0.9f, 0.9f, 0.9f);//translucent_grays(drbf, t, tf_type);

                                            //int t = __float_as_int(primInfo.y);
    const int t = __float_as_int(info.primInfo.y);

    int maxSteps = 10;
    float step = 2 * r / maxSteps;
    for (int i = 0; i < maxSteps; i++) {
        float3 pp2 = make_float3(hit_sample.x - pos.x, hit_sample.y - pos.y, hit_sample.z - pos.z);
        //float f = sdSphere(pp2, pr_rad);//getAtomVelocity(int(pos.w))* wScale*(sdSphere(pp2, getAtomRadius(int(pos.w))));
                                                                              //-0.25- -0.5 - red-yellow
        float scaleKoef = 0;                                                                      //1.7 -1.8 - green
        float t22 = pr_rad;
        if (t22 < 0) scaleKoef = -0.5;
        else scaleKoef = 1.7;
        //attribute function
        float f = scaleKoef* (sdSphere(pp2, pr_rad));//getAtomVelocity(int(pos.w))* wScale*(sdSphere(pp2, getAtomRadius(int(pos.w))));

                                                                              //check we are inside
        if (f <= 0) {
            float3 mainCol = transfer_function(t);
            float4 col = translucent_grays2(f, abs(t22)); //transfere function
            //col /= 3.0;
            col = make_float4(mainCol.x, mainCol.y, mainCol.z, col.w / 4);
            //0.1 - purple
            //0.3 - blue
            //0.4 - light blue+
            //0.6 - blue-green
            //0.7  -green
            //0.9 - yellow
            //0.0 - red
            // pre-multiply alpha
            col.x *= col.w*multOp;
            col.y *= col.w*multOp;
            col.z *= col.w*multOp;
            // "over" operator for front-to-back blending
            color_sample = color_sample + col*(1.0f - color_sample.w);
        }
        hit_sample += ray.direction * step;
    }

    //summ for ray
    prd.result += color_sample*(1.0f - prd.result.w);

    //perform Blin-Phong
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));
    prd.radiance += normal * 0.5f + 0.5f;

    rtIgnoreIntersection();
}

RT_PROGRAM void auditory_volume_any_hit()
{
    //auditory - write all info to array
    if (prd.cur_prim < MAX_PRIM_ALONG_RAY)
    {
        prd.primitives[prd.cur_prim] = make_float2(info.primInfo.x, float(info.type));// info.primInfo;
        prd.cur_prim++;

        // float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));
         //prd.radiance = normal * 0.5f + 0.5f;
        rtIgnoreIntersection();
    }
}

//auditory raytracing
RT_PROGRAM void auditory_raytrace_hit()
{
    //auditory - write all info to array
    if (prd.cur_prim < MAX_PRIM_ALONG_RAY)
    {
        prd.primitives[prd.cur_prim] = make_float2(info.primInfo.x, 5.0);//float(info.type));// info.primInfo;
        prd.cur_prim++;

        float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));
        float3 dir2 = optix::normalize(ray.direction);
        float3 dirNew = prd.dirCamera - dir2;//2 * optix::normalize(info.normal) + dir2;

        float trbf = info.primInfo.x;

        // int idx = __float_as_int(primInfo.y);
        float3 hit_point = ray.origin + ray.direction * trbf;

        //OptiX::Ray refl_ray( hit_point, R, radiance_ray_type,
        //scene_epsilon );
        //optix::Ray refl_ray = ray;
        //refl_ray.origin = hit_point;
        //refl_ray.direction = dirNew;
        //return optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);

        optix::Ray refl_ray = optix::make_Ray(hit_point, dirNew, 0, 0.00f, RT_DEFAULT_MAX);

        //continue ray tracing with reflected ray
        rtTrace(sysTopObject, refl_ray, prd);
		//printf("tracing done");
        //rtIgnoreIntersection();
    }
}