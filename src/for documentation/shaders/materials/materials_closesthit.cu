#include "hip/hip_runtime.h"
/*
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../renderer/per_ray_data.h"
#include "../basic_lights.h"
#include "../attributeInfo.h"
//#include "per_ray_data.h"

//rtBuffer<float3> TFBuffer;

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// Attributes.

//for sdf
rtDeclareVariable(float3, ambient_light_color, , );
rtDeclareVariable(attributeInfo, info, attribute info, );
rtDeclareVariable(int, useScalar, , );

//for triangulated objects
//TODO:probably now it is broken
rtDeclareVariable(float3, varGeoNormal, attribute GeoNormal, );

//array of lights
rtBuffer<BasicLight> lights;

typedef rtCallableProgramId<float3(int)> callTF;
rtDeclareVariable(callTF, tFunction, , );

//rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

// This closest hit program only uses the geometric normal and the shading normal attributes.
// OptiX will remove all code from the intersection programs for unused attributes automatically.

// Note that the matching between attribute outputs from the intersection program and
// the inputs in the closesthit and anyhit programs is done with the type (here float3) and
// the user defined attribute semantic (e.g. here NORMAL).
// The actual variable name doesn't need to match but it's recommended for clarity.

RT_PROGRAM void closesthit()
{
    // Transform the (unnormalized) object space normals into world space.
    float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));// varNormal));

    // Check if the ray hit the geometry on the frontface or the backface.
    // The geometric normal is always defined on the front face of the geometry.
    // In this implementation the coordinate systems are right-handed and the frontface triangle winding is counter-clockwise (matching OpenGL).

    // If theRay.direction and geometric normal are in the same hemisphere we're looking at a backface.
    if (0.0f < optix::dot(theRay.direction, geoNormal))
    {
        // Flip the shading normal to the backface, because only that is used below.
        // (See later examples for more intricate handling of the frontface condition.)
        normal = -normal;
    }

    // Visualize the resulting world space normal on the surface we're looking on.
    // Transform the normal components from [-1.0f, 1.0f] to the range [0.0f, 1.0f] to get colors for negative values.
    thePrd.radiance = normal * 0.5f + 0.5f;
}

RT_PROGRAM void closesthit_sdf()
{
    /**/
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));

    float Ka = 0.5;
    float Kd = 0.5;
    float Ks = 0.2;
    //todo implement primInfo type to color and a switch for mapping to color
    float3 color = Ka *  ambient_light_color;
    if (useScalar == 1) { //otherwise primInfo is not assigned a value
        if (info.type == 0)
        {
            color = info.useScalar;
        }
        else {
            int t = info.type;// __float_as_int(info.primInfo.y);
            color = tFunction(t);// transfer_function(t);
        }
        color *= Ka;
    }/**/
    //if(useScalar==2)
    //{
    // color = info.color;// transfer_function(t);
   //  color *= Ka;
    //}

    optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    for (int i = 0; i < lights.size(); ++i)
    {
        BasicLight light = lights[i];
        float3 L = optix::normalize(light.pos - hit_point);// varHit);
        float nDl = optix::dot(normal, L);

        //if (nDl > 0)
        //    color += Kd * nDl * light.color; // make_float3(1.0);//

        float phong_exp = 0.1;
        if (nDl > 0) {
            color += Kd * nDl * light.color;

            /* optix::float3 H = optix::normalize(L - theRay.direction);
             float nDh = optix::dot(normal, H);
             if (nDh > 0)
                 color += Ks * light.color * pow(nDh, phong_exp);*/
        }
    }
    thePrd.radiance = color;
}

RT_PROGRAM void closesthit_sdf2()
{
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));
    // Visualize the resulting world space normal on the surface we're looking on.
    // Transform the normal components from [-1.0f, 1.0f] to the range [0.0f, 1.0f] to get colors for negative values.
    thePrd.radiance *= 2.2f;

    //TODO: call two callable programs
    //1 - for optical model
    //2 - for auditory model
}