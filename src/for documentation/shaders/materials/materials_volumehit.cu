#include "hip/hip_runtime.h"
/*

 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../renderer/per_ray_data.h"
#include "../basic_lights.h"
#include "transferFunction.h"
 //#include "per_ray_data.h"
#include "../attributeInfo.h"
using namespace optix;

rtDeclareVariable(attributeInfo, info, attribute info, );
// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

//rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );
//rtDeclareVariable(optix::float3, varHit, attribute hit_point, );

//-----------
//for textures

rtTextureSampler<float, 3> tex0;
rtTextureSampler<float, 3> tex1;
rtTextureSampler<float, 3> tex2;

rtDeclareVariable(int, numTexDefined, , );

//
//for sdf
typedef rtCallableProgramId<float(float3, float3)> callT;
rtDeclareVariable(callT, sdfPrim, , );

//type of rendering
rtDeclareVariable(int, Type, , );

//array of lights
rtBuffer<BasicLight> lights;

//rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

// This closest hit program only uses the geometric normal and the shading normal attributes.
// OptiX will remove all code from the intersection programs for unused attributes automatically.

// Note that the matching between attribute outputs from the intersection program and
// the inputs in the closesthit and anyhit programs is done with the type (here float3) and
// the user defined attribute semantic (e.g. here NORMAL).
// The actual variable name doesn't need to match but it's recommended for clarity.
__device__ float computeVal(float3 p)
{
    float s1 = 0;

    switch (numTexDefined) {
    case 1:
    {
        s1 = tex3D<float>(tex0, p.x*0.5f + 0.5f, p.y*0.5f + 0.5f, p.z*0.5f + 0.5f);
        break;
    }
    case 2:
    {
        float s_min = tex3D(tex0, p.x*0.5f + 0.5f, p.y*0.5f + 0.5f, p.z*0.5f + 0.5f);
        float s_max = tex3D(tex1, p.x*0.5f + 0.5f, p.y*0.5f + 0.5f, p.z*0.5f + 0.5f);

        s1 = thePrd.TimeSound*s_max + (1.0 - thePrd.TimeSound)*s_min;//trace back to iso value and shift

                                                                     // return s1;
        break;
    }
    }
    return s1;
}

__device__ void render_Surface(float3 normal, float3 hit_point)
{
    float Ka = 0.5;
    float Kd = 0.5;
    float Ks = 0.2;

    float s1 = computeVal(hit_point) / 6;

    float4 col = translucent_grays(0.5, s1, 0);
    float3 color = Ka *  make_float3(col);// ambient_light_color;

//	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    for (int i = 0; i < lights.size(); ++i)
    {
        BasicLight light = lights[i];
        float3 L = optix::normalize(light.pos - hit_point);
        float nDl = optix::dot(normal, L);

        //if (nDl > 0)
        //    color += Kd * nDl * light.color; // make_float3(1.0);//

        float phong_exp = 0.1;
        if (nDl > 0) {
            color += Kd * nDl * light.color;

            /* optix::float3 H = optix::normalize(L - theRay.direction);
            float nDh = optix::dot(normal, H);
            if (nDh > 0)
            color += Ks * light.color * pow(nDh, phong_exp);*/
        }
    }

    thePrd.radiance = color;
}

__device__ void render_Volume(float3 normal, float3 hit_point)
{
    float tstep = 0.01;
    float3 pos = hit_point;// eyeRay.o + eyeRay.d*tnear;
    float3 step = theRay.direction*tstep;
    float t = theIntersectionDistance;

    //GetSDF function Value
    int jj = 0;
    float eps = 0.01;
    float4 sum = make_float4(0.2);

    bool stop = false;
    float s1 = sdfPrim(pos, make_float3(0.8));// interpolateSDF(time, pos, texSDF, texSDF_F);

    render_Surface(normal, hit_point);
    // blend
    sum = sum + make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.01)*(1 - sum.w);
    while (!stop)//we are inside the object

    {
        float sample = computeVal(pos) / 6;

        float4 col = translucent_grays(0.1 + s1 / 6, sample, 0);

        //kompute koef k_d
        int isoNum = 4;

        //isosurf highlightment
        //TODO: raymarch transmit;
        /*float rayMarch = 6.0;
        float koef =  (rayMarch - s1) / rayMarch;
        float k_d = fabsf(sinf(koef*3.14f*isoNum));
         k_d *= powf(2.6, koef);
         col.w *= (k_d + 0.2);*/

         // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        // "over" operator for front-to-back blending
        sum = sum + col*(1.0f - sum.w);

        //float op = sum.w;
        //isoNum = 2;

        //float4 currColor = col*(1.0f - sum.w);
        //computation of Blinn-Phong
        //if (k_d >= 0.7) {
        //computeColor(sum, pos, eyeRay, rayMarch, time, s1, currColor, k_d, texSDF, texSDF_F);

        //}

        // exit early if opaque
        if (sum.w > 1.1)
            stop = true;
        //	break;

        t += tstep;

        //get bounding box of the primitive here
        if (t > 20) stop = true;

        pos += step;
        s1 = sdfPrim(pos, make_float3(0.5)); //interpolateSDF(time, pos, texSDF, texSDF_F);
        if (s1 <= eps) stop = true;

        if (s1 > 2.0) {
            render_Surface(normal, hit_point);
            // blend
            sum = sum + make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.4)*(1.0f - sum.w);
        }
    }

    //float3 hitP = eyeRay.o;
    //hitP += eyeRay.d*tnear;
    //float3 cl = make_float3(sum.x, sum.y, sum.z);
    //float faceN =
    // computeColor(sum, hitP, eyeRay, rayMarch, time, rayMarch, sum, 2.01, texSDF, texSDF_F);

    thePrd.radiance = make_float3(sum); //+thePrd.radiance*0.8f;
}

RT_PROGRAM void anyhitvolume_sdf()
{
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));
    optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    switch (Type)
    {
    case 0: //surface
    {
        render_Surface(normal, hit_point);
        break;
    }
    case 1: //volume
    {
        render_Volume(normal, hit_point);
        break;
    }
    }
}