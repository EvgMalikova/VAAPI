#include "hip/hip_runtime.h"
/*
All basic variables for SDFs visual-auditory ray-tracing
 */
#include "sdfGeometryVariables.h"

using namespace optix;

rtBuffer<float3>    Positions;
rtBuffer<float3>    Positions2;
rtBuffer<float>    BSRadius;
rtBuffer<int>    BSType;

//TODO: remove them and think about something with prd or more complex
//variable
rtDeclareVariable(float3, pr_pos, attribute primitive_pos, );
rtDeclareVariable(float, pr_rad, attribute primitive_rad, );

//TODO:for SDF
//still have to be implemented as PTX should be generated automatically
//PTX can automatically generated for some primitives like spheres, boxes and etc
//consider further integration with python

//for SDF

typedef rtCallableProgramId<float(float3, float3, float)> callT;
rtDeclareVariable(callT, sdfPrim, , );

//rtDeclareVariable(float, TimeSound, , );

//------------------------------------------------------
//---intersection with dynamic molecule, use of morphing
//------------------------------------------------------

//intersection for molecular data

RT_PROGRAM void intersection_dyn_mol(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;

    const float3 pos = Positions[primIdx];
    const float3 pos2 = Positions2[primIdx];

    float3 leng = pos2 - pos;
    const float3 cent = leng / 2.0 + pos;
    const float rad = BSRadius[primIdx];
    const int type = BSType[primIdx];

    //const float3 pp=pos2*TimeSound+(1-TimeSound)*pos;
    float3 pp = 0.5*pos2 + 0.5*pos; //new center
    const float t = length(pp - theRay.origin);
    const float3 pos_along_ray = theRay.origin + theRay.direction * t;
    tmax = t + length(leng)*4.0; //to stop sphere tracing

   // if (length(pp - pos_along_ray) < length(leng) + rad)
    { //TDO: return it && rtPotentialIntersection(t)) {
        //tmin = t;
        // === Raymarching (Sphere Tracing) Procedure ===
        optix::float3 ray_direction = theRay.direction;
        optix::float3 eye = theRay.origin;
        //    eye.x -= global_t * 1.2f;
        optix::float3 x = eye;// +tmin * ray_direction;

        const float epsilon = 0.002;//delta;
        const float eps = 0.0001;
        float dist = 0;

        float totalDistance = 0.0;//Jitter * tea<4>(current_prd.seed, frame_number);
        int i = 0;
        bool stop = false;
        while (!stop)
        {
            dist = sdfPrim(x - pos, x - pos2, rad);

            // Step along the ray and accumulate the distance from the origin.
            x += abs(dist) * ray_direction;
            //dist_from_origin += dist * fudgeFactor;
            totalDistance += abs(dist);

            // Check if we're close enough or too far.
            if (dist < epsilon || totalDistance > tmax)
            {
                stop = true;
            }
            else i++;
        }

        // Found intersection?
        if (abs(dist) < epsilon)
        {
            if (rtPotentialIntersection(totalDistance))
            {
                //compute normal for primitive
                float dx = sdfPrim(x - pos + make_float3(eps, 0, 0), x - pos2 + make_float3(eps, 0, 0), rad) - sdfPrim(x - pos - make_float3(eps, 0, 0), x - pos2 - make_float3(eps, 0, 0), rad);
                float dy = sdfPrim(x - pos + make_float3(0, eps, 0), x - pos2 + make_float3(eps, 0, 0), rad) - sdfPrim(x - pos - make_float3(0, eps, 0), x - pos2 - make_float3(eps, 0, 0), rad);
                float dz = sdfPrim(x - pos + make_float3(0, 0, eps), x - pos2 + make_float3(eps, 0, 0), rad) - sdfPrim(x - pos - make_float3(0, 0, eps), x - pos2 - make_float3(eps, 0, 0), rad);

                //varNormal = normalize(make_float3(dx, dy, dz));

                //for material
                //fill attribute data for material
                float2 inf = make_float2(totalDistance, __int_as_float(type));
                info.primInfo = inf;
                info.type = type;
                info.normal = normalize(make_float3(dx, dy, dz));
                info.hit_point = eye + ray_direction * (totalDistance);
                pr_pos = pos;
                pr_rad = rad;
                //pr_type = type;

                rtReportIntersection(MaterialIndex);
            }
        }
    }
}

//bounding box
RT_PROGRAM void boundingbox_dyn_mol(int primIdx, float result[6])
{
    const float3 pos = Positions[primIdx];
    const float rad = BSRadius[primIdx];

    const float3 pos2 = Positions2[primIdx];

    float3 pos_min = fminf(pos, pos2);
    float3 pos_max = fmaxf(pos, pos2);

    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->m_min = pos_min - make_float3(rad);
    aabb->m_max = pos_max + make_float3(rad);
}