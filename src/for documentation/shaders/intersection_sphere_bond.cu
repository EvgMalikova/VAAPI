#include "hip/hip_runtime.h"
/*
All basic variables for SDFs visual-auditory ray-tracing
 */
#include "sdfGeometryVariables.h"


using namespace optix;

rtBuffer<float3>    Positions;
rtBuffer<int2>    Bonds;
rtBuffer<float>    BSRadius;
rtBuffer<int>    BSType;


rtDeclareVariable(float3, pr_pos, attribute primitive_pos, );
rtDeclareVariable(float, pr_rad, attribute primitive_rad, );
//rtDeclareVariable(int, pr_type, attribute primitive_type, );

//rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, ); //for direct tracing of sdf spheres, or defined set of primitives with definde BB


//TODO:for SDF
//still have to be implemented as PTX should be generated automatically
//PTX can automatically generated for some primitives like spheres, boxes and etc
//consider further integration with python

//for SDF

typedef rtCallableProgramId<float(float3, float3, float3, float, float)> callT;
rtDeclareVariable(callT, sdfPrim, , );

//for dynamic staff
rtDeclareVariable(int, PNum, , );
rtDeclareVariable(int, numFrames, , );
rtDeclareVariable(float, TimeSound, , );

//------------------------------------------------------
//---intersection with dynamic molecule, use of morphing
//------------------------------------------------------
inline __device__ float3 transfer_function(int t)
{
    // return TFBuffer[t];
    switch (t)
    {
    case 1: //H
        return make_float3(1, 1, 1);
        break;
    case 2: //C
        return make_float3(0.5);
        break;
    case 3: //N
        return make_float3(0, 0, 0.5);
        break;
    case 4: //S
        return make_float3(1, 1, 0);
        break;
    case 5: //O
        return make_float3(1, 0, 0);
        break;
    case 6: //P
        return make_float3(1, 0.5, 0);
        break;
    }
    return make_float3(0, 0, 0);
}
//intersection for molecular data

RT_PROGRAM void intersection_bond_mol(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;

    const int2 ids = Bonds[primIdx];
    const float rad1 = BSRadius[ids.x - 1];
    const float rad2 = BSRadius[ids.y - 1];

    //---------------

    int time = int(floorf(TimeSound)); //integer part
    if (time > numFrames) time = numFrames;

    float fract = TimeSound - float(time); //interpolation part

    //for bond interpolation
    float3 pos2 = make_float3(0);
    float3 pos = make_float3(0);

    //for frames
    float3 pos12 = make_float3(0);
    float3 pos11 = make_float3(0);

    float3 pos22 = make_float3(0);
    float3 pos21 = make_float3(0);

    //float3 pos1 = Positions[primIdx + time*PNum]; //getting correct frame

    pos11 = Positions[ids.x - 1 + time*PNum];
    pos12 = Positions[ids.y - 1 + time*PNum];

    if (numFrames > 0) //dynamic
    {
        if ((time + 1) < numFrames) //set frames count
        {
            pos21 = Positions[ids.x - 1 + (time + 1)*PNum];
            pos22 = Positions[ids.y - 1 + (time + 1)*PNum];
        }
        else {
            pos21 = pos11;
            pos22 = pos12;
        }
        pos = fract*pos12 + (1.0 - fract)*pos11; //time interpolation
        pos2 = fract*pos22 + (1.0 - fract)*pos21; //time interpolation
    }
    else
    {
        //pos = pos; //static

        pos = Positions[ids.x - 1 + time*PNum];
        pos2 = Positions[ids.y - 1 + time*PNum];
    }
    //--------------------

    float3 leng = pos2 - pos;
    const float3 cent = leng / 2.0 + pos;
    //const float rad = BSRadius[primIdx];
    const int type = BSType[ids.x - 1];
    const int type2 = BSType[ids.y - 1];

    float3 col1 = transfer_function(type);
    float3 col2 = transfer_function(type2);

    //const float3 pp=pos2*TimeSound+(1-TimeSound)*pos;
    float3 pp = 0.5*pos2 + 0.5*pos; //new center
    const float t = length(pp - theRay.origin);
    const float3 pos_along_ray = theRay.origin + theRay.direction * t;
    tmax = t + length(leng)*4.0; //to stop sphere tracing

   // if (length(pp - pos_along_ray) < length(leng) + rad)
    { //TDO: return it && rtPotentialIntersection(t)) {
        //tmin = t;
        // === Raymarching (Sphere Tracing) Procedure ===
        optix::float3 ray_direction = theRay.direction;
        optix::float3 eye = theRay.origin;
        //    eye.x -= global_t * 1.2f;
        optix::float3 x = eye;// +tmin * ray_direction;

        const float epsilon = 0.002;//delta;
        const float eps = 0.0001;
        float dist = 0;

        float totalDistance = 0.0;//Jitter * tea<4>(current_prd.seed, frame_number);
        int i = 0;
        bool stop = false;
        while (!stop)
        {
            dist = sdfPrim(x, pos, pos2, rad1, rad2);

            // Step along the ray and accumulate the distance from the origin.
            x += abs(dist) * ray_direction;
            //dist_from_origin += dist * fudgeFactor;
            totalDistance += abs(dist);

            // Check if we're close enough or too far.
            if (dist < epsilon || totalDistance > tmax)
            {
                stop = true;
            }
            else i++;
        }

        // Found intersection?
        if (abs(dist) < epsilon)
        {
            if (rtPotentialIntersection(totalDistance))
            {
                //compute normal for primitive
                float dx = sdfPrim(x + make_float3(eps, 0, 0), pos, pos2, rad1, rad2) - sdfPrim(x - make_float3(eps, 0, 0), pos, pos2, rad1, rad2);
                float dy = sdfPrim(x + make_float3(0, eps, 0), pos, pos2, rad1, rad2) - sdfPrim(x - make_float3(0, eps, 0), pos, pos2, rad1, rad2);
                float dz = sdfPrim(x + make_float3(0, 0, eps), pos, pos2, rad1, rad2) - sdfPrim(x - make_float3(0, 0, eps), pos, pos2, rad1, rad2);

                //varNormal = normalize(make_float3(dx, dy, dz));

                //compute color
                float d1 = length(x - pos) - rad1;
                float d2 = length(x - pos2) - rad2;
                float3 color;
                /*if (abs(d1) < 0.0001) color = col1;
                else if (abs(d2) < 0.0001) color = col2;
                else*/
                {
                    float d = abs(d1) + abs(d2);
                    color = (d1 / d)*col2 + (d2 / d)*col1;
                }
                //length sdfPrim(x, pos, pos2, rad1, rad2);

                //for material
                //fill attribute data for material
                float2 inf = make_float2(totalDistance, __int_as_float(type));
                info.primInfo = inf;
                info.type = 0; //don't use mapping
                info.useScalar = color;
                info.normal = normalize(make_float3(dx, dy, dz));
                info.hit_point = theRay.origin + theRay.direction * (totalDistance);
                pr_pos = pos;
                pr_rad = rad1;
                //pr_type = type;

                rtReportIntersection(MaterialIndex);
            }
        }
    }
}

//bounding box
RT_PROGRAM void boundingbox_bond_mol(int primIdx, float result[6])
{
    const int2 ids = Bonds[primIdx];
    const float rad1 = BSRadius[ids.x - 1];
    const float rad2 = BSRadius[ids.y - 1];

    float3 pos = Positions[ids.x - 1];
    float3 pos2 = Positions[ids.y - 1];

    float3 pos_min = fminf(pos, pos2);
    float3 pos_max = fmaxf(pos, pos2);

    if (numFrames > 0)
    {
        for (int i = 1; i < numFrames; i++)
        {
            pos = Positions[ids.x - 1 + i*PNum];
            pos2 = Positions[ids.y - 1 + i*PNum];

            pos_min = fminf(fminf(pos, pos2), pos_min);
            pos_max = fmaxf(fmaxf(pos, pos2), pos_max);
        }
    }

    float rad = fmaxf(rad1, rad2);
    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->m_min = pos_min - make_float3(rad);
    aabb->m_max = pos_max + make_float3(rad);
}