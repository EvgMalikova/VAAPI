#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //
#include <optix.h>
#include <optixu/optixu_math_namespace.h>


#include "material.h"
#include "../renderer/per_ray_data.h"
#include "sampling.h"

/*! the implicit state's ray we will intersect against */
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );
/*! the per ray data we operate on */
rtDeclareVariable(PerRayData, prd, rtPayload, );
rtDeclareVariable(rtObject, sysTopObject, , );


/*! the attributes we use to communicate between intersection programs and hit program */
rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );
rtDeclareVariable(optix::float3, varHit, attribute hit_point, );


/*! and finally - that particular material's parameters */
rtDeclareVariable(float3, albedo, , );


// Helper functions for sampling a cosine weighted hemisphere distrobution as needed for the Lambert shading model.

RT_FUNCTION void alignVector(float3 const& axis, float3& w)
{
    // Align w with axis.
    const float s = copysign(1.0f, axis.z);
    w.z *= s;
    const float3 h = make_float3(axis.x, axis.y, axis.z + s);
    const float  k = optix::dot(w, h) / (1.0f + fabsf(axis.z));
    w = k * h - w;
}

RT_FUNCTION void unitSquareToCosineHemisphere(const float2 sample, float3 const& axis, float3& w, float& pdf)
{
    // Choose a point on the local hemisphere coordinates about +z.
    const float theta = 2.0f * M_PIf * sample.x;
    const float r = sqrtf(sample.y);
    w.x = r * cosf(theta);
    w.y = r * sinf(theta);
    w.z = 1.0f - w.x * w.x - w.y * w.y;
    w.z = (0.0f < w.z) ? sqrtf(w.z) : 0.0f;

    pdf = w.z * M_1_PIf;

    // Align with axis.
    alignVector(axis, w);
}




/*! the actual scatter function - in Pete's reference code, that's a
  virtual function, but since we have a different function per program
  we do not need this here */
inline __device__ bool scatter(const optix::Ray &ray_in,
                               optix::float3 &attenuation,
                               optix::Ray &scattered)
{
  optix::float3 normal    = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));
  

 // optix::float3 jit = optix::make_float3(rng(prd.seed), rng(prd.seed), rng(prd.seed));// / prd.screen) * 2.0f - 1.0f;
 // jit = (jit / 10)*2-1.0;
  optix::float3 hit = ray_in.origin+ theIntersectionDistance*ray_in.direction;
  //+optix::make_float3(rng(seed))  - fixes the problem

  // Lambert sampling: Cosine weighted hemisphere sampling above the shading normal.
  // This calculates the ray.direction for the next path segment in wi and its probability density function value in pdf.
  unitSquareToCosineHemisphere(rng2(prd.seed), normal, prd.wi, prd.pdf);


  optix::float3 target = hit + normal + prd.wi*prd.pdf;
  //if (theIntersectionDistance != 0) printf("intersection %f,", theIntersectionDistance);
  scattered    = optix::Ray(hit,
                            (target- hit),
                            /*type*/0,
                            /*tmin*/1e-3f,
                            /*tmax*/RT_DEFAULT_MAX);
  attenuation  = albedo;
  return true;
}

RT_PROGRAM void closesthit_sdf()
{
  optix::Ray scattered;
  float3     attenuation=make_float3(0);
  if (prd.depth < 100 && scatter(theRay,attenuation,scattered)) {
    PerRayData rec;
    rec.depth = prd.depth+1;
    rec.seed = prd.seed;
    rtTrace(sysTopObject,scattered,rec);
    prd.radiance = attenuation * rec.radiance;
  } else {
    prd.radiance = make_float3(0);
  }
}
