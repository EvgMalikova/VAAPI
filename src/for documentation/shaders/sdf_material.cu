#include "hip/hip_runtime.h"
/*
*/
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "renderer/rt_function.h"
#include "renderer/per_ray_data.h"
#include "transferFunction.h"
#include "primitives.h"

using namespace optix;

rtDeclareVariable(float2, primInfo, attribute primInfo, ); // the attribute is shared with the material
                                                           //the main purpose of the material is to sort all primitives along the ray
                                                           //the actual mapping to material happens in raytracing function, 
                                                           //thus sdf surfaces and volume objects are rendered on the same basis
rtDeclareVariable(float3, pr_pos, attribute primitive_pos, );
rtDeclareVariable(float, pr_rad, attribute primitive_rad, );

rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, ); //for direct tracing of sdf spheres, or defined set of primitives with definde BB
rtDeclareVariable(PerRayData,  prd,            rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, rayDist, rtIntersectionDistance, );

typedef rtCallableProgramId<float2(float,float,float)> callT;
rtDeclareVariable(callT, dynamic, , );

RT_CALLABLE_PROGRAM optix::float2 highlight(float pr, float trbf, float TimeSound)
{
    //no perRayData or attributes
    float r = pr;
    float multOp = 1.0;
    // if (prd.isSoundRay) 
    {

        if (fabs(TimeSound - trbf / 10) < pr) //we hit this sphere in time
        {

            //Was particle reached
            float Dur = fabs(TimeSound - trbf / 10); // TimeSound / SoundTimeCoef - trbf;
                                                         //printf(" koef %f \n", TimeSound / SoundTimeCoef);
                                                         //float ArbDur = 4.0 / SoundTimeCoef; //duration of each atom oscilattion
                                                         //if (Dur > 0.0f)
            {
                float mult = 1.0;// exp(ArbDur - Dur);
                r *= cos(Dur)*1.2*mult;
                multOp = 5.4*(cos(Dur)*mult) + 0.1;
            }
            //multOp = 0.5; //highlightment coefficients
            //r *= 1.2;

            // printf("%f ", multOp);
        }
    }
    return optix::make_float2(r, multOp);
}

RT_PROGRAM void volume_any_hit()
{

        float trbf = primInfo.x;

        int idx = __float_as_int(primInfo.y);
        float3 hit_sample = ray.origin + ray.direction * trbf;

        float3 pos = pr_pos;
        float r = pr_rad;
        float multOp = 1.0;

        //highlightment in time
        //--------------------------------------------------
       float2 hi= dynamic(r, trbf, prd.TimeSound);
       r = hi.x;
       multOp = hi.y;
        //-----------------------------


        float4 color_sample = make_float4(0.2f);// , 0.9f, 0.9f, 0.9f);//translucent_grays(drbf, t, tf_type);
        int maxSteps = 10;
        float step = 2 * r / maxSteps;
        for (int i = 0; i < maxSteps; i++) {

            float3 pp2 = make_float3(hit_sample.x - pos.x, hit_sample.y - pos.y, hit_sample.z - pos.z);
            //float f = sdSphere(pp2, pr_rad);//getAtomVelocity(int(pos.w))* wScale*(sdSphere(pp2, getAtomRadius(int(pos.w))));
                                                                                  //-0.25- -0.5 - red-yellow
            float scaleKoef = 0;                                                                      //1.7 -1.8 - green 
            float t22 = pr_rad;
            if (t22 < 0) scaleKoef = -0.5;
            else scaleKoef = 1.7;
            //attribute function
            float f = scaleKoef* (sdSphere(pp2, pr_rad));//getAtomVelocity(int(pos.w))* wScale*(sdSphere(pp2, getAtomRadius(int(pos.w))));

                                                                                  //check we are inside
            if (f <= 0) {
                //f *= pos.w; //sign(r)
                float4 col = translucent_grays2(f, abs(t22)); //transfere function
                //0.1 - purple
                //0.3 - blue
                //0.4 - light blue+
                //0.6 - blue-green
                //0.7  -green
                //0.9 - yellow
                //0.0 - red
                // pre-multiply alpha
                col.x *= col.w*multOp;
                col.y *= col.w*multOp;
                col.z *= col.w*multOp;
                // "over" operator for front-to-back blending
                color_sample = color_sample + col*(1.0f - color_sample.w);
            }
            hit_sample += ray.direction * step;

        }

        //summ for ray
        prd.result += color_sample*(1.0f - prd.result.w);

        //perform Blin-Phong
        float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));
        prd.radiance+= normal * 0.5f + 0.5f;
       



       
        rtIgnoreIntersection();
   
}

RT_PROGRAM void auditory_volume_any_hit()
{
    //auditory - write all info to array
    if (prd.cur_prim < MAX_PRIM_ALONG_RAY)
    {
        prd.primitives[prd.cur_prim] = primInfo;
        prd.cur_prim++;
       

       // float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));
        //prd.radiance = normal * 0.5f + 0.5f;
        rtIgnoreIntersection();
    }
}
//D:\nvidia\volume\final\superbuild\build\jane-build\Release
//TODO: not finished
RT_PROGRAM void volume_texture_hit()
{
    float3 hit_sample = ray.origin + ray.direction * rayDist;

    
    //template
    float4 color_sample = make_float4(0.2f);// , 0.9f, 0.9f, 0.9f);//translucent_grays(drbf, t, tf_type);
    int maxSteps = 10;
    float step = 2 * 10.0 / maxSteps;
    for (int i = 0; i < maxSteps; i++) {

        
        //attribute function
        float f = 1.0;//scaleKoef* (sdSphere(pp2, pr_rad));//getAtomVelocity(int(pos.w))* wScale*(sdSphere(pp2, getAtomRadius(int(pos.w))));
        float t22 = 0.4;
                                                     //check we are inside
        {
            
            float4 col = translucent_grays2(f, abs(t22)); //transfere function
                                                          //0.1 - purple
                                                          //0.3 - blue
                                                          //0.4 - light blue+
                                                          //0.6 - blue-green
                                                          //0.7  -green
                                                          //0.9 - yellow
                                                          //0.0 - red
                                                          // pre-multiply alpha
            col.x *= col.w;
            col.y *= col.w;
            col.z *= col.w;
            // "over" operator for front-to-back blending
            color_sample = color_sample + col*(1.0f - color_sample.w);
        }
        hit_sample += ray.direction * step;

    }

    //summ for ray
    prd.radiance = make_float3(color_sample);// *(1.0f - prd.result.w);

    //perform Blin-Phong
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));
    prd.radiance += normal * 0.5f + 0.5f;
}

RT_PROGRAM void volume_closest_hit()
{
}

