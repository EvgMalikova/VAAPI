#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_math.h>
#include <optixu/optixu_matrix_namespace.h>
#include "../inc/data_structures/vertex_attributes.h"

rtBuffer<VertexAttributes> attributesBuffer;
rtBuffer<uint3>            indicesBuffer;

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, varTangent, attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord, attribute TEXCOORD, );

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

//for SDF

typedef rtCallableProgramId<float(float3, float)> callT;
rtDeclareVariable(callT, hit_hook, , );

// Intersection routine for indexed interleaved triangle data.
RT_PROGRAM void intersection_triangle_indexed(int primitiveIndex)
{
    const uint3 indices = indicesBuffer[primitiveIndex];

    VertexAttributes const& a0 = attributesBuffer[indices.x];
    VertexAttributes const& a1 = attributesBuffer[indices.y];
    VertexAttributes const& a2 = attributesBuffer[indices.z];

    const float3 v0 = a0.vertex;
    const float3 v1 = a1.vertex;
    const float3 v2 = a2.vertex;

    float3 n;
    float  t;
    float  beta;
    float  gamma;

    if (intersect_triangle(theRay, v0, v1, v2, n, t, beta, gamma))
    {
        if (rtPotentialIntersection(t))
        {
            // Barycentric interpolation:
            const float alpha = 1.0f - beta - gamma;

            // Note: No normalization on the TBN attributes here for performance reasons.
            //       It's done after the transformation into world space anyway.
            varGeoNormal = n;
            varTangent = a0.tangent  * alpha + a1.tangent  * beta + a2.tangent  * gamma;
            varNormal = a0.normal   * alpha + a1.normal   * beta + a2.normal   * gamma;
            varTexCoord = a0.texcoord * alpha + a1.texcoord * beta + a2.texcoord * gamma;

            rtReportIntersection(0);
        }
    }
}

__device__
inline float sdSphere(float3 p, float s)
{
    return length(p) - s;
}

RT_PROGRAM void intersection_sdf_sphere(int primIdx)
{
    bool shouldSphereTrace = false;
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;

    const float sqRadius = 100;

    float distance;

    //if (shouldSphereTrace)
    {
        //      Mandelbulb sdf(max_iterations);
        //      MengerSponge sdf(max_iterations);
        //      IFSTest sdf(max_iterations);
        //      sdf.setTime(global_t);
        //      sdf.evalParameters();

        // === Raymarching (Sphere Tracing) Procedure ===
        optix::float3 ray_direction = theRay.direction;
        optix::float3 eye = theRay.origin;
        //    eye.x -= global_t * 1.2f;
        optix::float3 x = eye + tmin * ray_direction;

        float dist_from_origin = tmin;

        const float epsilon = 0.000001;//delta;
        const float eps = 0.0001;
        float dist = 0;

        const float NonLinearPerspective = 1.1;

        const float Jitter = 0.05f;
        float totalDistance = 0.0;//Jitter * tea<4>(current_prd.seed, frame_number);
        int i = 0;
        bool stop = false;
        while (!stop)
        {
            //        if(current_prd.depth == 0)
            //        {
            //            float delta = sin( relative_t * 0.1f ) * 10 + tan(relative_t  * 0.001f) * 4;
            //            float2 rot = rotate( make_float2(originalDir.z, originalDir.y),
            //                                          radians(dist_from_origin * delta) ) * NonLinearPerspective;
            //            ray_direction.z = -rot.x;
            //            ray_direction.y = rot.y;
            //        }

            //      sdf.setTranslateHook(0, make_float3( -global_t * 1.0f, 0.0f, 0.0f ) );
            //      sdf.setRotateHook( 0, make_float3( radians(-global_t / 18.0f), 0.0f, 0.0f) );

            //      float scale = 1.0f;
            //      float3 offset = make_float3(0.92858,0.92858,0.32858);
            //      sdf.setScaleHook( 0, x * scale - offset * (scale - 1.0f));

            //float4 result = hit_hook(x, max_iterations, global_t);
            dist = sdSphere(x, 0.6);
            //float3 trapValue = make_float3(result.y, result.z, result.w);

            // Step along the ray and accumulate the distance from the origin.
            x += dist * ray_direction;
            //dist_from_origin += dist * fudgeFactor;
            totalDistance += dist;

            // Check if we're close enough or too far.
            if (dist < epsilon || totalDistance > tmax)
            {
                stop = true;
            }
            else i++;
        }

        // Found intersection?
        if (dist < epsilon)
        {
            if (rtPotentialIntersection(totalDistance))
            {
                //        sdf.setMaxIterations(14); // more iterations for normal estimate, to fake some more detail
                // varNormal        = calculateNormal(sdf, x, DEL);

                float dx = sdSphere(x + make_float3(eps, 0, 0), 0.6) - sdSphere(x - make_float3(eps, 0, 0), 0.6);
                float dy = sdSphere(x + make_float3(0, eps, 0), 0.6) - sdSphere(x - make_float3(0, eps, 0), 0.6);
                float dz = sdSphere(x + make_float3(0, 0, eps), 0.6) - sdSphere(x - make_float3(0, 0, eps), 0.6);

                varNormal = normalize(make_float3(dx, dy, dz));

                rtReportIntersection(0);
            }
        }
    }
}