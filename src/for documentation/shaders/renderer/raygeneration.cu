#include "hip/hip_runtime.h"
/*
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_math.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "random_number_generators.h"

#include "auditory_rendering.h"

 //optical output buffer
rtBuffer<float4, 2> sysOutputBuffer; // RGBA32F

//auditory output buffer
rtBuffer<auditoryPrim, 2> sysAuditoryOutputBuffer; // RGBA32F

//dynamic/static scene parameters
rtDeclareVariable(int, isDynamic, , ); //the scene is dynamic
rtDeclareVariable(int, computeAuditoryRendering, , ); //should we compute auditory rendering

rtDeclareVariable(rtObject, sysTopObject, , );

//optix vars similar to cuda kernel executions vars
rtDeclareVariable(uint2, theLaunchDim, rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

rtDeclareVariable(float3, sysCameraPosition, , );
rtDeclareVariable(float3, sysCameraU, , );
rtDeclareVariable(float3, sysCameraV, , );
rtDeclareVariable(float3, sysCameraW, , );

//rtDeclareVariable(int, renderWidget, , );
//rtDeclareVariable(float3, sysCords, , );

static __device__ __inline__  optix::Ray ComputeDirPos(PerRayData& prd)
{
    //---------------
    //Linking to cuda threads. This is implemented as in CUDA ADVANCED SAMPLES
    //link pixel number to thread
    const float2 pixel = make_float2(theLaunchIndex);

    //no antializing
    const float2 fragment = pixel + 0.5;

    // The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
    const float2 screen = make_float2(theLaunchDim);

    const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

    const float3 origin = sysCameraPosition;
    const float3 direction = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

    if (isDynamic)
        prd.TimeSound = TimeSound;

    //TODO: we now just compute optical_LaunchDim/auditory_LaunchDim ratio

   /* if (computeAuditoryRendering>0) {
        int numS = 0;
        prd.isSoundRay = isSoundRay(numS, ndc, pixel, screen);
        prd.numS = numS;
    }*/

    // Create ray
    return optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);
}

// Entry point for a pinhole camera.
RT_PROGRAM void raygeneration()
{
    PerRayData prd;
    // Initialize the random number generator seed from the linear pixel index and the iteration index.
    prd.seed = tea<16>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, 0);

    prd.radiance = make_float3(0.0f);
    prd.depth = 0;
    prd.result = make_float4(0.5);
    prd.cur_prim = 0;
    prd.isSoundRay = false;
    prd.rnd = rng(prd.seed);
    prd.TimeSound = 0.0f;
    prd.isDynamic = isDynamic;

    //compute normalized ray direction[-1,1]
    optix::Ray ray = ComputeDirPos(prd);

    // Start tracing ray from the camera and further
    rtTrace(sysTopObject, ray, prd);

    prd.radiance *= make_float3(prd.result);

    sysOutputBuffer[theLaunchIndex] = make_float4(prd.radiance, 1.0f);
}

RT_PROGRAM void auditory_raygeneration()
{
    PerRayData prd;
    // Initialize the random number generator seed from the linear pixel index and the iteration index.
    prd.seed = tea<16>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, 0);

    prd.radiance = make_float3(0.0f);
    prd.depth = 0;
    prd.result = make_float4(0.5);
    prd.cur_prim = 0;
    prd.isSoundRay = false;
    prd.rnd = rng(prd.seed);
    prd.TimeSound = 0.0f;

    for (int i = 0; i < MAX_PRIM_ALONG_RAY; i++)
    {
        prd.primitives[i] = make_float2(0);
    }

    //compute normalized ray direction[-1,1]
    optix::Ray ray = ComputeDirPos(prd);

    //trace for auditory rays
    //if (prd.isSoundRay) {
    prd.TimeSound = TimeSound;

    // Start tracing ray from the camera and further
    rtTrace(sysTopObject, ray, prd);
    //to this point prd is filled with intersection info

    //intf("launched");
    //TODO: write to auditory buffer
    //related to numS (raw_id)
    int num = 0;
    for (int i = 0; i < MAX_PRIM_ALONG_RAY - 1; i++)
    {
        sysAuditoryOutputBuffer[theLaunchIndex][i] = prd.primitives[i];
        if (prd.primitives[i].y > 0) num++;
    }
    sysAuditoryOutputBuffer[theLaunchIndex][MAX_PRIM_ALONG_RAY - 1] = make_float2(prd.cur_prim, num);
    //if (num > 0)
    //printf("%d ", num);
    //TODO: do something with outputsys buff as postprocessing
      //  }
}

//This one for interactive widget
//--------------------------------------------------------------------------------------

//Buffers

rtBuffer<float3, 2> movingPoints; // registered point cloud
rtBuffer<float3, 2> fixedPoints; // to be alligned with point cloud
//for SDF

typedef rtCallableProgramId<float(float3, float3)> callT;
rtDeclareVariable(callT, sdfPrim, , );

//TODO: sphere tracing of the widget
RT_CALLABLE_PROGRAM optix::Ray SphereTraceGeometry(optix::Ray  ray, bool& found)
{
    //TODO: consider case when there is no intersection with geometry
    optix::Ray  ray2 = ray;

    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;

    // === Raymarching (Sphere Tracing) Procedure ===

    float eps = 0.001;
    float t = 0.0002;
    float3 rad = make_float3(1.5);

    for (int i = 0; i < 100; i++)
    {
        optix::float3 p = ray.origin + t* ray.direction;
        float hit = abs(sdfPrim(p, rad)); //make it always positive so we can step to border
        if (hit < 0.0002) //|| t > 20.0)
        {
            //there is an intersectuib

            float dx = sdfPrim(p + make_float3(eps, 0, 0), rad) - sdfPrim(p - make_float3(eps, 0, 0), rad);
            float dy = sdfPrim(p + make_float3(0, eps, 0), rad) - sdfPrim(p - make_float3(0, eps, 0), rad);
            float dz = sdfPrim(p + make_float3(0, 0, eps), rad) - sdfPrim(p - make_float3(0, 0, eps), rad);

            //info.normal = normalize(make_float3(dx, dy, dz));
            ray2.origin = p;// ray.origin + t*ray.direction;
            //make it sphere normal
            //ray2.direction = normalize(make_float3(dx, dy, dz));

            found = true;
            break;
        }

        if (t > 20.0) {
            found = false;
            break; //no intersection
        }
        t += abs(hit); //only positive direction
    }

    return ray2;
}

rtDeclareVariable(float3, widgetCenter, , );
//-----------
//For widget Ray generation
static __device__ __inline__  optix::Ray ComputeDirPosWidget(PerRayData& prd)
{
    //---------------
    //Linking to cuda threads. This is implemented as in CUDA ADVANCED SAMPLES
    //link pixel number to thread
    const float2 pixel = make_float2(theLaunchIndex);

    //no antializing
    const float2 fragment = pixel + 0.5;

    // The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
    const float2 screen = make_float2(theLaunchDim);

    const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

    const float3 origin = widgetCenter;
    const float3 direction = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

    if (isDynamic)
        prd.TimeSound = TimeSound;

    //TODO: we now just compute optical_LaunchDim/auditory_LaunchDim ratio

   /* if (computeAuditoryRendering>0) {
        int numS = 0;
        prd.isSoundRay = isSoundRay(numS, ndc, pixel, screen);
        prd.numS = numS;
    }*/

    // Create ray
    return optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);
}

// Entry point for a widget.
//First we should sphere trace it's geometry
RT_PROGRAM void audio_ray_cast()
{
    PerRayData prd;
    // Initialize the random number generator seed from the linear pixel index and the iteration index.
    prd.seed = tea<16>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, 0);

    prd.radiance = make_float3(0.0f);
    prd.depth = 0;
    prd.result = make_float4(0.5);
    prd.cur_prim = 0;
    prd.isSoundRay = false;
    prd.rnd = rng(prd.seed);
    prd.TimeSound = 0.0f;
    prd.isDynamic = isDynamic;

	
	
	
	 prd.dirCamera=optix::normalize(widgetCenter-sysCameraPosition);
    //----------
    //set fail value by default
    movingPoints[theLaunchIndex] = make_float3(-1000);
    fixedPoints[theLaunchIndex] = make_float3(-1000);

    for (int i = 0; i < MAX_PRIM_ALONG_RAY; i++)
    {
        prd.primitives[i] = make_float2(0);
    }

    //compute normalized ray direction[-1,1]
    optix::Ray ray = ComputeDirPosWidget(prd);

    bool found = false;
    optix::Ray ray2 = SphereTraceGeometry(ray, found);

    if (found) {
        prd.TimeSound = TimeSound;

        // Start tracing ray from the camera and further
        rtTrace(sysTopObject, ray2, prd);
        //to this point prd is filled with intersection info

        //conventional gathering of distance information for auditory rendering
        int num = 0;
        for (int i = 0; i < MAX_PRIM_ALONG_RAY - 1; i++)
        {
            sysAuditoryOutputBuffer[theLaunchIndex][i] = prd.primitives[i];
            if (prd.primitives[i].y > 0) num++;
        }
        sysAuditoryOutputBuffer[theLaunchIndex][MAX_PRIM_ALONG_RAY - 1] = make_float2(prd.cur_prim, num);

        //movingPoints[theLaunchIndex] = make_float3(-1000);

        if (num > 0) //there is an intersection
        {
		printf("%d NUM", num);
            //add a moving point and fixed if there is an
            //intersection
            if (prd.primitives[0].x < 1.5) {
                movingPoints[theLaunchIndex] = ray2.origin;
                fixedPoints[theLaunchIndex] = ray2.origin + ray2.direction*prd.primitives[0].x;
            }
        }
        //   else //there is no intersection, identification point
        //   {
        //       movingPoints[theLaunchIndex] = make_float3(-1000);
        //       fixedPoints[theLaunchIndex] = make_float3(-1000);
        //   }
    } //if (num > 0)
    
    //TODO: do something with outputsys buff as postprocessing
      //  }
}