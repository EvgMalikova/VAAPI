#include "hip/hip_runtime.h"
/*
All basic variables for SDFs heterogeneous objects ray-tracing
*/
#include "sdfGeometryVariables.h"

using namespace optix;

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_math.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "attributeInfo.h"
#include "renderer/random_number_generators.h"

#include "sdfPrimPrograms.h"

rtBuffer<float3>    Positions;
rtBuffer<float>    BSRadius;
rtBuffer<int>    BSType;

//Connectivity info
rtBuffer<int4>    Tets;

rtDeclareVariable(float, MultiscaleParam, , );

rtDeclareVariable(float, sysSceneEpsilon, , );

//for dynamic staff
rtDeclareVariable(int, PNum, , );
rtDeclareVariable(int, numFrames, , );
rtDeclareVariable(float, TimeSound, , );
//-----------------

//for sphere tracing of various primitives

inline __device__ float evaluateFunction(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    float3 pos1 = descPrim.pos[0];
    float3 pos2 = descPrim.pos[1];
    float3 pos3 = descPrim.pos[2];
    float3 pos4 = descPrim.pos[3];

    //TODO: replace with prim
    f = sdfPrim4(x, pos1, pos2, pos3, pos4);

    return f;
}

inline __device__ float3 computeNormal(float eps, float3 x, primParamDesc descPrim)
{
    float dx = evaluateFunction(x + make_float3(eps, 0, 0), descPrim) - evaluateFunction(x - make_float3(eps, 0, 0), descPrim);
    float dy = evaluateFunction(x + make_float3(0, eps, 0), descPrim) - evaluateFunction(x - make_float3(0, eps, 0), descPrim);
    float dz = evaluateFunction(x + make_float3(0, 0, eps), descPrim) - evaluateFunction(x - make_float3(0, 0, eps), descPrim);

    return normalize(make_float3(dx, dy, dz));
}
inline __device__ float SphereTraceForward(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*t;

    float dist;

    float totalDistance = t;
    int i = 0;
    bool stop = false;
    while (!stop)
    {
        dist = evaluateFunction(x, descPrim);// sdfPrim1(x, pos, pos2, rad1, rad2);

                                        // Step along the ray and accumulate the distance from the origin.
        x += abs(dist) * ray_direction;
        totalDistance += abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance >= tmax)
        {
            stop = true;
        }
        if (dist < 0) {
            //x -= abs(dist) * ray_direction;
            totalDistance -= abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

inline __device__ float SphereTraceBack(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*tmax;

    float dist;

    //TODO: getType

    float totalDistance = tmax - t;
    int i = 0;
    bool stop = false;

    while (!stop)
    {
        //dist = sdfPrim1(x, pos, pos2, rad1, rad2);
        dist = evaluateFunction(x, descPrim);// sdfPrim1(x, pos, pos2, rad1, rad2);

                                        // Step along the ray and accumulate the distance from the origin.
        x -= abs(dist) * ray_direction;
        totalDistance -= abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance <= 0)
        {
            stop = true;
        }
        if (dist < 0) {
            x += abs(dist) * ray_direction;
            totalDistance += abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

//------------------------

inline __device__  primParamDesc getTimeData(int primIdx)
{
    int lower = int(floorf(TimeSound));
    int upper = int(ceilf(TimeSound));

    float timeS = TimeSound;
    if (upper > numFrames) upper = numFrames;

    float time = timeS - float(lower);

    float3 pos2 = make_float3(0);
    float3 pos = make_float3(0);

    int4 idx = Tets[primIdx];
    int ids[4];
    ids[0] = idx.x;
    ids[1] = idx.y;
    ids[2] = idx.z;
    ids[3] = idx.w;

    primParamDesc descPrim;
    descPrim.type = 4; //tetra

    float3 cent = make_float3(0);
    for (int j = 0; j < 4; j++)
    {
        float3 pos = Positions[ids[j]];
        descPrim.pos[j] = pos;
        cent += pos;
    }
    /*
    for (int j = 0; j < 4; j++)
    {
        int id = ids[j] + lower*PNum;
        float3 pos1 = Positions[id]; //getting correct frame

        if (numFrames > 0) //dynamic
        {
            id = ids[j] + lower*PNum;
            pos2 = Positions[id];
            pos = time*pos2 + (1.0 - time)*pos1; //time interpolation
        }
        else
        {
            pos = pos1; //static
        }
        descPrim.pos[j] = pos;
        cent += pos;
    }
    */
    cent /= 4;
    const int type = BSType[primIdx];

    descPrim.rad[0] = length(cent - descPrim.pos[0]);

    descPrim.types[0] = type;

    return descPrim;
}

inline __device__   float3 boundIntersection(primParamDesc  descPrim, float3 origin, float3 direction)
{
    float3 cent = make_float3(0);
    for (int j = 0; j < 4; j++)
    {
        cent += descPrim.pos[j];
    }
    cent /= 4.0;
    float rad = length(cent - descPrim.pos[0]);
    const float t = length(cent - origin);
    const float3 pos_along_ray = origin + direction * t;
    float tmax = abs(t + rad);

    float tmin = t;// fmaxf(0.0, t);

    float3 params = make_float3(tmin, tmax, 0.0);
    (length(cent - pos_along_ray) < (rad + 0.1)) ? params.z = 1.0 : params.z = 0.0; //within bounding sphere

    return params;
}

//Main program
RT_PROGRAM void intersection_mol(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;
    float epsilon = 0.01;//delta;
    if (epsilon > sysSceneEpsilon) epsilon = sysSceneEpsilon;
    float eps = 0.001;
    if (eps > sysSceneEpsilon) eps = sysSceneEpsilon;

    /* ------------------
    /* 1) Reading data and accessing current positions for current time
    /---------------------------------------------------------------*/

    primParamDesc descPrim = getTimeData(primIdx);

    //---------------
    float3 interSectParams = boundIntersection(descPrim, theRay.origin, theRay.direction);
    if (interSectParams.z > 0.0)
    {
        tmin = interSectParams.x;
        tmax = interSectParams.y;
        float total = tmin;

        if (rtPotentialIntersection(total))
        {
            //compute normal for primitive
            float3 x = theRay.origin + theRay.direction*total;

            float3 cNormal = computeNormal(eps, x, descPrim);

            //varNormal = normalize(make_float3(dx, dy, dz));

            //for material

            infoH.normal = cNormal;
            infoH.hit_point = theRay.origin + theRay.direction * (total);
            infoH.tmin = total;

            infoH.desc = descPrim;
            infoH.maxDist = tmax - total;
            rtReportIntersection(MaterialIndex);
        }
    }
}

//bounding box
RT_PROGRAM void primitive_bounds(int primIdx, float result[6])
{
    int4 idx = Tets[primIdx];
    int ids[4];
    ids[0] = idx.x;
    ids[1] = idx.y;
    ids[2] = idx.z;
    ids[3] = idx.w;

    float3 minX = Positions[idx.x];
    float3 maxX = Positions[idx.x];
    // for (int lower = 0; lower < PNum; lower++)
   //  {
    for (int j = 0; j < 4; j++)
    {
        int id = ids[j];// +lower*PNum;
        float3 pos = Positions[id]; //getting correct frame

        minX.x = min(minX.x, pos.x);
        maxX.x = max(maxX.x, pos.x);

        minX.y = min(minX.y, pos.y);
        maxX.y = max(maxX.y, pos.y);

        minX.z = min(minX.z, pos.z);
        maxX.z = max(maxX.z, pos.z);
        //      }
    }
    optix::Aabb *aabb = (optix::Aabb *) result;

    aabb->m_min = minX;
    aabb->m_max = maxX;
}