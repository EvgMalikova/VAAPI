#include "hip/hip_runtime.h"
/*
All basic variables for SDFs visual-auditory ray-tracing
*/
#include "sdfGeometryVariables.h"

using namespace optix;
rtDeclareVariable(float3, sysCameraPosition, , );
rtBuffer<float3>    Positions;
rtBuffer<int2>    Bonds;
rtBuffer<float>    BSRadius;
rtBuffer<int>    BSType;
rtDeclareVariable(float, MultiscaleParam, , );

rtDeclareVariable(float, sysSceneEpsilon, , );

//for dynamic staff
rtDeclareVariable(int, PNum, , );
rtDeclareVariable(int, numFrames, , );
rtDeclareVariable(float, TimeSound, , );
//-----------------

//for ray-casting approach structure
typedef rtCallableProgramX<float3(primParamDesc, float3, float3)> callBoundT;
rtDeclareVariable(callBoundT, boundIntersection, , );

typedef rtCallableProgramX<primParamDesc(int)> callReadDataT;
rtDeclareVariable(callReadDataT, getTimeData, , );

//for sphere tracing of various primitives
//for SDF
typedef rtCallableProgramId<float(float3, float3, float)> callTSp;
rtDeclareVariable(callTSp, sdfPrimSp, , );
typedef rtCallableProgramId<float(float3, float3, float3, float3, float3)> callT1;
rtDeclareVariable(callT1, sdfPrim1, , );

typedef rtCallableProgramId<float(float3, float3, float3, float, float)> callT2;
rtDeclareVariable(callT2, sdfPrim2, , );

typedef rtCallableProgramId<float(float3, float3, float3, float3, float, float, float)> callT3;
rtDeclareVariable(callT3, sdfPrim3, , );

inline __device__ float evaluateSDF(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    /* switch (type) {
    case 0: //sphere type
    {
    float3 pos = descPrim.pos[0];
    float rad1 = descPrim.rad[0];
    f = sdfPrim0(x, pos, rad1);
    break;
    }

    case 1://tetra type
    {
    float3 pos = descPrim.pos[0];
    float3 pos2 = descPrim.pos[1];
    float3 pos3 = descPrim.pos[2];
    float3 pos4 = descPrim.pos[3];
    f = sdfPrim1(x, pos, pos2, pos3, pos4);

    break;
    }
    case 2:
    {
    float3 pos = descPrim.pos[0];
    float3 pos2 = descPrim.pos[1];
    float rad1 = descPrim.rad[0];
    float rad2 = descPrim.rad[1];
    f = sdfPrim2(x, pos, pos2, rad1, rad2);

    break;
    }
    case 3:
    {
    float3 pos = descPrim.pos[0];
    float3 pos2 = descPrim.pos[1];
    float3 pos3 = descPrim.pos[2];
    float rad1 = descPrim.rad[0];
    float rad2 = descPrim.rad[1];
    float rad3 = descPrim.rad[2];
    f = sdfPrim3(x, pos, pos2, pos3, rad1, rad2, rad3);

    break;
    }
    }*/
    float3 pos = descPrim.pos[0];
    float rad1 = descPrim.rad[0];
    f = sdfPrimSp(x, pos, rad1);
    return f;
}

inline __device__ float3 computeNormal(float eps, float3 x, primParamDesc descPrim)
{
    /*float3 pos = descPrim.pos[0];
    float3 pos2 = descPrim.pos[1];
    float rad1 = descPrim.rad[0];
    float rad2 = descPrim.rad[1];*/

    float dx = evaluateSDF(x + make_float3(eps, 0, 0), descPrim) - evaluateSDF(x - make_float3(eps, 0, 0), descPrim);
    float dy = evaluateSDF(x + make_float3(0, eps, 0), descPrim) - evaluateSDF(x - make_float3(0, eps, 0), descPrim);
    float dz = evaluateSDF(x + make_float3(0, 0, eps), descPrim) - evaluateSDF(x - make_float3(0, 0, eps), descPrim);

    return normalize(make_float3(dx, dy, dz));
}

inline __device__ float SphereTraceForward(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*t;

    float dist;

    //TODO: getType

    /*float3 pos = descPrim.pos[0];
    float3 pos2 = descPrim.pos[1];
    float rad1 = descPrim.rad[0];
    float rad2 = descPrim.rad[1];*/

    float totalDistance = t;
    int i = 0;
    bool stop = false;
    while (!stop)
    {
        dist = evaluateSDF(x, descPrim);// sdfPrim(x, pos, pos2, rad1, rad2);

                                        // Step along the ray and accumulate the distance from the origin.
        x += abs(dist) * ray_direction;
        totalDistance += abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance >= tmax)
        {
            stop = true;
        }
        if (dist < 0) {
            //x -= abs(dist) * ray_direction;
            totalDistance -= abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

inline __device__ float SphereTraceBack(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*tmax;

    float dist;

    //TODO: getType

    float totalDistance = tmax - t;
    int i = 0;
    bool stop = false;

    while (!stop)
    {
        //dist = sdfPrim(x, pos, pos2, rad1, rad2);
        dist = evaluateSDF(x, descPrim);// sdfPrim(x, pos, pos2, rad1, rad2);

                                        // Step along the ray and accumulate the distance from the origin.
        x -= abs(dist) * ray_direction;
        totalDistance -= abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance <= 0)
        {
            stop = true;
        }
        if (dist < 0) {
            x += abs(dist) * ray_direction;
            totalDistance += abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

//------------------------------------------------------
//---intersection with dynamic molecule, use of morphing
//------------------------------------------------------

inline __device__ float3 transfer_function(int t, float d)
{
    // return TFBuffer[t];
    switch (t)
    {
    case 1: //H
        return make_float3(1, 0, 0);
        break;
    case 2: //C
        return make_float3(0, 0, 1);// 0.5);
        break;
    case 3: //N
        return make_float3(0, 0, 0.5);
        break;
    case 4: //S
        return make_float3(1, 1, 0);
        break;
    case 5: //O
        return make_float3(1, 0, d);
        break;
    case 6: //P
        return make_float3(1, 0.5, 0);
        break;
    case 7: //Si
        return make_float3(1, 1, 1);//255/236, 255/246, 0);
        break;
    }
    return make_float3(0, 0, 0);
}

/*---------------

----------------*/
RT_CALLABLE_PROGRAM primParamDesc ReadData(int primIdx)
{
    const int2 ids = Bonds[primIdx];
    const float rad1 = BSRadius[ids.x - 1];
    const float rad2 = BSRadius[ids.y - 1];

    const int type = BSType[ids.x - 1];
    const int type2 = BSType[ids.y - 1];

    //---------------
    int lower = int(floorf(TimeSound));
    int upper = int(ceilf(TimeSound));

    float timeS = TimeSound;
    if (upper > numFrames) upper = numFrames;

    float time = timeS - float(lower);

    //int time = int(floorf(TimeSound)); //integer part
    //int upper=int(time);

    //if (time > numFrames) time = numFrames;

    //for bond interpolation
    float3 pos2 = make_float3(0);
    float3 pos = make_float3(0);

    //for frames
    float3 pos12 = make_float3(0);
    float3 pos11 = make_float3(0);

    float3 pos22 = make_float3(0);
    float3 pos21 = make_float3(0);

    //float3 pos1 = Positions[primIdx + time*PNum]; //getting correct frame

    pos11 = Positions[ids.x - 1 + lower*PNum];
    pos12 = Positions[ids.y - 1 + lower*PNum];

    pos = Positions[ids.x - 1 + lower*PNum];
    pos2 = Positions[ids.y - 1 + lower*PNum];

    if (numFrames > 0) //dynamic
    {
        pos21 = Positions[ids.x - 1 + upper*PNum];
        pos22 = Positions[ids.y - 1 + upper*PNum];

        pos = time*pos21 + (1.0 - time)*pos11; //time interpolation
        pos2 = time*pos22 + (1.0 - time)*pos12; //time interpolation
    }
    primParamDesc descPrim;

    descPrim.type = 2;
    descPrim.pos[0] = pos;
    descPrim.pos[1] = pos2;
    descPrim.rad[0] = rad1;
    descPrim.rad[1] = rad2;
    descPrim.types[0] = type;
    descPrim.types[1] = type2;

    return descPrim;
}

RT_CALLABLE_PROGRAM  float3 BVInt(primParamDesc  descPrim, float3 origin, float3 direction)
{
    const float3 cent = (descPrim.pos[1] + descPrim.pos[0]) / 2;

    float leng = length(descPrim.pos[1] - cent);
    const float t = length(cent - origin);
    const float3 pos_along_ray = origin + direction * t;

    float boundEps = 0.1; //for extradynamic delta
    float maxRad = fmaxf(descPrim.rad[0], descPrim.rad[1]);
    float traced_bound = leng + maxRad + boundEps;
    float tmax = t + traced_bound; //to stop sphere tracing

    float tt = clamp(t - traced_bound, 0.0f, t - traced_bound);
    float totalDist = tt;

    float tmin = fmaxf(0.0, t - maxRad);
    float3 params = make_float3(tmin, tmax, 0.0);
    (length(cent - pos_along_ray) < traced_bound) ? params.z = 1.0 : params.z = 0.0; //within bounding sphere

    return params;
}
/*-----------
/* Note: Influence a performance seriously if defined as callable
-------------------------/*/
RT_CALLABLE_PROGRAM  float3 BoundingSubVolumesIntersect(primParamDesc descPrim)
{
    const float3 cent = (descPrim.pos[1] + descPrim.pos[0]) / 2;
    float leng = length(descPrim.pos[1] - cent);
    const float t = length(cent - theRay.origin);
    const float3 pos_along_ray = theRay.origin + theRay.direction * t;

    float boundEps = 0.1; //for extradynamic delta
    float maxRad = fmaxf(descPrim.rad[0], descPrim.rad[1]);
    float traced_bound = leng + maxRad + boundEps;
    float tmax = t + traced_bound; //to stop sphere tracing

    float tt = clamp(t - traced_bound, 0.0f, t - traced_bound);
    float totalDist = tt;

    float tmin = fmaxf(0.0, t - maxRad);
    float3 params = make_float3(tmin, tmax, 0.0);
    (length(cent - pos_along_ray) < traced_bound) ? params.z = 1.0 : params.z = 0.0; //within bounding sphere
    /*if (length(cent - pos_along_ray) < traced_bound)
        params.z = 1.0;
    else
        params.z = 0.0;*/
    return params;
}

RT_PROGRAM void intersection_bond_mol(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;
    float epsilon = 0.001;//delta;
    if (epsilon > sysSceneEpsilon) epsilon = sysSceneEpsilon;
    float eps = 0.0001;
    if (eps > sysSceneEpsilon) eps = sysSceneEpsilon;

    /* ------------------
    /* 1) Reading data and accessing current positions for current time
    /---------------------------------------------------------------*/

    primParamDesc descPrim = getTimeData(primIdx);
    //--------------------

    /*
    //---level of detail simple test
    float dist_cam = length(sysCameraPosition - (pos + pos2) / 2);
    float d;
    if (dist_cam < 10.0)
    {
        float interp = dist_cam / 10.0;
        d = optix::clamp(interp, 0.0, 1.0);
    }
    else d = 1;

    //block multiscale test in current implementation
    d = 1;
    */

    //----------------------------

       /* ------------------
    /* 2) Compute intersection with constructive tree subdivisions
    /* one or several bounding spheres
    /---------------------------------------------------------------*/

    float3 interSectParams = boundIntersection(descPrim, theRay.origin, theRay.direction);
    //if within bounding volume intersection
    //float3 interSectParams = BoundingSubVolumesIntersect(descPrim);
    if (interSectParams.z > 0.0)
    {
        tmin = interSectParams.x;
        tmax = interSectParams.y;
        float totalDistance = tmin;
        // === Raymarching (Sphere Tracing) Procedure ===

        totalDistance = SphereTraceForward(epsilon, tmin, tmax, descPrim);

        // Found potential intersection?
        if (totalDistance < tmax) //we found intersection
        {
            float totalDistance2 = SphereTraceBack(epsilon, totalDistance, tmax, descPrim);
            //------------
            if (totalDistance2 > epsilon) { //it is sufficiently large subvolume to ray-cast
                if (rtPotentialIntersection(totalDistance))
                {
                    //compute normal for primitive
                    float3 x = theRay.origin + theRay.direction*totalDistance;

                    float3 cNormal = computeNormal(eps, x, descPrim);

                    //compute color

                    //color at input point
                     /* ------------------
        /* DELETE: initial color
        /---------------------------------------------------------------*/
                    float d = 1;
                    float3 pos = descPrim.pos[0];
                    float3 pos2 = descPrim.pos[1];
                    float rad1 = descPrim.rad[0];
                    float rad2 = descPrim.rad[1];
                    float3 col1 = transfer_function(descPrim.types[0], d);
                    float3 col2 = transfer_function(descPrim.types[1], d);
                    //--------level of detail continue
                    //return mod1*d + (1.0 - d)*mod2;
                    float r1 = d*rad1 / 2 + (1 - d)*rad1;
                    float r2 = d*rad2 / 2 + (1 - d)*rad2;

                    float d1 = length(x - pos) - r1;
                    float d2 = length(x - pos2) - r2;
                    float3 color1;
                    float d_l = abs(d1) + abs(d2);
                    color1 = (d1 / d_l)*col2 + (d2 / d_l)*col1;

                    //color at output point
                    x = theRay.origin + theRay.direction*(totalDistance + totalDistance2);
                    d1 = length(x - pos) - r1;
                    d2 = length(x - pos2) - r2;
                    float3 color2;
                    d_l = abs(d1) + abs(d2);
                    color2 = (d1 / d_l)*col2 + (d2 / d_l)*col1;

                    //avaraging color
                    float3 color = (color1 + color2) / 2;

                    /*------------------
                    /* Fill attributes array for material
                    -------------------------*/

                    // float2 inf = make_float2(totalDistance, __int_as_float(type));
                    // info.primInfo = inf;
                    info.type = 0; //don't use mapping
                    info.useScalar = make_float4(color.x, color.y, color.z, 1.0);
                    info.normal = cNormal;
                    info.hit_point = theRay.origin + theRay.direction * (totalDistance);
                    info.types = make_int2(descPrim.types[0], descPrim.types[1]);
                    info.pos[0] = pos;
                    info.pos[1] = pos2;
                    info.rad[0] = rad1;
                    info.rad[1] = rad2;

                    info.maxDist = totalDistance2;

                    rtReportIntersection(MaterialIndex);
                }
            }
        }
    }
}

//bounding box
RT_PROGRAM void boundingbox_bond_mol(int primIdx, float result[6])
{
    const int2 ids = Bonds[primIdx];
    const float rad1 = BSRadius[ids.x - 1];
    const float rad2 = BSRadius[ids.y - 1];

    float3 pos = Positions[ids.x - 1];
    float3 pos2 = Positions[ids.y - 1];

    float3 pos_min = fminf(pos, pos2);
    float3 pos_max = fmaxf(pos, pos2);

    if (numFrames > 0)
    {
        for (int i = 1; i < numFrames; i++)
        {
            pos = Positions[ids.x - 1 + i*PNum];
            pos2 = Positions[ids.y - 1 + i*PNum];

            pos_min = fminf(fminf(pos, pos2), pos_min);
            pos_max = fmaxf(fmaxf(pos, pos2), pos_max);
        }
    }

    float rad = fmaxf(rad1, rad2);
    optix::Aabb* aabb = (optix::Aabb*)result;
    //increase for ao by 5
    aabb->m_min = pos_min - make_float3(rad + 0.1);
    aabb->m_max = pos_max + make_float3(rad + 0.1);
}