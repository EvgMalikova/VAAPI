#include "hip/hip_runtime.h"
/*
All basic variables for SDFs visual-auditory ray-tracing
 */
#include "sdfGeometryVariables.h"

#include "sdfPrimPrograms.h"
using namespace optix;
rtDeclareVariable(float3, sysCameraPosition, , );
rtBuffer<float3>    Positions;
rtBuffer<int2>    Bonds;
rtBuffer<int4>    Mols4;

rtBuffer<float>    BSRadius;
rtBuffer<int>    BSType;
rtDeclareVariable(float, MultiscaleParam, , );
rtDeclareVariable(float3, pr_pos, attribute primitive_pos, );
rtDeclareVariable(float, pr_rad, attribute primitive_rad, );
//rtDeclareVariable(int, pr_type, attribute primitive_type, );

rtDeclareVariable(float, sysSceneEpsilon, , );

rtDeclareVariable(int, MolSize, , );

//rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, ); //for direct tracing of sdf spheres, or defined set of primitives with definde BB

#include "renderer/per_ray_data.h"
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

//TODO:for SDF
//still have to be implemented as PTX should be generated automatically
//PTX can automatically generated for some primitives like spheres, boxes and etc
//consider further integration with python

//for SDF

//for dynamic staff
rtDeclareVariable(int, PNum, , );
rtDeclareVariable(int, numFrames, , );
rtDeclareVariable(float, TimeSound, , );

inline __device__ float evaluateFunction(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    float3 pos1 = descPrim.pos[0];
    float rad1 = descPrim.rad[0];

    float3 pos2 = descPrim.pos[1];
    float rad2 = descPrim.rad[1];

    float3 pos3 = descPrim.pos[2];
    float rad3 = descPrim.rad[2];

    float3 pos4 = descPrim.pos[3];
    float rad4 = descPrim.rad[3];

    //float3 pos5 = descPrim.pos[4];
    //float rad5 = descPrim.rad[4];

    float3 pos = (pos1 + pos2 + pos3 + pos4) / 4.0;
    f = length(x - pos) - rad1;
    //f = sdfPrim5(x, pos1, pos2, pos3, pos4, rad1, rad2);

    return f;
}

inline __device__ float3 computeNormal(float eps, float3 x, primParamDesc descPrim)
{
    float dx = evaluateFunction(x + make_float3(eps, 0, 0), descPrim) - evaluateFunction(x - make_float3(eps, 0, 0), descPrim);
    float dy = evaluateFunction(x + make_float3(0, eps, 0), descPrim) - evaluateFunction(x - make_float3(0, eps, 0), descPrim);
    float dz = evaluateFunction(x + make_float3(0, 0, eps), descPrim) - evaluateFunction(x - make_float3(0, 0, eps), descPrim);

    return normalize(make_float3(dx, dy, dz));
}

inline __device__ float SphereTraceForward(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*t;

    float dist;

    float totalDistance = t;
    int i = 0;
    bool stop = false;
    while (!stop)
    {
        dist = evaluateFunction(x, descPrim);// sdfPrim1(x, pos, pos2, rad1, rad2);

                                             // Step along the ray and accumulate the distance from the origin.
        x += abs(dist) * ray_direction;
        totalDistance += abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance >= tmax)
        {
            stop = true;
        }
        if (dist < 0) {
            //x -= abs(dist) * ray_direction;
            totalDistance -= abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

inline __device__ float SphereTraceBack(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*tmax;

    float dist;

    //TODO: getType

    float totalDistance = tmax - t;
    int i = 0;
    bool stop = false;

    while (!stop)
    {
        //dist = sdfPrim1(x, pos, pos2, rad1, rad2);
        dist = evaluateFunction(x, descPrim);// sdfPrim1(x, pos, pos2, rad1, rad2);

                                             // Step along the ray and accumulate the distance from the origin.
        x -= abs(dist) * ray_direction;
        totalDistance -= abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance <= 0)
        {
            stop = true;
        }
        if (dist < 0) {
            x += abs(dist) * ray_direction;
            totalDistance += abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

//------------------------

inline __device__  primParamDesc getTimeData(int primIdx)
{
    const int4 idsBonds = Mols4[primIdx];

    const int2 idsB1 = Bonds[idsBonds.x];
    const int2 idsB2 = Bonds[idsBonds.y];
    const int2 idsB3 = Bonds[idsBonds.z];
    const int2 idsB4 = Bonds[idsBonds.w];

    //idsB1.x==idsB2.x;

    //we know there are only 3 atoms - TODO:
    const float rad1 = BSRadius[idsB1.x - 1];
    const float rad2 = BSRadius[idsB1.y - 1];
    const float rad3 = BSRadius[idsB2.y - 1];
    const float rad4 = BSRadius[idsB3.y - 1];
    const float rad5 = BSRadius[idsB4.y - 1];

    const int type = BSType[idsB1.x - 1];
    const int type2 = BSType[idsB1.y - 1];
    const int type3 = BSType[idsB2.y - 1];
    const int type4 = BSType[idsB3.y - 1];
    const int type5 = BSType[idsB4.y - 1];

    //---------------
    int lower = int(floorf(TimeSound));
    int upper = int(ceilf(TimeSound));

    //0,1;1,2;2,3;

    float timeS = TimeSound;
    if (upper > numFrames) upper = numFrames;

    float time = timeS - float(lower);

    primParamDesc descPrim;

    descPrim.type = 5; //sphere data type
    descPrim.pos[0] = Positions[idsB4.y - 1]; //B1.x
    descPrim.pos[1] = Positions[idsB1.y - 1];
    descPrim.pos[2] = Positions[idsB2.y - 1];
    descPrim.pos[3] = Positions[idsB3.y - 1];
    //descPrim.pos[4] = Positions[idsB4.y - 1];

    descPrim.rad[0] = rad1;
    descPrim.rad[1] = rad2;
    descPrim.rad[2] = rad3;
    descPrim.rad[3] = rad4;
    // descPrim.rad[4] = rad5;

    descPrim.types[0] = type;
    descPrim.types[1] = type2;
    descPrim.types[2] = type3;
    descPrim.types[3] = type4;
    // descPrim.types[4] = type5;

    return descPrim;
}
//------------------------------------------------------
//---intersection with dynamic molecule, use of morphing
//------------------------------------------------------

inline __device__   float3 boundIntersection(primParamDesc  descPrim, float3 origin, float3 direction)
{
    // float t1, t2;
    float3 pos_along_ray1, per_ray_data2;
    float rayBoundSphere1, rayBoundSphere2;
    int caseN = -1; //ray miss

                    // float leng = length(pos2 - pos) + length(pos3 - pos);
    const float3 cent = (descPrim.pos[0] + descPrim.pos[1] + descPrim.pos[2] + descPrim.pos[3]) / 4.0;
    // const float delta = fmax(fmaxf(descPrim.rad[1], descPrim.rad[2]), descPrim.rad[0]);
    const float rayBoundSphere = 1.65; // length(cent - descPrim.pos[0]) + delta * 2;//include rad*2 for vibrations movement

    float t = length(cent - theRay.origin);
    float3 pos_along_ray = theRay.origin + theRay.direction * t;
    float tmax = t + 2 * rayBoundSphere; //to stop sphere tracing

                                   /* alternative splitting*/
                                   /*
                                   float3 cent1=(pos+pos2)/2;
                                   float3 cent2= (pos+pos3)/2;

                                   t1 = length(cent1 - theRay.origin);
                                   t2 = length(cent2 - theRay.origin);
                                   pos_along_ray1 = theRay.origin + theRay.direction * t1;
                                   pos_along_ray2 = theRay.origin + theRay.direction * t2;
                                   rayBoundSphere1 =length(cent1 - pos) + delta * 2;
                                   rayBoundSphere2 =length(cent1 - pos) + delta * 2;
                                   */

                                   /*float radiusB1 = length(descPrim.pos[0] - descPrim.pos[1]) / 2 + delta;
                                   float radiusB2 = length(descPrim.pos[0] - descPrim.pos[2]) / 2 + delta;
                                   float radiusB = fmaxf(radiusB1, radiusB2);*/

    float     tmin;

    if (length(cent - pos_along_ray) < rayBoundSphere)
        tmin = fmaxf(0.0, t - rayBoundSphere);
    /*
    t1 = length(descPrim.pos[1] - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t1;

    if (length(descPrim.pos[1] - pos_along_ray) < radiusB1)
        caseN = 1; //left

    t2 = length(descPrim.pos[2] - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t2;
    if (length(descPrim.pos[2] - pos_along_ray) < radiusB2)
        caseN = 2; //left

    t3 = length(descPrim.pos[0] - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t3;
    if (length(descPrim.pos[0] - pos_along_ray) < radiusB)
        caseN = 0; //central intersection

                   //get min intersection
    t = fminf(fminf(t1, t2), t3);

    float     tmin = fmaxf(0.0, t - radiusB);
    */
    float3 params = make_float3(tmin, tmax, 0.0);
    if (caseN >= 0)
    {
        params.z = 1.0;
    }
    else  params.z = 0.0; //within bounding sphere

    return params;
}
RT_PROGRAM void intersection_molecules(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;

    float epsilon = 0.001;//delta;
    if (epsilon > sysSceneEpsilon) epsilon = sysSceneEpsilon;
    float eps = 0.001;
    if (eps > sysSceneEpsilon) eps = sysSceneEpsilon;

    /* ------------------
    /* 1) Reading data and accessing current positions for current time
    /---------------------------------------------------------------*/

    primParamDesc descPrim = getTimeData(primIdx);

    float3 interSectParams = boundIntersection(descPrim, theRay.origin, theRay.direction);
    //if within bounding volume intersection
    //float3 interSectParams = BoundingSubVolumesIntersect(descPrim);
    if (interSectParams.z > 0.0)
    {
        tmin = interSectParams.x;
        tmax = interSectParams.y;
        float totalDistance = tmin;
        // === Raymarching (Sphere Tracing) Procedure ===

        //totalDistance = SphereTraceForward(epsilon, tmin, tmax, descPrim);

        // Found potential intersection?
        //if (totalDistance < tmax) //we found intersection
        {
            //  float totalDistance2 = SphereTraceBack(epsilon, totalDistance, tmax, descPrim);
              //------------
              //if (totalDistance2 > epsilon)
            { //it is sufficiently large subvolume to ray-cast
                if (rtPotentialIntersection(totalDistance))
                {
                    //compute normal for primitive
                    float3 x = theRay.origin + theRay.direction*totalDistance;

                    //float3 cNormal = computeNormal(eps, x, descPrim);

                    //infoH.normal = cNormal;
                    infoH.hit_point = theRay.origin + theRay.direction * (totalDistance);
                    infoH.tmin = tmin;// totalDistance;

                    infoH.desc = descPrim;
                    infoH.maxDist = tmax - tmin;// totalDistance2;
                    rtReportIntersection(MaterialIndex);
                }
            }
        }
    }
}

//bounding box
RT_PROGRAM void boundingbox_molecules(int primIdx, float result[6])
{
    const int4 idsBonds = Mols4[primIdx];

    const int2 idsB1 = Bonds[idsBonds.x];
    const int2 idsB2 = Bonds[idsBonds.y];
    const float3 pos = Positions[idsB1.x - 1];

    float rad = 1.65;
    optix::Aabb* aabb = (optix::Aabb*)result;
    //increase for ao by 5
    aabb->m_min = pos - make_float3(rad);
    aabb->m_max = pos + make_float3(rad);
}