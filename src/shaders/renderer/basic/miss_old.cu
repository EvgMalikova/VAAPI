#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../per_ray_data.h"
#include "../../basic_lights.h"

rtDeclareVariable(PerRayData, thePrd, rtPayload, );
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

//array of lights
//rtBuffer<BasicLight> lights;

rtDeclareVariable(float3, sysBackground, , );

//for sdf
typedef rtCallableProgramId<float(float3, float3)> callTBackSDF;
rtDeclareVariable(callTBackSDF, sdfPrimBack, , );

typedef rtCallableProgramId<float(float3, float3, float3, float, float)> callT4;
rtDeclareVariable(callT4, sdfPrim4, , );

inline __device__
float3 pal(float t, float3 a, float3 b, float3 c, float3 d)
{
    float3 x = 6.28318f*(c*t + d);
    x.x = cosf(x.x);
    x.y = cosf(x.y);
    x.z = cosf(x.z);

    return a + b*x;
}

__device__ void render_Surface21_old()
{
    float Ka = 0.5;
    float Kd = 0.5;
    float Ks = 0.2;
    float4 col = make_float4(0, 0, 0, 1);// translucent_grays(0.5, 0.1, 0);

    float tstep = 0.01;
    float3 pos = thePrd.last_hit_point - theRay.direction*tstep;// eyeRay.o + eyeRay.d*tnear;
    float3 step = theRay.direction*tstep;

    float4 sum = thePrd.result;// make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here
    BasicLight lights2[2];
    lights2[0].color = optix::make_float3(1.0);
    lights2[0].pos = optix::make_float3(10.0);

    lights2[1].color = optix::make_float3(1.0);
    lights2[1].pos = optix::make_float3(0, 0, 10.0);

    float s1 = sdfPrimBack(pos, make_float3(thePrd.maxDist));
    pos += step;
    float s2 = sdfPrimBack(pos, make_float3(thePrd.maxDist));
    float i = 0;
    float trp = 0.1;
    float max = thePrd.maxDist * 2 + 0.4; //bounding box size
    float3 color = Ka *  make_float3(0, 1, 0);// ambient_light_color;
    float3 color2 = Ka *  make_float3(0, 1, 0);                                          //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    for (int i = 0; i < 2; ++i)
    {
        BasicLight light = lights2[i];
        float3 L = optix::normalize(light.pos - thePrd.last_hit_point);
        float nDl = optix::dot(thePrd.normal, L);

        //if (nDl > 0)
        //    color += Kd * nDl * light.color; // make_float3(1.0);//

        float phong_exp = 0.1;
        if (nDl > 0) {
            color += Kd * nDl * light.color;

            /*  optix::float3 H = optix::normalize(L - theRay.direction);
            float nDh = optix::dot(normal, H);
            if (nDh > 0)
            color += Ks * light.color * pow(nDh, phong_exp);
            */
        }
    }

    while (i < max) //s2 < 0.01)
    {
        if (s2 < 0.01) {
            //col *= Ka;
            //col.w = 0.5; //s1 is very small

            //initial blend
            if (col.x == 0)
                col = make_float4(color);
            else
                col = make_float4(color2);
            col.w = trp;
            col.x *= col.w;
            col.y *= col.w;
            col.z *= col.w;
            // "over" operator for front-to-back blending
            sum = sum + col*(1.0f - sum.w);
        }
        i += tstep;
        s1 = s2;
        pos += step;

        if (sum.w >= 1.0) i = max + 1;
        else
            s2 = sdfPrimBack(pos, make_float3(thePrd.maxDist)); //interpolateSDF(time, pos, texSDF, texSDF_F);
                                                            // if (abs(s1) <= eps)
    }
    //or it should be
    // thePrd.radiance = make_float3(sum);
   // thePrd.radiance += make_float3(sum);
    thePrd.result = sum;
}
__device__ float  compute_SDF(optix::float3 pos, cellPrimDesc cell, int type)
{
    if (type == 0) {
        return sdfPrimBack(pos, make_float3(1.1));
    }
    else
    {
        return sdfPrim4(pos, cell.center, cell.center2, cell.rad1, cell.rad2);
    }
}
__device__ void  VolumeRaycast(int pN)
{
    float Ka = 0.5;
    float Kd = 0.2;
    float Ks = 0.2;
    float4 col = make_float4(0, 0, 0, 1);// translucent_grays(0.5, 0.1, 0);

    float tstep = 0.005;

    cellPrimDesc cell = thePrd.cellPrimitives[pN];
    float dist = cell.intersectionDist;
    //---------------
    float3 pos = theRay.origin + theRay.direction*dist;
    float3 step = theRay.direction*tstep;

    float4 sum = thePrd.result;// make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here

    if (sum.w >= 1.0) return;
    //compute color

    BasicLight lights2[2];
    lights2[0].color = optix::make_float3(1.0);
    lights2[0].pos = optix::make_float3(10.0);

    lights2[1].color = optix::make_float3(1.0);
    lights2[1].pos = optix::make_float3(0, 0, 10.0);

    float s1 = compute_SDF(pos, cell, cell.type); //sdfPrimBack(pos, make_float3(1.1));
    if (abs(s1) > tstep)
        step = theRay.direction*abs(s1);
    else step = theRay.direction*tstep;

    pos += step;
    float s2 = compute_SDF(pos, cell, cell.type); //sdfPrimBack(pos, make_float3(1.1));

    //creates a holes of the molecule form
    //max = abs(distF - dist);

    float3 color = Ka *  make_float3(cell.color.x, cell.color.y, cell.color.z);// ambient_light_color;
    float3 color2 = color;                                          //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    for (int i = 0; i < 2; ++i)
    {
        BasicLight light = lights2[i];
        float3 L = optix::normalize(light.pos - pos);
        float nDl = optix::dot(cell.normal, L);

        //if (nDl > 0)
        //    color += Kd * nDl * light.color; // make_float3(1.0);//

        float phong_exp = 0.1;
        if (nDl > 0) {
            color += Kd * nDl * light.color;
        }
    }

    float i = 0;
    float trp = cell.color.w;
    float max = cell.maxDist; //bounding box size

    //start volume integration
    while (i < max) //s2 < 0.01)
    {
        if (s2 < 0.0) {
            if (abs(s1) > tstep) //sum transparency
            {
                //col *= Ka;
               //col.w = 0.5; //s1 is very small
               // float3 color = Ka *  make_float3(0, 1, 0);// ambient_light_color;

    //initial blend
                if (col.x == 0)
                    col = make_float4(color);
                else
                    col = make_float4(color2);

                //Beer–Lambert law
                float F = exp(-trp*abs(s1) * 200);
                col = col*(1.0 - F);
                sum = sum + col*(1.0f - sum.w);

                // tracedDist += abs(s1);
            }
            else {
                //Beer–Lambert law
                //col = make_float4(color);
               // col = col*exp(-trp*abs(s1) / 100);
               // tracedDist = 0;
            }
        }

        s1 = s2;
        if (abs(s1) > tstep) {
            step = theRay.direction*abs(s1);
            i += abs(s1);
        }
        else
        {
            i += tstep;
            step = theRay.direction*tstep;
        }

        pos += step;
        if (sum.w >= 1.0) i = max + 1;
        else
            s2 = compute_SDF(pos, cell, cell.type);// sdfPrimBack(pos, make_float3(thePrd.maxDist)); //interpolateSDF(time, pos, texSDF, texSDF_F);
    }
    thePrd.result = sum;
}
__device__ void  VolumeRaycast(int pN, int fN)
{
    float Ka = 0.5;
    float Kd = 0.2;
    float Ks = 0.2;
    float4 col = make_float4(0, 0, 0, 1);// translucent_grays(0.5, 0.1, 0);

    float tstep = 0.005;

    cellPrimDesc cell = thePrd.cellPrimitives[pN];
    cellPrimDesc cellSec = thePrd.cellPrimitives[fN];
    float dist = cellSec.intersectionDist;
    //---------------
    float3 pos = theRay.origin + theRay.direction*dist;
    float3 step = theRay.direction*tstep;

    float4 sum = thePrd.result;// make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here

    if (sum.w >= 1.0) return;
    //compute color

    float s1 = compute_SDF(pos, cell, cell.type); //sdfPrimBack(pos, make_float3(1.1));

    if (abs(s1) > tstep)
        step = theRay.direction*abs(s1);
    else step = theRay.direction*tstep;

    pos += step;
    float s2 = compute_SDF(pos, cell, cell.type); //sdfPrimBack(pos, make_float3(1.1));

    float material = compute_SDF(pos, cellSec, cellSec.type); //sdfPrimBack(pos, make_float3(1.1));
                                                              //creates a holes of the molecule form
    float i = 0;
    float trp = cell.color.w*Ka;
    float max = cellSec.maxDist; //bounding box size

                              //start volume integration
    while (i < max) //s2 < 0.01)
    {
        if (s2 < 0.0) {
            if (abs(s1) > tstep) //sum transparency
            {
                //col *= Ka;
                //col.w = 0.5; //s1 is very small
                // float3 color = Ka *  make_float3(0, 1, 0);// ambient_light_color;

                //initial blend
                if (material > 0) {
                    col = (cellSec.color)*Ka; //(make_float4(0, 0, 1, 0.001) +
                    trp = col.w*Ka;
                }
                else
                    col = cell.color*Ka;

                //Beer–Lambert law
                float F = exp(-trp*abs(s1) * 200);
                col = col*(1.0 - F);
                sum = sum + col*(1.0f - sum.w);

                // tracedDist += abs(s1);
            }
            else {
                //Beer–Lambert law
                //col = make_float4(color);
                // col = col*exp(-trp*abs(s1) / 100);
                // tracedDist = 0;
            }
        }

        s1 = s2;

        if (abs(s1) > tstep) {
            step = theRay.direction*abs(s1);
            i += abs(s1);
        }
        else
        {
            i += tstep;
            step = theRay.direction*tstep;
        }

        pos += step;
        if (sum.w >= 1.0) i = max + 1;
        else
        {
            s2 = compute_SDF(pos, cell, cell.type);// sdfPrimBack(pos, make_float3(thePrd.maxDist)); //interpolateSDF(time, pos, texSDF, texSDF_F);
            material = compute_SDF(pos, cellSec, cellSec.type); //sdfPrimBack(pos, make_float3(1.1));
        }
    }
    thePrd.result = sum;
}

__device__ float  VolumeIntegration(float startDist, float maxDist, int pN)
{
    float Ka = 0.5;
    float Kd = 0.5;
    float Ks = 0.2;
    float4 col = make_float4(0, 0, 0, 1);// translucent_grays(0.5, 0.1, 0);

    float tstep = 0.005;

    float tracedDist = startDist;
    cellPrimDesc cell = thePrd.cellPrimitives[pN];
    float dist = cell.intersectionDist;
    //---------------
    float3 pos = theRay.origin + theRay.direction*startDist;
    float3 step = theRay.direction*tstep;

    float4 sum = thePrd.result;// make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here
    //break if there is 1 opacity
    if (sum.w >= 1.0) return;
    //compute color

    BasicLight lights2[2];
    lights2[0].color = optix::make_float3(1.0);
    lights2[0].pos = optix::make_float3(10.0);

    lights2[1].color = optix::make_float3(1.0);
    lights2[1].pos = optix::make_float3(0, 0, 10.0);

    float s1 = compute_SDF(pos, cell, cell.type); //sdfPrimBack(pos, make_float3(1.1));
    if (abs(s1) > tstep)
        step = theRay.direction*abs(s1);
    else step = theRay.direction*tstep;

    pos += step;
    float s2 = compute_SDF(pos, cell, cell.type); //sdfPrimBack(pos, make_float3(1.1));

                                                  //creates a holes of the molecule form
                                                  //max = abs(distF - dist);

    float3 color = Ka *  make_float3(cell.color.x, cell.color.y, cell.color.z);// ambient_light_color;
    float3 color2 = color;                                          //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    for (int i = 0; i < 2; ++i)
    {
        BasicLight light = lights2[i];
        float3 L = optix::normalize(light.pos - pos);
        float nDl = optix::dot(cell.normal, L);

        //if (nDl > 0)
        //    color += Kd * nDl * light.color; // make_float3(1.0);//

        float phong_exp = 0.1;
        if (nDl > 0) {
            color += Kd * nDl * light.color;
        }
    }

    float i = 0;
    float trp = cell.color.w*Ka;

    //start volume integration
    while (i < maxDist) //s2 < 0.01)
    {
        if (s2 < 0.0) {
            if (abs(s1) > tstep) //sum transparency
            {
                //save current position
                tracedDist = startDist + i;
                //col *= Ka;
                //col.w = 0.5; //s1 is very small
                // float3 color = Ka *  make_float3(0, 1, 0);// ambient_light_color;

                //initial blend
               // if (abs(startDist - dist) < 0.005)
               //     col = make_float4(color);
               // else
                col = make_float4(color2);

                //Beer–Lambert law
                float F = exp(-trp*abs(s1) * 100);
                col = col*(1.0 - F);
                sum = sum + col*(1.0f - sum.w);

                // tracedDist += abs(s1);
            }
            else {
                //Beer–Lambert law
                //col = make_float4(color);
                // col = col*exp(-trp*abs(s1) / 100);
                // tracedDist = 0;
            }
        }

        s1 = s2;
        if (abs(s1) > tstep) {
            step = theRay.direction*abs(s1);
            i += abs(s1);
        }
        else
        {
            i += tstep;
            step = theRay.direction*tstep;
        }

        pos += step;
        // if (sum.w >= 1.0) i = maxDist + 1;
        // else
        s2 = compute_SDF(pos, cell, cell.type);// sdfPrimBack(pos, make_float3(thePrd.maxDist)); //interpolateSDF(time, pos, texSDF, texSDF_F);
    }
    thePrd.result = sum;

    return tracedDist;
}
__device__ float  VolumeIntegrationInternalCell(float startDist, float maxDist, int pN, int fN)

{
    float Ka = 0.5;
    float4 col = make_float4(0, 0, 0, 1);// translucent_grays(0.5, 0.1, 0);

    float tstep = 0.01;

    float tracedDist = startDist;
    cellPrimDesc cellSec = thePrd.cellPrimitives[fN];
    cellPrimDesc cell = thePrd.cellPrimitives[pN];
    float dist = cell.intersectionDist;
    //---------------
    float3 pos = theRay.origin + theRay.direction*startDist;
    float3 step = theRay.direction*tstep;

    float4 sum = thePrd.result;// make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here
                               //break if there is 1 opacity
    if (sum.w >= 1.0) return;

    float s1 = compute_SDF(pos, cell, cell.type); //sdfPrimBack(pos, make_float3(1.1));
    float materialCol = compute_SDF(pos, cellSec, cellSec.type); //sdfPrimBack(pos, make_float3(1.1));

    if (abs(s1) > tstep)
        step = theRay.direction*abs(s1);
    else step = theRay.direction*tstep;

    pos += step;
    float s2 = compute_SDF(pos, cell, cell.type); //sdfPrimBack(pos, make_float3(1.1));

                                                  //creates a holes of the molecule form
                                                  //max = abs(distF - dist);

    float3 color = Ka *  make_float3(cell.color.x, cell.color.y, cell.color.z);// ambient_light_color;

    float i = 0;
    float trp = cell.color.w;

    //start volume integration
    while (i < maxDist) //s2 < 0.01)
    {
        if (s2 < 0.0) {
            if (abs(s1) > tstep) //sum transparency
            {
                //save current position
                tracedDist = startDist + i;
                //col *= Ka;
                //col.w = 0.5; //s1 is very small
                // float3 color = Ka *  make_float3(0, 1, 0);// ambient_light_color;

                //initial blend
                if (materialCol <= 0.0) //secondary material
                {
                    col = cellSec.color*Ka;
                    trp = cellSec.color.w*Ka;
                }
                else
                    col = cell.color;

                //Beer–Lambert law
                float F = exp(-trp*abs(s1) * 200);
                col = col*(1.0 - F);
                sum = sum + col*(1.0f - sum.w);

                // tracedDist += abs(s1);
            }
            else {
                //Beer–Lambert law
                //col = make_float4(color);
                // col = col*exp(-trp*abs(s1) / 100);
                // tracedDist = 0;
            }
        }

        s1 = s2;
        if (abs(s1) > tstep) {
            step = theRay.direction*abs(s1);
            i += abs(s1);
        }
        else
        {
            i += tstep;
            step = theRay.direction*tstep;
        }

        pos += step;

        // if (sum.w >= 1.0) i = maxDist + 1;
        // else
        {
            s2 = compute_SDF(pos, cell, cell.type);// sdfPrimBack(pos, make_float3(thePrd.maxDist)); //interpolateSDF(time, pos, texSDF, texSDF_F);
            materialCol = compute_SDF(pos, cellSec, cellSec.type); //sdfPrimBack(pos, make_float3(1.1));
        }
    }
    thePrd.result = sum;

    return tracedDist;
}

/*
__device ComputeCellsRelation(int i, int j)
{
    float relation_val = prd[i].dmax - prd[j].dmin;
int caseN = 0;
if (relation_val > 2 * eps) {
    //positive and big
    //inclusion
    caseN = 1;
}
else {
    if (relation_val < -2 * eps) {
        //negative and big
        //non adjucency
        caseN = 2;
    }
    //else case = 3; //adjucency =dist=eps;
}
}*/

/*
int i = pN + 1;
while (i < N)
{
cellPrimDesc curCell = thePrd.cellPrimitives[i];// .intersectionDist;

// if(prevCell.type==curCell.type)
// {
if (abs(prevCell.intersectionDist - curCell.intersectionDist) > 0.005)
{
//if (curCell.type > 0)
VolumeRaycast(i);
}
i++;
prevCell = curCell;
}
*/
__device__ void VolumeRayCasting2(float curDist, int pN, int pF, int NCell)
{
    float eps = 0.001;
    int i, j;
    i = pN;
    j = pF;

    if (pF >= NCell) //all cells are computed
    {
        if (pN < NCell) {
            float dmin1 = thePrd.cellPrimitives[pN].intersectionDist;
            float dmax1 = thePrd.cellPrimitives[pN].intersectionDist
                + thePrd.cellPrimitives[pN].maxDist;

            float curDist2 =
                VolumeIntegration(curDist, dmax1, pN);
        }

        return;
    }
    const cellPrimDesc prevCell = thePrd.cellPrimitives[pN];
    const cellPrimDesc curCell = thePrd.cellPrimitives[pF];

    //compute boundary values
    float dmax1 = thePrd.cellPrimitives[pN].intersectionDist + thePrd.cellPrimitives[pN].maxDist;
    float dmax2 = thePrd.cellPrimitives[pF].intersectionDist + thePrd.cellPrimitives[pF].maxDist;
    float dmin1 = thePrd.cellPrimitives[pN].intersectionDist;
    float dmin2 = thePrd.cellPrimitives[pF].intersectionDist;

    float curDist2 = curDist;
    if (curDist2 < dmin1) curDist2 = dmin1;
    /*----
    compute Cells relation
    ------*/

    float relation_val = dmax1 - dmin1;
    int caseN = 0;
    if (relation_val > eps) {
        //positive and big
        //inclusion
        caseN = 1;
    }
    else caseN = 2;

    /*----
    raycast cells for current ray-casting case
    ------*/

    switch (caseN) {
    case 1: {
        //Volume integration dist up to cell j boundary
        //curDist is set internally to prd[j].dmin
        //VolumeRaycast(pN);
        curDist2 = VolumeIntegration(curDist, dmin2, i);

        //curDist2 = VolumeIntegration(dmin1, dmax1, pN);

    //Volume integration within internal cell j with check of concave case
    //check concave case if volume ray-casting reaches prd[j].dmax according to current SDF function value
        //curDist2 = VolumeIntegrationInternalCell(dmin2, dmax2, pN, pF);
        //curDist2 = VolumeIntegration(dmin2, dmax2, pN);

        VolumeRaycast(i, j);
        curDist2 = dmax2;
        // curDist2 = VolumeIntegrationInternalCell(dmin2, dmax2, pN, pF);

    //increase j to check next internal cell

        j++;
        //check overlapping cells

        int ll = j - 1;
        int stop = 0;
        while (stop == 0) {
            if (j < NCell) {
                if (thePrd.cellPrimitives[ll].type == thePrd.cellPrimitives[j].type)
                {
                    if (abs(thePrd.cellPrimitives[ll].intersectionDist - thePrd.cellPrimitives[j].intersectionDist) < 0.01)
                    {
                        j++;
                    }
                    else stop = 1;
                }
                else stop = 1;
            }
            else stop = 1; //break;
        }

        break;
    }

    case 2:
    {
        //volume integration of cell i
        //curDist2 = VolumeIntegration(dmin1, dmax1, i);
        VolumeIntegration(curDist, dmax1, i);

        curDist2 = dmax1;
        //increase i and j to move to next neibouring cells
        i = j;
        //check overlapping cells

        int ll = j - 1;
        int stop = 0;
        while (stop == 0) {
            if (i < NCell) {
                if (thePrd.cellPrimitives[ll].type == thePrd.cellPrimitives[i].type)
                {
                    if (abs(thePrd.cellPrimitives[ll].intersectionDist - thePrd.cellPrimitives[i].intersectionDist) < 0.01)
                    {
                        i++;
                    }
                    else stop = 1;
                }
                else stop = 1;
            }
            else stop = 1; //break;
        }

        j = i + 1;
    }
    }

    VolumeRayCasting2(curDist2, i, j, NCell);
}

__device__ void render_Surface21_new()
{
    //-----------------
    //TODO: depth sort
    //---------------
    int N = thePrd.cur_prim - 1;
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N - i - 1; j++)
        {
            const cellPrimDesc tmp = thePrd.cellPrimitives[i];
            // const float2 tmp = prd.particles[i];
            if (tmp.intersectionDist < thePrd.cellPrimitives[j].intersectionDist) {
                thePrd.cellPrimitives[i] = thePrd.cellPrimitives[j];
                thePrd.cellPrimitives[j] = tmp;
            }
        }

    int j = 0;
    int pN = -1;

    while (j < N) {
        const cellPrimDesc cell = thePrd.cellPrimitives[j];
        if (cell.type == 0) //main
        {
            pN = j;
            j = N;
        }
        j++;
    }
    /* if (pN < 0)
    //     return;//to catch error

     int pF = pN + 1;
     cellPrimDesc prevCell = thePrd.cellPrimitives[pN];
     float dist = prevCell.intersectionDist + prevCell.maxDist;
     //call main prog
     */
    if (N > 0) {
        float dist = thePrd.cellPrimitives[0].intersectionDist;
        VolumeRayCasting2(dist, 0, 1, N);
    }
}

__device__ void render_Surface21()
{
    //-----------------
    //TODO: depth sort
    //---------------
    int N = thePrd.cur_prim;
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N - i - 1; j++)
        {
            const cellPrimDesc tmp = thePrd.cellPrimitives[i];
            // const float2 tmp = prd.particles[i];
            if (tmp.intersectionDist < thePrd.cellPrimitives[j].intersectionDist) {
                thePrd.cellPrimitives[i] = thePrd.cellPrimitives[j];
                thePrd.cellPrimitives[j] = tmp;
            }
        }

    int j = 0;
    int pN = -1;

    while (j < N) {
        const cellPrimDesc cell = thePrd.cellPrimitives[j];
        if (cell.type == 0) //main
        {
            pN = j;
            j = N;
        }
        j++;
    }
    if (pN < 0)
        return;//to catch error
    /*int finalN = thePrd.cur_prim - 2; //-1 doesn't cause an error
    if (finalN > 0)*/
    //float prevDist = (thePrd.cellPrimitives[pN]).intersectionDist;

   /* cellPrimDesc prevCell = thePrd.cellPrimitives[pN];
    int i = pN - 1;
    while (i > 0)
    {
        cellPrimDesc curCell = thePrd.cellPrimitives[i];// .intersectionDist;

       // if(prevCell.type==curCell.type)
       // {
        if (abs(prevCell.intersectionDist - curCell.intersectionDist) > 0.01)
        {
            VolumeRaycast(i);
        }
        i--;
        prevCell = curCell;
    }
   */

    int pF = pN + 1;
    cellPrimDesc prevCell = thePrd.cellPrimitives[pN];
    float dist = prevCell.intersectionDist + prevCell.maxDist;
    //call main prog

    VolumeRayCasting2(dist, 0, 1, N);
}

//int cellsType= computeCellRelation(pN, pN + 1);
/*
cellPrimDesc prevCell = thePrd.cellPrimitives[pN];

int i = pN + 1;
while (i < N)
{
    cellPrimDesc curCell = thePrd.cellPrimitives[i];// .intersectionDist;

                                                    // if(prevCell.type==curCell.type)
                                                    // {
    if (abs(prevCell.intersectionDist - curCell.intersectionDist) > 0.005)
    {
        //if (curCell.type > 0)
        VolumeRaycast(i);
    }
    i++;
    prevCell = curCell;
}
*/

RT_PROGRAM void miss_environment_constant()
{
    if (thePrd.renderType > 0) {
        render_Surface21_new();
        // printf("crossed prim %d ", thePrd.cur_prim);
    }
    if (thePrd.renderType == 0)
        thePrd.radiance += sysBackground; // Constant white emission.
}

RT_PROGRAM void auditory_miss_environment_constant()
{
    thePrd.result = make_float4(1.0f);
}