#include "hip/hip_runtime.h"
/*
 */

#include "default.h"
#include "../../basic_lights.h"
#include "../../materials/transferFunction.h"

 //rtDeclareVariable(float, TimeSound, , );
 //for sdf
typedef rtCallableProgramId<float(float3, float3)> callTBackSDF;
rtDeclareVariable(callTBackSDF, sdfPrimBack, , );

rtDeclareVariable(float3, bbox_min, , );
rtDeclareVariable(float3, bbox_max, , );
rtDeclareVariable(float3, bbox_center, , );

rtDeclareVariable(float3, sCell1, , );
rtDeclareVariable(float3, sCell2, , );
rtDeclareVariable(float3, sCell3, , );

typedef rtCallableProgramId<float(float3, primParamDesc)> callM;
rtDeclareVariable(callM, evalF, , );

typedef rtCallableProgramId<float3(float3, primParamDesc)> callC;
rtDeclareVariable(callC, evalCol, , );
__device__
inline float  plane(float3 p, float3 c, float3 n)
{
    return optix::dot(p - c, n);
}
__device__
inline float3 getNormal(float3 v1, float3 v2, float3 v3, float3 c, float3 ct)
{
    float3 a = v3 - v2;
    float3 b = v1 - v2;
    float3 n = cross(a, b);

    float3 nt = c - ct;

    //normalize(n);
    //normalize(nt);

    n = n*dot(n, nt);

    return normalize(n);
}
__device__
inline float3 getCenter(float3 p1, float3 p2, float3 p3)

{
    float3 center = (p1 + p2 + p3) / 3.0;
    return center;
}

__device__
inline float3 getCenterTetra(float4 p0, float4 p1, float4 p2, float4 p3)

{
    float3 center = make_float3((p0 + p1 + p2 + p3) / 4.0);
    return center;
}

__device__
inline float smaxp(float a, float b, float k)
{
    float h = optix::max(k - abs(a - b), 0.0) / k;
    return optix::max(a, b) - h*h*k*(1.0 / 4.0);
}
__device__
inline float sminp(float a, float b, float k)
{
    float h = optix::max(k - abs(a - b), 0.0) / k;
    return optix::min(a, b) - h*h*k*(1.0 / 4.0);
}
__device__
inline float bond(float3 p, float3 a, float3 b, float r)
{
    float3 pa = p - a;
    float3 ba = b - a;
    float h = clamp(dot(pa, ba) / dot(ba, ba), 0.0, 1.0);
    return length(pa - ba*h) - r;
}
__device__
inline float Tetra(float3 p, float3 v0, float3 v1, float3 v2, float3 v3)
{
    float3 c0 = getCenter(v0, v2, v1);
    float3 c1 = getCenter(v0, v3, v2);
    float3 c2 = getCenter(v1, v3, v0);
    float3 c3 = getCenter(v1, v2, v3);

    float3 ct = (v0 + v1 + v2 + v3) / 4.0f;
    //float rad1 = length(ct - c0);
    float rad = length(ct - v0);
    //rad = (rad + rad1) / (2.0*t);
    float3 n0 = getNormal(v0, v2, v1, c0, ct);
    float3 n1 = getNormal(v0, v3, v2, c1, ct);
    float3 n2 = getNormal(v1, v3, v0, c2, ct);
    float3 n3 = getNormal(v1, v2, v3, c3, ct);

    float a = plane(p, c0, n0);
    float b = plane(p, c1, n1);
    float c = plane(p, c2, n2);
    float d = plane(p, c3, n3);
    return fmaxf(fmaxf(a, b), fmaxf(c, d));
}
__device__
inline float TetraWire(float3 p, float3 v0, float3 v1, float3 v2, float3 v3)
{
    float f1 = bond(p, v0, v1, 0.4);
    float f2 = bond(p, v0, v2, 0.4);
    float f3 = bond(p, v0, v3, 0.4);
    float f4 = bond(p, v1, v2, 0.4);
    float f5 = bond(p, v1, v3, 0.4);

    f1 = sminp(f2, f1, 0.4);
    f1 = sminp(f3, f1, 0.4);
    f1 = sminp(f4, f1, 0.4);
    f1 = sminp(f5, f1, 0.4);
    return f1;
}

__device__
inline float TetraWire2(float3 p, float3 v0, float3 v1, float3 v2, float3 v3)
{
    float3 ct = (v0 + v1 + v2 + v3) / 4.0f;
    float3 c0 = getCenter(v0, v2, v1);
    float3 c1 = getCenter(v0, v3, v2);
    float3 c2 = getCenter(v1, v3, v0);
    float3 c3 = getCenter(v1, v2, v3);
    float r = length(ct - v0) / 20;

    float f1 = bond(p, ct, c1, r);
    float f2 = bond(p, ct, c2, r);
    float f3 = bond(p, ct, c3, r);
    float f4 = bond(p, ct, c0, r);

    f1 = sminp(f2, f1, 0.8);
    f1 = sminp(f3, f1, 0.8);
    f1 = sminp(f4, f1, 0.8);
    return f1;
}
__device__
inline float3 computeColTriangle(float3 p, float3 v0, float3 v1, float3 v2, float3 col10, float3 col11, float3 col12, float3 n0)
{
    float3 c0 = (v0 + v1 + v2) / 3;
    float dn = abs(plane(p, c0, n0));

    float3 p0 = v0 - p;
    float d0 = length(p0 - n0*dn);
    float3 p1 = v1 - p;
    float d1 = length(p1 - n0);

    float td = length(v0 - v1);
    //float cos2=normalize(dot(desc.pos[2] - desc.pos[0], dir));
    float3 colB1 = col10*d1 / td + col11*d0 / td;

    float3 p2 = v2 - p;
    float d2 = length(p2 - n0);

    float td2 = length(v0 - v2);
    float3 colB2 = col10*d2 / td2 + col12*d0 / td2;

    float td3 = length(v1 - v2);
    colB1 = colB1*d2 / td3 + colB2*d1 / td3;
    return colB1;
}
__device__
inline float computeTrTriangle(float3 p, float3 v0, float3 v1, float3 v2, float col10, float col11, float col12, float3 n0)
{
    float3 c0 = (v0 + v1 + v2) / 3;
    float dn = abs(plane(p, c0, n0));

    float3 p0 = v0 - p;
    float d0 = length(p0 - n0*dn);
    float3 p1 = v1 - p;
    float d1 = length(p1 - n0);

    float td = length(v0 - v1);
    //float cos2=normalize(dot(desc.pos[2] - desc.pos[0], dir));
    float colB1 = col10*d1 / td + col11*d0 / td;

    float3 p2 = v2 - p;
    float d2 = length(p2 - n0);

    float td2 = length(v0 - v2);
    float colB2 = col10*d2 / td2 + col12*d0 / td2;

    float td3 = length(v1 - v2);
    colB1 = colB1*d2 / td3 + colB2*d1 / td3;
    return colB1;
}

inline __device__  float3  GetColor(float3 x, float3 v0, float3 v1, float3 v2, float3 v3, float s0, float s1, float s2, float s3)
{
    float3 col10;
    float3 col11;
    float3 col12;
    float3 col13;
    float scale = 400;
    float4 col = translucent_grays(s0 / scale, s0 / scale, 1); //abs(f / 30), 1);////translucent_grays2(color_id, abs(f / 100));
    col10 = make_float3(col.x, col.y, col.z);
    col = translucent_grays(s1 / scale, s1 / scale, 1);
    col11 = make_float3(col.x, col.y, col.z);

    col = translucent_grays(s2 / scale, s2 / scale, 1);
    col12 = make_float3(col.x, col.y, col.z);

    col = translucent_grays(s3 / scale, s3 / scale, 1);
    col13 = make_float3(col.x, col.y, col.z);

    float3 c0 = getCenter(v0, v2, v1);
    float3 c1 = getCenter(v0, v3, v2);
    float3 c2 = getCenter(v1, v3, v0);
    float3 c3 = getCenter(v1, v2, v3);

    float3 ct = (v0 + v1 + v2 + v3) / 4.0f;
    //float rad1 = length(ct - c0);
    float rad = length(ct - v0);
    //rad = (rad + rad1) / (2.0*t);
    float3 n0 = getNormal(v0, v2, v1, c0, ct);
    float3 n1 = getNormal(v0, v3, v2, c1, ct);
    float3 n2 = getNormal(v1, v3, v0, c2, ct);
    float3 n3 = getNormal(v1, v2, v3, c3, ct);

    float3 col0 = computeColTriangle(x, v0, v1, v2, col10, col11, col12, n0);
    float3 col1 = computeColTriangle(x, v0, v2, v3, col10, col12, col13, n1);
    float3 col2 = computeColTriangle(x, v1, v0, v3, col11, col10, col13, n2);
    float3 col3 = computeColTriangle(x, v1, v2, v3, col11, col12, col13, n3);

    float d0 = abs(plane(x, c0, n0));
    float d1 = abs(plane(x, c1, n1));
    float d2 = abs(plane(x, c2, n2));
    float d3 = abs(plane(x, c3, n3));

    float dmax = max(d0, max(d1, (max(d2, d3))));
    float dmin = min(d0, min(d1, (min(d2, d3))));

    if (d0 >= 0.001) return col0;
    if (d1 >= 0.001) return col1;
    if (d2 >= 0.001) return col2;
    if (d3 >= 0.001) return col3;
    float3 colB1 = (col1 / d1 + col0 / d0 + col3 / d3 + col2 / d2) / (d1 + d0 + d3 + d2);

    return colB1;
}

inline __device__  float  GetIso(float3 x, float3 v0, float3 v1, float3 v2, float3 v3, float s0, float s1, float s2, float s3)
{
    float4 col;

    float scale = 450;
    col = make_float4(s0, s1, s2, s3);
    col /= scale;

    float3 c0 = getCenter(v0, v2, v1);
    float3 c1 = getCenter(v0, v3, v2);
    float3 c2 = getCenter(v1, v3, v0);
    float3 c3 = getCenter(v1, v2, v3);

    float3 ct = (v0 + v1 + v2 + v3) / 4.0f;
    //float rad1 = length(ct - c0);
    float rad = length(ct - v0);
    //rad = (rad + rad1) / (2.0*t);
    float3 n0 = getNormal(v0, v2, v1, c0, ct);
    float3 n1 = getNormal(v0, v3, v2, c1, ct);
    float3 n2 = getNormal(v1, v3, v0, c2, ct);
    float3 n3 = getNormal(v1, v2, v3, c3, ct);

    float col0 = computeTrTriangle(x, v0, v1, v2, col.x, col.y, col.z, n0);
    float col1 = computeTrTriangle(x, v0, v2, v3, col.x, col.z, col.w, n1);
    float col2 = computeTrTriangle(x, v1, v0, v3, col.y, col.x, col.w, n2);
    float col3 = computeTrTriangle(x, v1, v2, v3, col.y, col.z, col.w, n3);

    float d0 = abs(plane(x, c0, n0));
    float d1 = abs(plane(x, c1, n1));
    float d2 = abs(plane(x, c2, n2));
    float d3 = abs(plane(x, c3, n3));

    float dmax = max(d0, max(d1, (max(d2, d3))));
    float dmin = min(d0, min(d1, (min(d2, d3))));

    if (d0 >= 0.001) return col.x;
    if (d1 >= 0.001) return col.y;
    if (d2 >= 0.001) return col.z;
    if (d3 >= 0.001) return col.w;
    float colB1 = (col.y / d1 + col.x / d0 + col.w / d3 + col.z / d2) / (d1 + d0 + d3 + d2);

    return colB1;
}

inline __device__
float sdTetraD(float3 p, float3 v0, float3 v1, float3 v2, float3 v3)
{
    float f = Tetra(p, v0, v1, v2, v3); //compute tetra
    return f;
}

inline __device__
float sdTetra(float3 p, float3 v0, float3 v1, float3 v2, float3 v3)
{
    float t = TimeSound;// +1.0f;

    return Tetra(p, v0, v1, v2, v3); //compute tetra
}
inline __device__
float sdTetra2(float3 p, float3 v0, float3 v1, float3 v2, float3 v3)
{
    float t = TimeSound;// +1.0f;

    if (t < 2.0) {
        float f = Tetra(p, v0, v1, v2, v3); //compute tetra
        if (t < 1.0) {
            float3 ct = (v0 + v1 + v2 + v3) / 4.0f;
            float3 c0 = getCenter(v0, v2, v1);
            //float rad1 = length(ct - c0);
            float rad = length(ct - c0) / 2.0;
            float f1 = length(p - ct) - rad;
            //interpolation
            return (1 - t)*f1 + t*f;
        }
        else { //other time frame
            float f1 = TetraWire2(p, v0, v1, v2, v3);

            //interpolation
            float tt = t - 1;
            return (1 - (tt*tt*tt))*f + (tt*tt*tt)*f1;
        }
    }
    else {
        float f = TetraWire2(p, v0, v1, v2, v3);
        float3 ct = (v0 + v1 + v2 + v3) / 4.0f;
        float3 c0 = getCenter(v0, v2, v1);
        float rad = length(ct - c0);

        if (rad < 1.4) {
            //float f1 = Tetra(p, v0, v1, v2, v3);
            float3 ct = (v0 + v1 + v2 + v3) / 4.0f;
            float3 c0 = getCenter(v0, v2, v1);
            //float rad1 = length(ct - c0);
            float rad = length(ct - c0);
            float f1 = length(p - ct) - rad;
            //interpolation
            float tt = t - 2;
            return (1 - (tt))*f + (tt)*f1;
        }
        else {
            float f1 = TetraWire(p, v0, v1, v2, v3);
            float fs1 = length(p - v0) - 1.1;
            float fs2 = length(p - v1) - 1.1;
            float fs3 = length(p - v2) - 1.1;
            float fs4 = length(p - v3) - 1.1;

            float tt = (t - 2) * 4;

            float f2 = sminp(f, fs1, 0.8);
            f2 = sminp(f2, fs2, 0.8);
            f2 = sminp(f2, fs3, 0.8);
            f2 = sminp(f2, fs4, 0.8);
            f1 = sminp(f1, f2, 0.8);

            if (tt < 1.0)
            {
                return (1 - (tt))*f + (tt)*f2;
            }
            else {
                float tt2 = (tt - 1) / 3;
                //interpolation
                return (1 - (tt2*tt2))*f2 + (tt2*tt2)*f1;
            }
        }
        /*
       else {
           float3 c1 = getCenter(v0, v3, v2);
           float3 c2 = getCenter(v1, v3, v0);
           float3 c3 = getCenter(v1, v2, v3);

           float f1 = bond(p, ct, c1, 0.4);
           float f2 = bond(p, ct, c2, 0.4);
           float f3 = bond(p, ct, c3, 0.4);
           float f4 = bond(p, ct, c0, 0.4);

           f1 = sminp(f2, f1, 0.8);
           f1 = sminp(f3, f1, 0.8);
           f1 = sminp(f4, f1, 0.8);
           //interpolation
           return (1 - (t - 2))*f + (t - 2)*f1;
       }*/
    }
    /*if (TimeSound < 3.0) {
        float3 c0 = getCenter(v0, v2, v1);
        float3 c1 = getCenter(v0, v3, v2);
        float3 c2 = getCenter(v1, v3, v0);
        float3 c3 = getCenter(v1, v2, v3);

        float3 ct = (v0 + v1 + v2 + v3) / 4.0f;
        float rad1 = length(ct - c0);
        float rad = length(ct - v0);
        rad = (rad + rad1) / (2.0*t);
        float3 n0 = getNormal(v0, v2, v1, c0, ct);
        float3 n1 = getNormal(v0, v3, v2, c1, ct);
        float3 n2 = getNormal(v1, v3, v0, c2, ct);
        float3 n3 = getNormal(v1, v2, v3, c3, ct);

        float a = plane(p, c0, n0);
        float b = plane(p, c1, n1);
        float c = plane(p, c2, n2);
        float d = plane(p, c3, n3);
        float f1 = fmaxf(fmaxf(a, b), fmaxf(c, d));
        float3 ct2 = ct - (t - 1) / 10 * v0;
        float f = length(p - ct2) - rad;
        return smaxp(-f, f1, 1.8);
    }
    else*/
}

inline __device__  void sampleVolume(int pN, PerRayData& prd, float3 origin, float3 direction, float rayMin)
{
    float Ka = 0.5;
    float Kd = 0.9;
    float Ks = 0.9;
    cellPrimDesc cell = prd.cellPrimitives[pN];
    primParamDesc prim = prd.prims[pN];
    float initialStep = 0.1;
    float tstep = initialStep;
    float3 c = (prim.pos[0] + prim.pos[1] + prim.pos[2] + prim.pos[3]) / 4.0;
    float rad = length(c - prim.pos[0]);
    float primNorm = rad / ((TimeSound + 1) / 2);
    float dist = cell.intersectionDist - rad;

    if (dist >= rayMin)
    {
        float3 pos = origin + direction*dist;
        float3 step = direction*tstep;

        float4 sum = prd.result;// make_float4(prd.radiance.x, prd.radiance.y, prd.radiance.z, 0.1); //TODO: get background color here

        float s1 = sdTetra(pos, prim.pos[0], prim.pos[1], prim.pos[2], prim.pos[3]);
        //evalF(pos, prim);
        //int numSteps = 10;
        float i = 0.0;
        float max = cell.maxDist;// *2 + 0.4; //bounding box size

        while (i < max) //s2 < 0.01)
        {
            float ftr = GetIso(pos, prim.pos[0], prim.pos[1], prim.pos[2], prim.pos[3], prim.rad[0], prim.rad[1], prim.rad[2], prim.rad[3]);//cell.color;// make_float4(color2.x, color2.y, color2.z, trp*Ka);
            if (abs(ftr) < 2.2)
            {
                if (s1 < initialStep)
                {
                    //	optix::float3 hit_point = origin + theIntersectionDistance * direction;
                    //float3 cc = translucent_grays(ftr,ftr, 1); //GetColor(pos, prim.pos[0], prim.pos[1], prim.pos[2], prim.pos[3], prim.rad[0], prim.rad[1], prim.rad[2], prim.rad[3]);//cell.color;// make_float4(color2.x, color2.y, color2.z, trp*Ka);
                    float4 col = translucent_grays(ftr, ftr, 1); //make_float4(cc.x, cc.y, cc.z, cell.color.w);
                    col.w = cell.color.w;
                    if (tstep > initialStep)
                    {
                        float F = exp(-col.w*abs(s1) * 40 / primNorm);
                        col = col*(1.0 - F);
                        col.w = (1.0 - F);
                    }
                    else {
                        col.x *= col.w;
                        col.y *= col.w;
                        col.z *= col.w;
                    }

                    float t = sum.w;
                    // "over" operator for front-to-back blending
                    sum = sum + col*(1.0f - sum.w);
                }
            }
            tstep = fmax(initialStep, abs(s1));
            step = direction*tstep;
            i += tstep;
            pos += step;
            if (sum.w >= 1.0) {
                i = max + 1;
            }
            else
                s1 = sdTetra(pos, prim.pos[0], prim.pos[1], prim.pos[2], prim.pos[3]);
            //evalF(pos, prim);
        }

        prd.result = sum;
    }
}

inline __device__  void render_HeteroVolume(int pN, PerRayData& prd, float3 origin, float3 direction)
{
    float Ka = 0.5;
    float Kd = 0.9;
    float Ks = 0.5;
    cellPrimDesc cell = prd.cellPrimitives[pN];
    primParamDesc prim = prd.prims[pN];
    float tstep = 0.05;
    float dist = cell.intersectionDist;

    float3 pos = origin + direction*dist;
    float3 step = direction*tstep;

    float4 sum = prd.result;// make_float4(prd.radiance.x, prd.radiance.y, prd.radiance.z, 0.1); //TODO: get background color here
    float trp = 0.05;
    float trp0 = 0.1;

    float s1 = evalF(pos, prim);
    if (abs(s1) > tstep)
        step = direction*abs(s1);

    float i = 0.0;
    float max = cell.maxDist;// *2 + 0.4; //bounding box size
    float4 sumcol = make_float4(0.0);
    float tracedDist = 0;

    BasicLight lights2[2];
    lights2[0].color = optix::make_float3(1.0);
    lights2[0].pos = optix::make_float3(10.0);

    lights2[1].color = optix::make_float3(1.0);
    lights2[1].pos = optix::make_float3(0, 0, 10.0);

    //float4 col1 = translucent_grays(0.5, 0.01, 0);
    int VolInt = 1;

    while (i < max) //s2 < 0.01)
    {
        if (s1 < tstep)
        {
            // if (abs(s1) > tstep) //sum transparency
            {
                //is used to highlight isosurfaces
                //or create a more shell like effect
                //trp = trp0 + abs(s1) / 10;

                //----------------------
                //COLOR COMPUTATION
                //trp = trp0;

                VolInt = 1;
                //------------------

                //float3 color = Ka *  cell.color;// ambient_light_color;
                //float3 color2 = Ka *  cell.color;
                float3 c = evalCol(pos, prim);//	optix::float3 hit_point = origin + theIntersectionDistance * direction;
                float4 col = make_float4(c.x, c.y, c.z, cell.color.w);// make_float4(color2.x, color2.y, color2.z, trp*Ka);
                col *= Ka;
                if (i < tstep * 2) //surface
                {
                    float3 color = make_float3(col.x, col.y, col.z);                                          //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

                    for (int i = 0; i < 2; ++i)
                    {
                        BasicLight light = lights2[i];
                        float3 L = optix::normalize(light.pos - pos);
                        float nDl = optix::dot(cell.normal, L);

                        //if (nDl > 0)
                        //    color += Kd * nDl * light.color; // make_float3(1.0);//

                        float phong_exp = 0.1;
                        if (nDl > 0) {
                            color += Kd * nDl * light.color;

                            optix::float3 H = optix::normalize(L - direction);
                            float nDh = optix::dot(cell.normal, H);
                            if (nDh > 0)
                                color += Ks * light.color * pow(nDh, phong_exp);
                        }
                    }
                    col.x = color.x;
                    col.y = color.y;
                    col.z = color.z;
                }

                //Beer�Lambert law
                float F = exp(-cell.color.w*abs(s1) * 200);
                col = col*(1.0 - F);
                sum = sum + col*(1.0f - sum.w);

                // tracedDist += abs(s1);
            }
        }

        //s1 = s2;

        if (VolInt > 0) {
            if (abs(s1) > tstep) {
                step = direction*abs(s1);
                i += abs(s1);
            }
            else
            {
                i += tstep;
                step = direction*tstep;
            }
        }
        else { //volume sampling
            if (s1 > tstep) //employ space skipping
            {
                step = direction*abs(s1);
                i += abs(s1);
            }
            else {
                i += tstep;
                step = direction*tstep;
            }
        }

        pos += step;
        if (sum.w >= 1.0) {
            i = max + 1;
        }
        else
            s1 = evalF(pos, prim);
    }

    prd.result = sum;
}

inline __device__ void render_HeteroVolume_initial(int pN, PerRayData& prd, float3 origin, float3 direction)
{
    float Ka = 0.5;
    float Kd = 0.9;
    float Ks = 0.9;

    float tstep = 0.1;
    cellPrimDesc cell = prd.cellPrimitives[pN];
    primParamDesc prim = prd.prims[pN];
    float dist = cell.intersectionDist;

    if (prim.type == 4) {//tetra
        dist = fmaxf(0.0f, dist - prim.rad[0]);
    }
    //---------------
    float3 pos = origin + direction*dist;
    float3 step = direction*tstep;

    float4 sum = prd.result;// make_float4(prd.radiance.x, prd.radiance.y, prd.radiance.z, 0.1); //TODO: get background color here

    if (sum.w >= 1.0) return;
    //compute color

    float trp = 0.05;
    float segmLength = 0.0;

    float s1 = evalF(pos, prim); //sdTetra(pos,prim.pos[0],prim.pos[1],prim.pos[2],prim.pos[3]);
    if (abs(s1) > tstep)
        step = direction*abs(s1);

    float i = 0.0;
    float max = cell.maxDist;// *2 + 0.4; //bounding box size

    float tracedDist = 0;

    //float4 col1 = translucent_grays(0.5, 0.01, 0);
    int VolInt = 1;
    optix::float4 col = cell.color*Ka;

    while (i < max) //s2 < 0.01)
    {
        if (s1 < 0)
        {
            {
                VolInt = 1;
                //------------------
                float4 col2 = cell.color*Ka;// make_float4(col.x, col.y, col.z, trp*Ka);

                                            // if (abs(s1) > tstep / 2)
                {
                    //Beer�Lambert law
                    float F = exp(-col2.w*abs(s1) * 100);
                    col2 = col2*(1.0 - F);
                    col2.w = (1.0 - F);
                    sum = sum + col2*(1.0f - sum.w);
                }
                /*  else //sampling
                {
                trp = col2.w;
                col2 *= col2.w;
                col2.w = trp;
                sum = sum + col2*(1.0f - sum.w);
                }*/
                // tracedDist += abs(s1);
            }
        }

        //s1 = s2;

        if (abs(s1) > tstep) {
            step = direction*abs(s1);
            i += abs(s1);
        }
        else
        {
            i += tstep;
            step = direction*tstep;
        }

        pos += step;
        if (sum.w >= 1.0) {
            i = max + 1;
        }
        else
            s1 = evalF(pos, prim); //sdTetra(pos, prim);
    }

    prd.result = sum;
}

/* skips overlap up of two cells*/
__device__ int skipOverlap2(const int pN, PerRayData& prd)
{
    int i = pN;
    int j = pN + 1;

    int N = prd.cur_prim;
    if (j >= N) return i;

    const cellPrimDesc curCell = prd.cellPrimitives[i];
    const cellPrimDesc nextCell = prd.cellPrimitives[j];

    int type1 = prd.prims[i].type;
    int type2 = prd.prims[j].type;
    //compute boundary values
    float dmin1 = curCell.intersectionDist;
    float dmin2 = nextCell.intersectionDist;

    float dmax1 = curCell.maxDist;
    float dmax2 = nextCell.maxDist;
    if ((type1 == type2) && (type1 != 4))
    {
        //if ((curDist - dmin2 > 0.001f) && (dmax2 - curDist > 0.001f)) {//(abs(dmin1 - dmin2) < 0.1) &&
        if (abs(dmin1 - dmin2) < 0.3) //&& (abs(dmax1 - dmax2) < 0.001)) {
                                       //skip cell
        {
            if (dmax1 > dmax2) { //should trace the max cell
                prd.cellPrimitives[j] = prd.cellPrimitives[i];
                prd.prims[j] = prd.prims[i];
            }
            return j;
        }
    }
    return i;
}
/* volume ray cast*/

/*Sorts prim info and prim array by depth*/
__device__ void basic_sort(PerRayData& prd) {
    int N = prd.cur_prim;// -1;
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N - i - 1; j++)
        {
            const cellPrimDesc tmp = prd.cellPrimitives[i];
            // const float2 tmp = prd.particles[i];
            if (tmp.intersectionDist < prd.cellPrimitives[j].intersectionDist) {
                const primParamDesc tmp2 = prd.prims[i];

                prd.cellPrimitives[i] = prd.cellPrimitives[j];
                prd.cellPrimitives[j] = tmp;

                //sort prim info as well
                prd.prims[i] = prd.prims[j];
                prd.prims[j] = tmp2;
            }
        }
}

__device__ void render_Tetra(PerRayData& prd, float3 origin, float3 direction, float tmin)
{
    if (prd.result.w > 0.7)
        return;
    //printf("Render tetra");
    int VolInt = 1;
    //-----------------
    /*depth sort*/
    //---------------
    int N = prd.cur_prim;
    basic_sort(prd);

    //integrate over cells
    int i = 0;

    //prd.result += make_float4(1, 0, 0, 1);
    while (i < N)
    {
        //skip overlap
        //int j = skipOverlap2(i, prd);

        sampleVolume(i, prd, origin, direction, tmin);

        i++;
        if (prd.result.w > 0.7) i = N;
    }
}
__device__ void render_Mol(PerRayData& prd, float3 origin, float3 direction)
{
    if (prd.result.w > 0.7)
        return;
    //printf("Render tetra");
    int VolInt = 1;
    //-----------------
    /*depth sort*/
    //---------------
    int N = prd.cur_prim;
    basic_sort(prd);

    //integrate over cells
    int i = 0;

    //prd.result += make_float4(1, 0, 0, 1);
    while (i < N)
    {
        //skip overlap
        int j = skipOverlap2(i, prd);

        render_HeteroVolume(j, prd, origin, direction);

        i = j + 1;
        if (prd.result.w > 0.7) i = N;
    }
}

static __device__ __inline__  optix::Ray ComputeDirPos(PerRayData& prd)
{
    //---------------
    //Linking to cuda threads. This is implemented as in CUDA ADVANCED SAMPLES
    //link pixel number to thread
    const float2 pixel = make_float2(theLaunchIndex);

    //no antializing
    const float2 fragment = pixel + 0.5;

    // The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
    const float2 screen = make_float2(theLaunchDim);

    const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

    const float3 origin = sysCameraPosition;
    const float3 direction = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

    //if (isDynamic)
    prd.TimeSound = TimeSound;

    //TODO: we now just compute optical_LaunchDim/auditory_LaunchDim ratio

   /* if (computeAuditoryRendering>0) {
        int numS = 0;
        prd.isSoundRay = isSoundRay(numS, ndc, pixel, screen);
        prd.numS = numS;
    }*/

    // Create ray
    return optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);
}
// Entry point for a pinhole camera.
RT_PROGRAM void raygeneration0()
{
    PerRayData prd;
    // Initialize the random number generator seed from the linear pixel index and the iteration index.
    //    prd.seed = tea<16>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, 0);

    prd.radiance = make_float3(0.0f);
    prd.depth = 0;
    prd.result = make_float4(0.5);
    prd.cur_prim = 0;
    prd.isSoundRay = false;
    //   prd.rnd = rng(prd.seed);
    prd.TimeSound = 0.0f;
    prd.totalDist = 0;
    prd.radiance = sysBackground;
    prd.length = 0;
    prd.block = true;
    prd.renderType = 0;
    prd.result = make_float4(sysBackground);
    prd.result.w = 0.1;
    //compute normalized ray direction[-1,1]
    optix::Ray ray = ComputeDirPos(prd);

    // Start tracing ray from the camera and further
    rtTrace(sysTopObject, ray, prd);
    if (prd.renderType > 2)
    {
        //tetra or molecules
        render_Mol(prd, ray.origin, ray.direction);
    }
    //postprocessing and rendering
    //if (prd.renderType > 0)
    //    postRender(ray, prd, prd.normal, prd.last_hit_point);

    //prd.radiance *= 2.5 + make_float3(prd.result); //worked previously. instead
    // prd.result = prd.result + col*(1.0f - prd.result.w);

    //prd.result *= 1.5;
    //prd.result += col;
    prd.radiance = make_float3(prd.result.x, prd.result.y, prd.result.z);

    //float4 val = prd.result;// make_float4(prd.radiance, 0.0f);// +prd.result;//+ // *(1 - prd.result.w)//+make_float4(prd.radiance, 0.0f);// +;////;
    // val.w = 1;
    sysOutputBuffer[theLaunchIndex] = make_float4(prd.radiance, 1.0f);// +prd.result;//;
}
// Entry point for a pinhole camera.
RT_PROGRAM void raygeneration1()
{
    PerRayData prd;
    // Initialize the random number generator seed from the linear pixel index and the iteration index.
//    prd.seed = tea<16>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, 0);

    prd.radiance = make_float3(0.0f);
    prd.depth = 0;
    prd.result = make_float4(0.5);
    prd.cur_prim = 0;
    prd.isSoundRay = false;
    //   prd.rnd = rng(prd.seed);
    prd.TimeSound = 0.0f;
    prd.totalDist = 0;
    prd.radiance = sysBackground;
    prd.length = 0;
    prd.depth = 0;
    prd.block = true;
    prd.renderType = 0;
    prd.result = make_float4(sysBackground);
    prd.result.w = 0.01;
    //compute ray direction[-1,1]
    optix::Ray ray = ComputeDirPos(prd);

    float3 t0, t1, tmin, tmax;
    t0 = (bbox_max - ray.origin) / ray.direction;
    t1 = (bbox_min - ray.origin) / ray.direction;
    tmax = fmaxf(t0, t1);
    tmin = fminf(t0, t1);
    float tenter = fmaxf(0.f, fmaxf(tmin.x, fmaxf(tmin.y, tmin.z)));
    float texit = fminf(tmax.x, fminf(tmax.y, tmax.z));

    if (tenter < texit)
    {
        float tbuffer = 0.f;
        float step = 5.0;
        while (tbuffer < texit && prd.result.w < 0.7)
        {
            ray.tmin = tenter;
            ray.tmax = texit;
            ray.tmin = fmaxf(tenter, tbuffer);
            ray.tmax = fminf(texit, tbuffer + step);

            if (ray.tmax > tenter)    //doing this will keep rays more coherent
            {
                prd.cur_prim = 0;
                // Start tracing ray from the camera and further
                rtTrace(sysTopObject, ray, prd);

                //postprocessing and rendering

                if (prd.renderType > 2)
                {
                    //tetra or molecules
                    render_Tetra(prd, ray.origin, ray.direction, tenter);
                }

                //if (prd.renderType > 0)
                //    postRender(ray, prd, prd.normal, prd.last_hit_point);
            }

            tbuffer += step;
        }
    }

    float sampOp = 1.0;

    prd.radiance = make_float3(sampOp*prd.result.x, sampOp*prd.result.y, sampOp*prd.result.z);
    /* TODO:Antialising is not implemented as far. So some small distortions might present*/
    sysOutputBuffer[theLaunchIndex] = make_float4(prd.radiance, 1.0f);// +prd.result;//;
}

RT_PROGRAM void auditory_raygeneration()
{
    PerAudioRayData prd;
    // Initialize the random number generator seed from the linear pixel index and the iteration index.
//    prd.seed = tea<16>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, 0);

    prd.radiance = make_float3(0.0f);
    prd.depth = 0;
    prd.result = make_float4(0.5);
    prd.cur_prim = 0;
    prd.isSoundRay = false;
    //    prd.rnd = rng(prd.seed);
    prd.TimeSound = 0.0f;

    for (int i = 0; i < MAX_PRIM_ALONG_RAY; i++)
    {
        prd.primitives[i] = make_float2(0);
    }

    //compute normalized ray direction[-1,1]
/*    optix::Ray ray = ComputeDirPos(prd);

    prd.TimeSound = TimeSound;

    // Start tracing ray from the camera and further
    rtTrace(sysTopObject, ray, prd);

    int num = 0;
    for (int i = 0; i < MAX_PRIM_ALONG_RAY - 1; i++)
    {
        sysAuditoryOutputBuffer[theLaunchIndex][i] = prd.primitives[i];
        if (prd.primitives[i].y > 0) num++;
    }
    sysAuditoryOutputBuffer[theLaunchIndex][MAX_PRIM_ALONG_RAY - 1] = make_float2(prd.cur_prim, num);
    */
}

//This one for interactive widget
//--------------------------------------------------------------------------------------

//Buffers

rtBuffer<float3, 2> movingPoints; // registered point cloud
rtBuffer<float3, 2> fixedPoints; // to be alligned with point cloud
//for SDF

typedef rtCallableProgramId<float(float3, float3)> callT;
rtDeclareVariable(callT, sdfPrim, , );

//TODO: sphere tracing of the widget
RT_CALLABLE_PROGRAM optix::Ray SphereTraceGeometry(optix::Ray  ray, bool& found)
{
    //TODO: consider case when there is no intersection with geometry
    optix::Ray  ray2 = ray;

    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;

    // === Raymarching (Sphere Tracing) Procedure ===

    float eps = 0.001;
    float t = 0.0002;
    float3 rad = make_float3(1.5);

    for (int i = 0; i < 100; i++)
    {
        optix::float3 p = ray.origin + t* ray.direction;
        float hit = abs(sdfPrim(p, rad)); //make it always positive so we can step to border
        if (hit < 0.0002) //|| t > 20.0)
        {
            //there is an intersectuib

            float dx = sdfPrim(p + make_float3(eps, 0, 0), rad) - sdfPrim(p - make_float3(eps, 0, 0), rad);
            float dy = sdfPrim(p + make_float3(0, eps, 0), rad) - sdfPrim(p - make_float3(0, eps, 0), rad);
            float dz = sdfPrim(p + make_float3(0, 0, eps), rad) - sdfPrim(p - make_float3(0, 0, eps), rad);

            //info.normal = normalize(make_float3(dx, dy, dz));
            ray2.origin = p;// origin + t*direction;
            //make it sphere normal
            //ray2.direction = normalize(make_float3(dx, dy, dz));

            found = true;
            break;
        }

        if (t > 20.0) {
            found = false;
            break; //no intersection
        }
        t += abs(hit); //only positive direction
    }

    return ray2;
}

rtDeclareVariable(float3, widgetCenter, , );
//-----------
//For widget Ray generation
static __device__ __inline__  optix::Ray ComputeDirPosWidget(PerRayData& prd)
{
    //---------------
    //Linking to cuda threads. This is implemented as in CUDA ADVANCED SAMPLES
    //link pixel number to thread
    const float2 pixel = make_float2(theLaunchIndex);

    //no antializing
    const float2 fragment = pixel + 0.5;

    // The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
    const float2 screen = make_float2(theLaunchDim);

    const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

    const float3 origin = widgetCenter;
    const float3 direction = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

    if (isDynamic)
        prd.TimeSound = TimeSound;

    //TODO: we now just compute optical_LaunchDim/auditory_LaunchDim ratio

   /* if (computeAuditoryRendering>0) {
        int numS = 0;
        prd.isSoundRay = isSoundRay(numS, ndc, pixel, screen);
        prd.numS = numS;
    }*/

    // Create ray
    return optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);
}

// Entry point for a widget.
//First we should sphere trace it's geometry
RT_PROGRAM void audio_ray_cast()
{
    PerAudioRayData prd;
    // Initialize the random number generator seed from the linear pixel index and the iteration index.
//    prd.seed = tea<16>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, 0);

    prd.radiance = make_float3(0.0f);
    prd.depth = 0;
    prd.result = make_float4(0.5);
    prd.cur_prim = 0;
    prd.isSoundRay = false;
    //    prd.rnd = rng(prd.seed);
    prd.TimeSound = 0.0f;
    prd.isDynamic = isDynamic;

    prd.dirCamera = optix::normalize(widgetCenter - sysCameraPosition);
    //----------
    //set fail value by default
    movingPoints[theLaunchIndex] = make_float3(-1000);
    fixedPoints[theLaunchIndex] = make_float3(-1000);

    for (int i = 0; i < MAX_PRIM_ALONG_RAY; i++)
    {
        prd.primitives[i] = make_float2(0);
    }

    //compute normalized ray direction[-1,1]
/*    optix::Ray ray = ComputeDirPosWidget(prd);

    bool found = false;
    optix::Ray ray2 = SphereTraceGeometry(ray, found);

    if (found) {
        prd.TimeSound = TimeSound;

        // Start tracing ray from the camera and further
        rtTrace(sysTopObject, ray2, prd);
        //to this point prd is filled with intersection info

        //conventional gathering of distance information for auditory rendering
        int num = 0;
        for (int i = 0; i < MAX_PRIM_ALONG_RAY - 1; i++)
        {
            sysAuditoryOutputBuffer[theLaunchIndex][i] = prd.primitives[i];
            if (prd.primitives[i].y > 0) num++;
        }
        sysAuditoryOutputBuffer[theLaunchIndex][MAX_PRIM_ALONG_RAY - 1] = make_float2(prd.cur_prim, num);

        //movingPoints[theLaunchIndex] = make_float3(-1000);

        if (num > 0) //there is an intersection
        {
            printf("%d NUM", num);
            //add a moving point and fixed if there is an
            //intersection
            if (prd.primitives[0].x < 1.5) {
                movingPoints[theLaunchIndex] = ray2.origin;
                fixedPoints[theLaunchIndex] = ray2.origin + ray2.direction*prd.primitives[0].x;
            }
        }
        //   else //there is no intersection, identification point
        //   {
        //       movingPoints[theLaunchIndex] = make_float3(-1000);
        //       fixedPoints[theLaunchIndex] = make_float3(-1000);
        //   }
    } //if (num > 0)

    //TODO: do something with outputsys buff as postprocessing
      //  }
      */
}