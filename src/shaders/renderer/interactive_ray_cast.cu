#include "hip/hip_runtime.h"
/*
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "random_number_generators.h"

#include "auditory_rendering.h"

 //optical output buffer
rtBuffer<float4, 2> sysOutputBuffer; // RGBA32F

//auditory output buffer
rtBuffer<auditoryPrim, 2> sysAuditoryOutputBuffer; // RGBA32F

//dynamic/static scene parameters
rtDeclareVariable(int, isDynamic, , ); //the scene is dynamic
rtDeclareVariable(int, computeAuditoryRendering, , ); //should we compute auditory rendering

rtDeclareVariable(rtObject, sysTopObject, , );

//optix vars similar to cuda kernel executions vars
rtDeclareVariable(uint2, theLaunchDim, rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

rtDeclareVariable(float3, sysCameraPosition, , );
rtDeclareVariable(float3, sysCameraU, , );
rtDeclareVariable(float3, sysCameraV, , );
rtDeclareVariable(float3, sysCameraW, , );

//rtDeclareVariable(int, renderWidget, , );
//rtDeclareVariable(float3, sysCords, , );

static __device__ __inline__  optix::Ray ComputeDirPos(PerRayData& prd)
{
    //---------------
    //Linking to cuda threads. This is implemented as in CUDA ADVANCED SAMPLES
    //link pixel number to thread
    const float2 pixel = make_float2(theLaunchIndex);

    //no antializing
    const float2 fragment = pixel + 0.5;

    // The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
    const float2 screen = make_float2(theLaunchDim);

    const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

    const float3 origin = sysCameraPosition;
    const float3 direction = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

    if (isDynamic)
        prd.TimeSound = TimeSound;

    //TODO: we now just compute optical_LaunchDim/auditory_LaunchDim ratio

   /* if (computeAuditoryRendering>0) {
        int numS = 0;
        prd.isSoundRay = isSoundRay(numS, ndc, pixel, screen);
        prd.numS = numS;
    }*/

    // Create ray
    return optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);
}

// Entry point for a pinhole camera.
RT_PROGRAM void raygeneration()
{
    PerRayData prd;
    // Initialize the random number generator seed from the linear pixel index and the iteration index.
    prd.seed = tea<16>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, 0);

    prd.radiance = make_float3(0.0f);
    prd.depth = 0;
    prd.result = make_float4(0.5);
    prd.cur_prim = 0;
    prd.isSoundRay = false;
    prd.rnd = rng(prd.seed);
    prd.TimeSound = 0.0f;
    prd.isDynamic = isDynamic;

    //compute normalized ray direction[-1,1]
    optix::Ray ray = ComputeDirPos(prd);

    // Start tracing ray from the camera and further
    rtTrace(sysTopObject, ray, prd);

    prd.radiance *= make_float3(prd.result);

    sysOutputBuffer[theLaunchIndex] = make_float4(prd.radiance, 1.0f);
}

RT_PROGRAM void auditory_raygeneration()
{
    PerRayData prd;
    // Initialize the random number generator seed from the linear pixel index and the iteration index.
    prd.seed = tea<16>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, 0);

    prd.radiance = make_float3(0.0f);
    prd.depth = 0;
    prd.result = make_float4(0.5);
    prd.cur_prim = 0;
    prd.isSoundRay = false;
    prd.rnd = rng(prd.seed);
    prd.TimeSound = 0.0f;

    for (int i = 0; i < MAX_PRIM_ALONG_RAY; i++)
    {
        prd.primitives[i] = make_float2(0);
    }

    //compute normalized ray direction[-1,1]
    optix::Ray ray = ComputeDirPos(prd);

    //trace for auditory rays
    //if (prd.isSoundRay) {
    prd.TimeSound = TimeSound;

    // Start tracing ray from the camera and further
    rtTrace(sysTopObject, ray, prd);
    //to this point prd is filled with intersection info

    //intf("launched");
    //TODO: write to auditory buffer
    //related to numS (raw_id)
    int num = 0;
    for (int i = 0; i < MAX_PRIM_ALONG_RAY - 1; i++)
    {
        sysAuditoryOutputBuffer[theLaunchIndex][i] = prd.primitives[i];
        if (prd.primitives[i].y > 0) num++;
    }
    sysAuditoryOutputBuffer[theLaunchIndex][MAX_PRIM_ALONG_RAY - 1] = make_float2(prd.cur_prim, num);
    //if (num > 0)
    //printf("%d ", num);
    //TODO: do something with outputsys buff as postprocessing
      //  }
}