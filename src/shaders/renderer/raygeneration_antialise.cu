#include "hip/hip_runtime.h"
/* 
 */


#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "random_number_generators.h"
#include "per_ray_data.h"


//optical output buffer
rtBuffer<float4,  2> sysOutputBuffer; // RGBA32F

//auditory output buffer


//dynamic/static scene parameters
rtDeclareVariable(bool, isDynamic, , ); //the scene is dynamic
rtDeclareVariable(float, TimeSound, , );

rtDeclareVariable(float, MultiscaleParam, , );

rtDeclareVariable(rtObject, sysTopObject, , );
//rtDeclareVariable(int, sysIterationIndex, , ); //some staff for antializing


//optix vars similar to cuda kernel executions vars
rtDeclareVariable(uint2, theLaunchDim,   rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

rtDeclareVariable(float3, sysCameraPosition, , );
rtDeclareVariable(float3, sysCameraU, , );
rtDeclareVariable(float3, sysCameraV, , );
rtDeclareVariable(float3, sysCameraW, , );

/*static __device__ __inline__ optix::uchar4 make_color(const optix::float3& c)
{
    return optix::make_uchar4(static_cast<unsigned char>(__saturatef(c.z)*255.99f),  
        static_cast<unsigned char>(__saturatef(c.y)*255.99f),  
        static_cast<unsigned char>(__saturatef(c.x)*255.99f),  
        255u);                                                 
}*/

static __device__ __inline__  optix::float2 ComputeDirPos(unsigned int seed)
{
    //---------------
    //Linking to cuda threads. This is implemented as in CUDA ADVANCED SAMPLES 
    //link pixel number to thread
    const float2 pixel = make_float2(theLaunchIndex);

    //perform antializing
    const float2 fragment = pixel + rng2(seed);

    // The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
    const float2 screen = make_float2(theLaunchDim);

    return (fragment / screen) * 2.0f - 1.0f;

   
}

// Entry point for a pinhole camera.
RT_PROGRAM void raygeneration()
{
  PerRayData prd;
  // Initialize the random number generator seed from the linear pixel index and the iteration index.
  prd.seed = tea<16>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, 0);

  prd.radiance = make_float3(0.0f);
  prd.depth=1;
  prd.result = make_float4(0.5);
  prd.cur_prim = 0;
  prd.isSoundRay = false;
 
  //compute normalized ray direction[-1,1]
  float2 ndc = ComputeDirPos(prd.seed);
 const float3 origin = sysCameraPosition;
  const float3 direction = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

  // Create ray 
  optix::Ray ray = optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);

  // Start tracing ray from the camera and further
  rtTrace(sysTopObject, ray, prd);


  prd.radiance *= make_float3(prd.result);
  sysOutputBuffer[theLaunchIndex] = make_float4(prd.radiance, 1.0f);
}
