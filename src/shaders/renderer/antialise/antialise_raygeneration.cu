#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

 //#include "rt_function.h"
#include "../per_ray_data.h"
#include "shader_common.h"
#include "../random_number_generators.h"
//#include "rt_assert.h"

rtBuffer<float4, 2> sysOutputBuffer; // RGBA32F

rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float, sysSceneEpsilon, , );
rtDeclareVariable(int2, sysPathLengths, , );
rtDeclareVariable(int, sysIterationIndex, , );

rtDeclareVariable(float3, sysCameraPosition, , );
rtDeclareVariable(float3, sysCameraU, , );
rtDeclareVariable(float3, sysCameraV, , );
rtDeclareVariable(float3, sysCameraW, , );

rtDeclareVariable(uint2, theLaunchDim, rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );
RT_FUNCTION void integrator(PerRayData& prd, float3& radiance)
{
    radiance = make_float3(0.0f); // Start with black.

    float3 throughput = make_float3(1.0f); // The throughput for the next radiance, starts with 1.0f.

                                           // Russian Roulette path termination after a specified number of bounces needs the current depth.
    int depth = 0; // Path segment index. Primary ray is 0.

    prd.flags = 0;
    float ao = 1.0;
    while (depth < sysPathLengths.y)
    {
        prd.wo = -prd.wi;           // Direction to observer.
        prd.flags &= FLAG_CLEAR_MASK;   // Clear all non-persistent flags. In this demo only the last diffuse surface interaction stays.

        // Note that the primary rays wouldn't offset the ray t_min by sysSceneEpsilon.
        prd.ao = 0.0;
        optix::Ray ray = optix::make_Ray(prd.pos, prd.wi, 0, sysSceneEpsilon, RT_DEFAULT_MAX);
        rtTrace(sysTopObject, ray, prd);

        //accumulate it all
        float a1 = optix::clamp(float(1 - prd.ao), float(0.), float(1.0));

        //fix prd.radiance
        if (isnan(prd.radiance.x) || isnan(prd.radiance.y) || isnan(prd.radiance.z))
        {
            prd.radiance = make_float3(1.0f, 1.0f, 1.0f); // super red
            prd.flags |= FLAG_TERMINATE;
        }
        else if (isinf(prd.radiance.x) || isinf(prd.radiance.y) || isinf(prd.radiance.z))
        {
            prd.radiance = make_float3(1.0f, 1.0f, 1.0f); // super green
            prd.flags |= FLAG_TERMINATE;
        }
        else if (prd.radiance.x < 0.0f || prd.radiance.y < 0.0f || prd.radiance.z < 0.0f)
        {
            prd.radiance = make_float3(1.0f, 1.0f, 1.0f); // super blue
            prd.flags |= FLAG_TERMINATE;
        }

        //fix prd.f_over_pdf
        if (isnan(prd.f_over_pdf.x) || isnan(prd.f_over_pdf.y) || isnan(prd.f_over_pdf.z))
        {
            prd.f_over_pdf = make_float3(1.0f, 1.0f, 1.0f); // super red
            prd.flags |= FLAG_TERMINATE;
        }
        else if (isinf(prd.f_over_pdf.x) || isinf(prd.f_over_pdf.y) || isinf(prd.f_over_pdf.z))
        {
            prd.f_over_pdf = make_float3(1.0f, 1.0f, 1.0f); // super green
            prd.flags |= FLAG_TERMINATE;
        }
        else if (prd.f_over_pdf.x < 0.0f || prd.f_over_pdf.y < 0.0f || prd.f_over_pdf.z < 0.0f)
        {
            prd.f_over_pdf = make_float3(1.0f, 1.0f, 1.0f); // super blue
            prd.flags |= FLAG_TERMINATE;
        }

        radiance += throughput*prd.radiance;// / (depth + 1);

        // Path termination by miss shader or sample() routines.
        // If terminate is true, f_over_pdf and pdf might be undefined.
        if ((prd.flags & FLAG_TERMINATE) || prd.pdf <= 0.0f || isNull(prd.f_over_pdf))
        {
            break;
            depth = 13;
        }

        // PERF f_over_pdf already contains the proper throughput adjustment for diffuse materials: f * (fabsf(optix::dot(prd.wi, state.normal)) / prd.pdf);
        throughput *= prd.f_over_pdf;
        ao += prd.ao;
        // Unbiased Russian Roulette path termination.
        if (sysPathLengths.x <= depth) // Start termination after a minimum number of bounces.
        {
            const float probability = fmaxf(throughput); // DAR Other options: // intensity(throughput); // fminf(0.5f, intensity(throughput));
            if (probability < rng(prd.seed)) // Paths with lower probability to continue are terminated earlier.
            {
                break;
            }
            throughput /= probability; // Path isn't terminated. Adjust the throughput so that the average is right again.
        }

        ++depth; // Next path segment.
    }
    ao = optix::clamp(float(ao), float(0.), float(1.0));
    //radiance *= ao;
}

RT_FUNCTION void integrator3(PerRayData& prd, float3& radiance)
{
    radiance = make_float3(0.0f); // Start with black.

    float3 throughput = make_float3(1.0f); // The throughput for the next radiance, starts with 1.0f.

    // Russian Roulette path termination after a specified number of bounces needs the current depth.
    int depth = 0; // Path segment index. Primary ray is 0.

    prd.flags = 0;

    while (depth < sysPathLengths.y)
    {
        prd.wo = -prd.wi;           // Direction to observer.
        prd.flags &= FLAG_CLEAR_MASK;   // Clear all non-persistent flags. In this demo only the last diffuse surface interaction stays.

        // Note that the primary rays wouldn't offset the ray t_min by sysSceneEpsilon.
        optix::Ray ray = optix::make_Ray(prd.pos, prd.wi, 0, sysSceneEpsilon, RT_DEFAULT_MAX);
        rtTrace(sysTopObject, ray, prd);

        radiance += throughput * prd.radiance;

        // Path termination by miss shader or sample() routines.
        // If terminate is true, f_over_pdf and pdf might be undefined.
        if ((prd.flags & FLAG_TERMINATE) || prd.pdf <= 0.0f || isNull(prd.f_over_pdf))
        {
            break;
        }

        // PERF f_over_pdf already contains the proper throughput adjustment for diffuse materials: f * (fabsf(optix::dot(prd.wi, state.normal)) / prd.pdf);
        throughput *= prd.f_over_pdf;

        // Unbiased Russian Roulette path termination.
        if (sysPathLengths.x <= depth) // Start termination after a minimum number of bounces.
        {
            const float probability = fmaxf(throughput); // DAR Other options: // intensity(throughput); // fminf(0.5f, intensity(throughput));
            if (probability < rng(prd.seed)) // Paths with lower probability to continue are terminated earlier.
            {
                break;
            }
            throughput /= probability; // Path isn't terminated. Adjust the throughput so that the average is right again.
        }

        ++depth; // Next path segment.
    }
}

RT_FUNCTION void integrator2(PerRayData& prd, float3& radiance)
{
    radiance = make_float3(0.0f); // Start with black.

    float ao = 0.0; //ambient occlusion calculation

    // Russian Roulette path termination after a specified number of bounces needs the current depth.
    int depth = 0; // Path segment index. Primary ray is 0.

    prd.flags = 0;

    while (depth < sysPathLengths.y)
    {
        prd.wo = -prd.wi;           // Direction to observer.
        prd.flags &= FLAG_CLEAR_MASK;   // Clear all non-persistent flags. In this demo only the last diffuse surface interaction stays.
       //set initial ao to 0;
        prd.ao = 0.0;
        // shoot the new rays with adjusted direction and origin
        optix::Ray ray = optix::make_Ray(prd.pos, prd.wi, 0, sysSceneEpsilon, RT_DEFAULT_MAX);
        rtTrace(sysTopObject, ray, prd);

        //radiance += throughput * prd.radiance;

        // Path termination by miss shader or sample() routines.
        // If terminate is true, f_over_pdf and pdf might be undefined.
        if ((prd.flags & FLAG_TERMINATE))
        {
            break;
        }

        //summ ao
        radiance += prd.ao * prd.radiance / (depth + 1);//weight *i / depth;
        //ao += ;

        ++depth; // Next path segment.
    }
}

// Entry point for pinhole camera with manual accumulation, non-VCA.
RT_PROGRAM void raygeneration()
{
    PerRayData prd;

    prd.radiance = make_float3(0.0f);
    prd.depth = 0;
    prd.result = make_float4(0.5);
    prd.cur_prim = 0;
    prd.isSoundRay = false;
    //   prd.rnd = rng(prd.seed);
    prd.TimeSound = 0.0f;
    prd.isDynamic = false;
    prd.radiance = make_float3(0.0);// sysBackground;
    prd.length = 0;
    prd.ao = 0;

    //compute normalized ray direction[-1,1]
    //optix::Ray ray = ComputeDirPos(prd);

    // Initialize the random number generator seed from the linear pixel index and the iteration index.
    prd.seed = tea<8>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, sysIterationIndex);
    prd.ao = 0.0;
    prd.f_over_pdf = make_float3(1.0);
    prd.maxDist = 4.0;
    prd.length = 0;
    // Pinhole camera implementation:
    // The launch index is the pixel coordinate.
    // Note that launchIndex = (0, 0) is the bottom left corner of the image,
    // which matches the origin in the OpenGL texture used to display the result.
    const float2 pixel = make_float2(theLaunchIndex);
    // Sample the ray in the center of the pixel.
    const float2 fragment = pixel + 0.5;//rng2(prd.seed); // Random jitter of the fragment location in this pixel.
    // The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
    const float2 screen = make_float2(theLaunchDim);
    // Normalized device coordinates in range [-1, 1].
    const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

    // The integrator expects the next path segments ray.origin in prd.pos and the next ray.direction in prd.wi.
    prd.pos = sysCameraPosition;
    prd.wi = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

    float3 radiance;

    integrator(prd, radiance); // In this case a unidirectional path tracer.

  //#if USE_DEBUG_EXCEPTIONS
    // DAR DEBUG Highlight numerical errors.
    if (isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z))
    {
        radiance = make_float3(1.0f, 1.0f, 1.0f); // super red
    }
    else if (isinf(radiance.x) || isinf(radiance.y) || isinf(radiance.z))
    {
        radiance = make_float3(1.0f, 1.0f, 1.0f); // super green
    }
    else if (radiance.x < 0.0f || radiance.y < 0.0f || radiance.z < 0.0f)
    {
        radiance = make_float3(1.0f, 1.0f, 1.0f); // super blue
    }
    //#else
      // NaN values will never go away. Filter them out before they can arrive in the output buffer.
      // This only has an effect if the debug coloring above is off!
    //  if (!(isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z)))
    //#endif
    {
        if (0 < sysIterationIndex)
        {
            float4 dst = sysOutputBuffer[theLaunchIndex];  // RGBA32F
            sysOutputBuffer[theLaunchIndex] = optix::lerp(dst, make_float4(radiance, 1.0f), 1.0f / (float)(sysIterationIndex + 1));
            //printf("iteration done ");
        }
        else
        {
            // sysIterationIndex 0 will fill the buffer.
            // If this isn't done separately, the result of the lerp() above is undefined, e.g. dst could be NaN.
            sysOutputBuffer[theLaunchIndex] = make_float4(radiance, 1.0f);
        }
    }
}