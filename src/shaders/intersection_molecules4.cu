#include "hip/hip_runtime.h"
/*
All basic variables for SDFs visual-auditory ray-tracing
 */
#include "sdfGeometryVariables.h"

#include "sdfPrimPrograms.h"
using namespace optix;
rtDeclareVariable(float3, sysCameraPosition, , );
rtBuffer<float3>    Positions;
rtBuffer<int2>    Bonds;
rtBuffer<int4>    Mols;

rtBuffer<float>    BSRadius;
rtBuffer<int>    BSType;
rtDeclareVariable(float, MultiscaleParam, , );
rtDeclareVariable(float3, pr_pos, attribute primitive_pos, );
rtDeclareVariable(float, pr_rad, attribute primitive_rad, );
//rtDeclareVariable(int, pr_type, attribute primitive_type, );

rtDeclareVariable(float, sysSceneEpsilon, , );

//rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, ); //for direct tracing of sdf spheres, or defined set of primitives with definde BB

#include "renderer/per_ray_data.h"
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

//TODO:for SDF
//still have to be implemented as PTX should be generated automatically
//PTX can automatically generated for some primitives like spheres, boxes and etc
//consider further integration with python

//for SDF

//for dynamic staff
rtDeclareVariable(int, PNum, , );
rtDeclareVariable(int, numFrames, , );
rtDeclareVariable(float, TimeSound, , );

inline __device__ float evaluateFunction(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    float3 pos1 = descPrim.pos[0];
    float rad1 = descPrim.rad[0];

    float3 pos2 = descPrim.pos[1];
    float rad2 = descPrim.rad[1];

    float3 pos3 = descPrim.pos[2];
    float rad3 = descPrim.rad[2];
    float3 pos4 = descPrim.pos[3];

    float3 cent = (pos1 + pos2 + pos3) / 3.0;
    f = sdfPrim5(x, pos1, pos2, pos3, pos4, rad1, rad2); //length(x-cent)-rad1;//

    return f;
}

inline __device__ float3 computeNormal(float eps, float3 x, primParamDesc descPrim)
{
    float dx = evaluateFunction(x + make_float3(eps, 0, 0), descPrim) - evaluateFunction(x - make_float3(eps, 0, 0), descPrim);
    float dy = evaluateFunction(x + make_float3(0, eps, 0), descPrim) - evaluateFunction(x - make_float3(0, eps, 0), descPrim);
    float dz = evaluateFunction(x + make_float3(0, 0, eps), descPrim) - evaluateFunction(x - make_float3(0, 0, eps), descPrim);

    return normalize(make_float3(dx, dy, dz));
}

inline __device__ float SphereTraceForward(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*t;

    float dist;

    float totalDistance = t;
    int i = 0;
    bool stop = false;
    while (!stop)
    {
        dist = evaluateFunction(x, descPrim);// sdfPrim1(x, pos, pos2, rad1, rad2);

                                             // Step along the ray and accumulate the distance from the origin.
        x += abs(dist) * ray_direction;
        totalDistance += abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance >= tmax)
        {
            stop = true;
        }
        if (dist < 0) {
            //x -= abs(dist) * ray_direction;
            totalDistance -= abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

inline __device__ float SphereTraceBack(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*tmax;

    float dist;

    //TODO: getType

    float totalDistance = tmax - t;
    int i = 0;
    bool stop = false;

    while (!stop)
    {
        //dist = sdfPrim1(x, pos, pos2, rad1, rad2);
        dist = evaluateFunction(x, descPrim);// sdfPrim1(x, pos, pos2, rad1, rad2);

                                             // Step along the ray and accumulate the distance from the origin.
        x -= abs(dist) * ray_direction;
        totalDistance -= abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance <= 0)
        {
            stop = true;
        }
        if (dist < 0) {
            x += abs(dist) * ray_direction;
            totalDistance += abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

//------------------------

inline __device__  primParamDesc getTimeData(int primIdx)
{
    const int4 idsBonds = Mols[primIdx];

    const int2 idsB1 = Bonds[idsBonds.x];
    const int2 idsB2 = Bonds[idsBonds.y];
    const int2 idsB3 = Bonds[idsBonds.z];
    const int2 idsB4 = Bonds[idsBonds.w];

    //idsB1.x==idsB2.x;

    //we know there are only 3 atoms - TODO:
    const float rad1 = BSRadius[idsB1.x - 1];
    const float rad2 = BSRadius[idsB1.y - 1];
    const float rad3 = BSRadius[idsB2.y - 1];

    const int type = BSType[idsB1.x - 1];
    const int type2 = BSType[idsB1.y - 1];
    const int type3 = BSType[idsB2.y - 1];
    const int type4 = BSType[idsB3.y - 1];
    const int type5 = BSType[idsB4.y - 1];

    //---------------
    int lower = int(floorf(TimeSound));
    int upper = int(ceilf(TimeSound));

    //0,1;1,2;2,3;

    float timeS = TimeSound;
    if (upper > numFrames) upper = numFrames;

    float time = timeS - float(lower);

    //int time = int(floorf(TimeSound)); //integer part
    //int upper=int(time);

    //if (time > numFrames) time = numFrames;

    //for bond interpolation
    float3 pos2 = make_float3(0);
    float3 pos3 = make_float3(0);
    float3 pos = make_float3(0);
    float3 pos14, pos15, pos4, pos5, pos24, pos25;
    //for frames
    float3 pos12 = make_float3(0);
    float3 pos13 = make_float3(0);
    float3 pos11 = make_float3(0);

    float3 pos22 = make_float3(0);
    float3 pos23 = make_float3(0);
    float3 pos21 = make_float3(0);

    //float3 pos1 = Positions[primIdx + time*PNum]; //getting correct frame

    pos11 = Positions[idsB1.x - 1 + lower*PNum];
    pos12 = Positions[idsB1.y - 1 + lower*PNum];
    pos13 = Positions[idsB2.y - 1 + lower*PNum];
    pos14 = Positions[idsB3.y - 1 + lower*PNum];
    pos15 = Positions[idsB4.y - 1 + lower*PNum];

    pos = Positions[idsB1.x - 1 + lower*PNum];
    pos2 = Positions[idsB1.y - 1 + lower*PNum];
    pos3 = Positions[idsB2.y - 1 + lower*PNum];
    pos4 = Positions[idsB3.y - 1 + lower*PNum];
    pos5 = Positions[idsB4.y - 1 + lower*PNum];
    if (numFrames > 0) //dynamic
    {
        pos21 = Positions[idsB1.x - 1 + upper*PNum];
        pos22 = Positions[idsB1.y - 1 + upper*PNum];
        pos23 = Positions[idsB2.y - 1 + upper*PNum];
        pos24 = Positions[idsB3.y - 1 + upper*PNum];
        pos25 = Positions[idsB4.y - 1 + upper*PNum];

        pos = time*pos21 + (1.0 - time)*pos11; //time interpolation
        pos2 = time*pos22 + (1.0 - time)*pos12; //time interpolation
        pos3 = time*pos23 + (1.0 - time)*pos13; //time interpolation
        pos4 = time*pos24 + (1.0 - time)*pos14; //time interpolation
        pos5 = time*pos25 + (1.0 - time)*pos15; //time interpolation
    }

    primParamDesc descPrim;

    descPrim.type = 5; //sphere data type
    descPrim.pos[0] = pos2;
    descPrim.pos[1] = pos3;
    descPrim.pos[2] = pos4;
    descPrim.pos[3] = pos5;

    descPrim.rad[0] = rad1;
    descPrim.rad[1] = rad2;
    descPrim.rad[2] = rad3;

    descPrim.types[0] = type;
    descPrim.types[1] = type2;
    descPrim.types[2] = type3;

    return descPrim;
}
//------------------------------------------------------
//---intersection with dynamic molecule, use of morphing
//------------------------------------------------------

inline __device__   float3 boundIntersection(primParamDesc  descPrim, float3 origin, float3 direction)
{
    // float t1, t2;
    float3 pos_along_ray1, per_ray_data2;
    float rayBoundSphere1, rayBoundSphere2;
    int caseN = -1; //ray miss

                    // float leng = length(pos2 - pos) + length(pos3 - pos);
    const float3 cent = (descPrim.pos[2] + descPrim.pos[1] + descPrim.pos[0] + descPrim.pos[3]) / 4;
    const float delta = fmax(fmaxf(descPrim.rad[1], descPrim.rad[2]), descPrim.rad[0]);
    const float rayBoundSphere = length(cent - descPrim.pos[0]) + delta * 2 + descPrim.rad[0] * 4;//include rad*2 for vibrations movement

    float t = length(cent - theRay.origin);
    float3 pos_along_ray = theRay.origin + theRay.direction * t;
    float tmax = t + 2 * rayBoundSphere; //to stop sphere tracing

                                   /* alternative splitting*/
                                   /*
                                   float3 cent1=(pos+pos2)/2;
                                   float3 cent2= (pos+pos3)/2;

                                   t1 = length(cent1 - theRay.origin);
                                   t2 = length(cent2 - theRay.origin);
                                   pos_along_ray1 = theRay.origin + theRay.direction * t1;
                                   pos_along_ray2 = theRay.origin + theRay.direction * t2;
                                   rayBoundSphere1 =length(cent1 - pos) + delta * 2;
                                   rayBoundSphere2 =length(cent1 - pos) + delta * 2;
                                   */

    float radiusB = length(cent - descPrim.pos[1]) / 1.0 + delta;
    // float radiusB2 = length(cent - descPrim.pos[2]) / 2 + delta;
    // float radiusB2 = length(cent - descPrim.pos[2]) / 2 + delta;

   //  float radiusB = fmaxf(radiusB1, radiusB2);

     //compute center of attraction as there may be several intersections with central part
    float t1, t2, t3, t4, t5, t6, t7, t8, t9;
    t1 = 0;
    t2 = 0;
    t3 = 0;
    t4 = 0;
    t5 = 0;

    t1 = length(descPrim.pos[1] - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t1;

    if (length(descPrim.pos[1] - pos_along_ray) < radiusB)
        caseN = 1; //left

    t2 = length(descPrim.pos[2] - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t2;
    if (length(descPrim.pos[2] - pos_along_ray) < radiusB)
        caseN = 2; //left

    t3 = length(descPrim.pos[0] - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t3;
    if (length(descPrim.pos[0] - pos_along_ray) < radiusB)
        caseN = 0; //central intersection

    t4 = length(descPrim.pos[3] - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t4;
    if (length(descPrim.pos[3] - pos_along_ray) < radiusB)
        caseN = 4; //central intersection

    t5 = length(cent - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t5;
    if (length(cent - pos_along_ray) < radiusB)
        caseN = 5; //central intersection

    float3 c1 = cent;
    float3 c2 = cent;
    float3 c3 = cent;
    float3 c4 = cent;
    c1.y -= 3.3 / 2;
    c2.y += 3.3 / 2;
    c3.x -= 3.3 / 2;
    c4.x += 3.3 / 2;

    t6 = length(c1 - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t6;
    if (length(c1 - pos_along_ray) < radiusB)
        caseN = 6; //cell2 intersection

    t7 = length(c2 - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t7;
    if (length(c2 - pos_along_ray) < radiusB)
        caseN = 7; //cel2 intersection

    t8 = length(c3 - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t8;
    if (length(c3 - pos_along_ray) < radiusB)
        caseN = 8; //cell2 intersection

    t9 = length(c4 - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t9;
    if (length(c4 - pos_along_ray) < radiusB)
        caseN = 9; //cel2 intersection

                   //get min intersection
    t = fminf(fminf(t1, t2), t3);
    t = fminf(t, fminf(t4, t5));
    t = fminf(t, fminf(t6, t7));
    t = fminf(t, fminf(t8, t9));

    float     tmin = fmaxf(0.0, t - radiusB);
    float3 params = make_float3(tmin, tmax, 0.0);
    if (caseN >= 0)
    {
        params.z = 1.0;
    }
    else  params.z = 0.0; //within bounding sphere

    return params;
}
RT_PROGRAM void intersection_molecules(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;

    float epsilon = 0.001;//delta;
    if (epsilon > sysSceneEpsilon) epsilon = sysSceneEpsilon;
    float eps = 0.001;
    if (eps > sysSceneEpsilon) eps = sysSceneEpsilon;

    /* ------------------
    /* 1) Reading data and accessing current positions for current time
    /---------------------------------------------------------------*/

    primParamDesc descPrim = getTimeData(primIdx);

    //--------------------

    //---level of detail
    float dist_cam = length(sysCameraPosition - (descPrim.pos[0] + descPrim.pos[1] + descPrim.pos[2] + descPrim.pos[3]) / 4);
    float d;
    if (dist_cam < 20.0)
    {
        float interp = (dist_cam - 10) / 10.0;
        d = optix::clamp(interp, 0.0, 1.0);
    }
    else d = 1;

    //-----------------------------

    float3 interSectParams = boundIntersection(descPrim, theRay.origin, theRay.direction);
    //if within bounding volume intersection
    //float3 interSectParams = BoundingSubVolumesIntersect(descPrim);
    if (interSectParams.z > 0.0)
    {
        tmin = interSectParams.x;
        tmax = interSectParams.y;
        float totalDistance = tmin;
        // === Raymarching (Sphere Tracing) Procedure ===

        totalDistance = SphereTraceForward(epsilon, tmin, tmax, descPrim);

        // Found potential intersection?
        if (totalDistance < tmax) //we found intersection
        {
            float totalDistance2 = SphereTraceBack(epsilon, totalDistance, tmax, descPrim);
            //------------
            if (totalDistance2 > epsilon)
            { //it is sufficiently large subvolume to ray-cast
                if (rtPotentialIntersection(totalDistance))
                {
                    //compute normal for primitive
                    float3 x = theRay.origin + theRay.direction*totalDistance;

                    float3 cNormal = computeNormal(eps, x, descPrim);

                    infoH.normal = cNormal;
                    infoH.hit_point = theRay.origin + theRay.direction * (totalDistance);
                    infoH.tmin = totalDistance;

                    infoH.desc = descPrim;
                    infoH.maxDist = totalDistance2;
                    rtReportIntersection(MaterialIndex);
                }
            }
        }
    }
}

//bounding box
RT_PROGRAM void boundingbox_molecules(int primIdx, float result[6])
{
    const int4 idsBonds = Mols[primIdx];

    const int2 idsB1 = Bonds[idsBonds.x];
    const int2 idsB2 = Bonds[idsBonds.y];
    const int2 idsB3 = Bonds[idsBonds.z];
    const int2 idsB4 = Bonds[idsBonds.w];

    const float rad1 = BSRadius[idsB1.x - 1] * 5;
    const float rad2 = BSRadius[idsB1.y - 1] * 5;

    float3 pos = Positions[idsB1.x - 1];
    float3 pos2 = Positions[idsB1.y - 1];
    float3 pos3 = Positions[idsB2.y - 1];
    float3 pos4 = Positions[idsB3.y - 1];
    float3 pos5 = Positions[idsB4.y - 1];

    float3 pos_min = fminf(pos, pos2);
    pos_min = fminf(pos_min, pos3);
    pos_min = fminf(pos_min, pos4);
    pos_min = fminf(pos_min, pos5);

    float3 pos_max = fmaxf(pos, pos2);
    pos_max = fmaxf(pos_max, pos3);
    pos_max = fmaxf(pos_max, pos4);
    pos_max = fmaxf(pos_max, pos5);
    if (numFrames > 0)
    {
        for (int i = 1; i < numFrames; i++)
        {
            pos = Positions[idsB1.x - 1 + i*PNum];
            pos2 = Positions[idsB1.y - 1 + i*PNum];
            pos3 = Positions[idsB2.y - 1 + i*PNum];
            pos4 = Positions[idsB3.y - 1 + i*PNum];
            pos5 = Positions[idsB4.y - 1 + i*PNum];

            pos_min = fminf(fminf(pos, pos2), pos_min);
            pos_min = fminf(pos_min, pos3);
            pos_min = fminf(pos_min, pos4);
            pos_min = fminf(pos_min, pos5);

            pos_max = fmaxf(fmaxf(pos, pos2), pos_max);
            pos_max = fmaxf(pos_max, pos3);
            pos_max = fmaxf(pos_max, pos4);
            pos_max = fmaxf(pos_max, pos5);
        }
    }

    float rad = fmaxf(rad1, rad2);

    optix::Aabb* aabb = (optix::Aabb*)result;
    //increase for ao by 5
    aabb->m_min = pos_min - make_float3(rad);
    aabb->m_max = pos_max + make_float3(rad);
}