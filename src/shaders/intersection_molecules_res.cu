#include "hip/hip_runtime.h"
/*
All basic variables for SDFs visual-auditory ray-tracing
 */
#include "sdfGeometryVariables.h"
#include "renderer/random_number_generators.h"
using namespace optix;
rtDeclareVariable(float3, sysCameraPosition, , );
rtBuffer<float3>    Positions;
rtBuffer<int2>    Bonds;
rtBuffer<int2>    Mols;
rtBuffer<float>    BSRadius;
rtBuffer<int>    BSType;
rtDeclareVariable(float, MultiscaleParam, , );
rtDeclareVariable(float3, pr_pos, attribute primitive_pos, );
rtDeclareVariable(float, pr_rad, attribute primitive_rad, );
//rtDeclareVariable(int, pr_type, attribute primitive_type, );

rtDeclareVariable(float, sysSceneEpsilon, , );

//rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, ); //for direct tracing of sdf spheres, or defined set of primitives with definde BB

#include "renderer/per_ray_data.h"
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

//TODO:for SDF
//still have to be implemented as PTX should be generated automatically
//PTX can automatically generated for some primitives like spheres, boxes and etc
//consider further integration with python

//for SDF

typedef rtCallableProgramId<float(float3, float3, float3, float3, float, float, float)> callT;
rtDeclareVariable(callT, sdfPrim, , );

//for dynamic staff
rtDeclareVariable(int, PNum, , );
rtDeclareVariable(int, numFrames, , );
rtDeclareVariable(float, TimeSound, , );

//------------------------------------------------------
//---intersection with dynamic molecule, use of morphing
//------------------------------------------------------

inline __device__ float3 transfer_function(int t, float d)
{
    // return TFBuffer[t];
    switch (t)
    {
    case 1: //H
        return make_float3(1, 1, 1);
        break;
    case 2: //C
        return make_float3(0.5);
        break;
    case 3: //N
        return make_float3(0, 0, 0.5);
        break;
    case 4: //S
        return make_float3(1, 1, 0);
        break;
    case 5: //O
        return make_float3(1, 0, d);
        break;
    case 6: //P
        return make_float3(1, 0.5, 0);
        break;
    }
    return make_float3(0, 0, 0);
}

inline __device__ float3 vibTF(int t, float d, float3 pos)
{
    //float3 p=pos;
    switch (t)
    {
    case 1: //H
        return pos + make_float3(-0.41259998, 0, 0.538)*2.0*sin(TimeSound * M_PI_2 * 10);
        break;
    case 2: //C
        return pos;
        break;
    case 3: //N
        return pos;
        break;
    case 4: //S
        return pos;
        break;
    case 5: //O
        return pos;
        break;
    case 6: //P
        return pos;
        break;
    }
    return  pos;
}

inline __device__ float4  blendColor(float dMd, float3 x, float3 dir, float3 pos, float3 pos2, float3 pos3, float r1, float r2, float r3, float3 col1, float3 col2, float3 col3)
{
    float dNorm = 10;

    float3 vib_color[3];
    vib_color[0] = make_float3(0, 1.0, 0); //green
    vib_color[1] = make_float3(0, 0.0, 1.0); //blue
    vib_color[2] = make_float3(1, 1, 0); //yellow

    float3 wt_col = make_float3(0);
    dNorm = length(pos - pos2) + length(pos3 - pos2) + length(pos - pos3);
    dNorm /= 3;

    float d1 = (length(x - pos) - r1*(1 + dMd)) / dNorm;
    float d2 = (length(x - pos2) - r2*(1 + dMd)) / dNorm;
    float d3 = (length(x - pos3) - r3*(1 + dMd)) / dNorm;

    d1 = optix::clamp(d1, 0.01, 1.0);
    d2 = optix::clamp(d2, 0.01, 1.0);
    d3 = optix::clamp(d3, 0.01, 1.0);

    float3 color = make_float3(0);
    float tr = abs(dMd - 0.1);
    color += (1 - d1)*col1;
    color += (1 - d2)*col2;
    color += (1 - d3)*col3;
    //color += dMd*make_float3(1);
    //color*=tr;

    //dMd=1 - rep1;
    //dMd=0 -rep2
    //dMd*10

    float4 color_sample = make_float4(color.x*(1 - tr), color.y*(1 - tr), color.z*(1 - tr), tr);
    //float4 color_sample = make_float4(tr);
        //volume rendering
    float3 xx = x;
    int maxSteps = 7;
    float step = 2 * r1 / maxSteps;
    for (int i = 0; i < maxSteps; i++) {
        float d1 = (length(xx - pos) - r1);
        float d2 = (length(xx - pos2) - r2);
        float d3 = (length(xx - pos3) - r3);

        //  d1 = optix::clamp(d1, -0.9, 1.0);
        //  d2 = optix::clamp(d2, -0.9, 1.0);
        //  d3 = optix::clamp(d3, -0.9, 1.0);
        float3 col = make_float3(0);
        if (d1 < 0.1)
            col += abs(d1 / r1)*col1;
        if (d2 < 0.1)
            col += abs(d2 / r2)*col2;
        if (d3 < 0.1)
            col += abs(d3 / r3)*col3;

        //------------
        //interpolate between two models
        float3 coll = make_float3(0);
        d1 = optix::clamp(d1, -0.9, 0.0);
        d2 = optix::clamp(d2, -0.9, 0.0);
        d3 = optix::clamp(d3, -0.9, 0.0);
        coll += abs(1 - d1)*col1*abs(d1);
        coll += abs(1 - d2)*col2*abs(d2);
        coll += abs(1 - d3)*col3*abs(d3);
        //coll ;
        col = dMd*coll + (1 - dMd)*col;

        //col+=dMd*make_float3(1);

        //d1 += 0.1;
        //d2 += 0.1;
        //d3 += 0.1;
        tr = abs(1 - min(min(d1, d2), d3)) / 2;//abs(min(min(d1 / r1, d2 / r2), d3 / r3)) / 2 + 
        //tr /= 3;
        //tr=clamp(tr,0,1);
        // tr = (3 - d1 - d2 - d3) / 3;
        float4 col2 = make_float4(col.x, col.y, col.z, tr);
        // pre-multiply alpha
       // col2.w = (1 - d);
        col2.x *= col2.w;
        col2.y *= col2.w;
        col2.z *= col2.w;
        // "over" operator for front-to-back blending
        color_sample = color_sample + col2*(1.0f - color_sample.w);
        color_sample.w = tr;
        xx += theRay.direction*step;
    }

    //volume rendering
    //----------
//color_sample=make_float4(color,1.0);
    //return make_float3(color_sample.x, color_sample.y, color_sample.z);
    return color_sample;
}

RT_PROGRAM void intersection_molecules(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;

    const int2 idsBonds = Mols[primIdx];

    const int2 idsB1 = Bonds[idsBonds.x];
    const int2 idsB2 = Bonds[idsBonds.y];

    //idsB1.x==idsB2.x;

    //we know there are only 3 atoms - TODO:
    const float rad1 = BSRadius[idsB1.x - 1];
    const float rad2 = BSRadius[idsB1.y - 1];
    const float rad3 = BSRadius[idsB2.y - 1];

    const int type = BSType[idsB1.x - 1];
    const int type2 = BSType[idsB1.y - 1];
    const int type3 = BSType[idsB2.y - 1];

    //---------------
    int lower = int(floorf(TimeSound));
    int upper = int(ceilf(TimeSound));

    //0,1;1,2;2,3;

    float timeS = TimeSound;
    if (upper > numFrames) upper = numFrames;

    float time = timeS - float(lower);

    //int time = int(floorf(TimeSound)); //integer part
    //int upper=int(time);

    //if (time > numFrames) time = numFrames;

            //for bond interpolation
    float3 pos2 = make_float3(0);
    float3 pos3 = make_float3(0);
    float3 pos = make_float3(0);

    //for frames
    float3 pos12 = make_float3(0);
    float3 pos13 = make_float3(0);
    float3 pos11 = make_float3(0);

    float3 pos22 = make_float3(0);
    float3 pos23 = make_float3(0);
    float3 pos21 = make_float3(0);

    //float3 pos1 = Positions[primIdx + time*PNum]; //getting correct frame

    pos11 = Positions[idsB1.x - 1 + lower*PNum];
    pos12 = Positions[idsB1.y - 1 + lower*PNum];
    pos13 = Positions[idsB2.y - 1 + lower*PNum];

    pos = Positions[idsB1.x - 1 + lower*PNum];
    pos2 = Positions[idsB1.y - 1 + lower*PNum];
    pos3 = Positions[idsB2.y - 1 + lower*PNum];

    if (numFrames > 0) //dynamic
    {
        pos21 = Positions[idsB1.x - 1 + upper*PNum];
        pos22 = Positions[idsB1.y - 1 + upper*PNum];
        pos23 = Positions[idsB2.y - 1 + upper*PNum];

        pos = time*pos21 + (1.0 - time)*pos11; //time interpolation
        pos2 = time*pos22 + (1.0 - time)*pos12; //time interpolation
        pos3 = time*pos23 + (1.0 - time)*pos13; //time interpolation
    }

    //--------------------

    //---level of detail
    float dist_cam = length(sysCameraPosition - (pos + pos2 + pos3) / 3);
    float d;
    if (dist_cam < 20.0)
    {
        float interp = (dist_cam - 10) / 10.0;
        d = optix::clamp(interp, 0.0, 1.0);
    }
    else d = 1;

    //MultiscaleParam=d;

    float3 col1 = transfer_function(type, d);
    float3 col2 = transfer_function(type2, d);
    float3 col3 = transfer_function(type3, d);

    //-----------------------------

    float leng = length(pos2 - pos) + length(pos3 - pos);
    const float3 cent = (pos2 + pos3 + pos) / 3;
    //const float rad = BSRadius[primIdx];

    //const float3 pp=pos2*TimeSound+(1-TimeSound)*pos;
    //float3 pp = 0.5*pos2 + 0.5*pos; //new center
    const float t = length(cent - theRay.origin);
    const float3 pos_along_ray = theRay.origin + theRay.direction * t;
    tmax = t + leng*4.0; //to stop sphere tracing

   // if (length(pp - pos_along_ray) < length(leng) + rad)
    { //TDO: return it && rtPotentialIntersection(t)) {
        //tmin = t;
        // === Raymarching (Sphere Tracing) Procedure ===
        optix::float3 ray_direction = theRay.direction;
        optix::float3 eye = theRay.origin;
        //    eye.x -= global_t * 1.2f;
        optix::float3 x = eye;// +tmin * ray_direction;

        float epsilon = 0.002;//delta;
        if (epsilon > sysSceneEpsilon) epsilon = sysSceneEpsilon;
        float eps = 0.0001;
        if (eps > sysSceneEpsilon) eps = sysSceneEpsilon;
        float dist = 0;

        float totalDistance = 0.0;//Jitter * tea<4>(current_prd.seed, frame_number);
        int i = 0;
        bool stop = false;
        float dist1, dist2;

        while (!stop)
        {
            dist = sdfPrim(x, pos, pos2, pos3, rad1, rad2, rad3);
            // dist2 = sdfPrim(x, pos, pos3, rad1, rad3);
             //take a union of them
             //dist = min(dist1, dist2);
             // Step along the ray and accumulate the distance from the origin.
            x += abs(dist) * ray_direction;
            //dist_from_origin += dist * fudgeFactor;
            totalDistance += abs(dist);

            // Check if we're close enough or too far.
            if (dist < epsilon || totalDistance > tmax)
            {
                stop = true;
            }
            else i++;
        }

        // Found intersection?
        if (abs(dist) < epsilon)
        {
            if (rtPotentialIntersection(totalDistance))
            {
                //compute normal for primitive
               // float dx = min(sdfPrim(x + make_float3(eps, 0, 0), pos, pos2, rad1, rad2),sdfPrim(x + make_float3(eps, 0, 0), pos, pos3, rad1, rad3)) - min(sdfPrim(x - make_float3(eps, 0, 0), pos, pos2, rad1, rad2),sdfPrim(x + make_float3(eps, 0, 0), pos, pos3, rad1, rad3));
              //  float dy = min(sdfPrim(x + make_float3(0, eps, 0), pos, pos2, rad1, rad2),sdfPrim(x + make_float3( 0,eps, 0), pos, pos3, rad1, rad3)) - min(sdfPrim(x - make_float3(0, eps, 0), pos, pos2, rad1, rad2),sdfPrim(x + make_float3( 0,eps, 0), pos, pos3, rad1, rad3));
              //  float dz = min(sdfPrim(x + make_float3(0, 0, eps), pos, pos2, rad1, rad2),sdfPrim(x + make_float3( 0, 0,eps), pos, pos3, rad1, rad3)) - min(sdfPrim(x - make_float3(0, 0, eps), pos, pos2, rad1, rad2),sdfPrim(x + make_float3( 0, 0,eps), pos, pos3, rad1, rad3));

               //compute normal for primitive
                float dx = sdfPrim(x + make_float3(eps, 0, 0), pos, pos2, pos3, rad1, rad2, rad3) - sdfPrim(x - make_float3(eps, 0, 0), pos, pos2, pos3, rad1, rad2, rad3);
                float dy = sdfPrim(x + make_float3(0, eps, 0), pos, pos2, pos3, rad1, rad2, rad3) - sdfPrim(x - make_float3(0, eps, 0), pos, pos2, pos3, rad1, rad2, rad3);
                float dz = sdfPrim(x + make_float3(0, 0, eps), pos, pos2, pos3, rad1, rad2, rad3) - sdfPrim(x - make_float3(0, 0, eps), pos, pos2, pos3, rad1, rad2, rad3);

                //varNormal = normalize(make_float3(dx, dy, dz));

                //compute color

                    //vibrations for color

                   //--------level of detail continue

                float4 color = make_float4(0);
                float3 vib_color[3];
                vib_color[0] = make_float3(0, 1.0, 0); //green
                vib_color[1] = make_float3(0, 0.0, 1.0); //blue
                vib_color[2] = make_float3(1.0, 1.0, 0); //yellow

                float3 col21;
                float3 col31;

                //return mod1*d + (1.0 - d)*mod2;
                float r1 = d*rad1 / 2 + (1 - d)*rad1;
                float r2 = d*rad2 / 2 + (1 - d)*rad2;
                float r3 = d*rad3 / 2 + (1 - d)*rad3;

                float3 weights[3];

                if (d <= 1.0)
                {
                    //first vibration vector in molecule
                    float3 vib2[3];
                    float3 vib3[3];

                    vib2[0] = (pos2 - pos) / 3.5 * -cos(TimeSound * 20)*(1 - d);
                    vib3[0] = (pos3 - pos) / 3.5 * (cos(TimeSound * 20))*(1 - d);

                    vib2[1] = (pos2 - pos) / 3.5 * sin(TimeSound * 20)*(1 - d);
                    vib3[1] = (pos3 - pos) / 3.5 * (sin(TimeSound * 20))*(1 - d);

                    float3 vib_dir = pos2 - 2 * pos + pos3;

                    vib2[2] = vib_dir / 3.5 * sin(TimeSound * 20)*(1 - d);
                    vib3[2] = vib_dir / 3.5 * (sin(TimeSound * 20))*(1 - d);

                    float d_min = 0;
                    int vib_min = 0;

                    for (int i = 0; i < 2; i++)
                    {
                        float3 pp3 = pos3 + vib3[i];
                        float3 pp2 = pos2 + vib2[i];

                        col31 = d*col3 + (1 - d)*vib_color[i] *1.5;
                        col21 = d*col2 + (1 - d)*vib_color[i] * 1.5;

                        color += blendColor(d, x, theRay.direction, pos, pp2, pp3, r1, r2, r3, col1, col21, col31);
                    }
                    color;///= 3.0;
                }
                else {
                    color = blendColor(d, x, theRay.direction, pos, pos2, pos3, r1, r2, r3, col1, col2, col3);
                }

                //-----------
                //For multiscale
                       /*   float r1 = MultiscaleParam*rad1/ 2 + (1 - MultiscaleParam)*rad1 ;
                           float r2 = MultiscaleParam*rad2/2 + (1 - MultiscaleParam)*rad2 ;

                            float d1 = length(x - pos) - r1;
                            float d2 = length(x - pos2) - r2;
                            float3 color;

                            {
                                float d = abs(d1) + abs(d2);
                                color = (d1 / d)*col2 + (d2 / d)*col1;
                            }*/
                            //length sdfPrim(x, pos, pos2, rad1, rad2);

                            //for material
                            //fill attribute data for material
                float2 inf = make_float2(totalDistance, __int_as_float(type));
                info.primInfo = inf;
                info.type = 0; //don't use mapping
                info.useScalar = color;
                info.normal = normalize(make_float3(dx, dy, dz));
                info.hit_point = theRay.origin + theRay.direction * (totalDistance);
                info.types = make_int2(type, type2);

                pr_pos = pos;
                pr_rad = rad1;
                //pr_type = type;

                //write ao
                //info.ao = ao;//optix::clamp(float(1.0 - ao), float(0.), float(1.0));

                rtReportIntersection(MaterialIndex);
            }
        }
    }
}

//bounding box
RT_PROGRAM void boundingbox_molecules(int primIdx, float result[6])
{
    const int2 idsBonds = Mols[primIdx];

    const int2 idsB1 = Bonds[idsBonds.x];
    const int2 idsB2 = Bonds[idsBonds.y];
    const float rad1 = BSRadius[idsB1.x - 1] * 2;
    const float rad2 = BSRadius[idsB1.y - 1] * 2;
    const float rad3 = BSRadius[idsB2.y - 1] * 2;

    float3 pos = Positions[idsB1.x - 1];
    float3 pos2 = Positions[idsB1.y - 1];
    float3 pos3 = Positions[idsB2.y - 1];

    float3 pos_min = fminf(pos, pos2);
    pos_min = fminf(pos_min, pos3);
    float3 pos_max = fmaxf(pos, pos2);
    pos_max = fmaxf(pos_max, pos3);
    if (numFrames > 0)
    {
        for (int i = 1; i < numFrames; i++)
        {
            pos = Positions[idsB1.x - 1 + i*PNum];
            pos2 = Positions[idsB1.y - 1 + i*PNum];
            pos3 = Positions[idsB2.y - 1 + i*PNum];

            pos_min = fminf(fminf(pos, pos2), pos_min);
            pos_min = fminf(pos_min, pos3);

            pos_max = fmaxf(fmaxf(pos, pos2), pos_max);
            pos_max = fmaxf(pos_max, pos3);
        }
    }

    float rad = fmaxf(rad1, rad2);
    rad = fmaxf(rad, rad3);
    optix::Aabb* aabb = (optix::Aabb*)result;
    //increase for ao by 5
    aabb->m_min = pos_min - make_float3(rad);
    aabb->m_max = pos_max + make_float3(rad);
}