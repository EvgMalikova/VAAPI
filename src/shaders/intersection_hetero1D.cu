#include "hip/hip_runtime.h"
/*
All basic variables for SDFs heterogeneous objects ray-tracing
*/
#include "sdfGeometryVariables.h"

using namespace optix;

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_math.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "attributeInfo.h"
#include "renderer/random_number_generators.h"

#include "sdfPrimPrograms.h"

rtBuffer<float3>    Positions;
rtBuffer<float>    BSRadius;
rtBuffer<int>    BSType;

//Connectivity info
rtBuffer<int2>    Bonds;

rtDeclareVariable(float, MultiscaleParam, , );

rtDeclareVariable(float, sysSceneEpsilon, , );

//for dynamic staff
rtDeclareVariable(int, PNum, , );
rtDeclareVariable(int, numFrames, , );
rtDeclareVariable(float, TimeSound, , );

rtDeclareVariable(float, blendAdd, , );
//-----------------
//TODO:for SDF
//still have to be implemented as PTX should be generated automatically
//PTX can automatically generated for some primitives like spheres, boxes and etc
//consider further integration with python

inline __device__ float evaluateFunction(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    float3 pos1 = descPrim.pos[0];
    float rad1 = descPrim.rad[0];
    float3 pos2 = descPrim.pos[1];
    float rad2 = descPrim.rad[1];
    f = sdfPrim1(x, pos1, pos2, rad1, rad2);

    return f;
}

inline __device__ float3 computeNormal(float eps, float3 x, primParamDesc descPrim)
{
    float dx = evaluateFunction(x + make_float3(eps, 0, 0), descPrim) - evaluateFunction(x - make_float3(eps, 0, 0), descPrim);
    float dy = evaluateFunction(x + make_float3(0, eps, 0), descPrim) - evaluateFunction(x - make_float3(0, eps, 0), descPrim);
    float dz = evaluateFunction(x + make_float3(0, 0, eps), descPrim) - evaluateFunction(x - make_float3(0, 0, eps), descPrim);

    return normalize(make_float3(dx, dy, dz));
}

inline __device__ float SphereTraceForward(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*t;

    float dist;

    float totalDistance = t;
    int i = 0;
    bool stop = false;
    while (!stop)
    {
        dist = evaluateFunction(x, descPrim);// sdfPrim1(x, pos, pos2, rad1, rad2);

                                        // Step along the ray and accumulate the distance from the origin.
        x += abs(dist) * ray_direction;
        totalDistance += abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance >= tmax)
        {
            stop = true;
        }
        if (dist < 0) {
            //x -= abs(dist) * ray_direction;
            totalDistance -= abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

inline __device__ float SphereTraceBack(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*tmax;

    float dist;

    //TODO: getType

    float totalDistance = tmax - t;
    int i = 0;
    bool stop = false;

    while (!stop)
    {
        //dist = sdfPrim1(x, pos, pos2, rad1, rad2);
        dist = evaluateFunction(x, descPrim);// sdfPrim1(x, pos, pos2, rad1, rad2);

                                        // Step along the ray and accumulate the distance from the origin.
        x -= abs(dist) * ray_direction;
        totalDistance -= abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance <= 0)
        {
            stop = true;
        }
        if (dist < 0) {
            x += abs(dist) * ray_direction;
            totalDistance += abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

//------------------------

inline __device__  primParamDesc getTimeData(int primIdx)
{
    int numFr = 3;
    const int2 ids = Bonds[primIdx];
    const float rad1 = BSRadius[ids.x - 1];
    const float rad2 = BSRadius[ids.y - 1];

    const int type = BSType[ids.x - 1];
    const int type2 = BSType[ids.y - 1];

    //---------------
    int lower = int(floorf(TimeSound));
    int upper = int(ceilf(TimeSound));

    float timeS = TimeSound;
    if (upper > numFrames) upper = numFrames;

    float time = timeS - float(lower);

    //int time = int(floorf(TimeSound)); //integer part
    //int upper=int(time);

    //if (time > numFrames) time = numFrames;

    //for bond interpolation
    float3 pos2 = make_float3(0);
    float3 pos = make_float3(0);

    //for frames
    float3 pos12 = make_float3(0);
    float3 pos11 = make_float3(0);

    float3 pos22 = make_float3(0);
    float3 pos21 = make_float3(0);

    //float3 pos1 = Positions[primIdx + time*PNum]; //getting correct frame

    pos11 = Positions[ids.x - 1 + lower*PNum];
    pos12 = Positions[ids.y - 1 + lower*PNum];

    pos = Positions[ids.x - 1 + lower*PNum];
    pos2 = Positions[ids.y - 1 + lower*PNum];

    if (numFrames > 0) //dynamic
    {
        pos21 = Positions[ids.x - 1 + upper*PNum];
        pos22 = Positions[ids.y - 1 + upper*PNum];

        pos = time*pos21 + (1.0 - time)*pos11; //time interpolation
        pos2 = time*pos22 + (1.0 - time)*pos12; //time interpolation
    }
    primParamDesc descPrim;

    descPrim.type = 2;
    descPrim.pos[0] = pos;
    descPrim.pos[1] = pos2;
    descPrim.rad[0] = rad1;
    descPrim.rad[1] = rad2;
    descPrim.types[0] = type;
    descPrim.types[1] = type2;

    return descPrim;
}

inline __device__   float3 boundIntersection(primParamDesc  descPrim, float3 origin, float3 direction)
{
    float3 dir = normalize(descPrim.pos[0] - descPrim.pos[1]);
    float rad = (length(descPrim.pos[1] - descPrim.pos[0]) + descPrim.rad[1] + descPrim.rad[0]) / 2 + 2 * blendAdd;

    const float3 cent = descPrim.pos[0] + dir* (descPrim.rad[0] - rad);// (descPrim.pos[1] + descPrim.pos[0] + dir*descPrim.rad[0] - dir*descPrim.rad[1]) / 2;

    float radM = rad / 2;
    float3 c1 = cent + dir*radM;
    float3 c2 = cent - dir*radM;

    const float t1 = length(c1 - origin);
    const float t2 = length(c2 - origin);

    const float3 pos_along_ray1 = origin + direction * t1;
    const float3 pos_along_ray2 = origin + direction * t2;

    //float maxRad = fmaxf(descPrim.rad[0], descPrim.rad[1]);
    //float traced_bound = leng + maxRad;
    float tmax = t1 + radM; //to stop sphere tracing

    float tmin = fmaxf(0.0, t1 - radM);
    float3 params = make_float3(tmin, tmax, 0.0);
    if ((length(c1 - pos_along_ray1) < rad))
    {
        params.z = 1.0;
        params.y = t1 + rad;
        params.x = fmaxf(0.0, t1 - rad);
    }
    if (length(c2 - pos_along_ray2) < rad) {
        params.z = 1.0;
        params.y = fmaxf(params.y, t2 + rad);
        float r2 = fmaxf(0.0, t2 - rad);
        params.x = fminf(params.x, r2);
    }
    return params;
    /*
    float rad = fmax(descPrim.rad[0], descPrim.rad[1]) + blendAdd;
    const float3 cent = (descPrim.pos[0] + descPrim.pos[1]) / 2;
    float3 c1 = descPrim.pos[0];
    float3 c2 = descPrim.pos[1];

    const float t1 = length(c1 - origin);
    const float t2 = length(c2 - origin);

    const float3 pos_along_ray1 = origin + direction * t1;
    const float3 pos_along_ray2 = origin + direction * t2;

    float rad1 = length(c1 - cent) + rad;
    float rad2 = length(c2 - cent) + rad;

    float tmax = t1 + rad1; //to stop sphere tracing

    float tmin = fmaxf(0.0, t1 - rad1);
    float3 params = make_float3(tmin, tmax, 0.0);
    if ((length(c1 - pos_along_ray1) < rad1))
    {
        params.z = 1.0;
        params.y = t1 + rad1 * 2;
        params.x = fmaxf(0.0, t1 - rad1);
    }
    if (length(c2 - pos_along_ray2) < rad2) {
        params.z = 1.0;
        params.y = fmaxf(params.y, t2 + rad2 * 2);
        float r2 = fmaxf(0.0, t2 - rad2);
        params.x = fminf(params.x, r2);
    }
    return params;*/
}

//Main program
RT_PROGRAM void intersection_mol(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;
    float epsilon = 0.001;//delta;
    if (epsilon > sysSceneEpsilon) epsilon = sysSceneEpsilon;
    float eps = 0.001;
    if (eps > sysSceneEpsilon) eps = sysSceneEpsilon;

    /* ------------------
    /* 1) Reading data and accessing current positions for current time
    /---------------------------------------------------------------*/

    // primParamDesc descPrim = getTimeData(primIdx);
    int numFr = 3;
    const int2 ids = Bonds[primIdx];
    const float rad1 = BSRadius[ids.x - 1];
    const float rad2 = BSRadius[ids.y - 1];

    const int type = BSType[ids.x - 1];
    const int type2 = BSType[ids.y - 1];

    //---------------
    int lower = int(floorf(TimeSound));
    int upper = int(ceilf(TimeSound));

    float timeS = TimeSound;
    if (upper > numFrames) upper = numFrames;

    float time = timeS - float(lower);

    //int time = int(floorf(TimeSound)); //integer part
    //int upper=int(time);

    //if (time > numFrames) time = numFrames;

    //for bond interpolation
    float3 pos2 = make_float3(0);
    float3 pos = make_float3(0);

    //for frames
    float3 pos12 = make_float3(0);
    float3 pos11 = make_float3(0);

    float3 pos22 = make_float3(0);
    float3 pos21 = make_float3(0);

    //float3 pos1 = Positions[primIdx + time*PNum]; //getting correct frame

    pos11 = Positions[ids.x - 1 + lower*PNum];
    pos12 = Positions[ids.y - 1 + lower*PNum];

    pos = Positions[ids.x - 1 + lower*PNum];
    pos2 = Positions[ids.y - 1 + lower*PNum];

    if (numFrames > 0) //dynamic
    {
        pos21 = Positions[ids.x - 1 + upper*PNum];
        pos22 = Positions[ids.y - 1 + upper*PNum];

        pos = time*pos21 + (1.0 - time)*pos11; //time interpolation
        pos2 = time*pos22 + (1.0 - time)*pos12; //time interpolation
    }
    primParamDesc descPrim;

    descPrim.type = 2;
    descPrim.pos[0] = pos;
    descPrim.pos[1] = pos2;
    descPrim.rad[0] = rad1;
    descPrim.rad[1] = rad2;
    descPrim.types[0] = type;
    descPrim.types[1] = type2;

    //---------------
    float3 interSectParams = boundIntersection(descPrim, theRay.origin, theRay.direction);

    if (interSectParams.z > 0.0)
    {
        tmin = interSectParams.x;
        tmax = interSectParams.y;
        float totalDistance = tmin;
        // === Raymarching (Sphere Tracing) Procedure ===

        totalDistance = SphereTraceForward(epsilon, tmin, tmax, descPrim);

        // Found potential intersection?
        if (totalDistance < tmax) //we found intersection
        {
            float totalDistance2 = SphereTraceBack(epsilon, totalDistance, tmax, descPrim);
            //------------
            if (totalDistance2 > epsilon)
            { //it is sufficiently large subvolume to ray-cast
                if (rtPotentialIntersection(totalDistance))
                {
                    //compute normal for primitive
                    float3 x = theRay.origin + theRay.direction*totalDistance;

                    float3 cNormal = computeNormal(eps, x, descPrim);

                    //varNormal = normalize(make_float3(dx, dy, dz));

                    //for material

                    infoH.normal = cNormal;
                    infoH.hit_point = theRay.origin + theRay.direction * (totalDistance);
                    infoH.tmin = totalDistance;

                    infoH.desc = descPrim;
                    infoH.maxDist = totalDistance2;
                    rtReportIntersection(MaterialIndex);
                }
            }
        }
    }
}

//--balls and sticks
RT_PROGRAM void boundingbox_bond_mol(int primIdx, float result[6])
{
    const int2 ids = Bonds[primIdx];
    const float rad1 = BSRadius[ids.x - 1];
    const float rad2 = BSRadius[ids.y - 1];

    float3 pos = Positions[ids.x - 1];
    float3 pos2 = Positions[ids.y - 1];

    float3 pos_min = fminf(pos, pos2);
    float3 pos_max = fmaxf(pos, pos2);

    if (numFrames > 0)
    {
        for (int i = 1; i < numFrames; i++)
        {
            pos = Positions[ids.x - 1 + i*PNum];
            pos2 = Positions[ids.y - 1 + i*PNum];

            pos_min = fminf(fminf(pos, pos2), pos_min);
            pos_max = fmaxf(fmaxf(pos, pos2), pos_max);
        }
    }

    float rad = fmaxf(rad1, rad2);
    optix::Aabb* aabb = (optix::Aabb*)result;
    //increase for ao by 5
    aabb->m_min = pos_min - make_float3(rad + blendAdd);
    aabb->m_max = pos_max + make_float3(rad + blendAdd);
}