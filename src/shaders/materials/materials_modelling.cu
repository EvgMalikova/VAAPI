#include "hip/hip_runtime.h"
/*

 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../renderer/per_ray_data.h"
#include "../basic_lights.h"
 //#include "per_ray_data.h"
#include "../attributeInfo.h"
#include "../sdfPrimPrograms.h"
#include "transferFunction.h"
using namespace optix;

rtDeclareVariable(attributeInfo, info, attribute info, );

rtDeclareVariable(attributeInfo2, infoH, attribute infoH, );
// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

//type of rendering
rtDeclareVariable(int, Type, , );
rtDeclareVariable(int, HeteroObjType, , );

//array of lights
rtBuffer<BasicLight> lights;

//typedef rtCallableProgramX<float(float3, primParamDesc)> callM;
//rtDeclareVariable(callM, evalF, , );

typedef rtCallableProgramId<float(float3, primParamDesc)> callM;
rtDeclareVariable(callM, evalF, , );
/*
For heterogeneous objects
*/

//TODO: set as buffer
inline __device__ float3 transfer_function(int t, float d)
{
    // return TFBuffer[t];
    switch (t)
    {
    case 1: //H
        return make_float3(1, 1, 1);
        break;
    case 2: //C
        return make_float3(0.4);
        break;
    case 3: //N
        return make_float3(0, 0, 0.5);
        break;
    case 4: //S
        return make_float3(1, 1, 0);
        break;
    case 5: //O
        return make_float3(1, 0, 0);
        break;
    case 6: //P
        return make_float3(1, 0.5, 0);
        break;
    case 7: //Si
        return make_float3(1, 1, 1);//255/236, 255/246, 0);
        break;
    }
    return make_float3(0, 0, 0);
}

RT_CALLABLE_PROGRAM  float eval0(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    float3 pos1 = descPrim.pos[0];
    float rad1 = descPrim.rad[0];

    f = sdfPrim0(x - pos1, rad1);//length(x - pos1) - rad1; //sdfPrim1(x, pos1, pos2, rad1, rad2);
    return f;
}

RT_CALLABLE_PROGRAM  float eval1(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    float3 pos1 = descPrim.pos[0];
    float rad1 = descPrim.rad[0];
    float3 pos2 = descPrim.pos[1];
    float rad2 = descPrim.rad[1];

    f = sdfPrim1(x, pos1, pos2, rad1, rad2);//length(x - pos1) - rad1; //sdfPrim1(x, pos1, pos2, rad1, rad2);
    return f;
}

RT_CALLABLE_PROGRAM  float eval4(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    float3 pos1 = descPrim.pos[0];
    float3 pos2 = descPrim.pos[1];
    float3 pos3 = descPrim.pos[2];
    float3 pos4 = descPrim.pos[3];

    f = sdfPrim4(x, pos1, pos2, pos3, pos4);//length(x - pos1) - rad1; //sdfPrim1(x, pos1, pos2, rad1, rad2);
    return f;
}

RT_CALLABLE_PROGRAM  float evalDefault(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;

    f = sdfPrimDefault(x, descPrim);
    return f;
}

RT_CALLABLE_PROGRAM float3 GetColorBlend(float3 x, primParamDesc desc)
{
    float3 pos = desc.pos[0];
    float3 col1 = transfer_function(desc.types[0], 1.0);

    if (desc.type > 0) {
        float3 pos2 = desc.pos[1];
        float3 col2 = transfer_function(desc.types[1], 1.0);
        float r1 = desc.rad[0];
        float r2 = desc.rad[1];

        float d1 = length(x - pos) - r1 / 2;
        float d2 = length(x - pos2) - r2 / 2;
        float d = length(pos - pos2);
        if (d1 <= 0) return col1;
        if (d2 <= 0) return col2;
        float3 col = d1 / d*col2 + d2 / d*col1;

        return col;
    }
    else return col1;
}

inline __device__  float3  GetColor(float3 x)
{
    if (infoH.desc.type < 4) {
        float3 pos = infoH.desc.pos[0];
        float3 col1 = transfer_function(infoH.desc.types[0], 1.0);

        if (infoH.desc.type > 0) {
            float3 pos2 = infoH.desc.pos[1];
            float3 col2 = transfer_function(infoH.desc.types[1], 1.0);
            float r1 = infoH.desc.rad[0];
            float r2 = infoH.desc.rad[1];

            float d1 = length(x - pos) - r1 / 2;
            float d2 = length(x - pos2) - r2 / 2;
            float d = length(pos - pos2);
            if (d1 <= 0) return col1;
            if (d2 <= 0) return col2;
            float3 col = d1 / d*col2 + d2 / d*col1;

            return col;
        }
        else return col1;
    }
    else { //tetra
        float r = infoH.desc.rad[0];
        float4 col = translucent_grays(abs(0.01 / r) / 10, r / 8, 1); //abs(f / 30), 1);////translucent_grays2(color_id, abs(f / 100));
        return make_float3(col.x, col.y, col.z);
    }
}

__device__ void render_HeteroVolume(float3 normal, float3 hit_point)
{
    float Ka = 0.5;
    float Kd = 0.9;
    float Ks = 0.9;

    float4 col = make_float4(0, 0, 0, 1);// translucent_grays(0.5, 0.1, 0);

    float tstep = 0.1;
    float3 pos = hit_point;// eyeRay.o + eyeRay.d*tnear;
    float3 step = theRay.direction*tstep;

    float4 sum = thePrd.result;// make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here
    float trp = 0.05;
    float trp0 = 0.1;

    float s1 = evalF(pos, infoH.desc);
    if (abs(s1) > tstep)
        step = theRay.direction*abs(s1);

    float i = 0.0;
    float max = thePrd.maxDist;// *2 + 0.4; //bounding box size
    float4 sumcol = make_float4(0.0);
    float tracedDist = 0;

    //float4 col1 = translucent_grays(0.5, 0.01, 0);
    int VolInt = 1;

    while (i < max) //s2 < 0.01)
    {
        if (s1 < tstep / 2)
        {
            // if (abs(s1) > tstep) //sum transparency
            {
                //is used to highlight isosurfaces
                //or create a more shell like effect
                //trp = trp0 + abs(s1) / 10;

                //----------------------
                //COLOR COMPUTATION
                //trp = trp0;
                float3 colorm2 = GetColor(pos);
                VolInt = 1;
                //------------------

                float3 color = Ka *  colorm2;// ambient_light_color;
                float3 color2 = Ka *  colorm2;                                          //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;
                if (i < 0.02)//smooth boundary
                {
                    for (int i = 0; i < 2; ++i)
                    {
                        BasicLight light = lights[i];
                        float3 L = optix::normalize(light.pos - thePrd.last_hit_point);
                        float nDl = optix::dot(thePrd.normal, L);

                        //if (nDl > 0)
                        //    color += Kd * nDl * light.color; // make_float3(1.0);//

                        float phong_exp = 0.2;
                        if (nDl > 0) {
                            color += Kd * nDl * light.color;

                            /*  optix::float3 H = optix::normalize(L - theRay.direction);
                              float nDh = optix::dot(normal, H);
                              if (nDh > 0)
                                  color += Ks * light.color * pow(nDh, phong_exp);
                          */
                        }
                    }
                    //---------------

                    col = make_float4(color.x, color.y, color.z, trp*Ka);
                }
                else
                    col = make_float4(color2.x, color2.y, color2.z, trp*Ka);

                if (VolInt > 0) {
                    //Beer–Lambert law
                    float F = exp(-trp*abs(s1) * 200);
                    col = col*(1.0 - F);
                    sum = sum + col*(1.0f - sum.w);
                }
                else //conventional integration
                {
                    /*col.w = trp * 1.9;
                    col.x *= col.w;
                    col.y *= col.w;
                    col.z *= col.w;

                    float t = sum.w;
                    // "over" operator for front-to-back blending
                    sum = sum + col*(1.0f - t);
                    sum.w = t*(1.0 - trp);*/

                    float F = 1.0 - trp*1.9;
                    col = col*(1.0 - F);
                    sum = sum + col*(1.0f - sum.w);
                }
                // tracedDist += abs(s1);
            }
        }

        //s1 = s2;

        if (VolInt > 0) {
            if (abs(s1) > tstep) {
                step = theRay.direction*abs(s1);
                i += abs(s1);
            }
            else
            {
                i += tstep;
                step = theRay.direction*tstep;
            }
        }
        else { //volume sampling
            if (s1 > tstep) //employ space skipping
            {
                step = theRay.direction*abs(s1);
                i += abs(s1);
            }
            else {
                i += tstep;
                step = theRay.direction*tstep;
            }
        }

        pos += step;
        if (sum.w >= 1.0) {
            i = max + 1;
        }
        else
            s1 = evalF(pos, infoH.desc);
    }

    thePrd.result = sum;
}
/* Compiles various types of programs depending on primType*/
RT_PROGRAM void volume_hetero_close()
{
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, infoH.normal));
    optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;
    thePrd.maxDist = infoH.maxDist;
    thePrd.normal = normal;
    thePrd.last_hit_point = hit_point;

    float3 col = GetColor(hit_point);
    //thePrd.result += make_float4(col.x, col.y, col.z, 0.1);
    render_HeteroVolume(normal, hit_point); //for defalt rendering primitive

    thePrd.renderType = 0;
    thePrd.depth++;

    if (thePrd.totalDist > 20.0) //TODO set max trace depth as parameter
        thePrd.depth = 10;

    thePrd.totalDist += length(theRay.origin - hit_point);
    if (thePrd.result.w < 0.5) {
        if (thePrd.depth < 5) //TODO set max trace depth as parameter
        {
            //further ray
            optix::Ray ray = optix::make_Ray(hit_point + theRay.direction *(abs(infoH.maxDist) + 0.5), theRay.direction, 0, 0.0f, RT_DEFAULT_MAX);
            rtTrace(sysTopObject, ray, thePrd);
        }
    }
}

RT_PROGRAM void volume_hetero_any()
{
    //No volume integration
    //just saving preliminary data to array
    thePrd.renderType = 3; //for postprocessing
    if (thePrd.cur_prim < MAX_PRIM_ALONG_RAY)
    { //push intersections
        thePrd.cur_prim++;
        cellPrimDesc cell;
        cell.intersectionDist = infoH.tmin;
        cell.type = 3; //tracing with primitives and compute bounds
        cell.normal = infoH.normal;
        int typeC = int(infoH.desc.rad[0]);
        float3 col = GetColor(infoH.hit_point);

        //thePrd.result += make_float4(col.x, col.y, col.z, 0.1);
        cell.color = make_float4(col.x, col.y, col.z, 0.01);
        cell.maxDist = infoH.maxDist;

        thePrd.cellPrimitives[thePrd.cur_prim - 1] = cell;
        thePrd.prims[thePrd.cur_prim - 1] = infoH.desc;
        rtIgnoreIntersection();
    }
    else {
        rtTerminateRay();
    }
}