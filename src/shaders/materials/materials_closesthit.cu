#include "hip/hip_runtime.h"
/*
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../renderer/per_ray_data.h"
#include "../basic_lights.h"
#include "../attributeInfo.h"
#include "../renderer/rt_function.h"
#include "../renderer/random_number_generators.h"
//#include "per_ray_data.h"

//rtBuffer<float3> TFBuffer;

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float, MultiscaleParam, , );
// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// Attributes.

//for sdf
rtDeclareVariable(float3, ambient_light_color, , );
rtDeclareVariable(attributeInfo, info, attribute info, );
rtDeclareVariable(int, useScalar, , );

//for triangulated objects
//TODO:probably now it is broken
rtDeclareVariable(float3, varGeoNormal, attribute GeoNormal, );

//array of lights
rtBuffer<BasicLight> lights;

typedef rtCallableProgramId<float3(int)> callTF;
rtDeclareVariable(callTF, tFunction, , );

typedef rtCallableProgramId<float(float3, float3)> callTBackSDF;
rtDeclareVariable(callTBackSDF, sdfPrimBack, , );
typedef rtCallableProgramId<float(float3, float3, float, float)> callT;
rtDeclareVariable(callT, sdfPrim2, , );

typedef rtCallableProgramId<float(float3, float3, float3, float, float)> callT4;
rtDeclareVariable(callT4, sdfPrim4, , );

rtDeclareVariable(float, sysSceneEpsilon, , );
//rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

// This closest hit program only uses the geometric normal and the shading normal attributes.
// OptiX will remove all code from the intersection programs for unused attributes automatically.

// Note that the matching between attribute outputs from the intersection program and
// the inputs in the closesthit and anyhit programs is done with the type (here float3) and
// the user defined attribute semantic (e.g. here NORMAL).
// The actual variable name doesn't need to match but it's recommended for clarity.

// Helper functions for sampling a cosine weighted hemisphere distrobution as needed for the Lambert shading model.

//TODO: transmit as callable functions

RT_FUNCTION void render_Surface3(float3 normal, float3 hit_point, float4 color_inp, float max, float3 a, float3 b, float r1, float r2)
{
    float Ka = 0.2;
    float Kd = 0.5;
    float Ks = 0.2;
    float4 col = make_float4(0, 0, 0, 1);// translucent_grays(0.5, 0.1, 0);

    float tstep = 0.005;
    float3 pos = hit_point;// eyeRay.o + eyeRay.d*tnear;
    float3 step = theRay.direction*tstep;

    float4 sum = thePrd.result;// make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here

    if (sum.w >= 1.0)
    {
        return;
    }

    float trp = 0.1;
    float s1 = sdfPrim4(pos, a, b, r1, r2);
    if (abs(s1) > tstep)
        step = theRay.direction*abs(s1);

    pos += step;
    float s2 = sdfPrim4(pos, a, b, r1, r2);
    float i = 0;
    //bounding box size
    float4 sumcol = make_float4(0.0);
    float tracedDist = 0;

    //float4 col1 = translucent_grays(0.5, 0.01, 0);

    float3 color = Ka *  make_float3(color_inp.x, color_inp.y, color_inp.z);// ambient_light_color;
    float3 color2 = color;                                          //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    for (int i = 0; i < lights.size(); ++i)
    {
        BasicLight light = lights[i];
        float3 L = optix::normalize(light.pos - thePrd.last_hit_point);
        float nDl = optix::dot(thePrd.normal, L);

        //if (nDl > 0)
        //    color += Kd * nDl * light.color; // make_float3(1.0);//

        float phong_exp = 0.1;
        if (nDl > 0) {
            color += Kd * nDl * light.color;
        }
    }

    while (i < max) //s2 < 0.01)
    {
        if (s2 < 0.0) {
            if (abs(s1) > tstep) //sum transparency
            {
                //col *= Ka;
                //col.w = 0.5; //s1 is very small
                // float3 color = Ka *  make_float3(0, 1, 0);// ambient_light_color;

                //initial blend
                if (i == 0)
                    // if (ll < 5) {
                    trp = 0.001 + abs(s1) / 10;
                //     ll++;
                // }
                // else {
                //     trp = 0.001;
                //     ll = 0;
                // }
                if (i == 0)
                    col = make_float4(color.x, color.y, color.z, trp*Ka);

                else
                    col = make_float4(color2.x, color2.y, color2.z, trp*Ka);
                //else
               // col = make_float4(color2.x, color2.y, color2.z, trp*Ka);
                //initial blend
                // float4 col = make_float4(color);
                /*  float f = exp(-trp*abs(s1) * 0.01);
                float T = sum.w;
                sum = sum + sum.w*col*(1.0f - f);
                sum.w = T*f;*/
                /*col.w = trp*abs(s1) * 10;
                col.x *= col.w;
                col.y *= col.w;
                col.z *= col.w;*/

                // "over" operator for front-to-back blending

                //Beer�Lambert law
                float F = exp(-trp*abs(s1) * 200);
                col = col*(1.0 - F);
                sum = sum + col*(1.0f - sum.w);

                tracedDist += abs(s1);
            }
            else {
                //Beer�Lambert law
                //col = make_float4(color);
                // col = col*exp(-trp*abs(s1) / 100);
                tracedDist = 0;
            }
        }

        s1 = s2;
        if (abs(s1) > tstep) {
            step = theRay.direction*abs(s1);
            i += abs(s1);
        }
        else
        {
            step = theRay.direction*abs(tstep);
            i += abs(tstep);
        }
        pos += step;
        if (sum.w >= 1.0) i = max + 1;
        else
            s2 = sdfPrim4(pos, a, b, r1, r2); //interpolateSDF(time, pos, texSDF, texSDF_F);

                                                                //s2 = sdfPrimBack(pos, make_float3(1.1)); //interpolateSDF(time, pos, texSDF, texSDF_F);
                                                                // if (abs(s1) <= eps)
    }
    thePrd.result = sum;
}

RT_FUNCTION void alignVector(float3 const& axis, float3& w)
{
    // Align w with axis.
    const float s = copysign(1.0f, axis.z);
    w.z *= s;
    const float3 h = make_float3(axis.x, axis.y, axis.z + s);
    const float  k = optix::dot(w, h) / (1.0f + fabsf(axis.z));
    w = k * h - w;
}

RT_FUNCTION void unitSquareToCosineHemisphere(const float2 sample, float3 const& axis, float3& w, float& pdf)
{
    // Choose a point on the local hemisphere coordinates about +z.
    const float theta = 2.0f * M_PIf * sample.x;
    const float r = sqrtf(sample.y);
    w.x = r * cosf(theta);
    w.y = r * sinf(theta);
    w.z = 1.0f - w.x * w.x - w.y * w.y;
    w.z = (0.0f < w.z) ? sqrtf(w.z) : 0.0f;

    pdf = w.z * M_1_PIf;

    // Align with axis.
    alignVector(axis, w);

    //mix with normal for self-occlusion
    const float rad = 1.0 - 1.0 / 32;
    w = optix::normalize(axis + w*rad);
}

/*vaBasicMaterial program*/
RT_PROGRAM void closesthit_sdf()
{
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));

    float Ka = 0.3;
    float Kd = 0.2;
    float Ks = 0.2;
    //todo implement primInfo type to color and a switch for mapping to color
    float3 col = Ka *  ambient_light_color;
    float4 color = make_float4(col.x, col.y, col.z, 1.0);

    if (useScalar == 1) { //otherwise primInfo is not assigned a value
        if (info.type == 0)
        {
            color = info.useScalar;
        }
        else {
            int t = info.type;// __float_as_int(info.primInfo.y);
            col = tFunction(t);// transfer_function(t);
            color = make_float4(col.x, col.y, col.z, 1.0);
        }
        color *= Ka;
    }/**/
    //if(useScalar==2)
    //{
    // color = info.color;// transfer_function(t);
   //  color *= Ka;
    //}

    optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    for (int i = 0; i < lights.size(); ++i)
    {
        BasicLight light = lights[i];
        float3 L = optix::normalize(light.pos - hit_point);// varHit);
        float nDl = optix::dot(normal, L);

        //if (nDl > 0)
        //    color += Kd * nDl * light.color; // make_float3(1.0);//
        float4 lcol = make_float4(light.color.x, light.color.y, light.color.z, 1.0);
        float phong_exp = 0.5 *(1 - MultiscaleParam);
        if (nDl > 0) {
            color += Kd * nDl * lcol;
        }
        /* float3 H = optix::normalize(L - thePrd.wi);
         float nDh = optix::dot(normal, H);
         if (nDh > 0)
             color += Ks*(1 - MultiscaleParam) * light.color * pow(nDh, phong_exp);
     */
    }

    //blending of color

    thePrd.result = thePrd.result + color*(1.0f - thePrd.result.w);
    //thePrd.radiance = make_float3(color);
}

RT_PROGRAM void volumehit_sdf2()
{
    if (thePrd.cur_prim < MAX_PRIM_ALONG_RAY || thePrd.result.w >= 1.0)
    {
        //compute normal and color
        float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));
        float Ka = 0.2;

        //todo implement primInfo type to color and a switch for mapping to color
        float3 col = Ka *  ambient_light_color;
        float4 color = make_float4(col.x, col.y, col.z, 1.0);

        if (useScalar == 1) { //otherwise primInfo is not assigned a value
            if (info.type == 0)
            {
                color = info.useScalar;
            }
            else {
                int t = info.type;// __float_as_int(info.primInfo.y);
                col = tFunction(t);// transfer_function(t);
                color = make_float4(col.x, col.y, col.z, 1.0);
            }
            color *= Ka;
        }

        //todo:
        //avarage distance
        //avarage color for blending

        optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

        render_Surface3(normal, hit_point, color, info.maxDist, info.pos[0], info.pos[1], info.rad[0], info.rad[1]);
        // thePrd.result += make_float4(0.1, 0, 0, 0.1);

       //thePrd.renderType = 1;

       //if (thePrd.cur_prim < MAX_PRIM_ALONG_RAY)
        { //push intersections
            thePrd.cur_prim++;
            // thePrd.primitives[thePrd.cur_prim - 1] = make_float2(info.type, theIntersectionDistance);

             //fill second array
             //float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));
             //optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;
            cellPrimDesc cell;
            cell.intersectionDist = theIntersectionDistance;
            cell.type = 1; //not main
            cell.normal = normal;
            // cell.internalTypes = info.types;
            cell.maxDist = info.maxDist;
            //fake it as far
            /*cell.center = info.pos[0];// hit_point + theRay.direction*0.5;
            cell.center2 = info.pos[1];
            cell.rad1 = info.rad[0];
            cell.rad2 = info.rad[1];*/
            cell.color = make_float4(info.useScalar.x, info.useScalar.y, info.useScalar.z, 0.01);

            //cell.segLength=info.
            //cell.bmin= info.
            thePrd.cellPrimitives[thePrd.cur_prim - 1] = cell;
        }
        float skip = fmaxf(info.rad[0], info.rad[1]) * 2;
        skip = fmaxf(skip, info.maxDist) * 2;

        optix::Ray ray = optix::make_Ray(hit_point + theRay.direction * (skip + 0.01), theRay.direction, 0, sysSceneEpsilon, 10);

        // Start tracing ray from the camera and further
        rtTrace(sysTopObject, ray, thePrd);
    }
}

/*Multiscale molecule material
Initial assumptions:
//BVH doesn't overlap and we can perform volume
ray-casting without sorting

//Material doen't change significantly on ray-segment
//Thus it can be averaged and interpolated between certain
//ray-in  and ray-out values and
//regions of homogeneous material can be transmitted
*/
RT_PROGRAM void volumehit_sdf()
{
    //if (thePrd.cur_prim < 3)
    {
        float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));

        float Ka = 0.3;// + MultiscaleParam / 2;
        float Kd = 0.2;
        float Ks = 0.2;
        //todo implement primInfo type to color and a switch for mapping to color
        float3 col = Ka *  ambient_light_color;
        float4 color = make_float4(col.x, col.y, col.z, 1.0);

        if (useScalar == 1) { //otherwise primInfo is not assigned a value
            if (info.type == 0)
            {
                color = info.useScalar;
            }
            else {
                int t = info.type;// __float_as_int(info.primInfo.y);
                col = tFunction(t);// transfer_function(t);
                color = make_float4(col.x, col.y, col.z, 1.0);
            }
            color *= Ka;
        }/**/
         //if(useScalar==2)
         //{
         // color = info.color;// transfer_function(t);
         //  color *= Ka;
         //}

        optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

        for (int i = 0; i < lights.size(); ++i)
        {
            BasicLight light = lights[i];
            float3 L = optix::normalize(light.pos - hit_point);// varHit);
            float nDl = optix::dot(normal, L);

            //if (nDl > 0)
            //    color += Kd * nDl * light.color; // make_float3(1.0);//
            float4 lcol = make_float4(light.color.x, light.color.y, light.color.z, 1.0);
            float phong_exp = 0.5;//*(1 - MultiscaleParam);
            if (nDl > 0) {
                color += Kd * nDl * lcol;
            }
            /* float3 H = optix::normalize(L - thePrd.wi);
             float nDh = optix::dot(normal, H);
             if (nDh > 0)
                 color += Ks*(1 - MultiscaleParam) * lcol * pow(nDh, phong_exp);
              */
        }
        if (thePrd.cur_prim < 1) //first element
        {
            thePrd.radiance += make_float3(color.x*color.w, color.y*color.w, color.z*color.w);
        }
        thePrd.cur_prim++;
        //color.x *= color.w;
        //color.y *= color.w;
        //color.z *= color.w;

        thePrd.result += color *(1 - thePrd.result.w);
        //thePrd.result.w /= thePrd.cur_prim;
        // Create ray
        //rtIgnoreIntersection();

        //we consider that molecules don't intersect within bounding spheres
        //that type of data is just not possible
        optix::Ray ray = optix::make_Ray(hit_point + theRay.direction * info.maxDist, theRay.direction, 0, 0.0f, RT_DEFAULT_MAX);

        //remove from here
        thePrd.renderType = 1;

        // Start tracing ray from the camera and further
        rtTrace(sysTopObject, ray, thePrd);
    }
}
//
RT_PROGRAM void closesthit_sdf2()
{
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));
    // Visualize the resulting world space normal on the surface we're looking on.
    // Transform the normal components from [-1.0f, 1.0f] to the range [0.0f, 1.0f] to get colors for negative values.
    thePrd.radiance *= 2.2f;

    //TODO: call two callable programs
    //1 - for optical model
    //2 - for auditory model
}

__device__ void render_Surface2(float3 normal, float3 hit_point, float4 color)
{
    float Ka = 0.5;
    float Kd = 0.5;
    float Ks = 0.2;
    float tstep = 0.01;
    //scart tracing to the initial
    float3 pos = thePrd.last_hit_point;
    float3 step = theRay.direction*tstep;

    float max = optix::length(hit_point - thePrd.last_hit_point);
    float4 sum = thePrd.result;// make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here
    float totalD = max;
    //TODO still need to implement
    //start tracing the bounds itself
    float s1 = sdfPrimBack(pos, make_float3(thePrd.maxDist));
    pos += step;
    float s2 = sdfPrimBack(pos, make_float3(thePrd.maxDist));
    float i = 0;
    float trp = 0.01;
    float4 col;

    float3 color2 = Ka *  make_float3(0, 1, 0);// ambient_light_color;

    col = make_float4(color2);
    col += make_float4(0.5, 0, 0, 0);
    col.w = trp;
    col.x *= col.w;
    col.y *= col.w;
    col.z *= col.w;                                      //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    while (i < max) //s2 < 0.01)
    {
        if (s2 < 0.01) {
            //col *= Ka;
            //col.w = 0.5; //s1 is very small
            //float3 color = Ka *  make_float3(col);// ambient_light_color;

            //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

            //initial blend

            // "over" operator for front-to-back blending
            sum = sum + col*(1.0f - sum.w);

            //save the last output point
            thePrd.last_hit_point = pos;
        }
        i += tstep;
        s1 = s2;
        pos += step;

        if (sum.w >= 1.0) i = max + 1;
        else
            s2 = sdfPrimBack(pos, make_float3(thePrd.maxDist));
        // if (abs(s1) <= eps)
    }

    trp = 0.06;
    //----------------------
    pos = hit_point;// eyeRay.o + eyeRay.d*tnear;

    s1 = sdfPrim2(pos - info.pos[0], pos - info.pos[1], info.rad[0], info.rad[1]);
    float s3 = sdfPrimBack(pos, make_float3(8.1));
    pos += step;
    s2 = sdfPrim2(pos - info.pos[0], pos - info.pos[1], info.rad[0], info.rad[1]);
    i = 0;

    max = fmax(info.rad[0], info.rad[1]);
    max = max * 2 + 0.4; //bounding box size
    totalD += max;
    while (i < max) //s2 < 0.01)
    {
        if (s2 < 0.01) {
            if (s3 < 0.0001)
            {
                //col *= Ka;
                //col.w = 0.5; //s1 is very small
                //float3 color = Ka *  make_float3(col);// ambient_light_color;

                                                      //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

                //initial blend
                col = color;
                col.w = trp;
                col.x *= col.w;
                col.y *= col.w;
                col.z *= col.w;
                // "over" operator for front-to-back blending
                sum = sum + col*(1.0f - sum.w);

                //save the last output point
                thePrd.last_hit_point = pos;
            }
        }
        i += tstep;
        s1 = s2;
        s3 = sdfPrimBack(pos, make_float3(8.1));
        pos += step;

        if (sum.w >= 1.0) i = max + 1;
        else
            s2 = sdfPrim2(pos - info.pos[0], pos - info.pos[1], info.rad[0], info.rad[1]);
        // if (abs(s1) <= eps)
    }
    //or it should be
    // thePrd.radiance = make_float3(sum);
    //thePrd.radiance += make_float3(sum);

    thePrd.maxDist -= totalD;
    thePrd.result = sum;
}

//For molecule with bonds
RT_PROGRAM void anyhit_sdf_complex()
{
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));
    optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    if (thePrd.cur_prim < MAX_PRIM_ALONG_RAY)
    {
        thePrd.cur_prim++;
        //float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));

           //optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;
        cellPrimDesc cell;
        cell.intersectionDist = theIntersectionDistance;
        cell.type = 1; //not main
        cell.normal = normal;
        //        cell.internalTypes = info.types;
        cell.maxDist = info.maxDist;
        //fake it as far
       /* cell.center = info.pos[0];// hit_point + theRay.direction*0.5;
        cell.center2 = info.pos[1];
        cell.rad1 = info.rad[0];
        cell.rad2 = info.rad[1];*/
        cell.color = make_float4(info.useScalar.x, info.useScalar.y, info.useScalar.z, 0.01);

        //cell.segLength=info.
        //cell.bmin= info.
        thePrd.cellPrimitives[thePrd.cur_prim - 1] = cell;

        if (thePrd.renderType == 0)
            thePrd.renderType = 1;

        rtIgnoreIntersection();
    }

    else { //we have traced all primitives
        rtTerminateRay();
    }
}

RT_PROGRAM void anyhit_sdf_complex_old()
{
    if (thePrd.maxDist > 0) //some error here
    {
        if (thePrd.renderType > 0)
        {
            float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));

            float Ka = 0.3;// + MultiscaleParam / 2;
            float Kd = 0.2;
            float Ks = 0.2;
            //todo implement primInfo type to color and a switch for mapping to color
            float3 col = Ka *  ambient_light_color;
            float4 color = make_float4(col.x, col.y, col.z, 1.0);

            if (useScalar == 1) { //otherwise primInfo is not assigned a value
                if (info.type == 0)
                {
                    color = info.useScalar;
                }
                else {
                    int t = info.type;// __float_as_int(info.primInfo.y);
                    col = tFunction(t);// transfer_function(t);
                    color = make_float4(col.x, col.y, col.z, 1.0);
                }
                color *= Ka;
            }

            optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

            for (int i = 0; i < lights.size(); ++i)
            {
                BasicLight light = lights[i];
                float3 L = optix::normalize(light.pos - hit_point);// varHit);
                float nDl = optix::dot(normal, L);

                //if (nDl > 0)
                //    color += Kd * nDl * light.color; // make_float3(1.0);//
                float4 lcol = make_float4(light.color.x, light.color.y, light.color.z, 1.0);
                float phong_exp = 0.5 *(1 - MultiscaleParam);
                if (nDl > 0) {
                    color += Kd * nDl * lcol;
                }
            }

            //perform rendering
           // thePrd.result += make_float4(1, 0, 0, 0.5);

            thePrd.renderType = 1;
            render_Surface2(normal, hit_point, color);
            //thePrd.radiance += make_float3(color);
        }

        rtIgnoreIntersection();
    }
    else rtTerminateRay();
}