#include "hip/hip_runtime.h"
#include <optix_math.h>


rtDeclareVariable(int, numTexDefined, , );
rtDeclareVariable(float , TimeSound, , );
//for SDF




//array of lights
rtBuffer<float3> TFBuffer;



RT_CALLABLE_PROGRAM float3 transfer_function(int t)
{
    return TFBuffer[t];
}

