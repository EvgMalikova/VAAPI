#include "hip/hip_runtime.h"
/*

 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../renderer/per_ray_data.h"
#include "../basic_lights.h"
 //#include "per_ray_data.h"
#include "../attributeInfo.h"
#include "../sdfPrimPrograms.h"
#include "transferFunction.h"
using namespace optix;

rtDeclareVariable(attributeInfo2, infoH, attribute infoH, );
// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

rtDeclareVariable(float, MultiscaleParam, , );
rtDeclareVariable(float, TimeSound, , );
// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

//type of rendering
rtDeclareVariable(int, Type, , );
rtDeclareVariable(int, HeteroObjType, , );

//array of lights
rtBuffer<BasicLight> lights;

rtDeclareVariable(float3, sysCameraPosition, , );

typedef rtCallableProgramId<float(float3, primParamDesc)> callM;
rtDeclareVariable(callM, evalF, , );
/*
For heterogeneous objects
*/

//TODO: set as buffer
inline __device__ float3 transfer_function(int t, float d)
{
    // return TFBuffer[t];
    switch (t)
    {
    case 1: //H
        return make_float3(1, 1, 1);
        break;
    case 2: //C
        return make_float3(0.4);
        break;
    case 3: //N
        return make_float3(0, 0, 0.5);
        break;
    case 4: //S
        return make_float3(1, 1, 0);
        break;
    case 5: //O
        return make_float3(1, 0, 0);
        break;
    case 6: //P
        return make_float3(1, 0.5, 0);
        break;
    case 7: //Si
        return make_float3(1, 1, 1);//255/236, 255/246, 0);
        break;
    }
    return make_float3(0, 0, 0);
}

RT_CALLABLE_PROGRAM  float eval3(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    float3 pos1 = descPrim.pos[0];
    float3 pos2 = descPrim.pos[1];
    float3 pos3 = descPrim.pos[2];

    float rad1 = descPrim.rad[0];
    float rad2 = descPrim.rad[1];
    float rad3 = descPrim.rad[2];

    f = sdfPrim3(x, pos1, pos2, pos3, rad1, rad2, rad3);//length(x - pos1) - rad1; //sdfPrim1(x, pos1, pos2, rad1, rad2);
    return f;
}

inline __device__ float4  blendColor(float dMd, float3 x, float3 dir, float3 pos, float3 pos2, float3 pos3, float r1, float r2, float r3, float3 col1, float3 col2, float3 col3)
{
    float dNorm = 10;

    float3 vib_color[3];
    vib_color[0] = make_float3(0, 1.0, 0); //green
    vib_color[1] = make_float3(0, 0.0, 1.0); //blue
    vib_color[2] = make_float3(1, 1, 0); //yellow

    float3 wt_col = make_float3(0);
    dNorm = length(pos - pos2) + length(pos3 - pos2) + length(pos - pos3);
    dNorm /= 3;

    float d1 = (length(x - pos) - r1*(1 + dMd)) / dNorm;
    float d2 = (length(x - pos2) - r2*(1 + dMd)) / dNorm;
    float d3 = (length(x - pos3) - r3*(1 + dMd)) / dNorm;

    d1 = optix::clamp(d1, 0.01, 1.0);
    d2 = optix::clamp(d2, 0.01, 1.0);
    d3 = optix::clamp(d3, 0.01, 1.0);

    float3 color = make_float3(0);
    float tr = abs(dMd - 0.1);
    color += (1 - d1)*col1;
    color += (1 - d2)*col2;
    color += (1 - d3)*col3;
    //color += dMd*make_float3(1);
    //color*=tr;

    //dMd=1 - rep1;
    //dMd=0 -rep2
    //dMd*10

    d1 = (length(x - pos) - r1);
    d2 = (length(x - pos2) - r2);
    d3 = (length(x - pos3) - r3);

    //  d1 = optix::clamp(d1, -0.9, 1.0);
    //  d2 = optix::clamp(d2, -0.9, 1.0);
    //  d3 = optix::clamp(d3, -0.9, 1.0);
    float3 col = make_float3(0);
    if (d1 < 0.1)
        col += abs(d1 / r1)*col1;
    if (d2 < 0.1)
        col += abs(d2 / r2)*col2;
    if (d3 < 0.1)
        col += abs(d3 / r3)*col3;

    //------------
    //interpolate between two models
    float3 coll = make_float3(0);
    d1 = optix::clamp(d1, -0.9, 0.0);
    d2 = optix::clamp(d2, -0.9, 0.0);
    d3 = optix::clamp(d3, -0.9, 0.0);
    coll += abs(1 - d1)*col1*abs(d1);
    coll += abs(1 - d2)*col2*abs(d2);
    coll += abs(1 - d3)*col3*abs(d3);
    //coll ;
    col = dMd*coll + (1 - dMd)*col;

    //col+=dMd*make_float3(1);

    //d1 += 0.1;
    //d2 += 0.1;
    //d3 += 0.1;
    tr = abs(1 - min(min(d1, d2), d3)) / 2;//abs(min(min(d1 / r1, d2 / r2), d3 / r3)) / 2 +
                                           //tr /= 3;
                                           //tr=clamp(tr,0,1);
                                           // tr = (3 - d1 - d2 - d3) / 3;
    return make_float4(col.x, col.y, col.z, tr);
}

RT_CALLABLE_PROGRAM float3 GetColorBlend(float3 x, primParamDesc desc)
{
    //---level of detail
    float dist_cam = length(sysCameraPosition - (desc.pos[0] + desc.pos[1] + desc.pos[2]) / 3);
    float d;
    if (dist_cam < 20.0)
    {
        float interp = (dist_cam - 10) / 10.0;
        d = optix::clamp(interp, 0.0, 1.0);
    }
    else d = 1;

    //MultiscaleParam=d;

    float3 col1 = transfer_function(desc.types[0], d);
    float3 col2 = transfer_function(desc.types[1], d);
    float3 col3 = transfer_function(desc.types[2], d);

    //--------level of detail continue
    float rad1 = desc.rad[0];
    float rad2 = desc.rad[1];
    float rad3 = desc.rad[2];
    float4 color = make_float4(0);
    float3 vib_color[3];
    vib_color[0] = make_float3(0, 1.0, 0); //green
    vib_color[1] = make_float3(0, 0.0, 1.0); //blue
    vib_color[2] = make_float3(1.0, 1.0, 0); //yellow

    float3 col21;
    float3 col31;

    //return mod1*d + (1.0 - d)*mod2;
    float r1 = d*rad1 / 2 + (1 - d)*rad1;
    float r2 = d*rad2 / 2 + (1 - d)*rad2;
    float r3 = d*rad3 / 2 + (1 - d)*rad3;

    float3 weights[3];

    if (d <= 1.0)
    {
        float3 pos = desc.pos[0];
        float3 pos2 = desc.pos[1];
        float3 pos3 = desc.pos[2];
        //first vibration vector in molecule
        float3 vib2[3];
        float3 vib3[3];

        vib2[0] = (pos2 - pos) / 3.5 * -cos(TimeSound * 20)*(1 - d);
        vib3[0] = (pos3 - pos) / 3.5 * (cos(TimeSound * 20))*(1 - d);

        vib2[1] = (pos2 - pos) / 3.5 * sin(TimeSound * 20)*(1 - d);
        vib3[1] = (pos3 - pos) / 3.5 * (sin(TimeSound * 20))*(1 - d);

        float3 vib_dir = pos2 - 2 * pos + pos3;

        vib2[2] = vib_dir / 3.5 * sin(TimeSound * 20)*(1 - d);
        vib3[2] = vib_dir / 3.5 * (sin(TimeSound * 20))*(1 - d);

        float d_min = 0;
        int vib_min = 0;

        for (int i = 0; i < 2; i++)
        {
            float3 pp3 = pos3 + vib3[i];
            float3 pp2 = pos2 + vib2[i];

            col31 = d*col3 + (1 - d)*vib_color[i] * 1.5;
            col21 = d*col2 + (1 - d)*vib_color[i] * 1.5;

            color += blendColor(d, x, theRay.direction, pos, pp2, pp3, r1, r2, r3, col1, col21, col31);
        }
        color;///= 3.0;
    }
    else {
        color = blendColor(d, x, theRay.direction, desc.pos[0], desc.pos[1], desc.pos[2], r1, r2, r3, col1, col2, col3);
    }

    return make_float3(color.x, color.y, color.z);
}

inline __device__  float4  GetColor(float3 x)
{
    //---level of detail
    float dist_cam = length(sysCameraPosition - (infoH.desc.pos[0] + infoH.desc.pos[1] + infoH.desc.pos[2]) / 3);
    float d;
    if (dist_cam < 20.0)
    {
        float interp = (dist_cam - 10) / 10.0;
        d = optix::clamp(interp, 0.0, 1.0);
    }
    else d = 1;

    //MultiscaleParam=d;

    float3 col1 = transfer_function(infoH.desc.types[0], d);
    float3 col2 = transfer_function(infoH.desc.types[1], d);
    float3 col3 = transfer_function(infoH.desc.types[2], d);

    //--------level of detail continue
    float rad1 = infoH.desc.rad[0];
    float rad2 = infoH.desc.rad[1];
    float rad3 = infoH.desc.rad[2];
    float4 color = make_float4(0);
    float3 vib_color[3];
    vib_color[0] = make_float3(0, 1.0, 0); //green
    vib_color[1] = make_float3(0, 0.0, 1.0); //blue
    vib_color[2] = make_float3(1.0, 1.0, 0); //yellow

    float3 col21;
    float3 col31;

    //return mod1*d + (1.0 - d)*mod2;
    float r1 = d*rad1 / 2 + (1 - d)*rad1;
    float r2 = d*rad2 / 2 + (1 - d)*rad2;
    float r3 = d*rad3 / 2 + (1 - d)*rad3;

    float3 weights[3];

    if (d <= 1.0)
    {
        float3 pos = infoH.desc.pos[0];
        float3 pos2 = infoH.desc.pos[1];
        float3 pos3 = infoH.desc.pos[2];
        //first vibration vector in molecule
        float3 vib2[3];
        float3 vib3[3];

        vib2[0] = (pos2 - pos) / 3.5 * -cos(TimeSound * 20)*(1 - d);
        vib3[0] = (pos3 - pos) / 3.5 * (cos(TimeSound * 20))*(1 - d);

        vib2[1] = (pos2 - pos) / 3.5 * sin(TimeSound * 20)*(1 - d);
        vib3[1] = (pos3 - pos) / 3.5 * (sin(TimeSound * 20))*(1 - d);

        float3 vib_dir = pos2 - 2 * pos + pos3;

        vib2[2] = vib_dir / 3.5 * sin(TimeSound * 20)*(1 - d);
        vib3[2] = vib_dir / 3.5 * (sin(TimeSound * 20))*(1 - d);

        float d_min = 0;
        int vib_min = 0;

        for (int i = 0; i < 2; i++)
        {
            float3 pp3 = pos3 + vib3[i];
            float3 pp2 = pos2 + vib2[i];

            col31 = d*col3 + (1 - d)*vib_color[i] * 1.5;
            col21 = d*col2 + (1 - d)*vib_color[i] * 1.5;

            color += blendColor(d, x, theRay.direction, pos, pp2, pp3, r1, r2, r3, col1, col21, col31);
        }
        color;///= 3.0;
    }
    else {
        color = blendColor(d, x, theRay.direction, infoH.desc.pos[0], infoH.desc.pos[1], infoH.desc.pos[2], r1, r2, r3, col1, col2, col3);
    }

    return color;
}

__device__ void render_HeteroVolume(float3 normal, float3 hit_point)
{
    float Ka = 0.5;
    float Kd = 0.9;
    float Ks = 0.9;

    float4 col = make_float4(0, 0, 0, 1);// translucent_grays(0.5, 0.1, 0);

    float tstep = 0.1;
    float3 pos = hit_point;// eyeRay.o + eyeRay.d*tnear;
    float3 step = theRay.direction*tstep;

    float4 sum = thePrd.result;// make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here
    float trp = 0.05;
    float trp0 = 0.1;

    float s1 = evalF(pos, infoH.desc);
    if (abs(s1) > tstep)
        step = theRay.direction*abs(s1);

    float i = 0.0;
    float max = thePrd.maxDist;// *2 + 0.4; //bounding box size
    float4 sumcol = make_float4(0.0);
    float tracedDist = 0;

    //float4 col1 = translucent_grays(0.5, 0.01, 0);
    int VolInt = 1;

    while (i < max) //s2 < 0.01)
    {
        if (s1 < tstep / 2)
        {
            // if (abs(s1) > tstep) //sum transparency
            {
                VolInt = 1;
                //------------------
                                       //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;
                col = GetColor(pos);

                //Beer–Lambert law
                float F = exp(-trp*abs(s1) * 200);
                col = col*(1.0 - F);
                sum = sum + col*(1.0f - sum.w);
            }
        }

        if (abs(s1) > tstep) {
            step = theRay.direction*abs(s1);
            i += abs(s1);
        }
        else
        {
            i += tstep;
            step = theRay.direction*tstep;
        }

        pos += step;
        if (sum.w >= 1.0) {
            i = max + 1;
        }
        else
            s1 = evalF(pos, infoH.desc);
    }

    thePrd.result = sum;
}
/* Compiles various types of programs depending on primType*/
RT_PROGRAM void volume_hetero_close()
{
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, infoH.normal));
    optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;
    thePrd.maxDist = infoH.maxDist;
    thePrd.normal = normal;
    thePrd.last_hit_point = hit_point;

    thePrd.result = GetColor(hit_point);
    // render_HeteroVolume(normal, hit_point); //for defalt rendering primitive

    thePrd.renderType = 0;
    thePrd.depth++;

    if (thePrd.totalDist > 20.0) {//TODO set max trace depth as parameter
        thePrd.depth = 10;
    }

    thePrd.totalDist += length(theRay.origin - hit_point);
    /*if (thePrd.result.w < 0.5) {
        if (thePrd.depth < 5) //TODO set max trace depth as parameter
        {
            //further ray
            optix::Ray ray = optix::make_Ray(hit_point + theRay.direction *(abs(infoH.maxDist) + 0.1), theRay.direction, 0, 0.0f, RT_DEFAULT_MAX);
            rtTrace(sysTopObject, ray, thePrd);
        }
    }*/
}

RT_PROGRAM void volume_hetero_any()
{
    //No volume integration
    //just saving preliminary data to array
    thePrd.renderType = 3; //for postprocessing
    if (thePrd.cur_prim < MAX_PRIM_ALONG_RAY)
    { //push intersections
        thePrd.cur_prim++;
        cellPrimDesc cell;
        cell.intersectionDist = infoH.tmin;
        cell.type = 3; //tracing with primitives and compute bounds
        cell.normal = infoH.normal;
        int typeC = int(infoH.desc.rad[0]);
        //float3 col = GetColor(infoH.hit_point);

        //thePrd.result += make_float4(col.x, col.y, col.z, 0.1);
        cell.color = GetColor(infoH.hit_point);
        cell.maxDist = infoH.maxDist;

        thePrd.cellPrimitives[thePrd.cur_prim - 1] = cell;
        thePrd.prims[thePrd.cur_prim - 1] = infoH.desc;
        rtIgnoreIntersection();
    }
    else {
        rtTerminateRay();
    }
}