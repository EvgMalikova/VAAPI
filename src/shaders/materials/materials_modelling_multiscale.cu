#include "hip/hip_runtime.h"
/*

 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../renderer/per_ray_data.h"
#include "../basic_lights.h"
 //#include "per_ray_data.h"
#include "../attributeInfo.h"
#include "../sdfPrimPrograms.h"
#include "transferFunction.h"
using namespace optix;

rtDeclareVariable(attributeInfo2, infoH, attribute infoH, );
// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

rtDeclareVariable(float, MultiscaleParam, , );
rtDeclareVariable(float, TimeSound, , );
// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

//type of rendering
rtDeclareVariable(int, Type, , );
rtDeclareVariable(int, HeteroObjType, , );

//array of lights
rtBuffer<BasicLight> lights;

rtDeclareVariable(float3, sysCameraPosition, , );

typedef rtCallableProgramId<float(float3, primParamDesc)> callM;
rtDeclareVariable(callM, evalF, , );

rtDeclareVariable(float3, sCell1, , );
rtDeclareVariable(float3, sCell2, , );
rtDeclareVariable(float3, sCell3, , );
/*
For heterogeneous objects
*/

//TODO: set as buffer
inline __device__ float3 transfer_function(int t, float d)
{
    // return TFBuffer[t];
    switch (t)
    {
    case 1: //H
        return make_float3(1, 1, 1);
        break;
    case 2: //C
        return make_float3(0.5);
        break;
    case 3: //N
        return make_float3(0, 0, 0.5);
        break;
    case 4: //S
        return make_float3(1, 1, 0);
        break;
    case 5: //O
        return make_float3(1, 0, d);
        break;
    case 6: //P
        return make_float3(1, 0.5, 0);
        break;
    case 7: //Si
        return make_float3(1, 1, 1);//255/236, 255/246, 0);
        break;
    }
    return make_float3(0, 0, 0);
}

RT_CALLABLE_PROGRAM  float eval3(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    float3 pos1 = descPrim.pos[0];
    float3 pos2 = descPrim.pos[1];
    float3 pos3 = descPrim.pos[2];

    float rad1 = descPrim.rad[0];
    float rad2 = descPrim.rad[1];
    float rad3 = descPrim.rad[2];

    f = sdfPrim3(x, pos1, pos2, pos3, rad1, rad2, rad3);//length(x - pos1) - rad1; //sdfPrim1(x, pos1, pos2, rad1, rad2);
    return f;
}
RT_CALLABLE_PROGRAM  float eval5(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    float3 pos1 = descPrim.pos[0];
    float3 pos2 = descPrim.pos[1];
    float3 pos3 = descPrim.pos[2];
    float3 pos4 = descPrim.pos[3];

    float rad1 = descPrim.rad[0];
    float rad2 = descPrim.rad[1];

    f = sdfPrim5(x, pos1, pos2, pos3, pos4, rad1, rad2);//length(x - pos1) - rad1; //sdfPrim1(x, pos1, pos2, rad1, rad2);
    return f;
}

inline __device__ float4  blendColor(float dMd, float3 x, float3 dir, float3 pos, float3 pos2, float3 pos3, float r1, float r2, float r3, float3 col1, float3 col2, float3 col3)
{
    float dNorm = 10;

    float3 vib_color[3];
    vib_color[0] = make_float3(0, 1.0, 0); //green
    vib_color[1] = make_float3(0, 0.0, 1.0); //blue
    vib_color[2] = make_float3(1, 1, 0); //yellow

    float3 wt_col = make_float3(0);
    dNorm = length(pos - pos2) + length(pos3 - pos2) + length(pos - pos3);
    dNorm /= 3;

    float d1 = (length(x - pos) - r1*(1 + dMd)) / dNorm;
    float d2 = (length(x - pos2) - r2*(1 + dMd)) / dNorm;
    float d3 = (length(x - pos3) - r3*(1 + dMd)) / dNorm;

    d1 = optix::clamp(d1, 0.01, 1.0);
    d2 = optix::clamp(d2, 0.01, 1.0);
    d3 = optix::clamp(d3, 0.01, 1.0);

    float3 color = make_float3(0);
    float tr = abs(dMd - 0.1);
    color += (1 - d1)*col1;
    color += (1 - d2)*col2;
    color += (1 - d3)*col3;
    //color += dMd*make_float3(1);
    //color*=tr;

    //dMd=1 - rep1;
    //dMd=0 -rep2
    //dMd*10

    d1 = (length(x - pos) - r1);
    d2 = (length(x - pos2) - r2);
    d3 = (length(x - pos3) - r3);

    //  d1 = optix::clamp(d1, -0.9, 1.0);
    //  d2 = optix::clamp(d2, -0.9, 1.0);
    //  d3 = optix::clamp(d3, -0.9, 1.0);
    float3 col = make_float3(0);
    if (d1 < 0.1)
        col += abs(d1 / r1)*col1;
    if (d2 < 0.1)
        col += abs(d2 / r2)*col2;
    if (d3 < 0.1)
        col += abs(d3 / r3)*col3;

    //------------
    //interpolate between two models
    float3 coll = make_float3(0);
    d1 = optix::clamp(d1, -0.9, 0.0);
    d2 = optix::clamp(d2, -0.9, 0.0);
    d3 = optix::clamp(d3, -0.9, 0.0);
    coll += abs(1 - d1)*col1*abs(d1);
    coll += abs(1 - d2)*col2*abs(d2);
    coll += abs(1 - d3)*col3*abs(d3);
    //coll ;
    col = dMd*coll + (1 - dMd)*col;

    //col+=dMd*make_float3(1);

    //d1 += 0.1;
    //d2 += 0.1;
    //d3 += 0.1;
    tr = abs(1 - min(min(d1, d2), d3)) / 2;//abs(min(min(d1 / r1, d2 / r2), d3 / r3)) / 2 +
                                           //tr /= 3;
                                           //tr=clamp(tr,0,1);
                                           // tr = (3 - d1 - d2 - d3) / 3;
    return make_float4(col.x, col.y, col.z, tr);
}

inline __device__ float3 InterpolateColorRad(float3 x, float3 pos, float3 pos2, float r1, float r2, float3 col1, float3 col2)
{
    float d1 = length(x - pos) - r1;
    float d2 = length(x - pos2) - r2;
    float d = length(pos - pos2);
    if (d1 <= 0) return col1;
    if (d2 <= 0) return col2;
    float3 col = d1 / d*col2 + d2 / d*col1;

    return col;
}

RT_CALLABLE_PROGRAM float3 GetColorBlend(float3 x, primParamDesc desc)
{
    //---level of detail
    float dist_cam = length(sysCameraPosition - (desc.pos[0] + desc.pos[1] + desc.pos[2]) / 3);
    float d;
    if (dist_cam < 20.0)
    {
        float interp = (dist_cam - 10) / 10.0;
        d = optix::clamp(interp, 0.0, 1.0);
    }
    else d = 1;

    //MultiscaleParam=d;

    float3 col1 = transfer_function(desc.types[0], d);
    float3 col2 = transfer_function(desc.types[1], d);
    float3 col3 = transfer_function(desc.types[2], d);

    float3 col11 = InterpolateColorRad(x, desc.pos[0], desc.pos[1], desc.rad[0] / 2, desc.rad[1] / 2, col1, col2);
    float3 col12 = InterpolateColorRad(x, desc.pos[0], desc.pos[2], desc.rad[0] / 2, desc.rad[2] / 2, col1, col3);

    float3 dir = normalize(x - desc.pos[0]);
    float3 dir2 = normalize(desc.pos[1] - desc.pos[0]);
    float3 dir3 = normalize(desc.pos[2] - desc.pos[0]);
    float cos1 = abs(dot(dir2, dir));
    float cos2 = abs(dot(dir3, dir));

    float d1 = length(x - desc.pos[1]);
    float d2 = length(x - desc.pos[2]);
    float td = length(desc.pos[1] - desc.pos[2]);
    //float cos2=normalize(dot(desc.pos[2] - desc.pos[0], dir));
    float3 colB1 = col12*d1 / td + col11*d2 / td;
    float3 colB2 = make_float3(0);

    //return col;

    //--------level of detail continue
    float rad1 = desc.rad[0];
    float rad2 = desc.rad[1];
    float rad3 = desc.rad[2];
    float4 color = make_float4(0);
    float3 vib_color[3];
    vib_color[0] = make_float3(0, 1.0, 0); //green
    vib_color[1] = make_float3(0, 0.0, 1.0); //blue
    vib_color[2] = make_float3(1.0, 1.0, 0); //yellow

   // float3 col21;
   // float3 col31;

    float weights[3];

    float3 resCol = colB1;

    //block currently for cells
    if (desc.type != 5) {
        if (d <= 1.0)
        {
            float3 pos = desc.pos[0];
            float3 pos2 = desc.pos[1];
            float3 pos3 = desc.pos[2];
            //first vibration vector in molecule
            float3 vib2[3];
            float3 vib3[3];

            vib2[0] = (pos2 - pos) / 3.5 * -cos(TimeSound * 20)*(1 - d);
            vib3[0] = (pos3 - pos) / 3.5 * (cos(TimeSound * 20))*(1 - d);

            vib2[1] = (pos2 - pos) / 3.5 * sin(TimeSound * 20)*(1 - d);
            vib3[1] = (pos3 - pos) / 3.5 * (sin(TimeSound * 20))*(1 - d);

            float3 vib_dir = pos2 - 2 * pos + pos3;

            vib2[2] = vib_dir / 3.5 * sin(TimeSound * 20)*(1 - d);
            vib3[2] = vib_dir / 3.5 * (sin(TimeSound * 20))*(1 - d);

            float d_min = 0;
            int vib_min = 0;

            int l = 0;
            float dmax = 0;
            float dmin = 100;
            float dminprev = 100;

            float num = 3;
            if (MultiscaleParam <= 1)  num = 1;
            else
            {
                if (MultiscaleParam <= 2)
                    num = 2;
            }

            for (int i = 0; i < num; i++)
            {
                float3 pp3 = pos3 + vib3[i];
                float3 pp2 = pos2 + vib2[i];
                float d1 = length(x - pp2);
                float d2 = length(x - pp3);

                dmin = fminf(fminf(d1, dmin), d2);
                if (dmin < dminprev) {
                    l = i;
                    dminprev = dmin;
                }
                dmax = fmaxf(fmaxf(d1, dmin), d2);
            }
            weights[0] = 0.1;
            weights[1] = 0.1;
            weights[2] = 0.1;
            weights[l] = 1.5;

            float3 cols[3];
            for (int j = 0; j < num; j++)
            {
                cols[j] = make_float3(0);
                //Get current position
                float3 pp3 = pos3 + vib3[j];
                float3 pp2 = pos2 + vib2[j];

                float3 col21 = InterpolateColorRad(x, desc.pos[0], pp2, desc.rad[0] / 2, desc.rad[1] / 2, col1, vib_color[j]);
                float3 col31 = InterpolateColorRad(x, desc.pos[0], pp3, desc.rad[0] / 2, desc.rad[2] / 2, col1, vib_color[j]);

                //col31 = d*col3 + (1 - d)*vib_color[i] * 1.5;
                //col21 = d*col2 + (1 - d)*vib_color[i] * 1.5;

                float3 dir = normalize(x - desc.pos[0]);
                float3 dir2 = normalize(pp2 - desc.pos[0]);
                float3 dir3 = normalize(pp3 - desc.pos[0]);
                cos1 = dot(dir2, dir);
                float cos2 = dot(dir3, dir);
                //float cos2=normalize(dot(desc.pos[2] - desc.pos[0], dir));
                colB2 = (1 - cos2)*col31 + (1 - cos1)*col21;

                bool interpolate = true;
                float d1 = length(x - pp2);// -rad2 / 4;
                float d2 = length(x - pp3);// -rad3 / 4;
                if (d1 <= 0) {
                    cols[j] = col21;
                    interpolate = false;
                }
                if (d2 <= 0) {
                    cols[j] = col31;
                    interpolate = false;
                }

                if (interpolate) {
                    float dt = length(pp2 - pp3);

                    float dmin1 = fminf(d1, d2);
                    cols[j] = d1 / dt*col31 + d2 / dt*col21;
                    weights[j] = dmin / (dmin1*dmin1);
                }
                else {
                    //d1 = length(x - pp2);
                    //d2 = length(x - pp3);
                    weights[j] = 1.0;// dmin / fminf(d1, d2);
                }

                // colB2 += colB21;
                 //color += blendColor(d, x, theRay.direction, pos, pp2, pp3, r1, r2, r3, col1, col21, col31);
            }
            for (int j = 0; j < num; j++)
            {
                colB2 += weights[j] * cols[j];
            }
            colB2 /= 3.0;

            resCol = d*colB1 + (1 - d)*colB2;
        }
    }
    return resCol;
}

__device__
inline float  plane(float3 p, float3 c, float3 n)
{
    return optix::dot(p - c, n);
}
__device__
inline float3 getNormal(float3 v1, float3 v2, float3 v3, float3 c, float3 ct)
{
    float3 a = v3 - v2;
    float3 b = v1 - v2;
    float3 n = cross(a, b);

    float3 nt = c - ct;

    //normalize(n);
    //normalize(nt);

    n = n*dot(n, nt);

    return normalize(n);
}
__device__
inline float3 getCenter(float3 p1, float3 p2, float3 p3)

{
    float3 center = (p1 + p2 + p3) / 3.0;
    return center;
}

__device__
inline float3 getCenterTetra(float4 p0, float4 p1, float4 p2, float4 p3)

{
    float3 center = make_float3((p0 + p1 + p2 + p3) / 4.0);
    return center;
}

RT_CALLABLE_PROGRAM float3 GetColorBlendCell(float3 x, primParamDesc desc)
{
    //---level of detail
    float dist_cam = length(sysCameraPosition - (desc.pos[0] + desc.pos[1] + desc.pos[2]) / 3);
    float d;
    if (dist_cam < 20.0)
    {
        float interp = (dist_cam - 10) / 10.0;
        d = optix::clamp(interp, 0.0, 1.0);
    }
    else d = 1;

    float3 a = (desc.pos[0] + desc.pos[1] + desc.pos[2] + desc.pos[3]) / 4.0;

    //--------
    float cel1Rad = 3.3 * 2;// 1.65 + 3.3;
    float cel1Rad2 = 3.3;

    float t = clamp(TimeSound, 2.0, 4.0) - 1.0;
    float3 cCell3 = sCell3 - make_float3(3.3*t, 3.3, 3.3);
    float3 cCell2 = sCell2 + make_float3(3.3*t, 3.3, 3.3*t);
    float3 cCell1 = sCell1 - make_float3(3.3, 3.3*t, 3.3*t);

    int comp1 = 0;
    float3 addCol = make_float3(0);

    if ((length(x - cCell1) - cel1Rad) <= 0)
        addCol = make_float3(0.5, 0.5, 0);
    if ((length(x - cCell2) - cel1Rad2) <= 0)
        addCol = make_float3(0, 0.5, 0);
    if ((length(x - cCell3) - cel1Rad) <= 0)
        addCol = make_float3(0, 0, 0.5);

    addCol *= clamp(TimeSound, 0.0, 1.0f);

    //MultiscaleParam=d;

    float3 colE1 = transfer_function(desc.types[0], d);
    float3 colE2 = transfer_function(desc.types[1], d);

    float3 v0 = desc.pos[0];
    float3 v1 = desc.pos[1];
    float3 v2 = desc.pos[2];
    float3 v3 = desc.pos[3];

    float r = desc.rad[1];
    float radBB = length(a - desc.pos[0]) + r / 2;
    if (length(x - a) >= radBB) {
        // col10 = colE1;
        // col11 = colE1;
        // col12 = colE1;
        // col13 = colE1;

        return colE2;
    }

    float3 p0 = v0 - x;
    float d0 = length(p0);
    float3 p1 = v1 - x;
    float d1 = length(p1);
    float3 p2 = v2 - x;
    float d2 = length(p2);
    float3 p3 = v3 - x;
    float d3 = length(p3);

    float dmin = min(d0, min(d1, (min(d2, d3))));
    float3 col;
    if (d0 == dmin)
        col = InterpolateColorRad(x, a, desc.pos[0], desc.rad[0] / 2, desc.rad[1] / 2, colE1, colE2);
    if (d1 == dmin)
        col = InterpolateColorRad(x, a, desc.pos[1], desc.rad[0] / 2, desc.rad[1] / 2, colE1, colE2);

    if (d2 == dmin)
        col = InterpolateColorRad(x, a, desc.pos[2], desc.rad[0] / 2, desc.rad[1] / 2, colE1, colE2);

    if (d3 == dmin)
        col = InterpolateColorRad(x, a, desc.pos[3], desc.rad[0] / 2, desc.rad[1] / 2, colE1, colE2);

    /*float3 col11 = InterpolateColorRad(x, a, desc.pos[1], desc.rad[0], desc.rad[1], colE1, colE2);
    float3 col12 = InterpolateColorRad(x, a, desc.pos[2], desc.rad[0], desc.rad[1], colE1, colE2);
    float3 col13 = InterpolateColorRad(x, a, desc.pos[3], desc.rad[0], desc.rad[1], colE1, colE2);
    float3 col10 = InterpolateColorRad(x, a, desc.pos[0], desc.rad[0], desc.rad[1], colE1, colE2);
    */

    // float3 colB1 = (col11 / d1 + col10 / d0 + col13 / d3 + col12 / d2) / (d1 + d0 + d3 + d2);

    return col + addCol;
    /*
    float3 c0 = getCenter(v0, v2, v1);
    float3 c1 = getCenter(v0, v3, v2);
    float3 c2 = getCenter(v1, v3, v0);
    float3 c3 = getCenter(v1, v2, v3);

    float3 ct = (v0 + v1 + v2 + v3) / 4.0f;
    //float rad1 = length(ct - c0);
    float rad = length(ct - v0);
    //rad = (rad + rad1) / (2.0*t);
    float3 n0 = getNormal(v0, v2, v1, c0, ct);
    float3 n1 = getNormal(v0, v3, v2, c1, ct);
    float3 n2 = getNormal(v1, v3, v0, c2, ct);
    float3 n3 = getNormal(v1, v2, v3, c3, ct);

    float3 col0 = computeColTriangle(x, v0, v1, v2, col10, col11, col12, n0);
    float3 col1 = computeColTriangle(x, v0, v2, v3, col10, col12, col13, n1);
    float3 col2 = computeColTriangle(x, v1, v0, v3, col11, col10, col13, n2);
    float3 col3 = computeColTriangle(x, v1, v2, v3, col11, col12, col13, n3);

    float d0 = abs(plane(x, c0, n0));
    float d1 = abs(plane(x, c1, n1));
    float d2 = abs(plane(x, c2, n2));
    float d3 = abs(plane(x, c3, n3));

    float dmax = max(d0, max(d1, (max(d2, d3))));
    float dmin = min(d0, min(d1, (min(d2, d3))));

    if (d0 >= 0.001) return col0;
    if (d1 >= 0.001) return col1;
    if (d2 >= 0.001) return col2;
    if (d3 >= 0.001) return col3;
    */

    //    float3 colB1 = (col11 / d1 + col10 / d0 + col13 / d3 + col12 / d2) / (d1 + d0 + d3 + d2);

      //  return colB1;

        /*
        float d01 = length(d0*n0 - d1*n1);
        float d23 = length(d2*n2 - d3*n3);
        float d03 = length(d0*n0 - d3*n3);

        if (length )
        float3  colB1 = col1*d0 / (d01)+col0*d1 / (d01);
        float3 colB2 = col3*d2 / (d23)+col2*d3 / (d23);
        colB1 = colB1*d3 / d03 + colB2*d0 / d03;
        return colB1;*/
        /* float3 p0 = v0 - x;
        float d0 = length(p0 - n0);
        float3 p1 = v1 - x;
        float d1 = length(p1 - n1);
        float3 p2 = v2 - x;
        float d2 = length(p2 - n2);
        float3 p3 = v3 - x;
        float d3 = length(p3 - n3);

        //  triangle interp
        float dmax = max(d0, max(d1, (max(d2, d3))));

            float3 colB1 = col1*(1 - d1 / dmax) + col0*(1 - d0 / dmax) + col2*(1 - d2 / dmax) + col3*(1 - d3 / dmax);
            */
            /*
            float3 p0 = v0 - x;
            float d0 = length(p0);
            float3 p1 = v1 - x;
            float d1 = length(p1);
            float3 p2 = v2 - x;
            float d2 = length(p2);
            float3 p3 = v3 - x;
            float d3 = length(p3);
            float dmax = max(d0, max(d1, (max(d2, d3))));
            float dmin = min(d0, min(d1, (min(d2, d3))));
            float3 colB1 = make_float3(1, 1, 1);*/
            /* if (d0 == 0) return col0;
             if (d1 == 0) return col1;
             if (d2 == 0) return col2;
             if (d3 == 0) return col3;

             // colB1 = col1*(dmin / d1) + col0*(dmin / d2) + col2*(dmin / d2) + col3*(dmin / d3);
             // colB1 /= 4.0;

             float3  colB1 = col1*d0 / (d1 + d0) + col0*d1 / (d1 + d0);
             float3 colB2 = col3*d0 / (d3 + d0) + col0*d3 / (d3 + d0);
             float3 colB3 = colB1*d1 / (d3 + d1) + colB2*d3 / (d3 + d1);

             colB1 = col1*d2 / (d1 + d2) + col2*d1 / (d1 + d2);
             colB2 = col3*d2 / (d3 + d2) + col2*d3 / (d3 + d2);
             float3 colB4 = colB1*d1 / (d3 + d1) + colB2*d3 / (d3 + d1);

             colB1 = (colB3 + colB4) / 2;

             return colB3;*/
}

inline __device__  float4  GetColor(float3 x)
{
    //---level of detail
    float dist_cam = length(sysCameraPosition - (infoH.desc.pos[0] + infoH.desc.pos[1] + infoH.desc.pos[2]) / 3);
    float d;
    if (dist_cam < 20.0)
    {
        float interp = (dist_cam - 10) / 10.0;
        d = optix::clamp(interp, 0.0, 1.0);
    }
    else d = 1;

    //MultiscaleParam=d;

    float3 col1 = transfer_function(infoH.desc.types[0], d);
    float3 col2 = transfer_function(infoH.desc.types[1], d);
    float3 col3 = transfer_function(infoH.desc.types[2], d);

    //--------level of detail continue
    float rad1 = infoH.desc.rad[0];
    float rad2 = infoH.desc.rad[1];
    float rad3 = infoH.desc.rad[2];
    float4 color = make_float4(0);
    float3 vib_color[3];
    vib_color[0] = make_float3(0, 1.0, 0); //green
    vib_color[1] = make_float3(0, 0.0, 1.0); //blue
    vib_color[2] = make_float3(1.0, 1.0, 0); //yellow

    float3 col21;
    float3 col31;

    //return mod1*d + (1.0 - d)*mod2;
    float r1 = d*rad1 / 2 + (1 - d)*rad1;
    float r2 = d*rad2 / 2 + (1 - d)*rad2;
    float r3 = d*rad3 / 2 + (1 - d)*rad3;

    float3 weights[3];

    if (d <= 1.0)
    {
        float3 pos = infoH.desc.pos[0];
        float3 pos2 = infoH.desc.pos[1];
        float3 pos3 = infoH.desc.pos[2];
        //first vibration vector in molecule
        float3 vib2[3];
        float3 vib3[3];

        vib2[0] = (pos2 - pos) / 3.5 * -cos(TimeSound * 20)*(1 - d);
        vib3[0] = (pos3 - pos) / 3.5 * (cos(TimeSound * 20))*(1 - d);

        vib2[1] = (pos2 - pos) / 3.5 * sin(TimeSound * 20)*(1 - d);
        vib3[1] = (pos3 - pos) / 3.5 * (sin(TimeSound * 20))*(1 - d);

        float3 vib_dir = pos2 - 2 * pos + pos3;

        vib2[2] = vib_dir / 3.5 * sin(TimeSound * 20)*(1 - d);
        vib3[2] = vib_dir / 3.5 * (sin(TimeSound * 20))*(1 - d);

        float d_min = 0;
        int vib_min = 0;

        for (int i = 0; i < 2; i++)
        {
            float3 pp3 = pos3 + vib3[i];
            float3 pp2 = pos2 + vib2[i];

            col31 = d*col3 + (1 - d)*vib_color[i] * 1.5;
            col21 = d*col2 + (1 - d)*vib_color[i] * 1.5;

            color += blendColor(d, x, theRay.direction, pos, pp2, pp3, r1, r2, r3, col1, col21, col31);
        }
        color;///= 3.0;
    }
    else {
        color = blendColor(d, x, theRay.direction, infoH.desc.pos[0], infoH.desc.pos[1], infoH.desc.pos[2], r1, r2, r3, col1, col2, col3);
    }

    return color;
}

__device__ void render_HeteroVolume(float3 normal, float3 hit_point)
{
    float Ka = 0.5;
    float Kd = 0.9;
    float Ks = 0.9;

    float4 col = make_float4(0, 0, 0, 1);// translucent_grays(0.5, 0.1, 0);

    float tstep = 0.1;
    float3 pos = hit_point;// eyeRay.o + eyeRay.d*tnear;
    float3 step = theRay.direction*tstep;

    float4 sum = thePrd.result;// make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here
    float trp = 0.05;
    float trp0 = 0.1;

    float s1 = evalF(pos, infoH.desc);
    if (abs(s1) > tstep)
        step = theRay.direction*abs(s1);

    float i = 0.0;
    float max = thePrd.maxDist;// *2 + 0.4; //bounding box size
    float4 sumcol = make_float4(0.0);
    float tracedDist = 0;

    //float4 col1 = translucent_grays(0.5, 0.01, 0);
    int VolInt = 1;

    while (i < max) //s2 < 0.01)
    {
        if (s1 < tstep / 2)
        {
            // if (abs(s1) > tstep) //sum transparency
            {
                VolInt = 1;
                //------------------
                                       //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;
                col = GetColor(pos);

                //Beer–Lambert law
                float F = exp(-trp*abs(s1) * 200);
                col = col*(1.0 - F);
                sum = sum + col*(1.0f - sum.w);
            }
        }

        if (abs(s1) > tstep) {
            step = theRay.direction*abs(s1);
            i += abs(s1);
        }
        else
        {
            i += tstep;
            step = theRay.direction*tstep;
        }

        pos += step;
        if (sum.w >= 1.0) {
            i = max + 1;
        }
        else
            s1 = evalF(pos, infoH.desc);
    }

    thePrd.result = sum;
}
/* Compiles various types of programs depending on primType*/
RT_PROGRAM void volume_hetero_close()
{
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, infoH.normal));
    optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;
    thePrd.maxDist = infoH.maxDist;
    thePrd.normal = normal;
    thePrd.last_hit_point = hit_point;

    thePrd.result = GetColor(hit_point);
    // render_HeteroVolume(normal, hit_point); //for defalt rendering primitive

    thePrd.renderType = 0;
    thePrd.depth++;

    if (thePrd.totalDist > 20.0) {//TODO set max trace depth as parameter
        thePrd.depth = 10;
    }

    thePrd.totalDist += length(theRay.origin - hit_point);
    /*if (thePrd.result.w < 0.5) {
        if (thePrd.depth < 5) //TODO set max trace depth as parameter
        {
            //further ray
            optix::Ray ray = optix::make_Ray(hit_point + theRay.direction *(abs(infoH.maxDist) + 0.1), theRay.direction, 0, 0.0f, RT_DEFAULT_MAX);
            rtTrace(sysTopObject, ray, thePrd);
        }
    }*/
}

RT_PROGRAM void volume_hetero_any()
{
    //No volume integration
    //just saving preliminary data to array
    thePrd.renderType = 3; //for postprocessing
    if (thePrd.cur_prim < MAX_PRIM_ALONG_RAY)
    { //push intersections
        thePrd.cur_prim++;
        cellPrimDesc cell;
        cell.intersectionDist = infoH.tmin;
        cell.type = 3; //tracing with primitives and compute bounds
        cell.normal = infoH.normal;
        int typeC = int(infoH.desc.rad[0]);
        //float3 col = GetColor(infoH.hit_point);

        //thePrd.result += make_float4(col.x, col.y, col.z, 0.1);
        cell.color = GetColor(infoH.hit_point);
        cell.color.w = 0.009;
        cell.maxDist = infoH.maxDist;

        thePrd.cellPrimitives[thePrd.cur_prim - 1] = cell;
        thePrd.prims[thePrd.cur_prim - 1] = infoH.desc;
        rtIgnoreIntersection();
    }
    else {
        rtTerminateRay();
    }
}