#include "hip/hip_runtime.h"
/*

 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../renderer/per_ray_data.h"
#include "../basic_lights.h"
#include "transferFunction.h"
 //#include "per_ray_data.h"
#include "../attributeInfo.h"
using namespace optix;

rtDeclareVariable(attributeInfo, info, attribute info, );

rtDeclareVariable(attributeInfo2, infoH, attribute infoH, );
// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

//rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );
//rtDeclareVariable(optix::float3, varHit, attribute hit_point, );

//-----------
//for textures

rtTextureSampler<float, 3> tex0;
rtTextureSampler<float, 3> tex1;
rtTextureSampler<float, 3> tex2;

rtDeclareVariable(int, numTexDefined, , );

//
//for sdf
typedef rtCallableProgramId<float(float3, float3)> callT;
rtDeclareVariable(callT, sdfPrim, , );

//typedef rtCallableProgramId<float(float3, float3)> callT;
rtDeclareVariable(callT, sdfPrimBack, , );

//type of rendering
rtDeclareVariable(int, Type, , );

//array of lights
rtBuffer<BasicLight> lights;

//rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

// This closest hit program only uses the geometric normal and the shading normal attributes.
// OptiX will remove all code from the intersection programs for unused attributes automatically.

// Note that the matching between attribute outputs from the intersection program and
// the inputs in the closesthit and anyhit programs is done with the type (here float3) and
// the user defined attribute semantic (e.g. here NORMAL).
// The actual variable name doesn't need to match but it's recommended for clarity.
__device__ float computeVal(float3 p)
{
    float s1 = 0;

    switch (numTexDefined) {
    case 1:
    {
        s1 = tex3D<float>(tex0, p.x*0.5f + 0.5f, p.y*0.5f + 0.5f, p.z*0.5f + 0.5f);
        break;
    }
    case 2:
    {
        float s_min = tex3D(tex0, p.x*0.5f + 0.5f, p.y*0.5f + 0.5f, p.z*0.5f + 0.5f);
        float s_max = tex3D(tex1, p.x*0.5f + 0.5f, p.y*0.5f + 0.5f, p.z*0.5f + 0.5f);

        s1 = thePrd.TimeSound*s_max + (1.0 - thePrd.TimeSound)*s_min;//trace back to iso value and shift

                                                                     // return s1;
        break;
    }
    }
    return s1;
}

__device__ void render_Surface(float3 normal, float3 hit_point)
{
    float Ka = 0.5;
    float Kd = 0.5;
    float Ks = 0.2;

    float s1 = computeVal(hit_point) / 6;

    float4 col = translucent_grays(0.5, s1, 0);
    float3 color = Ka *  make_float3(col);// ambient_light_color;

//	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    for (int i = 0; i < lights.size(); ++i)
    {
        BasicLight light = lights[i];
        float3 L = optix::normalize(light.pos - hit_point);
        float nDl = optix::dot(normal, L);

        //if (nDl > 0)
        //    color += Kd * nDl * light.color; // make_float3(1.0);//

        float phong_exp = 0.1;
        if (nDl > 0) {
            color += Kd * nDl * light.color;

            /* optix::float3 H = optix::normalize(L - theRay.direction);
            float nDh = optix::dot(normal, H);
            if (nDh > 0)
            color += Ks * light.color * pow(nDh, phong_exp);*/
        }
    }

    thePrd.radiance = color;
}

__device__ void render_Volume(float3 normal, float3 hit_point)
{
    float tstep = 0.01;
    float3 pos = hit_point;// eyeRay.o + eyeRay.d*tnear;
    float3 step = theRay.direction*tstep;
    float t = theIntersectionDistance;

    //GetSDF function Value
    int jj = 0;
    float eps = 0.01;
    float4 sum = make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here

    bool stop = false;
    float s1 = sdfPrim(pos, make_float3(0.8));// interpolateSDF(time, pos, texSDF, texSDF_F);

    render_Surface(normal, hit_point);
    // blend
    sum = sum + make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.01)*(1 - sum.w);
    while (!stop)//we are inside the object

    {
        float sample = computeVal(pos) / 6;

        float4 col = translucent_grays(0.1 + s1 / 6, sample, 0);

        //kompute koef k_d
        int isoNum = 4;

        //isosurf highlightment
        //TODO: raymarch transmit;
        /*float rayMarch = 6.0;
        float koef =  (rayMarch - s1) / rayMarch;
        float k_d = fabsf(sinf(koef*3.14f*isoNum));
         k_d *= powf(2.6, koef);
         col.w *= (k_d + 0.2);*/

         // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        // "over" operator for front-to-back blending
        sum = sum + col*(1.0f - sum.w);

        //float op = sum.w;
        //isoNum = 2;

        //float4 currColor = col*(1.0f - sum.w);
        //computation of Blinn-Phong
        //if (k_d >= 0.7) {
        //computeColor(sum, pos, eyeRay, rayMarch, time, s1, currColor, k_d, texSDF, texSDF_F);

        //}

        // exit early if opaque
        if (sum.w > 1.1)
            stop = true;
        //	break;

        t += tstep;

        //get bounding box of the primitive here
        if (t > 20) stop = true;

        pos += step;
        s1 = sdfPrim(pos, make_float3(0.5)); //interpolateSDF(time, pos, texSDF, texSDF_F);
        if (s1 <= eps) stop = true;

        if (s1 > 2.0) {
            render_Surface(normal, hit_point);
            // blend
            sum = sum + make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.4)*(1.0f - sum.w);
        }
    }

    //float3 hitP = eyeRay.o;
    //hitP += eyeRay.d*tnear;
    //float3 cl = make_float3(sum.x, sum.y, sum.z);
    //float faceN =
    // computeColor(sum, hitP, eyeRay, rayMarch, time, rayMarch, sum, 2.01, texSDF, texSDF_F);

    thePrd.result = sum;
    // thePrd.radiance = make_float3(sum); //+thePrd.radiance*0.8f;
}
__device__ void render_Surface2(float3 normal, float3 hit_point)
{
    float Ka = 0.5;
    float Kd = 0.5;
    float Ks = 0.2;
    float4 col = make_float4(0, 1, 0, 1);// translucent_grays(0.5, 0.1, 0);

    float tstep = 0.01;
    float3 pos = hit_point;// eyeRay.o + eyeRay.d*tnear;
    float3 step = theRay.direction*tstep;

    float4 sum = make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here
    BasicLight lights2[2];
    lights2[0].color = optix::make_float3(1.0);
    lights2[0].pos = optix::make_float3(10.0);

    lights2[1].color = optix::make_float3(1.0);
    lights2[1].pos = optix::make_float3(0, 0, 10.0);

    float s1 = sdfPrim(pos, make_float3(thePrd.maxDist));
    pos += step;
    float s2 = sdfPrim(pos, make_float3(thePrd.maxDist));
    float i = 0;
    float trp = 0.01;
    float max = thePrd.maxDist * 2 + 0.4; //bounding box size
    while (i < max) //s2 < 0.01)
    {
        if (s2 < 0.01) {
            //col *= Ka;
           //col.w = 0.5; //s1 is very small
            float3 color = Ka *  make_float3(col);// ambient_light_color;

                                                  //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

            for (int i = 0; i < 2; ++i)
            {
                BasicLight light = lights2[i];
                float3 L = optix::normalize(light.pos - hit_point);
                float nDl = optix::dot(thePrd.normal, L);

                //if (nDl > 0)
                //    color += Kd * nDl * light.color; // make_float3(1.0);//

                float phong_exp = 0.1;
                if (nDl > 0) {
                    color += Kd * nDl * light.color;

                    /*  optix::float3 H = optix::normalize(L - theRay.direction);
                      float nDh = optix::dot(normal, H);
                      if (nDh > 0)
                          color += Ks * light.color * pow(nDh, phong_exp);
                  */
                }
            }
            //initial blend
            col = make_float4(color);
            col.w = trp;
            col.x *= col.w;
            col.y *= col.w;
            col.z *= col.w;
            // "over" operator for front-to-back blending
            sum = sum + col*(1.0f - sum.w);
        }
        i += tstep;
        s1 = s2;
        pos += step;

        if (sum.w >= 1.0) i = max + 1;
        else
            s2 = sdfPrim(pos, make_float3(thePrd.maxDist)); //interpolateSDF(time, pos, texSDF, texSDF_F);
           // if (abs(s1) <= eps)
    }
    //or it should be
    // thePrd.radiance = make_float3(sum);
    thePrd.radiance += make_float3(sum);
}
__device__ float fract(float x)
{
    return x - floor(x);
}
/* discontinuous pseudorandom uniformly distributed in [-0.5, +0.5]^3 */
/*__device__  float3 random3(float3 c) {
    float j = 4096.0*sin(dot(c, make_float3(17.0, 59.4, 15.0)));
    float3 r;
    r.z = fract(512.0*j);
    j *= 0.125;
    r.x = fract(512.0*j);
    j *= 0.125;
    r.y = fract(512.0*j);
    return r - 0.5;
}
*/
__device__ float noise3D(float3 p)
{
    return fract(sin(dot(p, make_float3(12.9898, 78.233, 126.7235))) * 43758.5453);
}

__device__ float evalNoise(float3 p)
{
    float r = 1.0;
    float3 f = floor(p);
    float3 x = make_float3(fract(p.x), fract(p.y), fract(p.z));
    for (int i = -1; i <= 1; i++)
    {
        for (int j = -1; j <= 1; j++)
        {
            for (int k = -1; k <= 1; k++)
            {
                float3 q = make_float3(float(i), float(j), float(k));
                float3 v = q + make_float3(noise3D((q + f)*1.11), noise3D((q + f)*1.14), noise3D((q + f)*1.17)) - x;
                float d = dot(v, v);
                r = fminf(r, d);
            }
        }
    }
    return sqrt(r);
}
__device__ float ComputeSDFCol(float3 pos)
{
    float radc = 5.0;
    float scale = 1.8;
    float3 cent1 = make_float3(radc, 0, 0);
    float3 cent2 = make_float3(-radc, 0, 0);
    float3 cent3 = make_float3(0, radc, 0);
    float3 cent4 = make_float3(0, -radc, 0);
    float d1 = length(pos - cent1) - radc / scale;
    float d2 = length(pos - cent2) - radc / scale;
    float d3 = length(pos - cent3) - radc / scale;
    float d4 = length(pos - cent4) - radc / scale;

    float d5 = min(d1, d2);
    d5 = min(d5, d3);
    d5 = min(d5, d4);
    d5 -= scale;
    d5 = -d5;

    float f = fmaxf(d5, d4);
    f = fmaxf(f, d3);
    f = fmaxf(f, d2);
    f = fmaxf(f, d1);

    return f;
}
__device__ float3 ComputeColor(float3 pos, int& VolInt, float&trp)
{
    //Compute color function
    //----------------
    float radc = 5.0;
    float scale = 1.8;
    float3 cent1 = make_float3(radc, 0, 0);
    float3 cent2 = make_float3(-radc, 0, 0);
    float3 cent3 = make_float3(0, radc, 0);
    float3 cent4 = make_float3(0, -radc, 0);
    float d1 = length(pos - cent1) - radc / scale;
    float d2 = length(pos - cent2) - radc / scale;
    float d3 = length(pos - cent3) - radc / scale;
    float d4 = length(pos - cent4) - radc / scale;

    float d5 = min(d1, d2);
    d5 = min(d5, d3);
    d5 = min(d5, d4);
    d5 -= scale * 1.5;
    d5 = -d5;

    float3 col1 = make_float3(1, 0, 0);
    float3 col2 = make_float3(0, 0, 1);
    float3 col3 = make_float3(1, 1, 0);
    float3 col4 = make_float3(0, 1, 1);
    float3 col5 = make_float3(0, 1, 0);
    //------------
    //interpolation with noise
   /* VolInt = 1;//ray-segment

                   //f = worley3D(vec3(p*0.25, 1.0)*f);;
                   //if (d4 < scale)
    {
        //VolInt = 0; //vol sampling for this region
        float f = 0;
        if (d4 <= 0) {
            f = evalNoise(pos);
            float3 p2 = pos*0.25;
            f = evalNoise(p2)*f;
            VolInt = 0;
        }
        else {
            if (d4 <= scale) {
                float3 p = pos - d4*thePrd.normal;
                f = evalNoise(p);
                float3 p2 = pos*0.25;
                f = evalNoise(p2)*f;
                VolInt = 0;
            }
            else f = 1 / 5;
            //trp *= f;
        }
        //trp += f;
        col4 = make_float3(5 * f);
        col4 *= make_float3(0.0, 1.0, 1.0)*f;
        //col4 *= expf(1.0f - col4);
    }
    */

    //-------------
    float3 colorm2;
    /* if (d1 <= scale) return col1;
     if (d2 <= scale) return col2;
     if (d3 <= scale) return col3;
     if (d4 <= scale) return col4;
     if (d5 <= scale) return col5;*/

    float d_l = abs(d1) + abs(d2);
    colorm2 = (d1 / d_l)*col2 + (d2 / d_l)*col1;
    d_l = abs(d3) + abs(d4);
    colorm2 += (d3 / d_l)*col4 + (d4 / d_l)*col3;
    d_l = abs(d1) + abs(d3);
    colorm2 += (d3 / d_l)*col1 + (d1 / d_l)*col3;
    d_l = abs(d3) + abs(d2);
    colorm2 += (d3 / d_l)*col2 + (d2 / d_l)*col3;
    d_l = abs(d1) + abs(d4);
    colorm2 += (d1 / d_l)*col4 + (d4 / d_l)*col1;
    d_l = abs(d2) + abs(d4);
    colorm2 += (d2 / d_l)*col4 + (d4 / d_l)*col2;

    //-----
    d_l = abs(d2) + abs(d5);
    colorm2 += (d2 / d_l)*col5 + (d5 / d_l)*col2;
    d_l = abs(d1) + abs(d5);
    colorm2 += (d1 / d_l)*col5 + (d5 / d_l)*col1;
    d_l = abs(d3) + abs(d5);
    colorm2 += (d3 / d_l)*col5 + (d5 / d_l)*col3;
    d_l = abs(d4) + abs(d5);
    colorm2 += (d4 / d_l)*col5 + (d5 / d_l)*col4;

    colorm2 /= 10.0;
    return colorm2;
}
__device__ void render_Surface3(float3 normal, float3 hit_point)
{
    float Ka = 0.5;
    float Kd = 0.9;
    float Ks = 0.9;
    float4 col = make_float4(0, 0, 0, 1);// translucent_grays(0.5, 0.1, 0);

    BasicLight lights2[2];
    lights2[0].color = optix::make_float3(1.0);
    lights2[0].pos = optix::make_float3(10.0);

    lights2[1].color = optix::make_float3(1.0);
    lights2[1].pos = optix::make_float3(0, 0, 10.0);

    float tstep = 0.01;
    float3 pos = hit_point;// eyeRay.o + eyeRay.d*tnear;
    float3 step = theRay.direction*tstep;

    float4 sum = thePrd.result;// make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here

    float trp = 0.01;
    float trp0 = 0.02;
    float s1 = sdfPrimBack(pos, make_float3(1.1));
    if (abs(s1) > tstep)
        step = theRay.direction*abs(s1);

    //pos += step;
    //float s2 = sdfPrimBack(pos, make_float3(1.1));
    float i = 0.0;
    float max = thePrd.maxDist;// *2 + 0.4; //bounding box size
    float4 sumcol = make_float4(0.0);
    float tracedDist = 0;

    //float4 col1 = translucent_grays(0.5, 0.01, 0);
    int VolInt = 1;

    while (i < max) //s2 < 0.01)
    {
        if (s1 < tstep / 2)
        {
            // if (abs(s1) > tstep) //sum transparency
            {
                //is used to highlight isosurfaces
                //or create a more shell like effect
                //trp = trp0 + abs(s1) / 10;

                //----------------------
                //COLOR COMPUTATION
                //trp = trp0;
                float3 colorm2 = ComputeColor(pos, VolInt, trp);
                VolInt = 1;
                //------------------

                float3 color = Ka *  colorm2;// ambient_light_color;
                float3 color2 = Ka *  colorm2;                                          //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;
                if (i < 0.02)//smooth boundary
                {
                    for (int i = 0; i < 2; ++i)
                    {
                        BasicLight light = lights2[i];
                        float3 L = optix::normalize(light.pos - thePrd.last_hit_point);
                        float nDl = optix::dot(thePrd.normal, L);

                        //if (nDl > 0)
                        //    color += Kd * nDl * light.color; // make_float3(1.0);//

                        float phong_exp = 0.2;
                        if (nDl > 0) {
                            color += Kd * nDl * light.color;

                            optix::float3 H = optix::normalize(L - theRay.direction);
                            float nDh = optix::dot(normal, H);
                            if (nDh > 0)
                                color += Ks * light.color * pow(nDh, phong_exp);
                        }
                    }
                    //---------------

                    col = make_float4(color.x, color.y, color.z, trp*Ka);
                }
                else
                    col = make_float4(color2.x, color2.y, color2.z, trp*Ka);

                if (VolInt > 0) {
                    //Beer–Lambert law
                    float F = exp(-trp*abs(s1) * 200);
                    col = col*(1.0 - F);
                    sum = sum + col*(1.0f - sum.w);
                }
                else //conventional integration
                {
                    /*col.w = trp * 1.9;
                    col.x *= col.w;
                    col.y *= col.w;
                    col.z *= col.w;

                    float t = sum.w;
                    // "over" operator for front-to-back blending
                    sum = sum + col*(1.0f - t);
                    sum.w = t*(1.0 - trp);*/

                    float F = 1.0 - trp*1.9;
                    col = col*(1.0 - F);
                    sum = sum + col*(1.0f - sum.w);
                }
                // tracedDist += abs(s1);
            }
        }

        //s1 = s2;

        if (VolInt > 0) {
            if (abs(s1) > tstep) {
                step = theRay.direction*abs(s1);
                i += abs(s1);
            }
            else
            {
                i += tstep;
                step = theRay.direction*tstep;
            }
        }
        else { //volume sampling
            if (s1 > tstep) //employ space skipping
            {
                step = theRay.direction*abs(s1);
                i += abs(s1);
            }
            else {
                i += tstep;
                step = theRay.direction*tstep;
            }
        }

        pos += step;
        if (sum.w >= 1.0) {
            i = max + 1;
        }
        else
            s1 = sdfPrimBack(pos, make_float3(thePrd.maxDist)); //interpolateSDF(time, pos, texSDF, texSDF_F);

                                                                //s2 = sdfPrimBack(pos, make_float3(1.1)); //interpolateSDF(time, pos, texSDF, texSDF_F);
                                                                // if (abs(s1) <= eps)
    }
    thePrd.result = sum;
}
__device__ void render_Surface4(float3 normal, float3 hit_point)
{
    float Ka = 0.5;
    float Kd = 0.5;
    float Ks = 0.2;

    float tstep = 0.01;
    float3 pos = hit_point;// eyeRay.o + eyeRay.d*tnear;
    float3 step = theRay.direction*tstep;

    float4 sum = make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here

    float trp = 0.1;
    float s1 = sdfPrim(pos, make_float3(1.1));
    if (abs(s1) > tstep)
        step = theRay.direction*abs(s1);

    pos += step;
    float s2 = sdfPrim(pos, make_float3(1.1));
    float i = 0;
    float max = 2.2 + 0.4; //bounding box size
    float4 sumcol = make_float4(0.0);
    float tracedDist = 0;

    float4 col1 = translucent_grays(0.5, 0.01, 0);
    float3 color = Ka *  make_float3(col1);// ambient_light_color;

                                           //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    for (int i = 0; i < lights.size(); ++i)
    {
        BasicLight light = lights[i];
        float3 L = optix::normalize(light.pos - hit_point);
        float nDl = optix::dot(normal, L);

        //if (nDl > 0)
        //    color += Kd * nDl * light.color; // make_float3(1.0);//

        float phong_exp = 0.1;
        if (nDl > 0) {
            color += Kd * nDl * light.color;

            optix::float3 H = optix::normalize(L - theRay.direction);
            float nDh = optix::dot(normal, H);
            if (nDh > 0)
                color += Ks * light.color * pow(nDh, phong_exp);
        }
    }

    while (i < max) //s2 < 0.01)
    {
        if (s2 < 0.0) {
            if (abs(s1) > tstep) //sum transparency
            {
                /*//col *= Ka;
                //col.w = 0.5; //s1 is very small

                //initial blend
               // float4 col = make_float4(color);

                //   col.w = trp*abs(s1) * 10;
                //Beer�Lambert law
                col.w = exp(-(trp)*abs(s1) * 1000);

                col.x *= col.w;
                col.y *= col.w;
                col.z *= col.w;

                //col = col*col.w;

               // "over" operator for front-to-back blending
                sum = sum + col*(1.0f - sum.w);*/
                tracedDist += abs(s1);
            }
        }
        else {
            float4 col = make_float4(color);
            col.w = exp(-trp*tracedDist * 100);

            col.x *= 1 - col.w;
            col.y *= 1 - col.w;
            col.z *= 1 - col.w;

            //col = col*col.w;

            // "over" operator for front-to-back blending
            sum = sum + col*(1.0f - sum.w);
            tracedDist = 0;
        }
        i += abs(s1);
        s1 = s2;
        if (abs(s1) > tstep)
            step = theRay.direction*abs(s1);

        pos += step;
        if (sum.w >= 1.0) i = max + 1;
        else
            s2 = sdfPrimBack(pos, make_float3(thePrd.maxDist)); //interpolateSDF(time, pos, texSDF, texSDF_F);

        //s2 = sdfPrim(pos, make_float3(1.1)); //interpolateSDF(time, pos, texSDF, texSDF_F);
                                             // if (abs(s1) <= eps)
    }
    thePrd.radiance = make_float3(sum);
}

__device__ void render_Surface21(float3 inp_normal, float3 inp_p)
{
    float Ka = 0.5;
    float Kd = 0.5;
    float Ks = 0.2;
    float4 col = make_float4(0, 0, 0, 1);// translucent_grays(0.5, 0.1, 0);

    float tstep = 0.01;
    float3 pos = thePrd.last_hit_point;// eyeRay.o + eyeRay.d*tnear;
    float3 step = theRay.direction*tstep;

    float4 sum = thePrd.result;// make_float4(thePrd.radiance.x, thePrd.radiance.y, thePrd.radiance.z, 0.1); //TODO: get background color here
    BasicLight lights2[2];
    lights2[0].color = optix::make_float3(1.0);
    lights2[0].pos = optix::make_float3(10.0);

    lights2[1].color = optix::make_float3(1.0);
    lights2[1].pos = optix::make_float3(0, 0, 10.0);

    float s1 = sdfPrimBack(pos, make_float3(thePrd.maxDist));
    pos += step;
    float s2 = sdfPrimBack(pos, make_float3(thePrd.maxDist));
    float i = 0;
    float trp = 0.01;
    float max = thePrd.maxDist * 2 + 0.4; //bounding box size
    float3 color = Ka *  make_float3(0, 1, 0);// ambient_light_color;
    float3 color2 = Ka *  make_float3(0, 1, 0);                                          //	optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;

    for (int i = 0; i < 2; ++i)
    {
        BasicLight light = lights2[i];
        float3 L = optix::normalize(light.pos - thePrd.last_hit_point);
        float nDl = optix::dot(thePrd.normal, L);

        //if (nDl > 0)
        //    color += Kd * nDl * light.color; // make_float3(1.0);//

        float phong_exp = 0.1;
        if (nDl > 0) {
            color += Kd * nDl * light.color;
        }
    }

    while (i < max) //s2 < 0.01)
    {
        if (s2 < 0.01) {
            //col *= Ka;
            //col.w = 0.5; //s1 is very small

            //initial blend
            if (col.x == 0)
                col = make_float4(color);
            else
                col = make_float4(color2);
            col.w = trp;
            col.x *= col.w;
            col.y *= col.w;
            col.z *= col.w;

            float t = sum.w;
            // "over" operator for front-to-back blending
            sum = sum + col*(1.0f - t);
            sum.w = t*(1.0 - trp);
        }
        i += tstep;
        s1 = s2;
        pos += step;

        if (sum.w >= 1.0) i = max + 1;
        else
            s2 = sdfPrimBack(pos, make_float3(thePrd.maxDist)); //interpolateSDF(time, pos, texSDF, texSDF_F);
                                                            // if (abs(s1) <= eps)
    }
    //or it should be
    // thePrd.radiance = make_float3(sum);
   // thePrd.radiance += make_float3(sum);
    thePrd.result = sum;
}

RT_PROGRAM void anyhitvolume_sdf()
{
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));
    optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;
    thePrd.maxDist = info.maxDist;
    thePrd.normal = normal;
    thePrd.last_hit_point = hit_point;
    switch (Type)
    {
    case 0: //surface
    {
        render_Surface(normal, hit_point);
        break;
    }
    case 1: //volume
    {
        render_Volume(normal, hit_point);
        break;
    }
    /*case 2: //volume
    {
        render_Transp(normal, hit_point);
        break;
    }*/
    }
    // For comparison
   //render_Surface21(info.normal, info.hit_point);
//    render_Surface3(info.normal, info.hit_point);
    //render_Volume(normal, hit_point); //for leap motion - don't remember
}

RT_PROGRAM void volumehit_complex_sdf()
{
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, info.normal));
    optix::float3 hit_point = theRay.origin + theIntersectionDistance * theRay.direction;
    //thePrd.normal = normal;
    //thePrd.last_hit_point = hit_point;
    //thePrd.maxDist = info.maxDist;
    /*
           thePrd.last_hit_point = hit_point - theRay.direction*0.4;
        thePrd.wo = theRay.direction;
        thePrd.maxDist = info.maxDist;

        //render_Surface21(info.normal, info.hit_point);
        rtIgnoreIntersection();

        thePrd.renderType = 2;
        //size compared to other rays
        optix::Ray ray = optix::make_Ray(hit_point + theRay.direction *1.2, theRay.direction, 0, 0.0f, RT_DEFAULT_MAX);
        rtTrace(sysTopObject, ray, thePrd);
    */

    if (thePrd.cur_prim < MAX_PRIM_ALONG_RAY)
    { //push intersections
        thePrd.cur_prim++;
        cellPrimDesc cell;
        cell.intersectionDist = theIntersectionDistance;
        cell.type = 0; //main
        cell.normal = normal;
        cell.color = make_float4(0, 1, 0, 0.005);
        cell.maxDist = info.maxDist;
        thePrd.cellPrimitives[thePrd.cur_prim - 1] = cell;

        //check this one after as it may lead to error in any hit
        thePrd.renderType = 2;
        rtIgnoreIntersection();
    }
    else {
        rtTerminateRay();
    }
}