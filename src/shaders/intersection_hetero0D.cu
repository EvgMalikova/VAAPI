#include "hip/hip_runtime.h"
/*
All basic variables for SDFs heterogeneous objects ray-tracing
*/
#include "sdfGeometryVariables.h"

using namespace optix;

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_math.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "attributeInfo.h"
#include "renderer/random_number_generators.h"


#include "sdfPrimPrograms.h"

rtBuffer<float3>    Positions;
rtBuffer<float>    BSRadius;
rtBuffer<int>    BSType;

//Connectivity info
rtBuffer<int2>    Bonds;

rtDeclareVariable(float, MultiscaleParam, , );

rtDeclareVariable(float, sysSceneEpsilon, , );

//for dynamic staff
rtDeclareVariable(int, PNum, , );
rtDeclareVariable(int, numFrames, , );
rtDeclareVariable(float, TimeSound, , );
//-----------------
//TODO:for SDF
//still have to be implemented as PTX should be generated automatically
//PTX can automatically generated for some primitives like spheres, boxes and etc
//consider further integration with python

//------------
//All inherited SDF functions
//for ray-casting approach structure
//typedef rtCallableProgramX<float3(primParamDesc, float3, float3)> callBoundT;
//rtDeclareVariable(callBoundT, boundIntersection, , );

//typedef rtCallableProgramX<primParamDesc(int)> callReadDataT;
//rtDeclareVariable(callReadDataT, getTimeData, , );

//for sphere tracing of various primitives


inline __device__ float evaluateFunction(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    float3 pos1 = descPrim.pos[0];
    float rad1 = descPrim.rad[0];

    f = sdfPrim0(x - pos1, rad1);

    return f;
}

inline __device__ float3 computeNormal(float eps, float3 x, primParamDesc descPrim)
{
    float dx = evaluateFunction(x + make_float3(eps, 0, 0), descPrim) - evaluateFunction(x - make_float3(eps, 0, 0), descPrim);
    float dy = evaluateFunction(x + make_float3(0, eps, 0), descPrim) - evaluateFunction(x - make_float3(0, eps, 0), descPrim);
    float dz = evaluateFunction(x + make_float3(0, 0, eps), descPrim) - evaluateFunction(x - make_float3(0, 0, eps), descPrim);

    return normalize(make_float3(dx, dy, dz));
}

inline __device__ float SphereTraceForward(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*t;

    float dist;

    float totalDistance = t;
    int i = 0;
    bool stop = false;
    while (!stop)
    {
        dist = evaluateFunction(x, descPrim);// sdfPrim1(x, pos, pos2, rad1, rad2);

                                        // Step along the ray and accumulate the distance from the origin.
        x += abs(dist) * ray_direction;
        totalDistance += abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance >= tmax)
        {
            stop = true;
        }
        if (dist < 0) {
            //x -= abs(dist) * ray_direction;
            totalDistance -= abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

inline __device__ float SphereTraceBack(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*tmax;

    float dist;

    //TODO: getType

    float totalDistance = tmax - t;
    int i = 0;
    bool stop = false;

    while (!stop)
    {
        //dist = sdfPrim1(x, pos, pos2, rad1, rad2);
        dist = evaluateFunction(x, descPrim);// sdfPrim1(x, pos, pos2, rad1, rad2);

                                        // Step along the ray and accumulate the distance from the origin.
        x -= abs(dist) * ray_direction;
        totalDistance -= abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance <= 0)
        {
            stop = true;
        }
        if (dist < 0) {
            x += abs(dist) * ray_direction;
            totalDistance += abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

//------------------------

inline __device__  primParamDesc getTimeData(int primIdx)
{
    int lower = int(floorf(TimeSound));
    int upper = int(ceilf(TimeSound));

    float timeS = TimeSound;
    if (upper > numFrames) upper = numFrames;

    float time = timeS - float(lower);

    float3 pos2 = make_float3(0);
    float3 pos = make_float3(0);
    float3 pos1 = Positions[primIdx + lower*PNum]; //getting correct frame

    if (numFrames > 0) //dynamic
    {
        pos2 = Positions[primIdx + upper*PNum];
        pos = time*pos2 + (1.0 - time)*pos1; //time interpolation
    }
    else
        pos = pos1; //static

    const float rad = BSRadius[primIdx];
    const int type = BSType[primIdx];
    primParamDesc descPrim;

    descPrim.type = 0; //sphere data type
    descPrim.pos[0] = pos;

    descPrim.rad[0] = rad;

    descPrim.types[0] = type;

    return descPrim;
}

inline __device__   float3 boundIntersection(primParamDesc  descPrim, float3 origin, float3 direction)
{
    const float t = length(descPrim.pos[0] - origin);
    const float3 pos_along_ray = origin + direction * t;
    float tmax = t + 2 * descPrim.rad[0];

    float tmin = fmaxf(0.0, t - descPrim.rad[0]);

    float3 params = make_float3(tmin, tmax, 0.0);
    (length(descPrim.pos[0] - pos_along_ray) < descPrim.rad[0]) ? params.z = 1.0 : params.z = 0.0; //within bounding sphere

    return params;
}

//Main program
RT_PROGRAM void intersection_mol(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;
    float epsilon = 0.001;//delta;
    if (epsilon > sysSceneEpsilon) epsilon = sysSceneEpsilon;
    float eps = 0.001;
    if (eps > sysSceneEpsilon) eps = sysSceneEpsilon;

    /* ------------------
    /* 1) Reading data and accessing current positions for current time
    /---------------------------------------------------------------*/

    primParamDesc descPrim = getTimeData(primIdx);

    //---------------
    float3 interSectParams = boundIntersection(descPrim, theRay.origin, theRay.direction);
    //if within bounding volume intersection
    //float3 interSectParams = BoundingSubVolumesIntersect(descPrim);
    if (interSectParams.z > 0.0)
    {
        tmin = interSectParams.x;
        tmax = interSectParams.y;
        float totalDistance = tmin;
        // === Raymarching (Sphere Tracing) Procedure ===

        totalDistance = SphereTraceForward(epsilon, tmin, tmax, descPrim);

        // Found potential intersection?
        if (totalDistance < tmax) //we found intersection
        {
            float totalDistance2 = SphereTraceBack(epsilon, totalDistance, tmax, descPrim);
            //------------
            if (totalDistance2 > epsilon)
            { //it is sufficiently large subvolume to ray-cast
                if (rtPotentialIntersection(totalDistance))
                {
                    //compute normal for primitive
                    float3 x = theRay.origin + theRay.direction*totalDistance;

                    float3 cNormal = computeNormal(eps, x, descPrim);

                    //varNormal = normalize(make_float3(dx, dy, dz));

                    //for material

                    infoH.normal = cNormal;
                    infoH.hit_point = theRay.origin + theRay.direction * (totalDistance);
					infoH.tmin=totalDistance;

                    infoH.desc = descPrim;
                    infoH.maxDist = totalDistance2;
                    rtReportIntersection(MaterialIndex);
                }
            }
        }
    }
}

//bounding box
RT_PROGRAM void primitive_bounds(int primIdx, float result[6])
{
    const float3 position = Positions[primIdx];
    //we compute only bounding sphere like parameters
    //the exact bounding box of SDF primitive can be much smaller
    const float radius = BSRadius[primIdx];

    optix::Aabb *aabb = (optix::Aabb *) result;

    aabb->m_min.x = position.x - radius;
    aabb->m_min.y = position.y - radius;
    aabb->m_min.z = position.z - radius;

    aabb->m_max.x = position.x + radius;
    aabb->m_max.y = position.y + radius;
    aabb->m_max.z = position.z + radius;
}