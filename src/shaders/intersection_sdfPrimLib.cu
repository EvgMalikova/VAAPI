#include "hip/hip_runtime.h"
#include <optix_math.h>
#include "primDesc.h"

rtDeclareVariable(float, spRad, , );
rtDeclareVariable(float3, center, , );

rtTextureSampler<float, 3> texSDF0;
rtDeclareVariable(float, shift0, , );
rtDeclareVariable(float, size0, , );

rtTextureSampler<float, 3> texSDF1;
rtDeclareVariable(float, shift1, , );
rtDeclareVariable(float, size1, , );

rtTextureSampler<float, 3> texSDF2;
rtDeclareVariable(float, shift2, , );
rtDeclareVariable(float, size2, , );

rtDeclareVariable(int, numTexDefined, , );
rtDeclareVariable(float, TimeSound, , );
//for SDF

typedef rtCallableProgramId<float(float3, float3)> callT;
rtDeclareVariable(callT, sdfOpPrim, , );

typedef rtCallableProgramId<float(float3, float3)> callT;
rtDeclareVariable(callT, sdfOpPrim2, , );

rtDeclareVariable(float3, sCell1, , );
rtDeclareVariable(float3, sCell2, , );
rtDeclareVariable(float3, sCell3, , );

//------------------------------------
// list of all used or unsued parameters

rtDeclareVariable(float3, varRadius, , );
rtDeclareVariable(float3, varCenter, , );

rtDeclareVariable(float3, varCenter0, , );
rtDeclareVariable(float3, varCenter1, , );
rtDeclareVariable(float3, varCenter2, , );
rtDeclareVariable(float3, varCenter3, , );
rtDeclareVariable(float3, varCenter4, , );

rtDeclareVariable(float, varK, , );
rtDeclareVariable(float2, varT, , );
rtDeclareVariable(float, MultiscaleParam, , );
rtDeclareVariable(float3, sysCameraPosition, , );
rtDeclareVariable(float, varBlob, , );

//#include "sdfGeometryVariables.h"

__device__
inline float3 max(float3 a, float3 b)
{
    return make_float3(max(a.x, b.x), max(a.y, b.y), max(a.z, b.z));
}

__device__
inline float3 abs(float3 a)
{
    return max(-a, a);
}

__device__
inline float  plane(float3 p, float3 c, float3 n)
{
    return dot(p - c, n);
}
__device__
inline float3 getNormal(float3 v1, float3 v2, float3 v3, float3 c, float3 ct)
{
    float3 a = v3 - v2;
    float3 b = v1 - v2;
    float3 n = cross(a, b);

    float3 nt = c - ct;

    n = n*dot(n, nt);

    return normalize(n);
}
__device__
inline float3 getCenter(float3 p1, float3 p2, float3 p3)

{
    float3 center = (p1 + p2 + p3) / 3.0;
    return center;
}

__device__
inline float3 getCenterTetra(float4 p0, float4 p1, float4 p2, float4 p3)

{
    float3 center = make_float3((p0 + p1 + p2 + p3) / 4.0);
    return center;
}

RT_CALLABLE_PROGRAM
float sdfTetra(float3 p, float3 v0, float3 v1, float3 v2, float3 v3)
{
    float3 c0 = getCenter(v0, v2, v1);
    float3 c1 = getCenter(v0, v3, v2);
    float3 c2 = getCenter(v1, v3, v0);
    float3 c3 = getCenter(v1, v2, v3);

    float3 ct = (v0 + v1 + v2 + v3) / 4.0;
    // float rad = length(ct - c0);

    //  return length(p - ct) - rad;

    float3 n0 = getNormal(v0, v2, v1, c0, ct);
    float3 n1 = getNormal(v0, v3, v2, c1, ct);
    float3 n2 = getNormal(v1, v3, v0, c2, ct);
    float3 n3 = getNormal(v1, v2, v3, c3, ct);

    float a = plane(p, c0, n0);
    float b = plane(p, c1, n1);
    float c = plane(p, c2, n2);
    float d = plane(p, c3, n3);
    return fmaxf(fmaxf(a, b), fmaxf(c, d));
}

RT_CALLABLE_PROGRAM float sdfSphere2(float3 p, float rad)
{
    return length(p) - rad;
}

//for molecules
RT_CALLABLE_PROGRAM float sdfSphere(float3 p, float3 rad)
{
    return length(p) - rad.x;
}
//for molecules
RT_CALLABLE_PROGRAM float bond(float3 p, float3 a, float3 b, float r)
{
    float3 pa = p - a;
    float3 ba = b - a;
    float h = clamp(dot(pa, ba) / dot(ba, ba), 0.0, 1.0);
    return length(pa - ba*h) - r;
}

RT_CALLABLE_PROGRAM float sdf_opSmoothUnion(float d1, float d2, float k) {
    //float k = 0.2; //0.7

    float val = 0.5f + 0.5f*(d2 - d1) / k;
    float h = clamp(val, 0.0f, 1.0f);
    float mix_res = (1.0 - h)*d2 + d1*h;
    return mix_res - k*h*(1.0 - h);
}

RT_CALLABLE_PROGRAM float sdfBondSphereBlob(float3 p, float3 a, float3 b, float rad1, float rad2)
{
    //float bf = bond(p, a, b, 0.1);
    float f1 = length(p - a) - rad1 / 2;
    float f2 = length(p - b) - rad2 / 2;

    // if (f1 <= 0) return f1;
    // if (f2 <= 0) return f2;
    // float3 dir = normalize(a - b);
    float rb = length((a - b) / 2);// + fmax(rad1, rad2);
    //float f = sdf_opSmoothUnion(f1, bf,0.8);
    //return sdf_opSmoothUnion(f2, f);
    float bf = bond(p, a, b, 0.1);
    if (f1 < rb)
        // bf = fminf(f1, bf);
        bf = sdf_opSmoothUnion(f1, bf, varBlob);
    if (f2 < rb)
        bf = sdf_opSmoothUnion(f2, bf, varBlob);
    //bf = fminf(f2, bf);//sdf_opSmoothUnion(f2, bf, 0.2);//
//float f11 = sdfMicrostructure2(p - a, rad1);*/
    return bf;
    /*bf = sdf_opSmoothUnion(f1, bf, 0.1);
    return sdf_opSmoothUnion(f2, bf, 0.1);*/
}

RT_CALLABLE_PROGRAM float sdfBondSphereMolBlob(float3 p, float3 a, float3 b, float3 c, float rad1, float rad2, float rad3)
{
    //float bf = bond(p, a, b, 0.1);
    float f1 = length(p - a) - rad1;
    float f2 = length(p - b) - rad2;
    float f3 = length(p - c) - rad3;

    //float f = sdf_opSmoothUnion(f1, bf,0.8);
    //return sdf_opSmoothUnion(f2, f);
    float f = sdf_opSmoothUnion(f2, f1, 1.0);
    return  sdf_opSmoothUnion(f3, f, 1.0);
}

RT_CALLABLE_PROGRAM float sminp(float a, float b, float k)
{
    float h = optix::max(k - abs(a - b), 0.0) / k;
    return optix::min(a, b) - h*h*k*(1.0 / 4.0);
}
RT_CALLABLE_PROGRAM float sdfMicrostructure4(float3 p, float rad1)
{
    float t = 2;

    float3 scale = 0.3*make_float3(abs(sin(t)), abs(cos(t)), abs(cos(t)));
    float dens_scale = abs(cos(t)) / 2;

    // float sphere1 = optix::length(p) - rad.x;// sdSphere(p, rad);
    // float sphere2 = optix::length(p) - rad.x / 2.2;// sdSphere(p, rad / 20);

    float rad2 = rad1 - 0.1;
    float sphere1 = optix::length(p) - rad2;
    float sphere2 = optix::length(p) - (rad2 - 0.1);
    float shell = max(sphere1, -sphere2);
    shell = max(shell, p.z);

    float3 tiled = make_float3(dens_scale);
    float3 tiled2 = 0.2 + tiled;

    float3 x = p + 0.5*tiled;
    // x - y * floor(x / y).
    float3 mod = x - tiled2*floor(x / tiled2);//modf(p + 0.5*tiled, tiled)
    float3 inX = mod - 0.5*tiled2;

    float3 c = make_float3(0., 0., 0.03 + 0.06*dens_scale);
    float cyly = length(make_float2(inX.x, inX.z) - make_float2(c.x, c.y)) - c.z;
    float cylx = length(make_float2(inX.y, inX.z) - make_float2(c.x, c.y)) - c.z;
    float cylz = length(make_float2(inX.x, inX.y) - make_float2(c.x, c.y)) - c.z;

    float mics = sminp(cylx, sminp(cyly, cylz, 0.08), 0.08);

    float res = sminp(shell, mics, 0.1);
    res = max(max(res, sphere1), x.z);
    // res = max(res, x.z);
    return res;
}

RT_CALLABLE_PROGRAM float sdfMicrostructure2(float3 p, float rad1)
{
    float t = TimeSound * 4;

    float3 scale = 0.3*make_float3(abs(sin(t)), abs(cos(t)), abs(cos(t)));
    float dens_scale = abs(cos(t)) / 2;

    // float sphere1 = optix::length(p) - rad.x;// sdSphere(p, rad);
    // float sphere2 = optix::length(p) - rad.x / 2.2;// sdSphere(p, rad / 20);

    float rad2 = rad1 - 0.1;
    float sphere1 = optix::length(p) - rad2;
    float sphere2 = optix::length(p) - (rad2 - 0.1);
    float shell = max(sphere1, -sphere2);
    shell = max(shell, p.z);

    float3 tiled = make_float3(dens_scale);
    float3 tiled2 = 0.2 + tiled;

    float3 x = p + 0.5*tiled;
    // x - y * floor(x / y).
    float3 mod = x - tiled2*floor(x / tiled2);//modf(p + 0.5*tiled, tiled)
    float3 inX = mod - 0.5*tiled2;

    float3 c = make_float3(0., 0., 0.03 + 0.06*dens_scale);
    float cyly = length(make_float2(inX.x, inX.z) - make_float2(c.x, c.y)) - c.z;
    float cylx = length(make_float2(inX.y, inX.z) - make_float2(c.x, c.y)) - c.z;
    float cylz = length(make_float2(inX.x, inX.y) - make_float2(c.x, c.y)) - c.z;

    float mics = sminp(cylx, sminp(cyly, cylz, 0.08), 0.08);

    float res = sminp(shell, mics, 0.1);
    res = max(max(res, sphere1), x.z);
    // res = max(res, x.z);
    return res;
}
RT_CALLABLE_PROGRAM float sdfBondSphere1(float3 p, float3 a, float3 b, float rad1, float rad2)
{
    //  float f11 = length(p - a + 0.2*make_float3(sin(TimeSound * 20), cos(TimeSound * 20), 0)) - rad1 / 2;
    //  float f21 = length(p - b + 0.2*make_float3(cos(TimeSound * 20), -sin(TimeSound), 0)) - rad2 / 2;

     // float f12 = length(p - a + 0.2*make_float3(-sin(TimeSound * 20), cos(TimeSound * 20), 0)) - rad1 / 2;
     // float f22 = length(p - b + 0.2*make_float3(-cos(TimeSound * 20), sin(TimeSound * 20), 0)) - rad2 / 2;

    //  float f1 = sdf_opSmoothUnion(f11, f12, 0.2);
    //  float f2 = sdf_opSmoothUnion(f21, f22, 0.2);

    float3 vib1 = make_float3(0);
    float3 vib2 = make_float3(0);
    float3 up = make_float3(0, 0, 1);

    float bf = bond(p, a, b, 0.1);

    float f1 = length(p - (a)) - rad1 / 2; //first sphere
    float f2 = length(p - (b)) - rad2 / 2; //second sphere

    //-----

    float f = sdf_opSmoothUnion(f1, bf, 0.2);
    float mod1 = sdf_opSmoothUnion(f2, f, 0.1);

    float mod2 = sdfBondSphereBlob(p, a, b, rad1, rad2);

    //manual interpolation
    //return mod1*MultiscaleParam +(1.0-MultiscaleParam)*mod2;

    return mod1;
    //return sdf_opSmoothUnion(f2, f1, 0.5);
    //return f2;
}

RT_CALLABLE_PROGRAM float sdfBondSphere_simple(float3 p, float3 a, float3 b, float rad1, float rad2)
{
    //  float f11 = length(p - a + 0.2*make_float3(sin(TimeSound * 20), cos(TimeSound * 20), 0)) - rad1 / 2;
    //  float f21 = length(p - b + 0.2*make_float3(cos(TimeSound * 20), -sin(TimeSound), 0)) - rad2 / 2;

    // float f12 = length(p - a + 0.2*make_float3(-sin(TimeSound * 20), cos(TimeSound * 20), 0)) - rad1 / 2;
    // float f22 = length(p - b + 0.2*make_float3(-cos(TimeSound * 20), sin(TimeSound * 20), 0)) - rad2 / 2;

    //  float f1 = sdf_opSmoothUnion(f11, f12, 0.2);
    //  float f2 = sdf_opSmoothUnion(f21, f22, 0.2);

    float3 vib1 = make_float3(0);
    float3 vib2 = make_float3(0);
    float3 up = make_float3(0, 0, 1);

    float t = sin(TimeSound * 4);
    vib2 = (b - a) / 2 * t;

    float3 c = cross(a, b);
    // vib2 += -t*optix::normalize(c) / 10;

    float3 cent1 = a - vib2;
    float3 cent2 = b + vib2;
    float len = length(b - a);
    // len = 1;

     //todo: evaluate withi sphre bounds
    // const float t = length(p - theRay.origin);
    // const float3 pos_along_ray = theRay.origin + theRay.direction * t;
    int eval = 0;
    float f = -100000.0;
    //if (length(p - (a + b) / 2) <= len / 2)// && rtPotentialIntersection(tt))
    f = bond(p, a, b, 0.1);

    //float f = bf;
    if (length(p - a) < rad1 / 1.5) {
        float f1 = length(p - (a)) - rad1 / 2;// *len* (abs(t)+1)/6; //first sphere //sdfMicrostructure2 (p-a, len*rad1 * (abs(t)+1)/6); //
       // if (f > -0.5)
        f = sdf_opSmoothUnion(f1, f, 0.1);
        //  else f = f1;
    }

    if (length(p - b) < rad2 / 1.5)
    {
        float f2 = length(p - (b)) - rad2 / 2;//*len* (abs(t)+1)/6; //second sphere   //sdfMicrostructure2 (p-b, len*rad1 * (abs(t)+1)/6);//
       // if (f > -0.5)
        f = sdf_opSmoothUnion(f2, f, 0.1);
        //  else f = f2;
    }
    //f1 = length(p - (a + vib2)) - rad1 / 1.2;
    return f;
}

RT_CALLABLE_PROGRAM float sdfCrazyBond(float3 p, float3 a, float3 b, float rad1, float rad2)
{
    float3 vib1 = make_float3(0);
    float3 vib2 = make_float3(0);
    float3 up = make_float3(0, 0, 1);

    float t = sin(TimeSound * 4);
    vib2 = (b - a) / 2.2 * t;

    float3 c = cross(a, b);
    // vib2 += -t*optix::normalize(c) / 10;

    float bf = bond(p, a - vib2, b + vib2, 0.1);
    float len = length(b - a);
    len = 1;

    float f1 = length(p - (a)) - rad1;// *len* (abs(t)+1)/6; //first sphere //sdfMicrostructure2 (p-a, len*rad1 * (abs(t)+1)/6); //
    float f2 = length(p - (b)) - rad2;//*len* (abs(t)+1)/6; //second sphere   //sdfMicrostructure2 (p-b, len*rad1 * (abs(t)+1)/6);//

    if (f1 < 0.5) {
        if (t > 1)
            f1 = sdfMicrostructure4(p - a, rad1);
    }
    if (f2 < 0.5) {
        if (t > 1)
            f2 = sdfMicrostructure4(p - b, rad2);
    }

    return sdf_opSmoothUnion(f1, f2, 0.2);
}
RT_CALLABLE_PROGRAM float sdfBondSphere(float3 p, float3 a, float3 b, float rad1, float rad2)
{
    float3 vib1 = make_float3(0);
    float3 vib2 = make_float3(0);
    float3 up = make_float3(0, 0, 1);

    float t2 = sin(TimeSound);
    float t = sin(TimeSound * 4);
    vib2 = (b - a) / 2.5 * t;

    float3 c = cross(a, b);
    // vib2 += -t*optix::normalize(c) / 10;

    float bf = bond(p, a - vib2, b + vib2, 0.1);
    float len = length(b - a);
    len = 1;

    float f1 = length(p - (a)) - rad1 / 2;// *len* (abs(t)+1)/6; //first sphere //sdfMicrostructure2 (p-a, len*rad1 * (abs(t)+1)/6); //
    float f2 = length(p - (b)) - rad2 / 2;//*len* (abs(t)+1)/6; //second sphere   //sdfMicrostructure2 (p-b, len*rad1 * (abs(t)+1)/6);//

   /* if (f1 < 0.5) {
        if (TimeSound > 2)
            f1 = sdfMicrostructure4(p - a, rad1);
    }
    if (f2 < 0.5) {
        if (TimeSound > 2)
            f2 = sdfMicrostructure4(p - b, rad2);
    }*/

    //-----
    float f3 = length(p - (b + vib2)) - rad2 / 4;
    float f4 = length(p - (a - vib2)) - rad1 / 4;

    //float f3 = sdfMicrostructure2(p - (b + vib2), rad2/2);
   //float f4 = sdfMicrostructure2(p - (a - vib2), rad1/2);

    f1 = abs(t)*f3 + (1 - abs(t))*f1;
    f2 = abs(t)*f4 + (1 - abs(t))*f2;

    float f = bf;
    if (f1 <= 0.5)
        f = sdf_opSmoothUnion(f1, f, 0.1);
    if (f2 <= 0.5)
        f = sdf_opSmoothUnion(f2, f, 0.1);

    float dB = length(p - (a + b) / 2) - rad2;
    if (dB <= 0.5) {
        float mod2 = sdfMicrostructure4(p - (a + b) / 2, rad2); //sdf_opSmoothUnion(f3, f4, 0.2); //sdfBondSphereBlob(p, a, b, rad1, rad2);
        f = abs(t2)*f + (1 - abs(t2))*mod2;
    }
    else {
        if (TimeSound < 1.0)
            f = abs(t2)*f + (1 - abs(t2))*dB;
    }
    return f;
}

/*------------*/
__device__
inline float Tetra(float3 p, float3 v0, float3 v1, float3 v2, float3 v3)
{
    float3 c0 = getCenter(v0, v2, v1);
    float3 c1 = getCenter(v0, v3, v2);
    float3 c2 = getCenter(v1, v3, v0);
    float3 c3 = getCenter(v1, v2, v3);

    float3 ct = (v0 + v1 + v2 + v3) / 4.0f;
    //float rad1 = length(ct - c0);
    float rad = length(ct - v0);
    //rad = (rad + rad1) / (2.0*t);
    float3 n0 = getNormal(v0, v2, v1, c0, ct);
    float3 n1 = getNormal(v0, v3, v2, c1, ct);
    float3 n2 = getNormal(v1, v3, v0, c2, ct);
    float3 n3 = getNormal(v1, v2, v3, c3, ct);

    float a = plane(p, c0, n0);
    float b = plane(p, c1, n1);
    float c = plane(p, c2, n2);
    float d = plane(p, c3, n3);
    return fmaxf(fmaxf(a, b), fmaxf(c, d));
}
__device__
inline float TetraWire(float3 p, float3 v0, float3 v1, float3 v2, float3 v3)
{
    float f1 = bond(p, v0, v1, 0.4);
    float f2 = bond(p, v0, v2, 0.4);
    float f3 = bond(p, v0, v3, 0.4);
    float f4 = bond(p, v1, v2, 0.4);
    float f5 = bond(p, v1, v3, 0.4);

    f1 = sminp(f2, f1, 0.4);
    f1 = sminp(f3, f1, 0.4);
    f1 = sminp(f4, f1, 0.4);
    f1 = sminp(f5, f1, 0.4);
    return f1;
}

__device__
inline float TetraWire2(float3 p, float3 v0, float3 v1, float3 v2, float3 v3)
{
    float3 ct = (v0 + v1 + v2 + v3) / 4.0f;
    float3 c0 = getCenter(v0, v2, v1);
    float3 c1 = getCenter(v0, v3, v2);
    float3 c2 = getCenter(v1, v3, v0);
    float3 c3 = getCenter(v1, v2, v3);
    float r = length(ct - v0) / 20;

    float f1 = bond(p, ct, c1, r);
    float f2 = bond(p, ct, c2, r);
    float f3 = bond(p, ct, c3, r);
    float f4 = bond(p, ct, c0, r);

    f1 = sminp(f2, f1, 0.8);
    f1 = sminp(f3, f1, 0.8);
    f1 = sminp(f4, f1, 0.8);
    return f1;
}

/* For multi-scale molecule*/
RT_CALLABLE_PROGRAM float sdfMicroCell(float3 p, float3 b, float3 c, float3 d, float3 e, float rad1, float rad2)
{
    float3 a = (b + c + d + e) / 4.0;
    float f1 = length(p - a) - rad1 / 2;
    float f;
    //if (TimeSound < 1.0) {
    float bf1 = bond(p, a, b, 0.1);
    float bf2 = bond(p, a, c, 0.1);
    float bf3 = bond(p, a, d, 0.1);
    float bf4 = bond(p, a, e, 0.1);

    float f2 = length(p - b) - rad2 / 2;
    float f3 = length(p - c) - rad2 / 2;
    float f4 = length(p - d) - rad2 / 2;
    float f5 = length(p - e) - rad2 / 2;

    f = sdf_opSmoothUnion(bf2, bf1, 0.1);
    f = sdf_opSmoothUnion(f, bf3, 0.1);
    f = sdf_opSmoothUnion(f, bf4, 0.1);
    f = sdf_opSmoothUnion(f, f1, 0.4);
    f = sdf_opSmoothUnion(f, f2, 0.1);
    f = sdf_opSmoothUnion(f, f3, 0.1);
    f = sdf_opSmoothUnion(f, f4, 0.1);
    f = sdf_opSmoothUnion(f, f5, 0.1);
    //  }
    //  else f = f1;

      //--------first frame
    float totalRad = length(sCell2 - sCell3) / 2 - 3.3;
    float inputRad = abs(abs(sCell2.z - sCell3.z) / 2 - 9.9);
    float cel1Rad = 3.3 * 2;// 1.65 + 3.3;
    float cel1Rad2 = 3.3 + 1.65;

    float t = clamp(TimeSound, 1.0, 4.0);// -1.0;
    float3 cCell3 = sCell3 - make_float3(3.3*t, 3.3, 3.3);
    float3 cCell2 = sCell2 + make_float3(3.3*t, 3.3, 3.3*t);
    float3 cCell1 = sCell1 - make_float3(3.3*t, 3.3, 3.3);

    float cel1Rad3 = 3.3 + 1.65*t;

    float3 cCell4 = (cCell3 + cCell1) / 2;

    int comp1 = 0;

    if ((length(p - cCell1) - cel1Rad) <= 0)
        comp1 = 1;
    if ((length(p - cCell2) - cel1Rad2) <= 0)
        comp1 = 2;
    if ((length(p - cCell3) - cel1Rad) <= 0)
        comp1 = 3;
    //if ((length(p - cCell4) - cel1Rad) <= 0)
    //    comp1 = 4;

    float rad = length(a - b);
    if ((length(a - cCell4) - cel1Rad3 / 2 - rad) <= 0) //-rad fix
        comp1 = 4;

    if ((length(a - cCell2) - cel1Rad2 + rad) <= 0)
        comp1 = 5;

    if (comp1 > 0)
    {
        //check for central only

        float bB = 3.3f / 2.0f;

        float rB = length(make_float3(bB));
        // float3 bmax = a + make_float3(bB);
         //float3 bmax = a + make_float3(bB);
         //float rB = length(bmax);
        float3 nb = normalize(b - a);
        float lb = length(b - a);
        float3 cb = a + nb*(rB - lb);

        float ft1 = length(p - cb) - rad2 / 2;
        float bt1 = bond(p, b, cb, 0.1);

        //----------
        float3 nc = normalize(c - a);
        float3 cc = a + nc*(rB - lb);

        float ft2 = length(p - cc) - rad2 / 2;
        float bt2 = bond(p, c, cc, 0.1);

        //----------
        float3 nd = normalize(d - a);
        float3 cd = a + nd*(rB - lb);

        float ft3 = length(p - cd) - rad2 / 2;
        float bt3 = bond(p, d, cd, 0.1);

        //----------
        float3 ne = normalize(e - a);
        float3 ce = a + ne*(rB - lb);

        float ft4 = length(p - ce) - rad2 / 2;
        float bt4 = bond(p, e, ce, 0.1);

        float fb = f;
        if (comp1 == 5) {
            fb = Tetra(p, b, c, d, e);
            fb = sdf_opSmoothUnion(fb, f, 0.4);
        }
        if (comp1 == 4)
        {
            float blScale = 1.0;
            float3 c1 = a;

            c1.x += bB;
            float rad = length(a - b);
            fb = length(p - a) - bB / 2;// sdfMicrostructure4(p - a, bB);

            if ((length(c1 - cCell4) - cel1Rad3 / 2) <= 0)
            {
                float f1 = length(p - c1) - bB / 4;
                fb = sdf_opSmoothUnion(fb, f1, blScale);
            }
            c1 = a;
            c1.x -= bB;
            if ((length(c1 - cCell4) - cel1Rad3 / 2) <= 0)
            {
                float f1 = length(p - c1) - bB / 4;
                fb = sdf_opSmoothUnion(fb, f1, blScale);
            }
            c1 = a;
            c1.y += bB;

            if ((length(c1 - cCell4) - cel1Rad3 / 2) <= 0)
            {
                float f1 = length(p - c1) - bB / 4;
                fb = sdf_opSmoothUnion(fb, f1, blScale);
            }

            c1 = a;
            c1.y -= bB;

            if ((length(c1 - cCell4) - cel1Rad3 / 2) <= 0)
            {
                float f1 = length(p - c1) - bB / 4;
                fb = sdf_opSmoothUnion(fb, f1, blScale);
            }
            c1 = a;
            c1.z += bB;

            if ((length(c1 - cCell4) - cel1Rad3 / 2) <= 0)
            {
                float f1 = length(p - c1) - bB / 4;
                fb = sdf_opSmoothUnion(fb, f1, blScale);
            }
            c1 = a;
            c1.z -= bB;

            if ((length(c1 - cCell4) - cel1Rad3 / 2) <= 0)
            {
                float f1 = length(p - c1) - bB / 4;
                fb = sdf_opSmoothUnion(fb, f1, blScale);
            }

            c1 = cb;

            if ((length(c1 - cCell4) - cel1Rad3 / 2) <= 0)
            {
                float f1 = length(p - c1) - bB / 4;
                fb = sdf_opSmoothUnion(fb, f1, blScale);
            }

            c1 = cd;

            if ((length(c1 - cCell4) - cel1Rad3 / 2) <= 0)
            {
                float f1 = length(p - c1) - bB / 4;
                fb = sdf_opSmoothUnion(fb, f1, blScale);
            }

            c1 = cc;

            if ((length(c1 - cCell4) - cel1Rad3 / 2) <= 0)
            {
                float f1 = length(p - c1) - bB / 4;
                fb = sdf_opSmoothUnion(fb, f1, blScale);
            }
            c1 = ce;

            if ((length(c1 - cCell4) - cel1Rad3 / 2) <= 0)
            {
                float f1 = length(p - c1) - bB / 4;
                fb = sdf_opSmoothUnion(fb, f1, blScale);
            }

            //---------------
            float sphere1 = fb;
            float tm = 2;

            float3 scale = 0.3*make_float3(abs(sin(tm)), abs(cos(tm)), abs(cos(tm)));
            float dens_scale = abs(cos(tm)) / 2;

            float fb2 = fb + 0.1;
            float shell = max(fb, -fb2);

            fb = shell;
            /*
            float3 tiled = make_float3(dens_scale);
            float3 tiled2 = 0.2 + tiled;

            float3 x = p + 0.5*tiled;
            // x - y * floor(x / y).
            float3 mod = x - tiled2*floor(x / tiled2);//modf(p + 0.5*tiled, tiled)
            float3 inX = mod - 0.5*tiled2;

            float3 c = make_float3(0., 0., 0.03 + 0.06*dens_scale);
            float cyly = length(make_float2(inX.x, inX.z) - make_float2(c.x, c.y)) - c.z;
            float cylx = length(make_float2(inX.y, inX.z) - make_float2(c.x, c.y)) - c.z;
            float cylz = length(make_float2(inX.x, inX.y) - make_float2(c.x, c.y)) - c.z;

            float mics = sminp(cylx, sminp(cyly, cylz, 0.08), 0.08);

            fb = sminp(shell, mics, 0.1);
            fb = max(max(fb, sphere1), x.z);*/
        }
        float dB = sdf_opSmoothUnion(fb, ft1, 0.1);
        dB = min(dB, bt1);

        dB = sdf_opSmoothUnion(dB, ft2, 0.1);
        dB = min(dB, bt2);

        dB = sdf_opSmoothUnion(dB, ft3, 0.1);
        dB = min(dB, bt3);

        dB = sdf_opSmoothUnion(dB, ft4, 0.1);
        dB = min(dB, bt4);

        if (TimeSound < 1.0) {
            float t2 = TimeSound;
            f = abs(t2)*dB + (1 - abs(t2))*f;
        }
        else f = dB;
    }

    if (comp1 > 1) {
        float bB = 3.3f / 2.0f;

        float3 cb1 = a;
        float3 cb = a;
        cb.y += bB;
        cb1.y -= bB;

        float bt1 = bond(p, cb1, cb, 0.1);
        float ft3 = length(p - cb1) - rad2 / 2;
        float ft2 = length(p - cb) - rad2 / 2;
        //float fb = Tetra(p, b, c, d, e);
        float dB = sdf_opSmoothUnion(f, bt1, 0.2);
        dB = sdf_opSmoothUnion(dB, ft2, 0.2);
        dB = sdf_opSmoothUnion(dB, ft3, 0.2);
        if (TimeSound < 1.0) {
            float t2 = TimeSound;
            f = abs(t2)*dB + (1 - abs(t2))*f;
        }
        else f = dB;
    }
    if (comp1 > 2) {
        float bB = 3.3f / 2.0f;

        float3 cb1 = a;
        float3 cb = a;
        cb.x += bB;
        cb1.x -= bB;

        float bt1 = bond(p, cb1, cb, 0.1);
        float ft3 = length(p - cb1) - rad2 / 2;
        float ft2 = length(p - cb) - rad2 / 2;
        float dB = sdf_opSmoothUnion(f, bt1, 0.2);
        dB = sdf_opSmoothUnion(dB, ft2, 0.2);
        dB = sdf_opSmoothUnion(dB, ft3, 0.2);
        if (TimeSound < 1.0) {
            float t2 = TimeSound;
            f = abs(t2)*dB + (1 - abs(t2))*f;
        }
        else f = dB;
    }

    return f;
}

RT_CALLABLE_PROGRAM float sdfMolBondSphere(float3 p, float3 a, float3 b, float3 c, float rad1, float rad2, float rad3)
{
    //first vibration vector in molecule
    float3 vib2[3];
    float3 vib3[3];
    float f1[3];
    float f2[3];
    float f3[3];

    float bf[3];
    float bf2[3];

    vib2[0] = (b - a) / 3.5 * -cos(TimeSound * 20);
    vib3[0] = (c - a) / 3.5 * (cos(TimeSound * 20));

    vib2[1] = (b - a) / 3.5 * sin(TimeSound * 20);
    vib3[1] = (c - a) / 3.5 * (sin(TimeSound * 20));

    float3 vib_dir = b - 2 * a + c;

    vib2[2] = vib_dir / 3.5 * sin(TimeSound * 20);
    vib3[2] = vib_dir / 3.5 * (sin(TimeSound * 20));

    float f[3];
    float num = 3;
    if (MultiscaleParam <= 1)  num = 1;
    else
    {
        if (MultiscaleParam <= 2)
            num = 2;
    }
    for (int i = 0; i < num; i++)
    {
        //computation of atoms
        f1[i] = length(p - a) - rad1 / 2;
        f2[i] = length(p - (b + vib2[i])) - rad2 / 2;
        f3[i] = length(p - (c + vib3[i])) - rad3 / 2;

        //chose subspace

        f[i] = fminf(fminf(f1[i], f2[i]), f3[i]);

        if (f[i] == f1[i]) //trace central part
        {
            //f[i]+=rad1/2; //get proper radius
            bf[i] = bond(p, a, (b + vib2[i]), 0.06);
            bf2[i] = bond(p, a, (c + vib3[i]), 0.06);
            f[i] = sdf_opSmoothUnion(f[i], bf[i], 0.09);
            f[i] = sdf_opSmoothUnion(f[i], bf2[i], 0.09);
        }
        else {
            if (f[i] == f2[i]) //trace left part
            {
                //f[i]+=rad2/2; //get proper radius
                bf[i] = bond(p, a, (b + vib2[i]), 0.06);
                f[i] = sdf_opSmoothUnion(f[i], bf[i], 0.09);
            }
            else {
                //f[i]+=rad3/2; //get proper radius
                bf2[i] = bond(p, a, (c + vib3[i]), 0.06);
                f[i] = sdf_opSmoothUnion(f[i], bf2[i], 0.09);
            }
        }
    }
    float mod1 = f[0];
    if (MultiscaleParam <= 1)
    {
        mod1 = f[0];
    }
    else
    {
        if (MultiscaleParam <= 2)
        {
            mod1 = sdf_opSmoothUnion(f[0], f[1], 0.1);
        }
        else
        {
            if (MultiscaleParam <= 3)
            {
                mod1 = sdf_opSmoothUnion(f[0], f[1], 0.1);
                mod1 = sdf_opSmoothUnion(mod1, f[2], 0.1);
            }
        }
    }

    // mod1 = sdf_opSmoothUnion(f[0], f[1], 0.1);
    // mod1 = sdf_opSmoothUnion(mod1, f[2], 0.1);

    float mod2 = sdfBondSphereMolBlob(p, a, b, c, rad1, rad2, rad3);

    //manual interpolation
    //return mod1*MultiscaleParam +(1.0-MultiscaleParam)*mod2;
    float dist = length(sysCameraPosition - (a + b + c) / 3);

    if (dist < 20.0)
    {
        //consider the center=0;
       // float3 camV = make_float3(0) - sysCameraPosition;
      //  float3
      //  if (dot(sysCameraPosition, (a + b) / 2) > 0)
        {
            float interp = (dist - 10) / 10.0;
            float d = optix::clamp(interp, 0.0, 1.0);
            return mod2*d + (1.0 - d)*mod1;
        }
        //    else return mod2;
    }
    else
        return mod2;
    //return sdf_opSmoothUnion(f2, f1, 0.2);
}

RT_CALLABLE_PROGRAM float sdfDynSphere(float3 p, float3 p2, float3 rad)
{
    //float3 pp = p2*TimeSound + (1.0 - TimeSound)*p;

    float f1 = length(p) - rad.x;
    float f2 = length(p2) - rad.x;// - make_float3(0.5, 0, 0)
    float f = sdf_opSmoothUnion(f1, f2, 0.7);
    for (int i = 0; i <= 10; i++)
    {
        float3 pp = p2*i / 10.0 + (1.0 - i / 10.0)*p;
        f1 = length(pp) - rad.x;
        f = sdf_opSmoothUnion(f, f1, 0.7);
    }
    return  f;//f2*TimeSound + (1.0 - TimeSound)*f1;
}

RT_CALLABLE_PROGRAM float sdfDynSphere2(float3 p, float3 p2, float3 rad)
{
    float3 pp = p2*TimeSound + (1.0 - TimeSound)*p;
    return  length(pp) - rad.x;
}

//primitives
RT_CALLABLE_PROGRAM float sdSphere(float3 p, float3 rad)
{
    return length(p - varCenter) - varRadius.x;
}

RT_CALLABLE_PROGRAM float sdfHand(float3 p, float3 rad)
{
    float3 shift = make_float3(-1, -1, -1);
    float f_m = length(p - (varCenter + shift)) - varRadius.x / 14;
    float f_0 = length(p - (varCenter0 + varCenter) / 2 - shift) - varRadius.x / 16;
    float f_1 = length(p - (varCenter1 + varCenter) / 2 - shift) - varRadius.x / 16;
    float f_2 = length(p - (varCenter2 + varCenter) / 2 - shift) - varRadius.x / 16;
    float f_3 = length(p - (varCenter3 + varCenter) / 2 - shift) - varRadius.x / 16;
    float f_4 = length(p - (varCenter4 + varCenter) / 2 - shift) - varRadius.x / 16;

    float3 b = (varCenter0 + varCenter) / 2.0 + shift;
    f_m = bond(p, varCenter + shift, b, 0.02);
    f_m = sdf_opSmoothUnion(f_m, f_0, 0.1);

    f_m = sdf_opSmoothUnion(f_m, f_1, 0.1);
    b = (varCenter1 + varCenter) / 2.0 + shift;
    f_m = sdf_opSmoothUnion(f_m, bond(p, varCenter + shift, b, 0.02), 0.1);

    f_m = sdf_opSmoothUnion(f_m, f_2, 0.1);
    b = (varCenter2 + varCenter) / 2.0 + shift;
    f_m = sdf_opSmoothUnion(f_m, bond(p, varCenter + shift, b, 0.02), 0.1);

    f_m = sdf_opSmoothUnion(f_m, f_3, 0.1);
    b = (varCenter3 + varCenter) / 2.0 + shift;
    f_m = sdf_opSmoothUnion(f_m, bond(p, varCenter + shift, b, 0.02), 0.1);

    f_m = sdf_opSmoothUnion(f_m, f_4, 0.1);
    b = (varCenter4 + varCenter) / 2.0 + shift;
    f_m = sdf_opSmoothUnion(f_m, bond(p, varCenter + shift, b, 0.02), 0.1);

    return f_m;
}

RT_CALLABLE_PROGRAM float sdfMicrostructure_test(float3 pos, float3 rad)
{
    float radc = 5.0;
    float scale = 1.8;
    float3 cent1 = make_float3(radc, 0, 0);
    float3 cent2 = make_float3(-radc, 0, 0);
    float3 cent3 = make_float3(0, radc, 0);
    float3 cent4 = make_float3(0, -radc, 0);
    float d1 = length(pos - cent1) - radc / scale;
    float d2 = length(pos - cent2) - radc / scale;
    float d3 = length(pos - cent3) - radc / scale;
    float d4 = length(pos - cent4) - radc / scale;

    float d5 = min(d1, d2);
    d5 = min(d5, d3);
    d5 = min(d5, d4);
    d5 -= scale * 1.5;
    d5 = -d5;

    float f = min(d5, d4);
    f = min(f, d3);
    f = min(f, d2);
    f = min(f, d1);

    float3 b = make_float3(5.0);// optix::make_float3(rad);
    float3 d = abs(pos) - b;
    float box = length(max(d, make_float3(0.0f)))
        + min(max(d.x, max(d.y, d.z)), 0.0f);
    f = max(f, box);
    return box;
}

RT_CALLABLE_PROGRAM float sdfMicrostructure(float3 pp, float3 rad)
{
    float t = TimeSound * 4;

    float3 p = pp - varCenter;
    float3 scale = 0.3*make_float3(abs(sin(t)), abs(cos(t)), abs(cos(t)));
    float dens_scale = abs(cos(t)) / 2;

    // float sphere1 = optix::length(p) - rad.x;// sdSphere(p, rad);
    // float sphere2 = optix::length(p) - rad.x / 2.2;// sdSphere(p, rad / 20);

    float rad2 = varRadius.x - 0.1;
    float sphere1 = optix::length(p) - rad2;
    float sphere2 = optix::length(p) - (rad2 - 0.1);
    float shell = max(sphere1, -sphere2);
    shell = max(shell, p.z);

    float3 tiled = make_float3(dens_scale);
    float3 tiled2 = 0.2 + tiled;

    float3 x = p + 0.5*tiled;
    // x - y * floor(x / y).
    float3 mod = x - tiled2*floor(x / tiled2);//modf(p + 0.5*tiled, tiled)
    float3 inX = mod - 0.5*tiled2;

    float3 c = make_float3(0., 0., 0.03 + 0.06*dens_scale);
    float cyly = length(make_float2(inX.x, inX.z) - make_float2(c.x, c.y)) - c.z;
    float cylx = length(make_float2(inX.y, inX.z) - make_float2(c.x, c.y)) - c.z;
    float cylz = length(make_float2(inX.x, inX.y) - make_float2(c.x, c.y)) - c.z;

    float mics = sminp(cylx, sminp(cyly, cylz, 0.08), 0.08);

    float res = sminp(shell, mics, 0.1);
    res = max(max(res, sphere1), x.z);
    // res = max(res, x.z);
    return res;
}

RT_CALLABLE_PROGRAM float sdfBox(float3 p, float3 rad)
{
    float3 b = varRadius;// optix::make_float3(rad);
    float3 d = abs(p - varCenter) - b;
    return length(max(d, make_float3(0.0f)))
        + min(max(d.x, max(d.y, d.z)), 0.0f); // remove this line for an only partially signed sdf
}
RT_CALLABLE_PROGRAM float sdfRoundBox(float3 p, float3 rad)
{
    float3 b = rad;// optix::make_float3(rad);
    float r = 0.1;
    float3 d = abs(p - varCenter) - b;
    return length(max(d, make_float3(0.0f))) - r
        + min(max(d.x, max(d.y, d.z)), 0.0f); // remove this line for an only partially signed sdf
}

RT_CALLABLE_PROGRAM float sdfTorus(float3  p, float3 rad)
{
    float2 t = varT;
    float3 pp = p - varCenter;
    float2 xz = make_float2(pp.x, pp.z);
    float2 q = make_float2(length(xz) - t.x, pp.y);
    return length(q) - t.y;
}

//operation
RT_CALLABLE_PROGRAM float opRound(float3 p, float3 rad)
{
    //TODO: the rad should be from primitive
    return sdfOpPrim(p, rad) - varK;
}

RT_CALLABLE_PROGRAM float opElongate(float3 p, float3 rad)
{
    float3 h = varRadius;
    float3 q = p - clamp(p, -h, h);
    //should be initial primitive Rad
    return sdfOpPrim(q, rad);
}

RT_CALLABLE_PROGRAM float opSmoothUnion(float3 p, float3 rad) {
    float k = varK;
    float d1 = sdfOpPrim(p, rad);
    float d2 = sdfOpPrim2(p, rad);
    float val = 0.5f + 0.5f*(d2 - d1) / k;
    float h = clamp(val, 0.0f, 1.0f);
    float mix_res = (1.0 - h)*d2 + d1*h;
    return mix_res - k*h*(1.0 - h);
}

RT_CALLABLE_PROGRAM float opSmoothIntersection(float3 p, float3 rad) {
    float k = varK;
    float d1 = sdfOpPrim(p, rad);
    float d2 = sdfOpPrim2(p, rad);
    float val = 0.5f - 0.5f*(d2 - d1) / k;
    float h = clamp(val, 0.0f, 1.0f);
    float mix_res = (1.0 - h)*d2 + d1*h;
    return mix_res + k*h*(1.0 - h);
}

RT_CALLABLE_PROGRAM float opSmoothSubtraction(float3 p, float3 rad) {
    float k = varK;
    float d1 = sdfOpPrim(p, rad);
    float d2 = sdfOpPrim2(p, rad);
    float val = 0.5f - 0.5f*(d2 - d1) / k;
    float h = clamp(val, 0.0f, 1.0f);
    float mix_res = (1.0 - h)*d2 - d1*h;
    return mix_res + k*h*(1.0 - h);
}

RT_CALLABLE_PROGRAM float opSubtraction(float3 p, float3 rad)
{
    float d1 = sdfOpPrim2(p, rad);
    float d2 = sdfOpPrim(p, rad);
    return max(-d1, d2);
}

RT_CALLABLE_PROGRAM float sdfField(float3 p, float3 rad) {
    float s1;

    switch (numTexDefined) {
    case 1:
    {
        s1 = tex3D<float>(texSDF0, p.x*0.5f + 0.5f, p.y*0.5f + 0.5f, p.z*0.5f + 0.5f);
        //trace back to iso value and shift

        s1 += shift0;
        //dist=index*spacing
        // and devide by texture size so 0,1 range

        //Chamber distance neglect spacing.
        //Distance is computed in voxels

        s1 /= size0;// 138.0f;
                    //s1 /= size;
        s1 *= 2.0f;
        break;
    }
    case 2:
    {
        float s_min = tex3D(texSDF0, p.x*0.5f + 0.5f, p.y*0.5f + 0.5f, p.z*0.5f + 0.5f);
        float s_max = tex3D(texSDF1, p.x*0.5f + 0.5f, p.y*0.5f + 0.5f, p.z*0.5f + 0.5f);

        s1 = TimeSound*s_max + (1.0 - TimeSound)*s_min;//trace back to iso value and shift
        float shift = TimeSound*shift1 + (1 - TimeSound)*shift0;
        float size = TimeSound*size1 + (1 - TimeSound)*size0;

        s1 += shift;
        //dist=index*spacing
        // and devide by texture size so 0,1 range

        //Chamber distance neglect spacing.
        //Distance is computed in voxels

        //s1 /= 139.0f;
        s1 /= size;
        s1 *= 2.0f;
        break;
    }
    }
    //float size of texture is currently unsused

    return s1;
}

/* ------------
/* for heterogeneous objects
--------------*/

//Molecules
RT_CALLABLE_PROGRAM float sdfAtom(float3 p, primParamDesc desc)
{
    float3 pos = desc.pos[0];
    float rad1 = desc.rad[0];
    return length(p - pos) - rad1;
}

//Microstructure like test
RT_CALLABLE_PROGRAM float sdfMicro(float3 pp, primParamDesc desc)
{
    float3 cent = desc.pos[0];
    float rad1 = desc.rad[0];

    float t = TimeSound * 4;

    float3 p = pp - cent;
    float3 scale = 0.3*make_float3(abs(sin(t)), abs(cos(t)), abs(cos(t)));
    float dens_scale = abs(cos(t)) / 2;

    // float sphere1 = optix::length(p) - rad.x;// sdSphere(p, rad);
    // float sphere2 = optix::length(p) - rad.x / 2.2;// sdSphere(p, rad / 20);

    float rad2 = rad1 - 0.1;
    float sphere1 = optix::length(p) - rad2;
    float sphere2 = optix::length(p) - (rad2 - 0.1);
    float shell = max(sphere1, -sphere2);
    shell = max(shell, p.z);

    float3 tiled = make_float3(dens_scale);
    float3 tiled2 = 0.2 + tiled;

    float3 x = p + 0.5*tiled;
    // x - y * floor(x / y).
    float3 mod = x - tiled2*floor(x / tiled2);//modf(p + 0.5*tiled, tiled)
    float3 inX = mod - 0.5*tiled2;

    float3 c = make_float3(0., 0., 0.03 + 0.06*dens_scale);
    float cyly = length(make_float2(inX.x, inX.z) - make_float2(c.x, c.y)) - c.z;
    float cylx = length(make_float2(inX.y, inX.z) - make_float2(c.x, c.y)) - c.z;
    float cylz = length(make_float2(inX.x, inX.y) - make_float2(c.x, c.y)) - c.z;

    float mics = sminp(cylx, sminp(cyly, cylz, 0.08), 0.08);

    float res = sminp(shell, mics, 0.1);
    res = max(max(res, sphere1), x.z);
    // res = max(res, x.z);
    return res;
}

//----Bonding
RT_CALLABLE_PROGRAM float sdfBondBlob(float3 p, primParamDesc desc)
{
    float3 a = desc.pos[0];
    float rad1 = desc.rad[0];
    float3 b = desc.pos[1];
    float rad2 = desc.rad[1];
    float bf = bond(p, a, b, 0.1);
    float f1 = length(p - a) - rad1 / 2;
    float f2 = length(p - b) - rad2 / 2;

    float f = sdf_opSmoothUnion(f1, bf, 0.4);
    return sdf_opSmoothUnion(f2, f, 0.4);
    //return sdf_opSmoothUnion(f2, f1, 0.8);
}