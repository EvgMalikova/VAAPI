#include "hip/hip_runtime.h"
/*
All basic variables for SDFs visual-auditory ray-tracing
 */
#include "sdfGeometryVariables.h"

#include "sdfPrimPrograms.h"
using namespace optix;
rtDeclareVariable(float3, sysCameraPosition, , );
rtBuffer<float3>    Positions;
rtBuffer<int2>    Bonds;
rtBuffer<int2>    Mols;

rtBuffer<float>    BSRadius;
rtBuffer<int>    BSType;
rtDeclareVariable(float, MultiscaleParam, , );
rtDeclareVariable(float3, pr_pos, attribute primitive_pos, );
rtDeclareVariable(float, pr_rad, attribute primitive_rad, );
//rtDeclareVariable(int, pr_type, attribute primitive_type, );

rtDeclareVariable(float, sysSceneEpsilon, , );

//rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, ); //for direct tracing of sdf spheres, or defined set of primitives with definde BB

#include "renderer/per_ray_data.h"
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

//TODO:for SDF
//still have to be implemented as PTX should be generated automatically
//PTX can automatically generated for some primitives like spheres, boxes and etc
//consider further integration with python

//for SDF

//for dynamic staff
rtDeclareVariable(int, PNum, , );
rtDeclareVariable(int, numFrames, , );
rtDeclareVariable(float, TimeSound, , );

inline __device__ float evaluateFunction(float3 x, primParamDesc descPrim)
{
    int type = descPrim.type;
    float f = 10000.0;
    float3 pos1 = descPrim.pos[0];
    float rad1 = descPrim.rad[0];

    float3 pos2 = descPrim.pos[1];
    float rad2 = descPrim.rad[1];

    float3 pos3 = descPrim.pos[2];
    float rad3 = descPrim.rad[2];

    f = sdfPrim3(x, pos1, pos2, pos3, rad1, rad2, rad3);

    return f;
}

inline __device__ float3 computeNormal(float eps, float3 x, primParamDesc descPrim)
{
    float dx = evaluateFunction(x + make_float3(eps, 0, 0), descPrim) - evaluateFunction(x - make_float3(eps, 0, 0), descPrim);
    float dy = evaluateFunction(x + make_float3(0, eps, 0), descPrim) - evaluateFunction(x - make_float3(0, eps, 0), descPrim);
    float dz = evaluateFunction(x + make_float3(0, 0, eps), descPrim) - evaluateFunction(x - make_float3(0, 0, eps), descPrim);

    return normalize(make_float3(dx, dy, dz));
}

inline __device__ float SphereTraceForward(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*t;

    float dist;

    float totalDistance = t;
    int i = 0;
    bool stop = false;
    while (!stop)
    {
        dist = evaluateFunction(x, descPrim);// sdfPrim1(x, pos, pos2, rad1, rad2);

                                             // Step along the ray and accumulate the distance from the origin.
        x += abs(dist) * ray_direction;
        totalDistance += abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance >= tmax)
        {
            stop = true;
        }
        if (dist < 0) {
            //x -= abs(dist) * ray_direction;
            totalDistance -= abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

inline __device__ float SphereTraceBack(float epsilon, float t, float tmax, primParamDesc descPrim)
{
    optix::float3 ray_direction = theRay.direction;
    optix::float3 x = theRay.origin + theRay.direction*tmax;

    float dist;

    //TODO: getType

    float totalDistance = tmax - t;
    int i = 0;
    bool stop = false;

    while (!stop)
    {
        //dist = sdfPrim1(x, pos, pos2, rad1, rad2);
        dist = evaluateFunction(x, descPrim);// sdfPrim1(x, pos, pos2, rad1, rad2);

                                             // Step along the ray and accumulate the distance from the origin.
        x -= abs(dist) * ray_direction;
        totalDistance -= abs(dist);

        // Check if we're close enough or too far.
        if (abs(dist) < epsilon || totalDistance <= 0)
        {
            stop = true;
        }
        if (dist < 0) {
            x += abs(dist) * ray_direction;
            totalDistance += abs(dist);
            stop = true;
        }
    }
    return totalDistance;
}

//------------------------

inline __device__  primParamDesc getTimeData(int primIdx)
{
    const int2 idsBonds = Mols[primIdx];

    const int2 idsB1 = Bonds[idsBonds.x];
    const int2 idsB2 = Bonds[idsBonds.y];

    //idsB1.x==idsB2.x;

    //we know there are only 3 atoms - TODO:
    const float rad1 = BSRadius[idsB1.x - 1];
    const float rad2 = BSRadius[idsB1.y - 1];
    const float rad3 = BSRadius[idsB2.y - 1];

    const int type = BSType[idsB1.x - 1];
    const int type2 = BSType[idsB1.y - 1];
    const int type3 = BSType[idsB2.y - 1];

    //---------------
    int lower = int(floorf(TimeSound));
    int upper = int(ceilf(TimeSound));

    //0,1;1,2;2,3;

    float timeS = TimeSound;
    if (upper > numFrames) upper = numFrames;

    float time = timeS - float(lower);

    //int time = int(floorf(TimeSound)); //integer part
    //int upper=int(time);

    //if (time > numFrames) time = numFrames;

    //for bond interpolation
    float3 pos2 = make_float3(0);
    float3 pos3 = make_float3(0);
    float3 pos = make_float3(0);

    //for frames
    float3 pos12 = make_float3(0);
    float3 pos13 = make_float3(0);
    float3 pos11 = make_float3(0);

    float3 pos22 = make_float3(0);
    float3 pos23 = make_float3(0);
    float3 pos21 = make_float3(0);

    //float3 pos1 = Positions[primIdx + time*PNum]; //getting correct frame

    pos11 = Positions[idsB1.x - 1 + lower*PNum];
    pos12 = Positions[idsB1.y - 1 + lower*PNum];
    pos13 = Positions[idsB2.y - 1 + lower*PNum];

    pos = Positions[idsB1.x - 1 + lower*PNum];
    pos2 = Positions[idsB1.y - 1 + lower*PNum];
    pos3 = Positions[idsB2.y - 1 + lower*PNum];

    if (numFrames > 0) //dynamic
    {
        pos21 = Positions[idsB1.x - 1 + upper*PNum];
        pos22 = Positions[idsB1.y - 1 + upper*PNum];
        pos23 = Positions[idsB2.y - 1 + upper*PNum];

        pos = time*pos21 + (1.0 - time)*pos11; //time interpolation
        pos2 = time*pos22 + (1.0 - time)*pos12; //time interpolation
        pos3 = time*pos23 + (1.0 - time)*pos13; //time interpolation
    }

    primParamDesc descPrim;

    descPrim.type = 3; //sphere data type
    descPrim.pos[0] = pos;
    descPrim.pos[1] = pos2;
    descPrim.pos[2] = pos3;

    descPrim.rad[0] = rad1;
    descPrim.rad[1] = rad2;
    descPrim.rad[2] = rad3;

    descPrim.types[0] = type;
    descPrim.types[1] = type2;
    descPrim.types[2] = type3;

    return descPrim;
}
//------------------------------------------------------
//---intersection with dynamic molecule, use of morphing
//------------------------------------------------------

inline __device__ float3 transfer_function(int t, float d)
{
    // return TFBuffer[t];
    switch (t)
    {
    case 1: //H
        return make_float3(1, 1, 1);
        break;
    case 2: //C
        return make_float3(0.5);
        break;
    case 3: //N
        return make_float3(0, 0, 0.5);
        break;
    case 4: //S
        return make_float3(1, 1, 0);
        break;
    case 5: //O
        return make_float3(1, 0, d);
        break;
    case 6: //P
        return make_float3(1, 0.5, 0);
        break;
    }
    return make_float3(0, 0, 0);
}

inline __device__ float4  blendColor(float dMd, float3 x, float3 dir, float3 pos, float3 pos2, float3 pos3, float r1, float r2, float r3, float3 col1, float3 col2, float3 col3)
{
    float dNorm = 10;

    float3 vib_color[3];
    vib_color[0] = make_float3(0, 1.0, 0); //green
    vib_color[1] = make_float3(0, 0.0, 1.0); //blue
    vib_color[2] = make_float3(1, 1, 0); //yellow

    float3 wt_col = make_float3(0);
    dNorm = length(pos - pos2) + length(pos3 - pos2) + length(pos - pos3);
    dNorm /= 3;

    float d1 = (length(x - pos) - r1*(1 + dMd)) / dNorm;
    float d2 = (length(x - pos2) - r2*(1 + dMd)) / dNorm;
    float d3 = (length(x - pos3) - r3*(1 + dMd)) / dNorm;

    d1 = optix::clamp(d1, 0.01, 1.0);
    d2 = optix::clamp(d2, 0.01, 1.0);
    d3 = optix::clamp(d3, 0.01, 1.0);

    float3 color = make_float3(0);
    float tr = abs(dMd - 0.1);
    color += (1 - d1)*col1;
    color += (1 - d2)*col2;
    color += (1 - d3)*col3;
    //color += dMd*make_float3(1);
    //color*=tr;

    //dMd=1 - rep1;
    //dMd=0 -rep2
    //dMd*10

    float4 color_sample = make_float4(color.x*(1 - tr), color.y*(1 - tr), color.z*(1 - tr), tr);
    //float4 color_sample = make_float4(tr);
        //volume rendering
    float3 xx = x;
    int maxSteps = 7;
    float step = 2 * r1 / maxSteps;
    for (int i = 0; i < maxSteps; i++) {
        float d1 = (length(xx - pos) - r1);
        float d2 = (length(xx - pos2) - r2);
        float d3 = (length(xx - pos3) - r3);

        //  d1 = optix::clamp(d1, -0.9, 1.0);
        //  d2 = optix::clamp(d2, -0.9, 1.0);
        //  d3 = optix::clamp(d3, -0.9, 1.0);
        float3 col = make_float3(0);
        if (d1 < 0.1)
            col += abs(d1 / r1)*col1;
        if (d2 < 0.1)
            col += abs(d2 / r2)*col2;
        if (d3 < 0.1)
            col += abs(d3 / r3)*col3;

        //------------
        //interpolate between two models
        float3 coll = make_float3(0);
        d1 = optix::clamp(d1, -0.9, 0.0);
        d2 = optix::clamp(d2, -0.9, 0.0);
        d3 = optix::clamp(d3, -0.9, 0.0);
        coll += abs(1 - d1)*col1*abs(d1);
        coll += abs(1 - d2)*col2*abs(d2);
        coll += abs(1 - d3)*col3*abs(d3);
        //coll ;
        col = dMd*coll + (1 - dMd)*col;

        //col+=dMd*make_float3(1);

        //d1 += 0.1;
        //d2 += 0.1;
        //d3 += 0.1;
        tr = abs(1 - min(min(d1, d2), d3)) / 2;//abs(min(min(d1 / r1, d2 / r2), d3 / r3)) / 2 +
        //tr /= 3;
        //tr=clamp(tr,0,1);
        // tr = (3 - d1 - d2 - d3) / 3;
        float4 col2 = make_float4(col.x, col.y, col.z, tr);
        // pre-multiply alpha
       // col2.w = (1 - d);
        col2.x *= col2.w;
        col2.y *= col2.w;
        col2.z *= col2.w;
        // "over" operator for front-to-back blending
        color_sample = color_sample + col2*(1.0f - color_sample.w);
        color_sample.w = tr;
        xx += theRay.direction*step;
    }

    //volume rendering
    //----------
//color_sample=make_float4(color,1.0);
    //return make_float3(color_sample.x, color_sample.y, color_sample.z);
    return color_sample;
}

inline __device__   float3 boundIntersection(primParamDesc  descPrim, float3 origin, float3 direction)
{
    // float t1, t2;
    float3 pos_along_ray1, per_ray_data2;
    float rayBoundSphere1, rayBoundSphere2;
    int caseN = -1; //ray miss

                    // float leng = length(pos2 - pos) + length(pos3 - pos);
    const float3 cent = (descPrim.pos[2] + descPrim.pos[1] + descPrim.pos[0]) / 3;
    const float delta = fmax(fmaxf(descPrim.rad[1], descPrim.rad[2]), descPrim.rad[0]);
    const float rayBoundSphere = length(cent - descPrim.pos[0]) + delta * 2;//include rad*2 for vibrations movement

    float t = length(cent - theRay.origin);
    float3 pos_along_ray = theRay.origin + theRay.direction * t;
    float tmax = t + 2 * rayBoundSphere; //to stop sphere tracing

                                   /* alternative splitting*/
                                   /*
                                   float3 cent1=(pos+pos2)/2;
                                   float3 cent2= (pos+pos3)/2;

                                   t1 = length(cent1 - theRay.origin);
                                   t2 = length(cent2 - theRay.origin);
                                   pos_along_ray1 = theRay.origin + theRay.direction * t1;
                                   pos_along_ray2 = theRay.origin + theRay.direction * t2;
                                   rayBoundSphere1 =length(cent1 - pos) + delta * 2;
                                   rayBoundSphere2 =length(cent1 - pos) + delta * 2;
                                   */

    float radiusB1 = length(descPrim.pos[0] - descPrim.pos[1]) / 2 + delta;
    float radiusB2 = length(descPrim.pos[0] - descPrim.pos[2]) / 2 + delta;
    float radiusB = fmaxf(radiusB1, radiusB2);

    //compute center of attraction as there may be several intersections with central part
    float t1, t2, t3;
    t1 = 0;
    t2 = 0;
    t3 = 0;

    t1 = length(descPrim.pos[1] - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t1;

    if (length(descPrim.pos[1] - pos_along_ray) < radiusB1)
        caseN = 1; //left

    t2 = length(descPrim.pos[2] - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t2;
    if (length(descPrim.pos[2] - pos_along_ray) < radiusB2)
        caseN = 2; //left

    t3 = length(descPrim.pos[0] - theRay.origin);
    pos_along_ray = theRay.origin + theRay.direction * t3;
    if (length(descPrim.pos[0] - pos_along_ray) < radiusB)
        caseN = 0; //central intersection

                   //get min intersection
    t = fminf(fminf(t1, t2), t3);

    float     tmin = fmaxf(0.0, t - radiusB);
    float3 params = make_float3(tmin, tmax, 0.0);
    if (caseN >= 0)
    {
        params.z = 1.0;
    }
    else  params.z = 0.0; //within bounding sphere

    return params;
}
RT_PROGRAM void intersection_molecules(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;

    float epsilon = 0.001;//delta;
    if (epsilon > sysSceneEpsilon) epsilon = sysSceneEpsilon;
    float eps = 0.001;
    if (eps > sysSceneEpsilon) eps = sysSceneEpsilon;

    /* ------------------
    /* 1) Reading data and accessing current positions for current time
    /---------------------------------------------------------------*/

    primParamDesc descPrim = getTimeData(primIdx);

    //--------------------

    //---level of detail
    float dist_cam = length(sysCameraPosition - (descPrim.pos[0] + descPrim.pos[1] + descPrim.pos[2]) / 3);
    float d;
    if (dist_cam < 20.0)
    {
        float interp = (dist_cam - 10) / 10.0;
        d = optix::clamp(interp, 0.0, 1.0);
    }
    else d = 1;

    //-----------------------------

    float3 interSectParams = boundIntersection(descPrim, theRay.origin, theRay.direction);
    //if within bounding volume intersection
    //float3 interSectParams = BoundingSubVolumesIntersect(descPrim);
    if (interSectParams.z > 0.0)
    {
        tmin = interSectParams.x;
        tmax = interSectParams.y;
        float totalDistance = tmin;
        // === Raymarching (Sphere Tracing) Procedure ===

        totalDistance = SphereTraceForward(epsilon, tmin, tmax, descPrim);

        // Found potential intersection?
        if (totalDistance < tmax) //we found intersection
        {
            float totalDistance2 = SphereTraceBack(epsilon, totalDistance, tmax, descPrim);
            //------------
            if (totalDistance2 > epsilon)
            { //it is sufficiently large subvolume to ray-cast
                if (rtPotentialIntersection(totalDistance))
                {
                    //compute normal for primitive
                    float3 x = theRay.origin + theRay.direction*totalDistance;

                    float3 cNormal = computeNormal(eps, x, descPrim);

                    infoH.normal = cNormal;
                    infoH.hit_point = theRay.origin + theRay.direction * (totalDistance);
                    infoH.tmin = totalDistance;

                    infoH.desc = descPrim;
                    infoH.maxDist = totalDistance2;
                    rtReportIntersection(MaterialIndex);
                }
            }
        }
    }
}

//bounding box
RT_PROGRAM void boundingbox_molecules(int primIdx, float result[6])
{
    const int2 idsBonds = Mols[primIdx];

    const int2 idsB1 = Bonds[idsBonds.x];
    const int2 idsB2 = Bonds[idsBonds.y];
    const float rad1 = BSRadius[idsB1.x - 1] * 2;
    const float rad2 = BSRadius[idsB1.y - 1] * 2;
    const float rad3 = BSRadius[idsB2.y - 1] * 2;

    float3 pos = Positions[idsB1.x - 1];
    float3 pos2 = Positions[idsB1.y - 1];
    float3 pos3 = Positions[idsB2.y - 1];

    float3 pos_min = fminf(pos, pos2);
    pos_min = fminf(pos_min, pos3);
    float3 pos_max = fmaxf(pos, pos2);
    pos_max = fmaxf(pos_max, pos3);
    if (numFrames > 0)
    {
        for (int i = 1; i < numFrames; i++)
        {
            pos = Positions[idsB1.x - 1 + i*PNum];
            pos2 = Positions[idsB1.y - 1 + i*PNum];
            pos3 = Positions[idsB2.y - 1 + i*PNum];

            pos_min = fminf(fminf(pos, pos2), pos_min);
            pos_min = fminf(pos_min, pos3);

            pos_max = fmaxf(fmaxf(pos, pos2), pos_max);
            pos_max = fmaxf(pos_max, pos3);
        }
    }

    float rad = fmaxf(rad1, rad2);
    rad = fmaxf(rad, rad3);
    optix::Aabb* aabb = (optix::Aabb*)result;
    //increase for ao by 5
    aabb->m_min = pos_min - make_float3(rad);
    aabb->m_max = pos_max + make_float3(rad);
}