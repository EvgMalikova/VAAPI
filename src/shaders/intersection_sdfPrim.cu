#include "hip/hip_runtime.h"
/*
All basic variables for SDFs visual-auditory ray-tracing
 */
#include "sdfGeometryVariables.h"

 //for SDF spehre and others
rtDeclareVariable(optix::float3, varCenter, , );
rtDeclareVariable(optix::float3, varRadius, , );

//for SDF

typedef rtCallableProgramId<float(float3, float3)> callT;
rtDeclareVariable(callT, sdfPrim, , );

//TODO:
//or create a callT Buffer

//second buffer of parameters for the  buffer of called programs
//should be created as well

RT_PROGRAM void intersection_sdf_sphere(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;

    const float sqRadius = 100;

    float distance;
    // === Raymarching (Sphere Tracing) Procedure ===
    optix::float3 rd = theRay.direction;
    optix::float3 eye = theRay.origin;
    float eps = 0.001;
    float t = 0.002;
    float material = -1;
    if (varRadius.x > 0.0) //responsible for show/hide
    {
        for (int i = 0; i < 1000; i++)
        {
            optix::float3 p = eye + t* rd;
            float hit = sdfPrim(p, varRadius);
            if (hit < 0.002) //|| t > 20.0)
            {
                //printf("was hitted\n");
                if (rtPotentialIntersection(t))
                {
                    //        sdf.setMaxIterations(14); // more iterations for normal estimate, to fake some more detail
                    // varNormal        = calculateNormal(sdf, x, DEL);

                    float dx = sdfPrim(p + make_float3(eps, 0, 0), varRadius) - sdfPrim(p - make_float3(eps, 0, 0), varRadius);
                    float dy = sdfPrim(p + make_float3(0, eps, 0), varRadius) - sdfPrim(p - make_float3(0, eps, 0), varRadius);
                    float dz = sdfPrim(p + make_float3(0, 0, eps), varRadius) - sdfPrim(p - make_float3(0, 0, eps), varRadius);

                    info.normal = normalize(make_float3(dx, dy, dz));
                    info.hit_point = theRay.origin + t*theRay.direction;
                    info.type = 1;
                    info.maxDist = varRadius.x*2;
                    //rtReportIntersection(0);
                    rtReportIntersection(MaterialIndex);
                }
                break;
            }

            if (t > 20.0) {
                break; //no intersection
            }
            t += abs(hit); //only positive direction
        }
    }
}

RT_PROGRAM void boundingbox_sdf_sphere(int, float result[6])
{
    optix::Aabb* aabb = (optix::Aabb*)result;

    aabb->m_min = varCenter - varRadius;// optix::make_float3(varCenter.x - varRadius, varCenter.y - varRadius, varCenter.z - varRadius);
    aabb->m_max = varCenter + varRadius; // optix::make_float3(varCenter.x + varRadius, varCenter.y + varRadius, varCenter.z + varRadius);
}