#include "hip/hip_runtime.h"
/*
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_math.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "attributeInfo.h"
#include "renderer/random_number_generators.h"
using namespace optix;

rtBuffer<float3>    Positions;
rtBuffer<float>    BSRadius;
rtBuffer<int>    BSType;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, pr_pos, attribute primitive_pos, );
rtDeclareVariable(float, pr_rad, attribute primitive_rad, );
//rtDeclareVariable(int, pr_type, attribute primitive_type, );

//rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, ); //for direct tracing of sdf spheres, or defined set of primitives with definde BB

//sets Material index to call
//0 - optical type
//1 - auditory type

rtDeclareVariable(int, MaterialIndex, , );
rtDeclareVariable(int, PNum, , );
rtDeclareVariable(int, numFrames, , );
rtDeclareVariable(float, TimeSound, , );
rtDeclareVariable(attributeInfo, info, attribute info, );

//TODO:for SDF
//still have to be implemented as PTX should be generated automatically
//PTX can automatically generated for some primitives like spheres, boxes and etc
//consider further integration with python

typedef rtCallableProgramId<float(float3, float)> callT;
rtDeclareVariable(callT, sdfPrim, , );

//for accel build
RT_PROGRAM void primitive_bounds(int primIdx, float result[6])
{
    const float3 position = Positions[primIdx];
    //we compute only bounding sphere like parameters
    //the exact bounding box of SDF primitive can be much smaller
    const float radius = BSRadius[primIdx];

    optix::Aabb *aabb = (optix::Aabb *) result;

    aabb->m_min.x = position.x - radius;
    aabb->m_min.y = position.y - radius;
    aabb->m_min.z = position.z - radius;

    aabb->m_max.x = position.x + radius;
    aabb->m_max.y = position.y + radius;
    aabb->m_max.z = position.z + radius;
}

//intersection for molecular data

RT_PROGRAM void intersection_mol(int primIdx)
{
    float tmin, tmax;
    tmin = 0;
    tmax = RT_DEFAULT_MAX;

    //---------------
    int lower = int(floorf(TimeSound));
    int upper = int(ceilf(TimeSound));

    float timeS = TimeSound;
    if (upper > numFrames) upper = numFrames;

    float time = timeS - float(lower);

    float3 pos2 = make_float3(0);
    float3 pos = make_float3(0);
    float3 pos1 = Positions[primIdx + lower*PNum]; //getting correct frame

    if (numFrames > 0) //dynamic
    {
        pos2 = Positions[primIdx + upper*PNum];
        pos = time*pos2 + (1.0 - time)*pos1; //time interpolation
    }
    else
        pos = pos1; //static

    const float rad = BSRadius[primIdx];
    const int type = BSType[primIdx];
    const float t = length(pos - ray.origin);
    const float3 pos_along_ray = ray.origin + ray.direction * t;
    tmax = t + 2 * rad;
    if (length(pos - pos_along_ray) < (rad))
    {
        tmin = fmaxf(0.0, t - rad);
        // === Raymarching (Sphere Tracing) Procedure ===
        optix::float3 ray_direction = ray.direction;
        optix::float3 eye = ray.origin;
        //    eye.x -= global_t * 1.2f;
        optix::float3 x = eye + tmin * ray_direction;

        const float epsilon = 0.002;//delta;
        const float eps = 0.0001;
        float dist = 0;

        float totalDistance = tmin;//fmaxf(0.0, t - rad);//Jitter * tea<4>(current_prd.seed, frame_number);
        int i = 0;
        bool stop = false;
        while (!stop)
        {
            dist = sdfPrim(x - pos, rad);
            // Step along the ray and accumulate the distance from the origin.
            x += abs(dist) * ray_direction;
            totalDistance += abs(dist);

            // Check if we're close enough or too far.
            if (dist < epsilon || totalDistance > tmax)
            {
                stop = true;
            }
            else i++;
        }

        // Found intersection?
        if (abs(dist) < epsilon)
        {
            if (rtPotentialIntersection(totalDistance))
            {
                //compute normal for primitive
                float dx = sdfPrim(x - pos + make_float3(eps, 0, 0), rad) - sdfPrim(x - pos - make_float3(eps, 0, 0), rad);
                float dy = sdfPrim(x - pos + make_float3(0, eps, 0), rad) - sdfPrim(x - pos - make_float3(0, eps, 0), rad);
                float dz = sdfPrim(x - pos + make_float3(0, 0, eps), rad) - sdfPrim(x - pos - make_float3(0, 0, eps), rad);

                //varNormal = normalize(make_float3(dx, dy, dz));

                //for material
                //fill attribute data for material
                float2 inf = make_float2(totalDistance, __int_as_float(type));
                info.primInfo = inf;
                info.type = type;
                info.normal = normalize(make_float3(dx, dy, dz));
                info.hit_point = ray.origin + ray.direction * (totalDistance);
                info.pos[0] = pos;
                info.rad[0] = rad;
                pr_pos = pos;
                pr_rad = rad;

                rtReportIntersection(MaterialIndex);
            }
        }
    }
}

//bounding box
RT_PROGRAM void boundingbox_mol(int primIdx, float result[6])
{
    float3 posMin = Positions[primIdx];
    float3 posMax = Positions[primIdx];
    float rad = BSRadius[primIdx];

    if (numFrames > 0)//dynamic
    {
        for (int i = 1; i < numFrames; i++) {
            float3 pos1 = Positions[primIdx + i*PNum]; //getting correct frame
            posMin.x = fminf(pos1.x, posMin.x);
            posMin.y = fminf(pos1.y, posMin.y);
            posMin.z = fminf(pos1.z, posMin.z);

            posMax.x = fmaxf(pos1.x, posMax.x);
            posMax.y = fmaxf(pos1.y, posMax.y);
            posMax.z = fmaxf(pos1.z, posMax.z);
        }
    }

    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->m_min = posMin - make_float3(rad);
    aabb->m_max = posMax + make_float3(rad);
}